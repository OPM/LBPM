#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

#define NBLOCKS 1024
#define NTHREADS 256

/*
1. constants that are known at compile time should be defined using preprocessor macros (e.g. #define) or via C/C++ const variables at global/file scope.
2. Usage of __constant__ memory may be beneficial for programs who use certain values that don't change for the duration of the kernel and for which certain access patterns are present (e.g. all threads access the same value at the same time). This is not better or faster than constants that satisfy the requirements of item 1 above.
3. If the number of choices to be made by a program are relatively small in number, and these choices affect kernel execution, one possible approach for additional compile-time optimization would be to use templated code/kernels
 */

__constant__ __device__ double mrt_V1=0.05263157894736842;
__constant__ __device__ double mrt_V2=0.012531328320802;
__constant__ __device__ double mrt_V3=0.04761904761904762;
__constant__ __device__ double mrt_V4=0.004594820384294068;
__constant__ __device__ double mrt_V5=0.01587301587301587;
__constant__ __device__ double mrt_V6=0.0555555555555555555555555;
__constant__ __device__ double mrt_V7=0.02777777777777778;
__constant__ __device__ double mrt_V8=0.08333333333333333;
__constant__ __device__ double mrt_V9=0.003341687552213868;
__constant__ __device__ double mrt_V10=0.003968253968253968;
__constant__ __device__ double mrt_V11=0.01388888888888889;
__constant__ __device__ double mrt_V12=0.04166666666666666;


// functionality for parallel reduction in Flux BC routines -- probably should be re-factored to another location
// functions copied from https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/

//__shared__ double Transform[722]=
//	   {};

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val) { 
   unsigned long long int* address_as_ull = (unsigned long long int*)address;
   unsigned long long int old = *address_as_ull, assumed;

   do {
      assumed = old;
      old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val+__longlong_as_double(assumed)));
   } while (assumed != old);
   return __longlong_as_double(old);
}
#endif

using namespace cooperative_groups;
__device__ double reduce_sum(thread_group g, double *temp, double val)
{
    int lane = g.thread_rank();

    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2)
    {
        temp[lane] = val;
        g.sync(); // wait for all threads to store
        if(lane<i) val += temp[lane + i];
        g.sync(); // wait for all threads to load
    }
    return val; // note: only thread 0 will return full sum
}

__device__ double thread_sum(double *input, double n) 
{
    double sum = 0;

    for(int i = blockIdx.x * blockDim.x + threadIdx.x;
        i < n / 4; 
        i += blockDim.x * gridDim.x)
    {
        int4 in = ((int4*)input)[i];
        sum += in.x + in.y + in.z + in.w;
    }
    return sum;
}

__global__ void sum_kernel_block(double *sum, double *input, int n)
{
	double my_sum = thread_sum(input, n);

    extern __shared__ double temp[];
    thread_group g = this_thread_block();
    double block_sum = reduce_sum(g, temp, my_sum);

    if (g.thread_rank() == 0) atomicAdd(sum, block_sum);
}

__inline__ __device__
double warpReduceSum(double val) {
	for (int offset = warpSize/2; offset > 0; offset /= 2)
		val += __shfl_down_sync(0xFFFFFFFF, val, offset, 32);
	return val;
}

__inline__ __device__
double blockReduceSum(double val) {

	static __shared__ double shared[32]; // Shared mem for 32 partial sums
	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;

	val = warpReduceSum(val);     // Each warp performs partial reduction

	if (lane==0) shared[wid]=val; // Write reduced value to shared memory

	__syncthreads();              // Wait for all partial reductions

	//read from shared memory only if that warp existed
	val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

	if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

	return val;
}

__global__ void deviceReduceKernel(double *in, double* out, int N) {
	double sum = 0;
	//reduce multiple elements per thread
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
			i < N;
			i += blockDim.x * gridDim.x) {
		sum += in[i];
	}
	sum = blockReduceSum(sum);
	if (threadIdx.x==0)
		out[blockIdx.x]=sum;
}

__global__ void dvc_ScaLBL_D3Q19_Pack(int q, int *list, int start, int count, double *sendbuf, double *dist, int N){
	//....................................................................................
	// Pack distribution q into the send buffer for the listed lattice sites
	// dist may be even or odd distributions stored by stream layout
	//....................................................................................
	int idx,n;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<count){
		n = list[idx];
		sendbuf[start+idx] = dist[q*N+n];
		//printf("%f \n",dist[q*N+n]);
	}

}

__global__ void dvc_ScaLBL_D3Q19_Unpack(int q,  int *list,  int start, int count,
		double *recvbuf, double *dist, int N){
	//....................................................................................
	// Unpack distribution from the recv buffer
	// Distribution q matche Cqx, Cqy, Cqz
	// swap rule means that the distributions in recvbuf are OPPOSITE of q
	// dist may be even or odd distributions stored by stream layout
	//....................................................................................
	int n,idx;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<count){
		// Get the value from the list -- note that n is the index is from the send (non-local) process
		n = list[start+idx];
		// unpack the distribution to the proper location
		if (!(n<0)) { dist[q*N+n] = recvbuf[start+idx];
		//printf("%f \n",,dist[q*N+n]);
		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_Init(char *ID, double *f_even, double *f_odd, int Nx, int Ny, int Nz)
{
	int n,N;
	N = Nx*Ny*Nz;
	char id;
	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N ){
			id = ID[n];
			if (id > 0 ){
				f_even[n] = 0.3333333333333333;
				f_odd[n] = 0.055555555555555555;		//double(100*n)+1.f;
				f_even[N+n] = 0.055555555555555555;	//double(100*n)+2.f;
				f_odd[N+n] = 0.055555555555555555;	//double(100*n)+3.f;
				f_even[2*N+n] = 0.055555555555555555;	//double(100*n)+4.f;
				f_odd[2*N+n] = 0.055555555555555555;	//double(100*n)+5.f;
				f_even[3*N+n] = 0.055555555555555555;	//double(100*n)+6.f;
				f_odd[3*N+n] = 0.0277777777777778;   //double(100*n)+7.f;
				f_even[4*N+n] = 0.0277777777777778;   //double(100*n)+8.f;
				f_odd[4*N+n] = 0.0277777777777778;   //double(100*n)+9.f;
				f_even[5*N+n] = 0.0277777777777778;  //double(100*n)+10.f;
				f_odd[5*N+n] = 0.0277777777777778;  //double(100*n)+11.f;
				f_even[6*N+n] = 0.0277777777777778;  //double(100*n)+12.f;
				f_odd[6*N+n] = 0.0277777777777778;  //double(100*n)+13.f;
				f_even[7*N+n] = 0.0277777777777778;  //double(100*n)+14.f;
				f_odd[7*N+n] = 0.0277777777777778;  //double(100*n)+15.f;
				f_even[8*N+n] = 0.0277777777777778;  //double(100*n)+16.f;
				f_odd[8*N+n] = 0.0277777777777778;  //double(100*n)+17.f;
				f_even[9*N+n] = 0.0277777777777778;  //double(100*n)+18.f;
			}
			else{
				for(int q=0; q<9; q++){
					f_even[q*N+n] = -1.0;
					f_odd[q*N+n] = -1.0;
				}
				f_even[9*N+n] = -1.0;
			}
		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_AA_Init(double *f_even, double *f_odd, int Np)
{
	int n;
	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<Np ){
			f_even[n] = 0.3333333333333333;
			f_odd[n] = 0.055555555555555555;                //double(100*n)+1.f;
			f_even[Np+n] = 0.055555555555555555;    //double(100*n)+2.f;
			f_odd[Np+n] = 0.055555555555555555;     //double(100*n)+3.f;
			f_even[2*Np+n] = 0.055555555555555555;  //double(100*n)+4.f;
			f_odd[2*Np+n] = 0.055555555555555555;   //double(100*n)+5.f;
			f_even[3*Np+n] = 0.055555555555555555;  //double(100*n)+6.f;
			f_odd[3*Np+n] = 0.0277777777777778;   //double(100*n)+7.f;
			f_even[4*Np+n] = 0.0277777777777778;   //double(100*n)+8.f;
			f_odd[4*Np+n] = 0.0277777777777778;   //double(100*n)+9.f;
			f_even[5*Np+n] = 0.0277777777777778;  //double(100*n)+10.f;
			f_odd[5*Np+n] = 0.0277777777777778;  //double(100*n)+11.f;
			f_even[6*Np+n] = 0.0277777777777778;  //double(100*n)+12.f;
			f_odd[6*Np+n] = 0.0277777777777778;  //double(100*n)+13.f;
			f_even[7*Np+n] = 0.0277777777777778;  //double(100*n)+14.f;
			f_odd[7*Np+n] = 0.0277777777777778;  //double(100*n)+15.f;
			f_even[8*Np+n] = 0.0277777777777778;  //double(100*n)+16.f;
			f_odd[8*Np+n] = 0.0277777777777778;  //double(100*n)+17.f;
			f_even[9*Np+n] = 0.0277777777777778;  //double(100*n)+18.f;
		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_Init(double *dist, int Np)
{
	int n;
	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<Np ){
			dist[n] = 0.3333333333333333;
			dist[Np+n] = 0.055555555555555555;		//double(100*n)+1.f;
			dist[2*Np+n] = 0.055555555555555555;	//double(100*n)+2.f;
			dist[3*Np+n] = 0.055555555555555555;	//double(100*n)+3.f;
			dist[4*Np+n] = 0.055555555555555555;	//double(100*n)+4.f;
			dist[5*Np+n] = 0.055555555555555555;	//double(100*n)+5.f;
			dist[6*Np+n] = 0.055555555555555555;	//double(100*n)+6.f;
			dist[7*Np+n] = 0.0277777777777778;   //double(100*n)+7.f;
			dist[8*Np+n] = 0.0277777777777778;   //double(100*n)+8.f;
			dist[9*Np+n] = 0.0277777777777778;   //double(100*n)+9.f;
			dist[10*Np+n] = 0.0277777777777778;  //double(100*n)+10.f;
			dist[11*Np+n] = 0.0277777777777778;  //double(100*n)+11.f;
			dist[12*Np+n] = 0.0277777777777778;  //double(100*n)+12.f;
			dist[13*Np+n] = 0.0277777777777778;  //double(100*n)+13.f;
			dist[14*Np+n] = 0.0277777777777778;  //double(100*n)+14.f;
			dist[15*Np+n] = 0.0277777777777778;  //double(100*n)+15.f;
			dist[16*Np+n] = 0.0277777777777778;  //double(100*n)+16.f;
			dist[17*Np+n] = 0.0277777777777778;  //double(100*n)+17.f;
			dist[18*Np+n] = 0.0277777777777778;  //double(100*n)+18.f;
		}
	}
}

//*************************************************************************
__global__  void dvc_ScaLBL_D3Q19_Swap_Compact(int *neighborList, double *disteven, double *distodd, int Np, int q){
	int n,nn;
	double f1,f2;
	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<Np){
			nn = neighborList[q*Np+n];
			if (!(nn<0)){
				f1 = distodd[q*Np+n];
				f2 = disteven[(q+1)*Np+nn];
				disteven[(q+1)*Np+nn] = f1;
				distodd[q*Np+n] = f2;
			}
		}
	}
}

//__launch_bounds__(512,4)

__global__ void 
dvc_ScaLBL_AAodd_Compact(int *d_neighborList, double *dist, int Np) {

	int n;
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int nread;
	int S = Np/NBLOCKS/NTHREADS+1;

	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<Np) {

			f0 = dist[n];

			nread = d_neighborList[n];
			f1 = dist[nread];

			nread = d_neighborList[n+2*Np];
			f3 = dist[nread];

			nread = d_neighborList[n+4*Np];
			f5 = dist[nread];

			nread = d_neighborList[n+6*Np];
			f7 = dist[nread];

			nread = d_neighborList[n+8*Np];
			f9 = dist[nread];

			nread = d_neighborList[n+10*Np];
			f11 = dist[nread];

			nread = d_neighborList[n+12*Np];
			f13 = dist[nread];

			nread = d_neighborList[n+14*Np];
			f15 = dist[nread];

			nread = d_neighborList[n+16*Np];
			f17 = dist[nread];


			nread = d_neighborList[n+Np];
			f2 = dist[nread];

			nread = d_neighborList[n+3*Np];
			f4 = dist[nread];

			nread = d_neighborList[n+5*Np];
			f6 = dist[nread];

			nread = d_neighborList[n+7*Np];
			f8 = dist[nread];

			nread = d_neighborList[n+9*Np];
			f10 = dist[nread];

			nread = d_neighborList[n+11*Np];
			f12 = dist[nread];

			nread = d_neighborList[n+13*Np];
			f14 = dist[nread];

			nread = d_neighborList[n+15*Np];
			f16 = dist[nread];

			nread = d_neighborList[n+17*Np];
			f18 = dist[nread];

			// ORIGINAL CORRECT WRITES
			//                              nwrite = d_neighborList[n];      naccess = 10*Np;
			//                              if (nwrite<0) { nwrite=n;        naccess = Np;  }
			//                              dist[nwrite + naccess]   = f1;

			//                              nwrite = d_neighborList[n+Np];   naccess = Np;
			//                              if (nwrite<0) { nwrite=n;        naccess = 10*Np; }
			//                              dist[nwrite + naccess]   = f2;

			nread = d_neighborList[n];
			dist[nread] = f2;

			nread = d_neighborList[n+2*Np];
			dist[nread] = f4;

			nread = d_neighborList[n+4*Np];
			dist[nread] = f6;

			nread = d_neighborList[n+6*Np];
			dist[nread] = f8;

			nread = d_neighborList[n+8*Np];
			dist[nread] = f10;

			nread = d_neighborList[n+10*Np];
			dist[nread] = f12;

			nread = d_neighborList[n+12*Np];
			dist[nread] = f14;

			nread = d_neighborList[n+14*Np];
			dist[nread] = f16;

			nread = d_neighborList[n+16*Np];
			dist[nread] = f18;


			nread = d_neighborList[n+Np];
			dist[nread] = f1;

			nread = d_neighborList[n+3*Np];
			dist[nread] = f3;

			nread = d_neighborList[n+5*Np];
			dist[nread] = f5;

			nread = d_neighborList[n+7*Np];
			dist[nread] = f7;

			nread = d_neighborList[n+9*Np];
			dist[nread] = f9;

			nread = d_neighborList[n+11*Np];
			dist[nread]= f11;

			nread = d_neighborList[n+13*Np];
			dist[nread] = f13;

			nread = d_neighborList[n+15*Np];
			dist[nread] = f15;

			nread = d_neighborList[n+17*Np];
			dist[nread] = f17;

		}
	}
}


__global__ void 
dvc_ScaLBL_AAodd_MRT(int *neighborList, double *dist, int start, int finish, int Np, double rlx_setA, double rlx_setB, double Fx, double Fy, double Fz) {

	int n;
	double fq;
	// conserved momemnts
	double rho,jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	int nread;
	int S = Np/NBLOCKS/NTHREADS+1;

	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			// q=0
			fq = dist[n];
			rho = fq;
			m1  = -30.0*fq;
			m2  = 12.0*fq;

			// q=1
			nread = neighborList[n]; // neighbor 2 ( > 10Np => odd part of dist)
			fq = dist[nread]; // reading the f1 data into register fq
			//fp = dist[10*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jx = fq;
			m4 = -4.0*fq;
			m9 = 2.0*fq;
			m10 = -4.0*fq;

			// f2 = dist[10*Np+n];
			nread = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
			fq = dist[nread];  // reading the f2 data into register fq
			//fq = dist[Np+n];
			rho += fq;
			m1 -= 11.0*(fq);
			m2 -= 4.0*(fq);
			jx -= fq;
			m4 += 4.0*(fq);
			m9 += 2.0*(fq);
			m10 -= 4.0*(fq);

			// q=3
			nread = neighborList[n+2*Np]; // neighbor 4
			fq = dist[nread];
			//fq = dist[11*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy = fq;
			m6 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 = fq;
			m12 = -2.0*fq;

			// q = 4
			nread = neighborList[n+3*Np]; // neighbor 3
			fq = dist[nread];
			//fq = dist[2*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy -= fq;
			m6 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 += fq;
			m12 -= 2.0*fq;

			// q=5
			nread = neighborList[n+4*Np];
			fq = dist[nread];
			//fq = dist[12*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz = fq;
			m8 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;


			// q = 6
			nread = neighborList[n+5*Np];
			fq = dist[nread];
			//fq = dist[3*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz -= fq;
			m8 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q=7
			nread = neighborList[n+6*Np];
			fq = dist[nread];
			//fq = dist[13*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy += fq;
			m6 += fq;
			m9  += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 = fq;
			m16 = fq;
			m17 = -fq;

			// q = 8
			nread = neighborList[n+7*Np];
			fq = dist[nread];
			//fq = dist[4*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 += fq;
			m16 -= fq;
			m17 += fq;

			// q=9
			nread = neighborList[n+8*Np];
			fq = dist[nread];
			//fq = dist[14*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 += fq;
			m17 += fq;

			// q = 10
			nread = neighborList[n+9*Np];
			fq = dist[nread];
			//fq = dist[5*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy += fq;
			m6 += fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 -= fq;
			m17 -= fq;

			// q=11
			nread = neighborList[n+10*Np];
			fq = dist[nread];
			//fq = dist[15*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 = fq;
			m16 -= fq;
			m18 = fq;

			// q=12
			nread = neighborList[n+11*Np];
			fq = dist[nread];
			//fq = dist[6*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 += fq;
			m16 += fq;
			m18 -= fq;

			// q=13
			nread = neighborList[n+12*Np];
			fq = dist[nread];
			//fq = dist[16*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 -= fq;
			m18 -= fq;

			// q=14
			nread = neighborList[n+13*Np];
			fq = dist[nread];
			//fq = dist[7*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 += fq;
			m18 += fq;

			// q=15
			nread = neighborList[n+14*Np];
			fq = dist[nread];
			//fq = dist[17*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 = fq;
			m17 += fq;
			m18 -= fq;

			// q=16
			nread = neighborList[n+15*Np];
			fq = dist[nread];
			//fq = dist[8*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 += fq;
			m17 -= fq;
			m18 += fq;

			// q=17
			//fq = dist[18*Np+n];
			nread = neighborList[n+16*Np];
			fq = dist[nread];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 += fq;
			m18 += fq;

			// q=18
			nread = neighborList[n+17*Np];
			fq = dist[nread];
			//fq = dist[9*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 -= fq;
			m18 -= fq;

			//..............incorporate external force................................................
			//..............carry out relaxation process...............................................
			m1 = m1 + rlx_setA*((19*(jx*jx+jy*jy+jz*jz)/rho - 11*rho) - m1);
			m2 = m2 + rlx_setA*((3*rho - 5.5*(jx*jx+jy*jy+jz*jz)/rho) - m2);
			m4 = m4 + rlx_setB*((-0.6666666666666666*jx) - m4);
			m6 = m6 + rlx_setB*((-0.6666666666666666*jy) - m6);
			m8 = m8 + rlx_setB*((-0.6666666666666666*jz) - m8);
			m9 = m9 + rlx_setA*(((2*jx*jx-jy*jy-jz*jz)/rho) - m9);
			m10 = m10 + rlx_setA*(-0.5*((2*jx*jx-jy*jy-jz*jz)/rho) - m10);
			m11 = m11 + rlx_setA*(((jy*jy-jz*jz)/rho) - m11);
			m12 = m12 + rlx_setA*(-0.5*((jy*jy-jz*jz)/rho) - m12);
			m13 = m13 + rlx_setA*((jx*jy/rho) - m13);
			m14 = m14 + rlx_setA*((jy*jz/rho) - m14);
			m15 = m15 + rlx_setA*((jx*jz/rho) - m15);
			m16 = m16 + rlx_setB*( - m16);
			m17 = m17 + rlx_setB*( - m17);
			m18 = m18 + rlx_setB*( - m18);
			//.......................................................................................................
			//.................inverse transformation......................................................

			// q=0
			fq = mrt_V1*rho-mrt_V2*m1+mrt_V3*m2;
			dist[n] = fq;

			// q = 1
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10)+0.16666666*Fx;
			nread = neighborList[n+Np];
			dist[nread] = fq;

			// q=2
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10) -  0.16666666*Fx;
			nread = neighborList[n];
			dist[nread] = fq;

			// q = 3
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) + 0.16666666*Fy;
			nread = neighborList[n+3*Np];
			dist[nread] = fq;

			// q = 4
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) - 0.16666666*Fy;
			nread = neighborList[n+2*Np];
			dist[nread] = fq;

			// q = 5
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) + 0.16666666*Fz;
			nread = neighborList[n+5*Np];
			dist[nread] = fq;

			// q = 6
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) - 0.16666666*Fz;
			nread = neighborList[n+4*Np];
			dist[nread] = fq;

			// q = 7
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+mrt_V7*m9+mrt_V11*m10+
					mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17) + 0.08333333333*(Fx+Fy);
			
			nread = neighborList[n+7*Np];
			dist[nread] = fq;

			// q = 8
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11
					+mrt_V12*m12+0.25*m13+0.125*(m17-m16) - 0.08333333333*(Fx+Fy);
			nread = neighborList[n+6*Np];
			dist[nread] = fq;

			// q = 9
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+mrt_V7*m9+mrt_V11*m10+
					mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17) + 0.08333333333*(Fx-Fy);
			nread = neighborList[n+9*Np];
			dist[nread] = fq;

			// q = 10
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+mrt_V7*m9+mrt_V11*m10+
					mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17)- 0.08333333333*(Fx-Fy);
			nread = neighborList[n+8*Np];
			dist[nread] = fq;

			// q = 11
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12+0.25*m15+0.125*(m18-m16) + 0.08333333333*(Fx+Fz);
			nread = neighborList[n+11*Np];
			dist[nread] = fq;

			// q = 12
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+
					mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18) - 0.08333333333*(Fx+Fz);
			nread = neighborList[n+10*Np];
			dist[nread]= fq;

			// q = 13
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15-0.125*(m16+m18) + 0.08333333333*(Fx-Fz);
			nread = neighborList[n+13*Np];
			dist[nread] = fq;

			// q= 14
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15+0.125*(m16+m18) - 0.08333333333*(Fx-Fz);
			nread = neighborList[n+12*Np];
			dist[nread] = fq;


			// q = 15
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18) + 0.08333333333*(Fy+Fz);
			nread = neighborList[n+15*Np];
			dist[nread] = fq;

			// q = 16
			fq =  mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17)- 0.08333333333*(Fy+Fz);
			nread = neighborList[n+14*Np];
			dist[nread] = fq;


			// q = 17
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18) + 0.08333333333*(Fy-Fz);
			nread = neighborList[n+17*Np];
			dist[nread] = fq;

			// q = 18
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18) - 0.08333333333*(Fy-Fz);
			nread = neighborList[n+16*Np];
			dist[nread] = fq;

		}
	}
}


//__launch_bounds__(512,1)
__global__ void 
dvc_ScaLBL_AAeven_MRT(double *dist, int start, int finish, int Np, double rlx_setA, double rlx_setB, double Fx, double Fy, double Fz) {

	int n;
	double fq;
	// conserved momemnts
	double rho,jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;

		if ( n<finish ){

			// q=0
			fq = dist[n];
			rho = fq;
			m1  = -30.0*fq;
			m2  = 12.0*fq;

			// q=1
			fq = dist[2*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jx = fq;
			m4 = -4.0*fq;
			m9 = 2.0*fq;
			m10 = -4.0*fq;

			// q=2
			fq = dist[1*Np+n];
			rho += fq;
			m1 -= 11.0*(fq);
			m2 -= 4.0*(fq);
			jx -= fq;
			m4 += 4.0*(fq);
			m9 += 2.0*(fq);
			m10 -= 4.0*(fq);

			// q=3
			fq = dist[4*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy = fq;
			m6 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 = fq;
			m12 = -2.0*fq;

			// q = 4
			fq = dist[3*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy -= fq;
			m6 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 += fq;
			m12 -= 2.0*fq;

			// q=5
			fq = dist[6*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz = fq;
			m8 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q = 6
			fq = dist[5*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz -= fq;
			m8 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q=7
			fq = dist[8*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy += fq;
			m6 += fq;
			m9  += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 = fq;
			m16 = fq;
			m17 = -fq;

			// q = 8
			fq = dist[7*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 += fq;
			m16 -= fq;
			m17 += fq;

			// q=9
			fq = dist[10*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 += fq;
			m17 += fq;

			// q = 10
			fq = dist[9*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy += fq;
			m6 += fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 -= fq;
			m17 -= fq;

			// q=11
			fq = dist[12*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 = fq;
			m16 -= fq;
			m18 = fq;

			// q=12
			fq = dist[11*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 += fq;
			m16 += fq;
			m18 -= fq;

			// q=13
			fq = dist[14*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 -= fq;
			m18 -= fq;

			// q=14
			fq = dist[13*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 += fq;
			m18 += fq;

			// q=15
			fq = dist[16*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 = fq;
			m17 += fq;
			m18 -= fq;

			// q=16
			fq = dist[15*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 += fq;
			m17 -= fq;
			m18 += fq;

			// q=17
			fq = dist[18*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 += fq;
			m18 += fq;

			// q=18
			fq = dist[17*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 -= fq;
			m18 -= fq;

			//........................................................................
			//					READ THE DISTRIBUTIONS
			//		(read from opposite array due to previous swap operation)
			//........................................................................

			//..............incorporate external force................................................
			//..............carry out relaxation process...............................................
			m1 = m1 + rlx_setA*((19*(jx*jx+jy*jy+jz*jz)/rho - 11*rho) - m1);
			m2 = m2 + rlx_setA*((3*rho - 5.5*(jx*jx+jy*jy+jz*jz)/rho) - m2);
			m4 = m4 + rlx_setB*((-0.6666666666666666*jx) - m4);
			m6 = m6 + rlx_setB*((-0.6666666666666666*jy) - m6);
			m8 = m8 + rlx_setB*((-0.6666666666666666*jz) - m8);
			m9 = m9 + rlx_setA*(((2*jx*jx-jy*jy-jz*jz)/rho) - m9);
			m10 = m10 + rlx_setA*(-0.5*((2*jx*jx-jy*jy-jz*jz)/rho) - m10);
			m11 = m11 + rlx_setA*(((jy*jy-jz*jz)/rho) - m11);
			m12 = m12 + rlx_setA*(-0.5*((jy*jy-jz*jz)/rho) - m12);
			m13 = m13 + rlx_setA*((jx*jy/rho) - m13);
			m14 = m14 + rlx_setA*((jy*jz/rho) - m14);
			m15 = m15 + rlx_setA*((jx*jz/rho) - m15);
			m16 = m16 + rlx_setB*( - m16);
			m17 = m17 + rlx_setB*( - m17);
			m18 = m18 + rlx_setB*( - m18);
			//.......................................................................................................
			//.................inverse transformation......................................................

			// q=0
			fq = mrt_V1*rho-mrt_V2*m1+mrt_V3*m2;
			dist[n] = fq;

			// q = 1
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10) + 0.16666666*Fx;
			dist[1*Np+n] = fq;

			// q=2
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10) -  0.16666666*Fx;
			dist[2*Np+n] = fq;

			// q = 3
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) + 0.16666666*Fy;
			dist[3*Np+n] = fq;

			// q = 4
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) - 0.16666666*Fy;
			dist[4*Np+n] = fq;

			// q = 5
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) + 0.16666666*Fz;
			dist[5*Np+n] = fq;

			// q = 6
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) - 0.16666666*Fz;
			dist[6*Np+n] = fq;

			// q = 7
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17) + 
					0.08333333333*(Fx+Fy);
			dist[7*Np+n] = fq;


			// q = 8
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11
					+mrt_V12*m12+0.25*m13+0.125*(m17-m16) - 0.08333333333*(Fx+Fy);
			dist[8*Np+n] = fq;

			// q = 9
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17)+
					0.08333333333*(Fx-Fy);
			dist[9*Np+n] = fq;

			// q = 10
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17)-
					0.08333333333*(Fx-Fy);
			dist[10*Np+n] = fq;


			// q = 11
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12+0.25*m15+0.125*(m18-m16) + 0.08333333333*(Fx+Fz);
			dist[11*Np+n] = fq;

			// q = 12
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+
					mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18)-
					0.08333333333*(Fx+Fz);
			dist[12*Np+n] = fq;

			// q = 13
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15-0.125*(m16+m18) + 0.08333333333*(Fx-Fz);
			dist[13*Np+n] = fq;

			// q= 14
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15+0.125*(m16+m18) - 0.08333333333*(Fx-Fz);

			dist[14*Np+n] = fq;

			// q = 15
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18) + 0.08333333333*(Fy+Fz);
			dist[15*Np+n] = fq;

			// q = 16
			fq =  mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17)- 0.08333333333*(Fy+Fz);
			dist[16*Np+n] = fq;


			// q = 17
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18) + 0.08333333333*(Fy-Fz);
			dist[17*Np+n] = fq;

			// q = 18
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18) - 0.08333333333*(Fy-Fz);
			dist[18*Np+n] = fq;
			//........................................................................
		}
	}
}

//__launch_bounds__(512,4)

__global__ void dvc_ScaLBL_AAeven_Compact( double *dist, int Np) {

	int n;
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;

		if ( n<Np ){

			//........................................................................
			//					READ THE DISTRIBUTIONS
			//		(read from opposite array due to previous swap operation)
			//........................................................................
			// even
			f2 = dist[10*Np+n];
			f4 = dist[11*Np+n];
			f6 = dist[12*Np+n];
			f8 = dist[13*Np+n];
			f10 = dist[14*Np+n];
			f12 = dist[15*Np+n];
			f14 = dist[16*Np+n];
			f16 = dist[17*Np+n];
			f18 = dist[18*Np+n];
			f0 = dist[n];
			// odd
			f1 = dist[Np+n];
			f3 = dist[2*Np+n];
			f5 = dist[3*Np+n];
			f7 = dist[4*Np+n];
			f9 = dist[5*Np+n];
			f11 = dist[6*Np+n];
			f13 = dist[7*Np+n];
			f15 = dist[8*Np+n];
			f17 = dist[9*Np+n];

			//........................................................................
			//					WRITE THE DISTRIBUTIONS
			// even
			//disteven[n] = f0;
			dist[Np+n] = f2;
			dist[2*Np+n] = f4;
			dist[3*Np+n] = f6;
			dist[4*Np+n] = f8;
			dist[5*Np+n] = f10;
			dist[6*Np+n] = f12;
			dist[7*Np+n] = f14;
			dist[8*Np+n] = f16;
			dist[9*Np+n] = f18;
			// odd
			dist[10*Np+n] = f1;
			dist[11*Np+n] = f3;
			dist[12*Np+n] = f5;
			dist[13*Np+n] = f7;
			dist[14*Np+n] = f9;
			dist[15*Np+n] = f11;
			dist[16*Np+n] = f13;
			dist[17*Np+n] = f15;
			dist[18*Np+n] = f17;
			//........................................................................
		}
	}
}


__global__  void dvc_ScaLBL_D3Q19_Swap(char *ID, double *disteven, double *distodd, int Nx, int Ny, int Nz)
{
	int i,j,k,n,nn,N;
	// distributions
	char id;
	double f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;

	N = Nx*Ny*Nz;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N){ 
			id = ID[n];
			if (id > 0){
				//.......Back out the 3-D indices for node n..............
				k = n/(Nx*Ny);
				j = (n-Nx*Ny*k)/Nx;
				i = n-Nx*Ny*k-Nx*j;
				//........................................................................
				// Retrieve even distributions from the local node (swap convention)
				//		f0 = disteven[n];  // Does not particupate in streaming
				f1 = distodd[n];
				f3 = distodd[N+n];
				f5 = distodd[2*N+n];
				f7 = distodd[3*N+n];
				f9 = distodd[4*N+n];
				f11 = distodd[5*N+n];
				f13 = distodd[6*N+n];
				f15 = distodd[7*N+n];
				f17 = distodd[8*N+n];
				//........................................................................

				//........................................................................
				// Retrieve odd distributions from neighboring nodes (swap convention)
				//........................................................................
				nn = n+1;							// neighbor index (pull convention)
				if (!(i+1<Nx))	nn -= Nx;			// periodic BC along the x-boundary
				//if (i+1<Nx){
				f2 = disteven[N+nn];					// pull neighbor for distribution 2
				if (f2 > 0.0){
					distodd[n] = f2;
					disteven[N+nn] = f1;
				}
				//}
				//........................................................................
				nn = n+Nx;							// neighbor index (pull convention)
				if (!(j+1<Ny))	nn -= Nx*Ny;		// Perioidic BC along the y-boundary
				//if (j+1<Ny){
				f4 = disteven[2*N+nn];				// pull neighbor for distribution 4
				if (f4 > 0.0){
					distodd[N+n] = f4;
					disteven[2*N+nn] = f3;
					//	}
				}
				//........................................................................
				nn = n+Nx*Ny;						// neighbor index (pull convention)
				if (!(k+1<Nz))	nn -= Nx*Ny*Nz;		// Perioidic BC along the z-boundary
				//if (k+1<Nz){
				f6 = disteven[3*N+nn];				// pull neighbor for distribution 6
				if (f6 > 0.0){
					distodd[2*N+n] = f6;
					disteven[3*N+nn] = f5;
					//	}
				}
				//........................................................................
				nn = n+Nx+1;						// neighbor index (pull convention)
				if (!(i+1<Nx))		nn -= Nx;		// periodic BC along the x-boundary
				if (!(j+1<Ny))		nn -= Nx*Ny;	// Perioidic BC along the y-boundary
				//if ((i+1<Nx) && (j+1<Ny)){
				f8 = disteven[4*N+nn];				// pull neighbor for distribution 8
				if (f8 > 0.0){
					distodd[3*N+n] = f8;
					disteven[4*N+nn] = f7;
					//	}
				}
				//........................................................................
				nn = n-Nx+1;						// neighbor index (pull convention)
				if (!(i+1<Nx))	nn -= Nx;		// periodic BC along the x-boundary
				if (j-1<0)		nn += Nx*Ny;	// Perioidic BC along the y-boundary
				//if (!(i-1<0) && (j+1<Ny)){
				f10 = disteven[5*N+nn];					// pull neighbor for distribution 9
				if (f10 > 0.0){
					distodd[4*N+n] = f10;
					disteven[5*N+nn] = f9;
					//	}
				}
				//........................................................................
				nn = n+Nx*Ny+1;						// neighbor index (pull convention)
				if (!(i+1<Nx))	nn -= Nx;		// periodic BC along the x-boundary
				if (!(k+1<Nz))	nn -= Nx*Ny*Nz;	// Perioidic BC along the z-boundary
				//if ( !(i-1<0) && !(k-1<0)){
				f12 = disteven[6*N+nn];				// pull distribution 11
				if (f12 > 0.0){
					distodd[5*N+n] = f12;
					disteven[6*N+nn] = f11;
					//	}
				}
				//........................................................................
				nn = n-Nx*Ny+1;						// neighbor index (pull convention)
				if (!(i+1<Nx))	nn -= Nx;		// periodic BC along the x-boundary
				if (k-1<0)		nn += Nx*Ny*Nz;	// Perioidic BC along the z-boundary
				//if (!(i-1<0) && (k+1<Nz)){
				f14 = disteven[7*N+nn];				// pull neighbor for distribution 13
				if (f14 > 0.0){
					distodd[6*N+n] = f14;
					disteven[7*N+nn] = f13;
					//	}
				}
				//........................................................................
				nn = n+Nx*Ny+Nx;					// neighbor index (pull convention)
				if (!(j+1<Ny))	nn -= Nx*Ny;		// Perioidic BC along the y-boundary
				if (!(k+1<Nz))	nn -= Nx*Ny*Nz;		// Perioidic BC along the z-boundary
				//if (!(j-1<0) && !(k-1<0)){
				f16 = disteven[8*N+nn];				// pull neighbor for distribution 15
				if (f16 > 0.0){
					distodd[7*N+n] = f16;
					disteven[8*N+nn] = f15;
					//	}
				}
				//........................................................................
				nn = n-Nx*Ny+Nx;					// neighbor index (pull convention)
				if (!(j+1<Ny))	nn -= Nx*Ny;		// Perioidic BC along the y-boundary
				if (k-1<0)		nn += Nx*Ny*Nz;		// Perioidic BC along the z-boundary
				//if (!(j-1<0) && (k+1<Nz)){
				f18 = disteven[9*N+nn];				// pull neighbor for distribution 17
				if (f18 > 0.0){
				distodd[8*N+n] = f18;
					disteven[9*N+nn] = f17;
					//	}
				}
				//........................................................................

			}
		}
	}
}


__global__  void dvc_ScaLBL_D3Q19_Momentum(double *dist, double *vel, int N)
{
	int n;
	// distributions
	double f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double vx,vy,vz;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N){

			f2 = dist[2*N+n];
			f4 = dist[4*N+n];
			f6 = dist[6*N+n];
			f8 = dist[8*N+n];
			f10 = dist[10*N+n];
			f12 = dist[12*N+n];
			f14 = dist[14*N+n];
			f16 = dist[16*N+n];
			f18 = dist[18*N+n];
			//........................................................................
			f1 = dist[N+n];
			f3 = dist[3*N+n];
			f5 = dist[5*N+n];
			f7 = dist[7*N+n];
			f9 = dist[9*N+n];
			f11 = dist[11*N+n];
			f13 = dist[13*N+n];
			f15 = dist[15*N+n];
			f17 = dist[17*N+n];			

			//.................Compute the velocity...................................
			vx = f1-f2+f7-f8+f9-f10+f11-f12+f13-f14;
			vy = f3-f4+f7-f8-f9+f10+f15-f16+f17-f18;
			vz = f5-f6+f11-f12-f13+f14+f15-f16-f17+f18;
			//..................Write the velocity.....................................
			vel[n] = vx;
			vel[N+n] = vy;
			vel[2*N+n] = vz;
			//........................................................................
		}
	}
}

__global__  void dvc_ScaLBL_D3Q19_Pressure(const double *dist, double *Pressure, int N)
{
	int n;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N){				//.......................................................................
			// Registers to store the distributions
			//........................................................................
			//........................................................................
			// Registers to store the distributions
			//........................................................................
			f0 = dist[n];
			f2 = dist[2*N+n];
			f4 = dist[4*N+n];
			f6 = dist[6*N+n];
			f8 = dist[8*N+n];
			f10 = dist[10*N+n];
			f12 = dist[12*N+n];
			f14 = dist[14*N+n];
			f16 = dist[16*N+n];
			f18 = dist[18*N+n];
			//........................................................................
			f1 = dist[N+n];
			f3 = dist[3*N+n];
			f5 = dist[5*N+n];
			f7 = dist[7*N+n];
			f9 = dist[9*N+n];
			f11 = dist[11*N+n];
			f13 = dist[13*N+n];
			f15 = dist[15*N+n];
			f17 = dist[17*N+n];
			//.................Compute the velocity...................................
			Pressure[n] = 0.3333333333333333*(f0+f2+f1+f4+f3+f6+f5+f8+f7+f10+
					f9+f12+f11+f14+f13+f16+f15+f18+f17);
		}
	}
}

__global__  void dvc_ScaLBL_D3Q19_AAeven_Pressure_BC_z(int *list, double *dist, double din, int count, int Np)
{
	int idx, n;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double ux,uy,uz,Cyz,Cxz;
	ux = uy = 0.0;

	idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < count){

		n = list[idx];
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f6 = dist[5*Np+n];
		f7 = dist[8*Np+n];
		f8 = dist[7*Np+n];
		f9 = dist[10*Np+n];
		f10 = dist[9*Np+n];
		f12 = dist[11*Np+n];
		f13 = dist[14*Np+n];
		f16 = dist[15*Np+n];
		f17 = dist[18*Np+n];
		//...................................................
		// Determine the inlet flow velocity
		//ux = (f1-f2+f7-f8+f9-f10+f11-f12+f13-f14);
		//uy = (f3-f4+f7-f8-f9+f10+f15-f16+f17-f18);
		uz = din - (f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f6+f12+f13+f16+f17));

		Cxz = 0.5*(f1+f7+f9-f2-f10-f8) - 0.3333333333333333*ux;
		Cyz = 0.5*(f3+f7+f10-f4-f9-f8) - 0.3333333333333333*uy;

		f5 = f6 + 0.33333333333333338*uz;
		f11 = f12 + 0.16666666666666678*(uz+ux)-Cxz;
		f14 = f13 + 0.16666666666666678*(uz-ux)+Cxz;
		f15 = f16 + 0.16666666666666678*(uy+uz)-Cyz;
		f18 = f17 + 0.16666666666666678*(uz-uy)+Cyz;
		//........Store in "opposite" memory location..........
		dist[6*Np+n] = f5;
		dist[12*Np+n] = f11;
		dist[13*Np+n] = f14;
		dist[16*Np+n] = f15;
		dist[17*Np+n] = f18;
	}
}

__global__  void dvc_ScaLBL_D3Q19_AAeven_Pressure_BC_Z(int *list, double *dist, double dout, int count, int Np)
{
	int idx,n;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double ux,uy,uz,Cyz,Cxz;
	ux = uy = 0.0;

	idx = blockIdx.x*blockDim.x + threadIdx.x;

	// Loop over the boundary - threadblocks delineated by start...finish
	if ( idx < count ){

		n = list[idx];
		//........................................................................
		// Read distributions 
		//........................................................................
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f5 = dist[6*Np+n];
		f7 = dist[8*Np+n];
		f8 = dist[7*Np+n];
		f9 = dist[10*Np+n];
		f10 = dist[9*Np+n];
		f11 = dist[12*Np+n];
		f14 = dist[13*Np+n];
		f15 = dist[16*Np+n];
		f18 = dist[17*Np+n];
		
		// Determine the outlet flow velocity
		//ux = f1-f2+f7-f8+f9-f10+f11-f12+f13-f14;
		//uy = f3-f4+f7-f8-f9+f10+f15-f16+f17-f18;
		uz = -dout + (f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f5+f11+f14+f15+f18));

		Cxz = 0.5*(f1+f7+f9-f2-f10-f8) - 0.3333333333333333*ux;
		Cyz = 0.5*(f3+f7+f10-f4-f9-f8) - 0.3333333333333333*uy;

		f6 = f5 - 0.33333333333333338*uz;
		f12 = f11 - 0.16666666666666678*(uz+ux)+Cxz;
		f13 = f14 - 0.16666666666666678*(uz-ux)-Cxz;
		f16 = f15 - 0.16666666666666678*(uy+uz)+Cyz;
		f17 = f18 - 0.16666666666666678*(uz-uy)-Cyz;

		dist[5*Np+n] = f6;
		dist[11*Np+n] = f12;
		dist[14*Np+n] = f13;
		dist[15*Np+n] = f16;
		dist[18*Np+n] = f17;
		//...................................................
	}
}
__global__  void dvc_ScaLBL_D3Q19_Reflection_BC_z(int *list, double *dist, int count, int Np){
	int idx, n;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		double f5 = 0.111111111111111111111111 - dist[6*Np+n];
		double f11 = 0.05555555555555555555556 - dist[12*Np+n];
		double f14 = 0.05555555555555555555556 - dist[13*Np+n];
		double f15 = 0.05555555555555555555556 - dist[16*Np+n];
		double f18 = 0.05555555555555555555556 - dist[17*Np+n];
		
		dist[6*Np+n] = f5;
		dist[12*Np+n] = f11;
		dist[13*Np+n] = f14;
		dist[16*Np+n] = f15;
		dist[17*Np+n] = f18;
	}
}

__global__  void dvc_ScaLBL_D3Q19_Reflection_BC_Z(int *list, double *dist, int count, int Np){
	int idx, n;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		double f6 = 0.111111111111111111111111 - dist[5*Np+n];
		double f12 = 0.05555555555555555555556 - dist[11*Np+n];
		double f13 = 0.05555555555555555555556 - dist[14*Np+n] ;
		double f16 = 0.05555555555555555555556 - dist[15*Np+n];
		double f17 = 0.05555555555555555555556 - dist[18*Np+n];
		
		dist[5*Np+n] = f6;
		dist[11*Np+n] = f12;
		dist[14*Np+n] = f13;
		dist[15*Np+n] = f16;
		dist[18*Np+n] = f17;
	}
}

__global__  void dvc_ScaLBL_D3Q19_AAodd_Pressure_BC_z(int *d_neighborList, int *list, double *dist, double din, int count, int Np)
{
	int idx, n;
	int nread;
	int nr5,nr11,nr14,nr15,nr18;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double ux,uy,uz,Cyz,Cxz;
	ux = uy = 0.0;

	idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < count){
		
		n = list[idx];
		f0 = dist[n];
				
		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+6*Np];
		f7 = dist[nread];

		nread = d_neighborList[n+8*Np];
		f9 = dist[nread];

		nread = d_neighborList[n+12*Np];
		f13 = dist[nread];

		nread = d_neighborList[n+16*Np];
		f17 = dist[nread];

		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

		nread = d_neighborList[n+5*Np];
		f6 = dist[nread];

		nread = d_neighborList[n+7*Np];
		f8 = dist[nread];

		nread = d_neighborList[n+9*Np];
		f10 = dist[nread];

		nread = d_neighborList[n+11*Np];
		f12 = dist[nread];

		nread = d_neighborList[n+15*Np];
		f16 = dist[nread];

		// Unknown distributions
		nr5 = d_neighborList[n+4*Np];
		nr11 = d_neighborList[n+10*Np];
		nr15 = d_neighborList[n+14*Np];
		nr14 = d_neighborList[n+13*Np];
		nr18 = d_neighborList[n+17*Np];
		
		//...................................................
		//........Determine the inlet flow velocity.........
		//ux = (f1-f2+f7-f8+f9-f10+f11-f12+f13-f14);
		//uy = (f3-f4+f7-f8-f9+f10+f15-f16+f17-f18);
		uz = din - (f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f6+f12+f13+f16+f17));

		Cxz = 0.5*(f1+f7+f9-f2-f10-f8) - 0.3333333333333333*ux;
		Cyz = 0.5*(f3+f7+f10-f4-f9-f8) - 0.3333333333333333*uy;

		f5 = f6 + 0.33333333333333338*uz;
		f11 = f12 + 0.16666666666666678*(uz+ux)-Cxz;
		f14 = f13 + 0.16666666666666678*(uz-ux)+Cxz;
		f15 = f16 + 0.16666666666666678*(uy+uz)-Cyz;
		f18 = f17 + 0.16666666666666678*(uz-uy)+Cyz;
		//........Store in "opposite" memory location..........
		dist[nr5] = f5;
		dist[nr11] = f11;
		dist[nr14] = f14;
		dist[nr15] = f15;
		dist[nr18] = f18;
	}
}

__global__  void dvc_ScaLBL_D3Q19_AAodd_Pressure_BC_Z(int *d_neighborList, int *list, double *dist, double dout, int count, int Np)
{
	int idx,n,nread;
	int nr6,nr12,nr13,nr16,nr17;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double ux,uy,uz,Cyz,Cxz;
	ux = uy = 0.0;

	idx = blockIdx.x*blockDim.x + threadIdx.x;

	// Loop over the boundary - threadblocks delineated by start...finish
	if ( idx < count ){

		n = list[idx];
		//........................................................................
		// Read distributions 
		//........................................................................
		f0 = dist[n];

		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+4*Np];
		f5 = dist[nread];

		nread = d_neighborList[n+6*Np];
		f7 = dist[nread];

		nread = d_neighborList[n+8*Np];
		f9 = dist[nread];

		nread = d_neighborList[n+10*Np];
		f11 = dist[nread];

		nread = d_neighborList[n+14*Np];
		f15 = dist[nread];


		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

		nread = d_neighborList[n+7*Np];
		f8 = dist[nread];

		nread = d_neighborList[n+9*Np];
		f10 = dist[nread];

		nread = d_neighborList[n+13*Np];
		f14 = dist[nread];

		nread = d_neighborList[n+17*Np];
		f18 = dist[nread];
		
		// unknown distributions
		nr6 = d_neighborList[n+5*Np];
		nr12 = d_neighborList[n+11*Np];
		nr16 = d_neighborList[n+15*Np];
		nr17 = d_neighborList[n+16*Np];
		nr13 = d_neighborList[n+12*Np];

		
		//........Determine the outlet flow velocity.........
		//ux = f1-f2+f7-f8+f9-f10+f11-f12+f13-f14;
		//uy = f3-f4+f7-f8-f9+f10+f15-f16+f17-f18;
		uz = -dout + (f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f5+f11+f14+f15+f18));

		Cxz = 0.5*(f1+f7+f9-f2-f10-f8) - 0.3333333333333333*ux;
		Cyz = 0.5*(f3+f7+f10-f4-f9-f8) - 0.3333333333333333*uy;

		f6 = f5 - 0.33333333333333338*uz;
		f12 = f11 - 0.16666666666666678*(uz+ux)+Cxz;
		f13 = f14 - 0.16666666666666678*(uz-ux)-Cxz;
		f16 = f15 - 0.16666666666666678*(uy+uz)+Cyz;
		f17 = f18 - 0.16666666666666678*(uz-uy)-Cyz;

		//........Store in "opposite" memory location..........
		dist[nr6] = f6;
		dist[nr12] = f12;
		dist[nr13] = f13;
		dist[nr16] = f16;
		dist[nr17] = f17;
		//...................................................
	}
}


__global__  void dvc_ScaLBL_D3Q19_AAeven_Flux_BC_z(int *list, double *dist, double flux, double Area, 
		double *dvcsum, int count, int Np)
{
	int idx, n;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double factor = 1.f/(Area);
	double sum = 0.f;

	idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < count){
		
		n = list[idx];
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f6 = dist[5*Np+n];
		f7 = dist[8*Np+n];
		f8 = dist[7*Np+n];
		f9 = dist[10*Np+n];
		f10 = dist[9*Np+n];
		f12 = dist[11*Np+n];
		f13 = dist[14*Np+n];
		f16 = dist[15*Np+n];
		f17 = dist[18*Np+n];
		sum = factor*(f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f6+f12+f13+f16+f17));
	}

	//sum = blockReduceSum(sum);
	//if (threadIdx.x==0)
	//   atomicAdd(dvcsum, sum);
	
    extern __shared__ double temp[];
    thread_group g = this_thread_block();
    double block_sum = reduce_sum(g, temp, sum);

    if (g.thread_rank() == 0) atomicAdd(dvcsum, block_sum);
}


__global__  void dvc_ScaLBL_D3Q19_AAodd_Flux_BC_z(int *d_neighborList, int *list, double *dist, double flux, 
		double Area, double *dvcsum, int count, int Np)
{
	int idx, n;
	int nread;

	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double factor = 1.f/(Area);
	double sum = 0.f;

	idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < count){
		
		n = list[idx];
				
		f0 = dist[n];
		
		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+6*Np];
		f7 = dist[nread];

		nread = d_neighborList[n+8*Np];
		f9 = dist[nread];

		nread = d_neighborList[n+12*Np];
		f13 = dist[nread];

		nread = d_neighborList[n+16*Np];
		f17 = dist[nread];

		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

		nread = d_neighborList[n+5*Np];
		f6 = dist[nread];

		nread = d_neighborList[n+7*Np];
		f8 = dist[nread];

		nread = d_neighborList[n+9*Np];
		f10 = dist[nread];

		nread = d_neighborList[n+11*Np];
		f12 = dist[nread];

		nread = d_neighborList[n+15*Np];
		f16 = dist[nread];

		sum = factor*(f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f6+f12+f13+f16+f17));

	}

	//sum = blockReduceSum(sum);
	//if (threadIdx.x==0)
	//   atomicAdd(dvcsum, sum);
	
    extern __shared__ double temp[];
    thread_group g = this_thread_block();
    double block_sum = reduce_sum(g, temp, sum);

    if (g.thread_rank() == 0) atomicAdd(dvcsum, block_sum);
}


__global__  void dvc_D3Q19_Velocity_BC_z(double *disteven, double *distodd, double uz,
		int Nx, int Ny, int Nz)
{
	int n,N;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double din;

	N = Nx*Ny*Nz;
	n = Nx*Ny +  blockIdx.x*blockDim.x + threadIdx.x;

	if (n < 2*Nx*Ny){
		//........................................................................
		// Read distributions from "opposite" memory convention
		//........................................................................
		//........................................................................
		f1 = distodd[n];
		f3 = distodd[N+n];
		f5 = distodd[2*N+n];
		f7 = distodd[3*N+n];
		f9 = distodd[4*N+n];
		f11 = distodd[5*N+n];
		f13 = distodd[6*N+n];
		f15 = distodd[7*N+n];
		f17 = distodd[8*N+n];
		//........................................................................
		f0 = disteven[n];
		f2 = disteven[N+n];
		f4 = disteven[2*N+n];
		f6 = disteven[3*N+n];
		f8 = disteven[4*N+n];
		f10 = disteven[5*N+n];
		f12 = disteven[6*N+n];
		f14 = disteven[7*N+n];
		f16 = disteven[8*N+n];
		f18 = disteven[9*N+n];
		//...................................................

		// Determine the outlet flow velocity
		//	uz = 1.0 - (f0+f4+f3+f2+f1+f8+f7+f9+f10 +
		//			2*(f5+f15+f18+f11+f14))/din;
		din = (f0+f4+f3+f2+f1+f8+f7+f9+f10+2*(f5+f15+f18+f11+f14))/(1.0-uz);
		// Set the unknown distributions:
		f6 = f5 + 0.3333333333333333*din*uz;
		f16 = f15 + 0.1666666666666667*din*uz;
		f17 = f16 + f4 - f3-f15+f18+f8-f7	+f9-f10;
		f12= (din*uz+f5+ f15+f18+f11+f14-f6-f16-f17-f2+f1-f14+f11-f8+f7+f9-f10)*0.5;
		f13= din*uz+f5+ f15+f18+f11+f14-f6-f16-f17-f12;

		//........Store in "opposite" memory location..........
		disteven[3*N+n] = f6;
		disteven[6*N+n] = f12;
		distodd[6*N+n] = f13;
		disteven[8*N+n] = f16;
		distodd[8*N+n] = f17;
		//...................................................
	}
}

__global__ void dvc_D3Q19_Velocity_BC_Z(double *disteven, double *distodd, double uz,
		int Nx, int Ny, int Nz, int outlet){
	int n,N;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double dout;

	N = Nx*Ny*Nz;
	n = outlet +  blockIdx.x*blockDim.x + threadIdx.x;

	// Loop over the boundary - threadblocks delineated by start...finish
	if ( n<N-Nx*Ny ){
		// Read distributions from "opposite" memory convention
		//........................................................................
		f1 = distodd[n];
		f3 = distodd[N+n];
		f5 = distodd[2*N+n];
		f7 = distodd[3*N+n];
		f9 = distodd[4*N+n];
		f11 = distodd[5*N+n];
		f13 = distodd[6*N+n];
		f15 = distodd[7*N+n];
		f17 = distodd[8*N+n];
		//........................................................................
		f0 = disteven[n];
		f2 = disteven[N+n];
		f4 = disteven[2*N+n];
		f6 = disteven[3*N+n];
		f8 = disteven[4*N+n];
		f10 = disteven[5*N+n];
		f12 = disteven[6*N+n];
		f14 = disteven[7*N+n];
		f16 = disteven[8*N+n];
		f18 = disteven[9*N+n];
		//uz = -1.0 + (f0+f4+f3+f2+f1+f8+f7+f9+f10 + 2*(f6+f16+f17+f12+f13))/dout;
		dout = (f0+f4+f3+f2+f1+f8+f7+f9+f10 + 2*(f6+f16+f17+f12+f13))/(1.0+uz);
		f5 = f6 - 0.33333333333333338*dout* uz;
		f15 = f16 - 0.16666666666666678*dout* uz;
		f18 = f15 - f4 + f3-f16+f17-f8+f7-f9+f10;
		f11 = (-dout*uz+f6+ f16+f17+f12+f13-f5-f15-f18+f2-f1-f13+f12+f8-f7-f9+f10)*0.5;
		f14 = -dout*uz+f6+ f16+f17+f12+f13-f5-f15-f18-f11;
		//........Store in "opposite" memory location..........
		distodd[2*N+n] = f5;
		distodd[5*N+n] = f11;
		disteven[7*N+n] = f14;
		distodd[7*N+n] = f15;
		disteven[9*N+n] = f18;
		//...................................................
	}
}

__global__ void dvc_D3Q19_Flux_BC_z(double *disteven, double *distodd, double flux, double *dvcsum,
		int Nx, int Ny, int Nz){
	// Note that this routine assumes the distributions are stored "opposite"
	// odd distributions in disteven and even distributions in distodd.
	int n,N;
	// distributions
	double f0,f1,f2,f3,f4,f6,f7,f8,f9;
	double f10,f12,f13,f16,f17;

	//double A = 1.f*double(Nx*Ny);
	double factor = 1.f/(double(Nx*Ny)*(1.0-flux));

	double sum = 0.f;

	N = Nx*Ny*Nz;
	n = Nx*Ny +  blockIdx.x*blockDim.x + threadIdx.x;

	if (n < 2*Nx*Ny){

		//........................................................................
		f1 = distodd[n];
		f3 = distodd[N+n];
//		f5 = distodd[2*N+n];
		f7 = distodd[3*N+n];
		f9 = distodd[4*N+n];
//		f11 = distodd[5*N+n];
		f13 = distodd[6*N+n];
//		f15 = distodd[7*N+n];
		f17 = distodd[8*N+n];
		//........................................................................
		f0 = disteven[n];
		f2 = disteven[N+n];
		f4 = disteven[2*N+n];
		f6 = disteven[3*N+n];
		f8 = disteven[4*N+n];
		f10 = disteven[5*N+n];
		f12 = disteven[6*N+n];
//		f14 = disteven[7*N+n];
		f16 = disteven[8*N+n];
//		f18 = disteven[9*N+n];
		//...................................................
		// compute local sum to determine the density value to set pressure
		//sum = (f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f6+f12+f13+f16+f17))/(A*(1.0-flux));
		sum = factor*(f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f6+f12+f13+f16+f17));
		//localsum[n]=sum;
	}

	//sum = warpReduceSum(sum);
	//if (threadIdx.x & (warpSize-1) == 0 ){
	//   atomicAdd(dvcsum,sum);
	//}

	sum = blockReduceSum(sum);
	if (threadIdx.x==0)
	   atomicAdd(dvcsum, sum);
}

__global__ void dvc_D3Q19_Flux_BC_Z(double *disteven, double *distodd, double flux, double *dvcsum,
		int Nx, int Ny, int Nz, int outlet){
	int n,N;
	// distributions
	double f0,f1,f2,f3,f4,f5,f7,f8,f9;
	double f10,f11,f14,f15,f18;

	N = Nx*Ny*Nz;
	n = outlet +  blockIdx.x*blockDim.x + threadIdx.x;

	double factor = 1.f/(double(Nx*Ny)*(1.0+flux));
	double sum = 0.f;

	// Loop over the boundary - threadblocks delineated by start...finish
	if ( n<N-Nx*Ny ){
		//........................................................................
		// Read distributions from "opposite" memory convention
		//........................................................................
		f1 = distodd[n];
		f3 = distodd[N+n];
		f5 = distodd[2*N+n];
		f7 = distodd[3*N+n];
		f9 = distodd[4*N+n];
		f11 = distodd[5*N+n];
//		f13 = distodd[6*N+n];
		f15 = distodd[7*N+n];
//		f17 = distodd[8*N+n];
		//........................................................................
		f0 = disteven[n];
		f2 = disteven[N+n];
		f4 = disteven[2*N+n];
//		f6 = disteven[3*N+n];
		f8 = disteven[4*N+n];
		f10 = disteven[5*N+n];
//		f12 = disteven[6*N+n];
		f14 = disteven[7*N+n];
//		f16 = disteven[8*N+n];
		f18 = disteven[9*N+n];

		// Local sum (based on the consistency condition)
		//sum = (f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f5+f11+f14+f15+f18))/(A*(1.0+flux));
		sum = factor*(f0+f1+f2+f3+f4+f7+f8+f9+f10 + 2*(f5+f11+f14+f15+f18));
		//localsum[n]=sum;
	}

	sum = blockReduceSum(sum);
	if (threadIdx.x==0)
		atomicAdd(dvcsum, sum);

}

__global__ void dvc_ScaLBL_D3Q19_Init_Simple(char *ID, double *f_even, double *f_odd, int Nx, int Ny, int Nz)
{
	int n,N;
	N = Nx*Ny*Nz;
	char id;
	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N ){
			id = ID[n];
			if (id > 0 ){
				f_even[n] = 0 + 0.01*0;
				f_odd[n] = 0+ 0.01*1;		//double(100*n)+1.f;
				f_even[N+n] = 1+ 0.01*2;	//double(100*n)+2.f;
				f_odd[N+n] = 1+ 0.01*3;	//double(100*n)+3.f;
				f_even[2*N+n] = 2+ 0.01*4;	//double(100*n)+4.f;
				f_odd[2*N+n] = 2+ 0.01*5;	//double(100*n)+5.f;
				f_even[3*N+n] = 3+ 0.01*6;	//double(100*n)+6.f;
				f_odd[3*N+n] = 3+ 0.01*7;   //double(100*n)+7.f;
				f_even[4*N+n] = 4+ 0.01*8;   //double(100*n)+8.f;
				f_odd[4*N+n] = 4+ 0.01*9;   //double(100*n)+9.f;
				f_even[5*N+n] = 5+ 0.01*10;  //double(100*n)+10.f;
				f_odd[5*N+n] = 5+ 0.01*11;  //double(100*n)+11.f;
				f_even[6*N+n] = 6+ 0.01*12;  //double(100*n)+12.f;
				f_odd[6*N+n] = 6+ 0.01*13;  //double(100*n)+13.f;
				f_even[7*N+n] = 7+ 0.01*14;  //double(100*n)+14.f;
				f_odd[7*N+n] = 7+ 0.01*15;  //double(100*n)+15.f;
				f_even[8*N+n] = 8+ 0.01*16;  //double(100*n)+16.f;
				f_odd[8*N+n] = 8+ 0.01*17;  //double(100*n)+17.f;
				f_even[9*N+n] = 9+ 0.01*18;  //double(100*n)+18.f;
			}
			else{
				for(int q=0; q<9; q++){
					f_even[q*N+n] = -1.0;
					f_odd[q*N+n] = -1.0;
				}
				f_even[9*N+n] = -1.0;
			}
		}
	}
}


//*************************************************************************

//extern "C" void ScaLBL_D3Q19_MapRecv(int q, int Cqx, int Cqy, int Cqz, int *list,  int start, int count,
//			int *d3q19_recvlist, int Nx, int Ny, int Nz){
//	int GRID = count / 512 + 1;
//	dvc_ScaLBL_D3Q19_Unpack <<<GRID,512 >>>(q, Cqx, Cqy, Cqz, list, start, count, d3q19_recvlist, Nx, Ny, Nz);
//}

extern "C" void ScaLBL_D3Q19_Pack(int q, int *list, int start, int count, double *sendbuf, double *dist, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_Pack <<<GRID,512 >>>(q, list, start, count, sendbuf, dist, N);
}

extern "C" void ScaLBL_D3Q19_Unpack(int q, int *list,  int start, int count, double *recvbuf, double *dist, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_Unpack <<<GRID,512 >>>(q, list, start, count, recvbuf, dist, N);
}
//*************************************************************************

extern "C" void ScaLBL_D3Q19_AA_Init(double *f_even, double *f_odd, int Np){
	dvc_ScaLBL_D3Q19_AA_Init<<<NBLOCKS,NTHREADS >>>(f_even, f_odd, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AA_Init: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_Init(double *dist, int Np){
	dvc_ScaLBL_D3Q19_Init<<<NBLOCKS,NTHREADS >>>(dist, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Init: %s \n",hipGetErrorString(err));
	}
}


extern "C" void ScaLBL_D3Q19_Swap(char *ID, double *disteven, double *distodd, int Nx, int Ny, int Nz){
	dvc_ScaLBL_D3Q19_Swap<<<NBLOCKS,NTHREADS >>>(ID, disteven, distodd, Nx, Ny, Nz);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Swap: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_Swap_Compact(int *neighborList, double *disteven, double *distodd, int Np)
{

	const int Q = 9;
	//	hipStream_t streams[Q];
	// Launch the swap operation as different kernels
	for (int q=0; q<Q; q++){
		dvc_ScaLBL_D3Q19_Swap_Compact<<<NBLOCKS,NTHREADS >>>(neighborList, disteven, distodd, Np, q);
	}
	// cpu should wait for all kernels to finish (to avoid launch of dependent kernels)
	//hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Swap: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAeven_Compact( double *d_dist,  int Np) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(dvc_ScaLBL_AAeven_Compact), hipFuncCachePreferL1);
	dvc_ScaLBL_AAeven_Compact<<<NBLOCKS,NTHREADS>>>(d_dist, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Init: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAodd_Compact( int *d_neighborList, double *d_dist, int Np) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(dvc_ScaLBL_AAodd_Compact), hipFuncCachePreferL1);
	dvc_ScaLBL_AAodd_Compact<<<NBLOCKS,NTHREADS>>>(d_neighborList, d_dist,Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Init: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_Momentum(double *dist, double *vel, int Np){

	dvc_ScaLBL_D3Q19_Momentum<<<NBLOCKS,NTHREADS >>>(dist, vel, Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Velocity: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_Pressure(double *fq, double *Pressure, int Np){
	dvc_ScaLBL_D3Q19_Pressure<<< NBLOCKS,NTHREADS >>>(fq, Pressure, Np);
}

extern "C" void ScaLBL_D3Q19_Velocity_BC_z(double *disteven, double *distodd, double uz,int Nx, int Ny, int Nz){
	int GRID = Nx*Ny / 512 + 1;
	dvc_D3Q19_Velocity_BC_z<<<GRID,512>>>(disteven,distodd, uz, Nx, Ny, Nz);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Velocity_BC_z: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_Velocity_BC_Z(double *disteven, double *distodd, double uz, int Nx, int Ny, int Nz, int outlet){
	int GRID = Nx*Ny / 512 + 1;
	dvc_D3Q19_Velocity_BC_Z<<<GRID,512>>>(disteven, distodd, uz, Nx, Ny, Nz, outlet);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Velocity_BC_Z: %s \n",hipGetErrorString(err));
	}
}

extern "C" double ScaLBL_D3Q19_Flux_BC_z(double *disteven, double *distodd, double flux,int Nx, int Ny, int Nz){

	int GRID = Nx*Ny / 512 + 1;

	// IMPORTANT -- this routine may fail if Nx*Ny > 512*512
	if (Nx*Ny > 512*512){
		printf("WARNING (ScaLBL_D3Q19_Flux_BC_z): CUDA reduction operation may fail if Nx*Ny > 512*512");
	}

	// Allocate memory to store the sums
	double din;
	double sum[1];
 	double *dvcsum;
	int sharedBytes = NTHREADS*sizeof(double);
	hipMalloc((void **)&dvcsum,sizeof(double)*Nx*Ny);
	hipMemset(dvcsum,0,sizeof(double)*Nx*Ny);
	
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Flux_BC_z (memory allocation): %s \n",hipGetErrorString(err));
	}

	// compute the local flux and store the result
	dvc_D3Q19_Flux_BC_z<<<GRID,512,sharedBytes>>>(disteven, distodd, flux, dvcsum, Nx, Ny, Nz);
	
	err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Flux_BC_z (flux calculation, step 1): %s \n",hipGetErrorString(err));
	}

	// Now read the total flux
	hipMemcpy(&sum[0],dvcsum,sizeof(double),hipMemcpyDeviceToHost);
	din=sum[0];
	
	err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Flux_BC_z (flux calculation, step 2): %s \n",hipGetErrorString(err));
	}

	// free the memory needed for reduction
	hipFree(dvcsum);

	return din;
}


extern "C" void ScaLBL_D3Q19_AAeven_Pressure_BC_z(int *list, double *dist, double din, int count, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_AAeven_Pressure_BC_z<<<GRID,512>>>(list, dist, din, count, N);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_Pressure_BC_z (kernel): %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAeven_Pressure_BC_Z(int *list, double *dist, double dout, int count, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_AAeven_Pressure_BC_Z<<<GRID,512>>>(list, dist, dout, count, N);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_Pressure_BC_Z (kernel): %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAodd_Pressure_BC_z(int *neighborList, int *list, double *dist, double din, int count, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_AAodd_Pressure_BC_z<<<GRID,512>>>(neighborList, list, dist, din, count, N);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_Pressure_BC_z (kernel): %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAodd_Pressure_BC_Z(int *neighborList, int *list, double *dist, double dout, int count, int N){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_AAodd_Pressure_BC_Z<<<GRID,512>>>(neighborList, list, dist, dout, count, N);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_Pressure_BC_Z (kernel): %s \n",hipGetErrorString(err));
	}
}


extern "C" double ScaLBL_D3Q19_AAeven_Flux_BC_z(int *list, double *dist, double flux, double area, 
		 int count, int N){

	int GRID = count / 512 + 1;

	// IMPORTANT -- this routine may fail if Nx*Ny > 512*512
	if (count > 512*512){
		printf("WARNING (ScaLBL_D3Q19_Flux_BC_Z): CUDA reduction operation may fail if count > 512*512");
	}

	// Allocate memory to store the sums
	double din;
	double sum[1];
 	double *dvcsum;
	hipMalloc((void **)&dvcsum,sizeof(double)*count);
	hipMemset(dvcsum,0,sizeof(double)*count);
	int sharedBytes = 512*sizeof(double);
	
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_Flux_BC_z (memory allocation): %s \n",hipGetErrorString(err));
	}

	// compute the local flux and store the result
	dvc_ScaLBL_D3Q19_AAeven_Flux_BC_z<<<GRID,512,sharedBytes>>>(list, dist, flux, area, dvcsum, count, N);
	err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_Flux_BC_z (kernel): %s \n",hipGetErrorString(err));
	}

	// Now read the total flux
	hipMemcpy(&sum[0],dvcsum,sizeof(double),hipMemcpyDeviceToHost);
	din=sum[0];
	err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_Flux_BC_z (reduction): %s \n",hipGetErrorString(err));
	}

	// free the memory needed for reduction
	hipFree(dvcsum);

	return din;
}

extern "C" double ScaLBL_D3Q19_AAodd_Flux_BC_z(int *neighborList, int *list, double *dist, double flux, 
		double area, int count, int N){

	int GRID = count / 512 + 1;

	// IMPORTANT -- this routine may fail if Nx*Ny > 512*512
	if (count > 512*512){
		printf("WARNING (ScaLBL_D3Q19_AAodd_Flux_BC_z): CUDA reduction operation may fail if count > 512*512");
	}

	// Allocate memory to store the sums
	double din;
	double sum[1];
 	double *dvcsum;
	hipMalloc((void **)&dvcsum,sizeof(double)*count);
	hipMemset(dvcsum,0,sizeof(double)*count);
	int sharedBytes = 512*sizeof(double);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_Flux_BC_z (memory allocation): %s \n",hipGetErrorString(err));
	}

	// compute the local flux and store the result
	dvc_ScaLBL_D3Q19_AAodd_Flux_BC_z<<<GRID,512,sharedBytes>>>(neighborList, list, dist, flux, area, dvcsum, count, N);
	err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_Flux_BC_z (kernel): %s \n",hipGetErrorString(err));
	}
	// Now read the total flux
	hipMemcpy(&sum[0],dvcsum,sizeof(double),hipMemcpyDeviceToHost);
	din=sum[0];
	err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_Flux_BC_z (reduction): %s \n",hipGetErrorString(err));
	}

	// free the memory needed for reduction
	hipFree(dvcsum);

	return din;
}

extern "C" double ScaLBL_D3Q19_Flux_BC_Z(double *disteven, double *distodd, double flux, int Nx, int Ny, int Nz, int outlet){

	int GRID = Nx*Ny / 512 + 1;

	// IMPORTANT -- this routine may fail if Nx*Ny > 512*512
	if (Nx*Ny > 512*512){
		printf("WARNING (ScaLBL_D3Q19_Flux_BC_Z): CUDA reduction operation may fail if Nx*Ny > 512*512");
	}

	// Allocate memory to store the sums
	double dout;
	double sum[1];
 	double *dvcsum;
	hipMalloc((void **)&dvcsum,sizeof(double)*Nx*Ny);
	hipMemset(dvcsum,0,sizeof(double)*Nx*Ny);

	// compute the local flux and store the result
	dvc_D3Q19_Flux_BC_Z<<<GRID,512>>>(disteven, distodd, flux, dvcsum, Nx, Ny, Nz, outlet);

	// Now read the total flux
	hipMemcpy(&sum[0],dvcsum,sizeof(double),hipMemcpyDeviceToHost);

	// free the memory needed for reduction

	dout = sum[0];

	hipFree(dvcsum);

	return dout;

}

extern "C" double deviceReduce(double *in, double* out, int N) {
	int threads = 512;
	int blocks = min((N + threads - 1) / threads, 1024);

	double sum = 0.f;
	deviceReduceKernel<<<blocks, threads>>>(in, out, N);
	deviceReduceKernel<<<1, 1024>>>(out, out, blocks);
	return sum;
}

extern "C" void ScaLBL_D3Q19_Reflection_BC_z(int *list, double *dist, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_Reflection_BC_z<<<GRID,512>>>(list, dist, count, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Reflection_BC_z (kernel): %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_Reflection_BC_Z(int *list, double *dist, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q19_Reflection_BC_Z<<<GRID,512>>>(list, dist, count, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_Reflection_BC_Z (kernel): %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAeven_MRT(double *dist, int start, int finish, int Np, double rlx_setA, double rlx_setB, double Fx,
       double Fy, double Fz){
       
       dvc_ScaLBL_AAeven_MRT<<<NBLOCKS,NTHREADS >>>(dist,start,finish,Np,rlx_setA,rlx_setB,Fx,Fy,Fz);

       hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_MRT: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAodd_MRT(int *neighborlist, double *dist, int start, int finish, int Np, double rlx_setA, double rlx_setB, double Fx,
       double Fy, double Fz){
       
       dvc_ScaLBL_AAodd_MRT<<<NBLOCKS,NTHREADS >>>(neighborlist,dist,start,finish,Np,rlx_setA,rlx_setB,Fx,Fy,Fz);

       hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_MRT: %s \n",hipGetErrorString(err));
	}
}

