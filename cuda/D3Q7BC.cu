/*
  Copyright 2013--2018 James E. McClure, Virginia Polytechnic & State University
  Copyright Equnior ASA

  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#define NBLOCKS 1024
#define NTHREADS 256


#define CHECK_ERROR(KERNEL)                                         \
    do {                                                            \
        auto err = hipGetLastError();                              \
        if ( hipSuccess != err ){                                  \
            auto errString = hipGetErrorString(err);               \
            printf("error in %s (kernel): %s \n",KERNEL,errString); \
        }                                                           \
    } while(0)


__global__ void dvc_ScaLBL_Solid_Dirichlet_D3Q7(double *dist, double *BoundaryValue, int *BounceBackDist_list, int *BounceBackSolid_list, int count)
{

    int idx;
    int iq,ib;
    double value_b,value_q;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		iq = BounceBackDist_list[idx];
        ib = BounceBackSolid_list[idx];
		value_b = BoundaryValue[ib];//get boundary value from a solid site
        value_q = dist[iq];
		dist[iq] = -1.0*value_q + value_b*0.25;//NOTE 0.25 is the speed of sound for D3Q7 lattice
	}
}

__global__ void dvc_ScaLBL_Solid_Neumann_D3Q7(double *dist, double *BoundaryValue, int *BounceBackDist_list, int *BounceBackSolid_list, int count)
{

    int idx;
    int iq,ib;
    double value_b,value_q;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		iq = BounceBackDist_list[idx];
        ib = BounceBackSolid_list[idx];
		value_b = BoundaryValue[ib];//get boundary value from a solid site
        value_q = dist[iq];
		dist[iq] = value_q + value_b;
	}
}

__global__ void dvc_ScaLBL_Solid_DirichletAndNeumann_D3Q7(double *dist, double *BoundaryValue,int *BoundaryLabel, int *BounceBackDist_list, int *BounceBackSolid_list, int count)
{

    int idx;
    int iq,ib;
    double value_b,value_b_label,value_q;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		iq = BounceBackDist_list[idx];
        ib = BounceBackSolid_list[idx];
		value_b = BoundaryValue[ib];//get boundary value from a solid site
		value_b_label = BoundaryLabel[ib];//get boundary label (i.e. type of BC) from a solid site
        value_q = dist[iq];
        if (value_b_label==1){//Dirichlet BC
		    dist[iq] = -1.0*value_q + value_b*0.25;//NOTE 0.25 is the speed of sound for D3Q7 lattice
        }
        if (value_b_label==2){//Neumann BC
		    dist[iq] = value_q + value_b;
        }
	}
}

__global__ void dvc_ScaLBL_Solid_SlippingVelocityBC_D3Q19(double *dist, double *zeta_potential, double *ElectricField, double *SolidGrad,
                                                          double epsilon_LB, double tau, double rho0,double den_scale, double h, double time_conv,
                                                          int *BounceBackDist_list, int *BounceBackSolid_list, int *FluidBoundary_list,
                                                          double *lattice_weight, float *lattice_cx, float *lattice_cy, float *lattice_cz,
                                                          int count, int Np)
{
    int idx;
    int iq,ib,ifluidBC;
    double value_b,value_q;
    double Ex,Ey,Ez;
    double Etx,Ety,Etz;//tangential part of electric field
    double E_mag_normal;
    double nsx,nsy,nsz;//unit normal solid gradient
    double ubx,uby,ubz;//slipping velocity at fluid boundary nodes
    float cx,cy,cz;//lattice velocity (D3Q19)
    double LB_weight;//lattice weighting coefficient (D3Q19)
    double cs2_inv = 3.0;//inverse of cs^2 for D3Q19
    double nu_LB = (tau-0.5)/cs2_inv;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		iq       = BounceBackDist_list[idx];
        ib       = BounceBackSolid_list[idx];
        ifluidBC = FluidBoundary_list[idx];
		value_b = zeta_potential[ib];//get zeta potential from a solid site
        value_q = dist[iq];

        //Load electric field and compute its tangential componet
        Ex = ElectricField[ifluidBC+0*Np]; 
        Ey = ElectricField[ifluidBC+1*Np];
        Ez = ElectricField[ifluidBC+2*Np];
        nsx = SolidGrad[ifluidBC+0*Np]; 
        nsy = SolidGrad[ifluidBC+1*Np];
        nsz = SolidGrad[ifluidBC+2*Np];
        E_mag_normal = Ex*nsx+Ey*nsy+Ez*nsz;//magnitude of electric field in the direction normal to solid nodes
        //compute tangential electric field
        Etx = Ex - E_mag_normal*nsx;
        Ety = Ey - E_mag_normal*nsy;
        Etz = Ez - E_mag_normal*nsz;
        ubx = -epsilon_LB*value_b*Etx/(nu_LB*rho0)*time_conv*time_conv/(h*h*1.0e-12)/den_scale;                                                                                                        
        uby = -epsilon_LB*value_b*Ety/(nu_LB*rho0)*time_conv*time_conv/(h*h*1.0e-12)/den_scale;                                                                                                        
        ubz = -epsilon_LB*value_b*Etz/(nu_LB*rho0)*time_conv*time_conv/(h*h*1.0e-12)/den_scale;                                                                                                        

        //compute bounce-back distribution
        LB_weight = lattice_weight[idx];
        cx = lattice_cx[idx];
        cy = lattice_cy[idx];
        cz = lattice_cz[idx];
		dist[iq] = value_q - 2.0*LB_weight*rho0*cs2_inv*(cx*ubx+cy*uby+cz*ubz);
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAeven_Poisson_Potential_BC_z(int *list, double *dist, double Vin, int count, int Np)
{
    int idx,n;
	double f0,f1,f2,f3,f4,f5,f6;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f6 = dist[5*Np+n];
		//...................................................
		f5 = Vin - (f0+f1+f2+f3+f4+f6);
		dist[6*Np+n] = f5;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAeven_Poisson_Potential_BC_Z(int *list, double *dist, double Vout, int count, int Np)
{
    int idx,n;
	double f0,f1,f2,f3,f4,f5,f6;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f5 = dist[6*Np+n];
		//...................................................
		f6 = Vout - (f0+f1+f2+f3+f4+f5);
		dist[5*Np+n] = f6;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAodd_Poisson_Potential_BC_z(int *d_neighborList, int *list, double *dist, double Vin, int count, int Np)
{
	int idx, n;
    int nread,nr5;
	double f0,f1,f2,f3,f4,f5,f6;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];

		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

		nread = d_neighborList[n+5*Np];
		f6 = dist[nread];

		// Unknown distributions
		nr5 = d_neighborList[n+4*Np];
		f5 = Vin - (f0+f1+f2+f3+f4+f6);
		dist[nr5] = f5;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAodd_Poisson_Potential_BC_Z(int *d_neighborList, int *list, double *dist, double Vout, int count, int Np)
{
	int idx, n;
    int nread,nr6;
	double f0,f1,f2,f3,f4,f5,f6;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];

		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+4*Np];
		f5 = dist[nread];

		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

		// unknown distributions
		nr6 = d_neighborList[n+5*Np];
		f6 = Vout - (f0+f1+f2+f3+f4+f5);
		dist[nr6] = f6;
	}
}

__global__ void dvc_ScaLBL_Poisson_D3Q7_BC_z(int *list, int *Map, double *Psi, double Vin, int count)
{
	int idx,n,nm;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		nm = Map[n];
		Psi[nm] = Vin;
	}
}


__global__ void dvc_ScaLBL_Poisson_D3Q7_BC_Z(int *list, int *Map, double *Psi, double Vout, int count)
{
	int idx,n,nm;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		nm = Map[n];
		Psi[nm] = Vout;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAeven_Ion_Concentration_BC_z(int *list, double *dist, double Cin, int count, int Np)
{
    int idx,n;
	double f0,f1,f2,f3,f4,f5,f6;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f6 = dist[5*Np+n];
		//...................................................
		f5 = Cin - (f0+f1+f2+f3+f4+f6);
		dist[6*Np+n] = f5;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAeven_Ion_Concentration_BC_Z(int *list, double *dist, double Cout, int count, int Np)
{
    int idx,n;
	double f0,f1,f2,f3,f4,f5,f6;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f5 = dist[6*Np+n];
		//...................................................
		f6 = Cout - (f0+f1+f2+f3+f4+f5);
		dist[5*Np+n] = f6;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAodd_Ion_Concentration_BC_z(int *d_neighborList, int *list, double *dist, double Cin, int count, int Np)
{
	int idx, n;
    int nread,nr5;
	double f0,f1,f2,f3,f4,f5,f6;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];

		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

		nread = d_neighborList[n+5*Np];
		f6 = dist[nread];

		// Unknown distributions
		nr5 = d_neighborList[n+4*Np];
		f5 = Cin - (f0+f1+f2+f3+f4+f6);
		dist[nr5] = f5;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAodd_Ion_Concentration_BC_Z(int *d_neighborList, int *list, double *dist, double Cout, int count, int Np)
{
	int idx, n;
    int nread,nr6;
	double f0,f1,f2,f3,f4,f5,f6;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];

		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+4*Np];
		f5 = dist[nread];

		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

		// unknown distributions
		nr6 = d_neighborList[n+5*Np];
		f6 = Cout - (f0+f1+f2+f3+f4+f5);
		dist[nr6] = f6;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAeven_Ion_Flux_Diff_BC_z(int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np)
{
    //NOTE: FluxIn is the inward flux
    int idx,n;
	double f0,f1,f2,f3,f4,f5,f6;
    double fsum_partial;
    double uz;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f6 = dist[5*Np+n];
        fsum_partial = f0+f1+f2+f3+f4+f6;
        uz = VelocityZ[n];
		//...................................................
        f5 =(FluxIn+(1.0-0.5/tau)*(f6+uz*fsum_partial))/(1.0-0.5/tau)/(1.0-uz); 
		dist[6*Np+n] = f5;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAeven_Ion_Flux_Diff_BC_Z(int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np)
{
    //NOTE: FluxIn is the inward flux
    int idx,n;
	double f0,f1,f2,f3,f4,f5,f6;
    double fsum_partial;
    double uz;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f5 = dist[6*Np+n];
        fsum_partial = f0+f1+f2+f3+f4+f5;
        uz = VelocityZ[n];
		//...................................................
        f6 =(FluxIn+(1.0-0.5/tau)*(f5-uz*fsum_partial))/(1.0-0.5/tau)/(1.0+uz); 
		dist[5*Np+n] = f6;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAodd_Ion_Flux_Diff_BC_z(int *d_neighborList, int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np)
{
    //NOTE: FluxIn is the inward flux
	int idx, n;
    int nread,nr5;
	double f0,f1,f2,f3,f4,f5,f6;
    double fsum_partial;
    double uz;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];

		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

		nread = d_neighborList[n+5*Np];
		f6 = dist[nread];

        fsum_partial = f0+f1+f2+f3+f4+f6;
        uz = VelocityZ[n];
		//...................................................
        f5 =(FluxIn+(1.0-0.5/tau)*(f6+uz*fsum_partial))/(1.0-0.5/tau)/(1.0-uz); 

		// Unknown distributions
		nr5 = d_neighborList[n+4*Np];
		dist[nr5] = f5;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAodd_Ion_Flux_Diff_BC_Z(int *d_neighborList, int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np)
{
    //NOTE: FluxIn is the inward flux
	int idx, n;
    int nread,nr6;
	double f0,f1,f2,f3,f4,f5,f6;
    double fsum_partial;
    double uz;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];

		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+4*Np];
		f5 = dist[nread];

		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

        fsum_partial = f0+f1+f2+f3+f4+f5;
        uz = VelocityZ[n];
		//...................................................
        f6 =(FluxIn+(1.0-0.5/tau)*(f5-uz*fsum_partial))/(1.0-0.5/tau)/(1.0+uz); 

		// unknown distributions
		nr6 = d_neighborList[n+5*Np];
		dist[nr6] = f6;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvc_BC_z(int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np)
{
    //NOTE: FluxIn is the inward flux
    int idx,n;
	double f0,f1,f2,f3,f4,f5,f6;
    double fsum_partial;
    double uz;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f6 = dist[5*Np+n];
        fsum_partial = f0+f1+f2+f3+f4+f6;
        uz = VelocityZ[n];
		//...................................................
        f5 =(FluxIn+(1.0-0.5/tau)*f6-0.5*uz*fsum_partial/tau)/(1.0-0.5/tau+0.5*uz/tau); 
		dist[6*Np+n] = f5;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvc_BC_Z(int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np)
{
    //NOTE: FluxIn is the inward flux
    int idx,n;
	double f0,f1,f2,f3,f4,f5,f6;
    double fsum_partial;
    double uz;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f5 = dist[6*Np+n];
        fsum_partial = f0+f1+f2+f3+f4+f5;
        uz = VelocityZ[n];
		//...................................................
        f6 =(FluxIn+(1.0-0.5/tau)*f5+0.5*uz*fsum_partial/tau)/(1.0-0.5/tau-0.5*uz/tau); 
		dist[5*Np+n] = f6;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvc_BC_z(int *d_neighborList, int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np)
{
    //NOTE: FluxIn is the inward flux
	int idx, n;
    int nread,nr5;
	double f0,f1,f2,f3,f4,f5,f6;
    double fsum_partial;
    double uz;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];

		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

		nread = d_neighborList[n+5*Np];
		f6 = dist[nread];

        fsum_partial = f0+f1+f2+f3+f4+f6;
        uz = VelocityZ[n];
		//...................................................
        f5 =(FluxIn+(1.0-0.5/tau)*f6-0.5*uz*fsum_partial/tau)/(1.0-0.5/tau+0.5*uz/tau); 

		// Unknown distributions
		nr5 = d_neighborList[n+4*Np];
		dist[nr5] = f5;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvc_BC_Z(int *d_neighborList, int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np)
{
    //NOTE: FluxIn is the inward flux
	int idx, n;
    int nread,nr6;
	double f0,f1,f2,f3,f4,f5,f6;
    double fsum_partial;
    double uz;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];

		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+4*Np];
		f5 = dist[nread];

		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

        fsum_partial = f0+f1+f2+f3+f4+f5;
        uz = VelocityZ[n];
		//...................................................
        f6 =(FluxIn+(1.0-0.5/tau)*f5+0.5*uz*fsum_partial/tau)/(1.0-0.5/tau-0.5*uz/tau); 

		// unknown distributions
		nr6 = d_neighborList[n+5*Np];
		dist[nr6] = f6;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvcElec_BC_z(int *list, double *dist, double FluxIn, double tau, double *VelocityZ, double *ElectricField_Z,
                                                                  double Di, double zi, double Vt, int count, int Np)
{
    //NOTE: FluxIn is the inward flux
    int idx,n;
	double f0,f1,f2,f3,f4,f5,f6;
    double fsum_partial;
    double uz;
    double uEPz;//electrochemical induced velocity
    double Ez;//electrical field
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f6 = dist[5*Np+n];
        fsum_partial = f0+f1+f2+f3+f4+f6;
        uz = VelocityZ[n];
        Ez = ElectricField_Z[n];
        uEPz=zi*Di/Vt*Ez;
		//...................................................
        f5 =(FluxIn+(1.0-0.5/tau)*f6-(0.5*uz/tau+uEPz)*fsum_partial)/(1.0-0.5/tau+0.5*uz/tau+uEPz); 
		dist[6*Np+n] = f5;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvcElec_BC_Z(int *list, double *dist, double FluxIn, double tau, double *VelocityZ, double *ElectricField_Z, 
                                                                  double Di, double zi, double Vt, int count, int Np)
{
    //NOTE: FluxIn is the inward flux
    int idx,n;
	double f0,f1,f2,f3,f4,f5,f6;
    double fsum_partial;
    double uz;
    double uEPz;//electrochemical induced velocity
    double Ez;//electrical field
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f5 = dist[6*Np+n];
        fsum_partial = f0+f1+f2+f3+f4+f5;
        uz = VelocityZ[n];
        Ez = ElectricField_Z[n];
        uEPz=zi*Di/Vt*Ez;
		//...................................................
        f6 =(FluxIn+(1.0-0.5/tau)*f5+(0.5*uz/tau+uEPz)*fsum_partial)/(1.0-0.5/tau-0.5*uz/tau-uEPz); 
		dist[5*Np+n] = f6;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvcElec_BC_z(int *d_neighborList, int *list, double *dist, double FluxIn, double tau, double *VelocityZ, double *ElectricField_Z,
                                                                 double Di, double zi, double Vt, int count, int Np)
{
    //NOTE: FluxIn is the inward flux
	int idx, n;
    int nread,nr5;
	double f0,f1,f2,f3,f4,f5,f6;
    double fsum_partial;
    double uz;
    double uEPz;//electrochemical induced velocity
    double Ez;//electrical field
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];

		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

		nread = d_neighborList[n+5*Np];
		f6 = dist[nread];

        fsum_partial = f0+f1+f2+f3+f4+f6;
        uz = VelocityZ[n];
        Ez = ElectricField_Z[n];
        uEPz=zi*Di/Vt*Ez;
		//...................................................
        f5 =(FluxIn+(1.0-0.5/tau)*f6-(0.5*uz/tau+uEPz)*fsum_partial)/(1.0-0.5/tau+0.5*uz/tau+uEPz); 

		// Unknown distributions
		nr5 = d_neighborList[n+4*Np];
		dist[nr5] = f5;
	}
}

__global__ void dvc_ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvcElec_BC_Z(int *d_neighborList, int *list, double *dist, double FluxIn, double tau, double *VelocityZ, double *ElectricField_Z,
                                                                 double Di, double zi, double Vt, int count, int Np)
{
    //NOTE: FluxIn is the inward flux
	int idx, n;
    int nread,nr6;
	double f0,f1,f2,f3,f4,f5,f6;
    double fsum_partial;
    double uz;
    double uEPz;//electrochemical induced velocity
    double Ez;//electrical field
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		f0 = dist[n];

		nread = d_neighborList[n];
		f1 = dist[nread];

		nread = d_neighborList[n+2*Np];
		f3 = dist[nread];

		nread = d_neighborList[n+4*Np];
		f5 = dist[nread];

		nread = d_neighborList[n+Np];
		f2 = dist[nread];

		nread = d_neighborList[n+3*Np];
		f4 = dist[nread];

        fsum_partial = f0+f1+f2+f3+f4+f5;
        uz = VelocityZ[n];
        Ez = ElectricField_Z[n];
        uEPz=zi*Di/Vt*Ez;
		//...................................................
        f6 =(FluxIn+(1.0-0.5/tau)*f5+(0.5*uz/tau+uEPz)*fsum_partial)/(1.0-0.5/tau-0.5*uz/tau-uEPz); 

		// unknown distributions
		nr6 = d_neighborList[n+5*Np];
		dist[nr6] = f6;
	}
}
//*************************************************************************

extern "C" void ScaLBL_Solid_Dirichlet_D3Q7(double *dist, double *BoundaryValue, int *BounceBackDist_list, int *BounceBackSolid_list, int count){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_Solid_Dirichlet_D3Q7<<<GRID,512>>>(dist, BoundaryValue, BounceBackDist_list, BounceBackSolid_list, count);
    CHECK_ERROR("ScaLBL_Solid_Dirichlet_D3Q7");
}

extern "C" void ScaLBL_Solid_Neumann_D3Q7(double *dist, double *BoundaryValue, int *BounceBackDist_list, int *BounceBackSolid_list, int count){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_Solid_Neumann_D3Q7<<<GRID,512>>>(dist, BoundaryValue, BounceBackDist_list, BounceBackSolid_list, count);
    CHECK_ERROR("ScaLBL_Solid_Neumann_D3Q7");
}

extern "C" void ScaLBL_Solid_DirichletAndNeumann_D3Q7(double *dist, double *BoundaryValue,int *BoundaryLabel, int *BounceBackDist_list, int *BounceBackSolid_list, int count){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_Solid_DirichletAndNeumann_D3Q7<<<GRID,512>>>(dist, BoundaryValue, BoundaryLabel, BounceBackDist_list, BounceBackSolid_list, count);
    CHECK_ERROR("ScaLBL_Solid_DirichletAndNeumann_D3Q7");
}

extern "C" void ScaLBL_Solid_SlippingVelocityBC_D3Q19(double *dist, double *zeta_potential, double *ElectricField, double *SolidGrad,
                                                      double epsilon_LB, double tau, double rho0,double den_scale, double h, double time_conv,
                                                      int *BounceBackDist_list, int *BounceBackSolid_list, int *FluidBoundary_list,
                                                      double *lattice_weight, float *lattice_cx, float *lattice_cy, float *lattice_cz,
                                                      int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_Solid_SlippingVelocityBC_D3Q19<<<GRID,512>>>(dist, zeta_potential, ElectricField, SolidGrad,
                                                            epsilon_LB, tau, rho0, den_scale, h, time_conv,
                                                            BounceBackDist_list, BounceBackSolid_list, FluidBoundary_list,
                                                            lattice_weight, lattice_cx, lattice_cy, lattice_cz,
                                                            count, Np);
    CHECK_ERROR("ScaLBL_Solid_SlippingVelocityBC_D3Q19");
}

extern "C" void ScaLBL_D3Q7_AAeven_Poisson_Potential_BC_z(int *list, double *dist, double Vin, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAeven_Poisson_Potential_BC_z<<<GRID,512>>>(list, dist, Vin, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAeven_Poisson_Potential_BC_z");
}

extern "C" void ScaLBL_D3Q7_AAeven_Poisson_Potential_BC_Z(int *list, double *dist, double Vout, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAeven_Poisson_Potential_BC_Z<<<GRID,512>>>(list, dist, Vout, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAeven_Poisson_Potential_BC_Z");
}

extern "C" void ScaLBL_D3Q7_AAodd_Poisson_Potential_BC_z(int *d_neighborList, int *list, double *dist, double Vin, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAodd_Poisson_Potential_BC_z<<<GRID,512>>>(d_neighborList, list, dist, Vin, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAodd_Poisson_Potential_BC_z");
}

extern "C" void ScaLBL_D3Q7_AAodd_Poisson_Potential_BC_Z(int *d_neighborList, int *list, double *dist, double Vout, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAodd_Poisson_Potential_BC_Z<<<GRID,512>>>(d_neighborList, list, dist, Vout, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAodd_Poisson_Potential_BC_Z");
}

extern "C" void ScaLBL_Poisson_D3Q7_BC_z(int *list, int *Map, double *Psi, double Vin, int count){
	int GRID = count / 512 + 1;
    dvc_ScaLBL_Poisson_D3Q7_BC_z<<<GRID,512>>>(list, Map, Psi, Vin, count);
    CHECK_ERROR("ScaLBL_Poisson_D3Q7_BC_z");
}

extern "C" void ScaLBL_Poisson_D3Q7_BC_Z(int *list, int *Map, double *Psi, double Vout, int count){
	int GRID = count / 512 + 1;
    dvc_ScaLBL_Poisson_D3Q7_BC_Z<<<GRID,512>>>(list, Map, Psi, Vout, count);
    CHECK_ERROR("ScaLBL_Poisson_D3Q7_BC_Z");
}

extern "C" void ScaLBL_D3Q7_AAeven_Ion_Concentration_BC_z(int *list, double *dist, double Cin, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAeven_Ion_Concentration_BC_z<<<GRID,512>>>(list, dist, Cin, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAeven_Ion_Concentration_BC_z");
}

extern "C" void ScaLBL_D3Q7_AAeven_Ion_Concentration_BC_Z(int *list, double *dist, double Cout, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAeven_Ion_Concentration_BC_Z<<<GRID,512>>>(list, dist, Cout, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAeven_Ion_Concentration_BC_Z");
}

extern "C" void ScaLBL_D3Q7_AAodd_Ion_Concentration_BC_z(int *d_neighborList, int *list, double *dist, double Cin, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAodd_Ion_Concentration_BC_z<<<GRID,512>>>(d_neighborList, list, dist, Cin, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAodd_Ion_Concentration_BC_z");
}

extern "C" void ScaLBL_D3Q7_AAodd_Ion_Concentration_BC_Z(int *d_neighborList, int *list, double *dist, double Cout, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAodd_Ion_Concentration_BC_Z<<<GRID,512>>>(d_neighborList, list, dist, Cout, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAodd_Ion_Concentration_BC_Z");
}
//------------Diff-----------------
extern "C" void ScaLBL_D3Q7_AAeven_Ion_Flux_Diff_BC_z(int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAeven_Ion_Flux_Diff_BC_z<<<GRID,512>>>(list, dist, FluxIn, tau, VelocityZ, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAeven_Ion_Flux_Diff_BC_z");
}

extern "C" void ScaLBL_D3Q7_AAeven_Ion_Flux_Diff_BC_Z(int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAeven_Ion_Flux_Diff_BC_Z<<<GRID,512>>>(list, dist, FluxIn, tau, VelocityZ, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAeven_Ion_Flux_Diff_BC_Z");
}

extern "C" void ScaLBL_D3Q7_AAodd_Ion_Flux_Diff_BC_z(int *d_neighborList, int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAodd_Ion_Flux_Diff_BC_z<<<GRID,512>>>(d_neighborList, list, dist, FluxIn, tau, VelocityZ, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAodd_Ion_Flux_Diff_BC_z");
}

extern "C" void ScaLBL_D3Q7_AAodd_Ion_Flux_Diff_BC_Z(int *d_neighborList, int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAodd_Ion_Flux_Diff_BC_Z<<<GRID,512>>>(d_neighborList, list, dist, FluxIn, tau, VelocityZ, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAodd_Ion_Flux_Diff_BC_Z");
}
//----------DiffAdvc-------------
extern "C" void ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvc_BC_z(int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvc_BC_z<<<GRID,512>>>(list, dist, FluxIn, tau, VelocityZ, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvc_BC_z");
}

extern "C" void ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvc_BC_Z(int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvc_BC_Z<<<GRID,512>>>(list, dist, FluxIn, tau, VelocityZ, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvc_BC_Z");
}

extern "C" void ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvc_BC_z(int *d_neighborList, int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvc_BC_z<<<GRID,512>>>(d_neighborList, list, dist, FluxIn, tau, VelocityZ, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvc_BC_z");
}

extern "C" void ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvc_BC_Z(int *d_neighborList, int *list, double *dist, double FluxIn, double tau, double *VelocityZ, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvc_BC_Z<<<GRID,512>>>(d_neighborList, list, dist, FluxIn, tau, VelocityZ, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvc_BC_Z");
}
//----------DiffAdvcElec-------------
extern "C" void ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvcElec_BC_z(int *list, double *dist, double FluxIn, double tau, double *VelocityZ, double *ElectricField_Z,
                                                              double Di, double zi, double Vt, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvcElec_BC_z<<<GRID,512>>>(list, dist, FluxIn, tau, VelocityZ, ElectricField_Z, Di, zi, Vt, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvcElec_BC_z");
}

extern "C" void ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvcElec_BC_Z(int *list, double *dist, double FluxIn, double tau, double *VelocityZ, double *ElectricField_Z,
                                                              double Di, double zi, double Vt, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvcElec_BC_Z<<<GRID,512>>>(list, dist, FluxIn, tau, VelocityZ, ElectricField_Z, Di, zi, Vt, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAeven_Ion_Flux_DiffAdvcElec_BC_Z");
}

extern "C" void ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvcElec_BC_z(int *d_neighborList, int *list, double *dist, double FluxIn, double tau, double *VelocityZ, double *ElectricField_Z,
                                                             double Di, double zi, double Vt, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvcElec_BC_z<<<GRID,512>>>(d_neighborList, list, dist, FluxIn, tau, VelocityZ, ElectricField_Z, Di, zi, Vt, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvcElec_BC_z");
}

extern "C" void ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvcElec_BC_Z(int *d_neighborList, int *list, double *dist, double FluxIn, double tau, double *VelocityZ, double *ElectricField_Z,
                                                             double Di, double zi, double Vt, int count, int Np){
	int GRID = count / 512 + 1;
	dvc_ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvcElec_BC_Z<<<GRID,512>>>(d_neighborList, list, dist, FluxIn, tau, VelocityZ, ElectricField_Z, Di, zi, Vt, count, Np);
    CHECK_ERROR("ScaLBL_D3Q7_AAodd_Ion_Flux_DiffAdvcElec_BC_Z");
}
//-------------------------------
