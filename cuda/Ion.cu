/*
  Copyright 2013--2018 James E. McClure, Virginia Polytechnic & State University
  Copyright Equnior ASA

  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
//#include <cuda_profiler_api.h>

#define NBLOCKS 1024
#define NTHREADS 512


/***** pH equilibrium ******/
__global__  void dvc_ScaLBL_D3Q7_AAodd_pH_ionization(int *neighborList, double *dist,
                                      double *Den, double *ElectricField, double *Velocity,
                                      double Di, double Vt,
                                      int pH_ion, int start, int finish, int Np) {
    int n;
    double Ex, Ey, Ez;       //electrical field
    double ux, uy, uz;
    double uEPx, uEPy, uEPz; //electrochemical induced velocity
    double Ca, Cb;
    double A0, A1, A2, A3, A4, A5, A6;
    double B0, B1, B2, B3, B4, B5, B6;
    double f0, f1, f2, f3, f4, f5, f6;
    int nr1, nr2, nr3, nr4, nr5, nr6;
    double rhoe, tmp;


    int S = Np/NBLOCKS/NTHREADS + 1;
    for (int s=0; s<S; s++){
    	//........Get 1-D index for this thread....................
    	n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
    	if (n<finish) {

    		//Load data
    		//Ci = Den[n];
    		Ex = ElectricField[n + 0 * Np];
    		Ey = ElectricField[n + 1 * Np];
    		Ez = ElectricField[n + 2 * Np];

    		ux = Velocity[n + 0 * Np];
    		uy = Velocity[n + 1 * Np];
    		uz = Velocity[n + 2 * Np];

    		uEPx = Di / Vt * Ex;
    		uEPy = Di / Vt * Ey;
    		uEPz = Di / Vt * Ez;

    		// q=0
    				// q=1
    		nr1 = neighborList[n]; // neighbor 2 ( > 10Np => odd part of dist)
    		// q=2
    		nr2 = neighborList[n + Np]; // neighbor 1 ( < 10Np => even part of dist)
    		// q=3
    		nr3 = neighborList[n + 2 * Np]; // neighbor 4
    		// q=4
    		nr4 = neighborList[n + 3 * Np]; // neighbor 3
    		// q=5
    		nr5 = neighborList[n + 4 * Np];
    		// q=6
    		nr6 = neighborList[n + 5 * Np];

    		A0 = dist[pH_ion*7*Np + n];
    		A1 = dist[pH_ion*7*Np + nr1];        // reading the A1 data into register Aq
    		A2 = dist[pH_ion*7*Np + nr2];        // reading the A2 data into register Aq
    		A3 = dist[pH_ion*7*Np + nr3];
    		A4 = dist[pH_ion*7*Np + nr4];
    		A5 = dist[pH_ion*7*Np + nr5];
    		A6 = dist[pH_ion*7*Np + nr6];

    		// charge density
    		rhoe = A0 + A1 + A2 + A3 + A4 + A5 + A6;
    		//rhoe = Ca - Cb;
    		// new equilibrium
    		tmp = sqrt(rhoe*rhoe + 4.04e-14);
    		Ca = rhoe + tmp;
    		Cb = Ca - rhoe;

    		Den[pH_ion*Np + n] = Ca - Cb;

    		// proton production
    		A1 = 0.125 * Ca * (1.0 + 4.0 * (ux + uEPx));
    		A2 = 0.125 * Ca * (1.0 - 4.0 * (ux + uEPx));
    		A3 = 0.125 * Ca * (1.0 + 4.0 * (uy) + uEPy);
    		A4 = 0.125 * Ca * (1.0 - 4.0 * (uy) + uEPy);
    		A5 = 0.125 * Ca * (1.0 + 4.0 * (uz) + uEPz);
    		A6 = 0.125 * Ca * (1.0 - 4.0 * (uz) + uEPz);  

    		A0 = Ca - (A1+A2+A3+A4+A5+A6);

    		// hydroxide ions created by water ionization (no net charge increase)
    		//Cb += (f1 + f2 + f3 + f4 + f5 + f6);
    		// use relative mass of hydroxide + momentum conservation
    		B1 = 0.125 * Cb * (1.0 + 4.0 * (ux - uEPx));
    		B2 = 0.125 * Cb * (1.0 - 4.0 * (ux - uEPx));
    		B3 = 0.125 * Cb * (1.0 + 4.0 * (uy - uEPy));
    		B4 = 0.125 * Cb * (1.0 - 4.0 * (uy - uEPy));
    		B5 = 0.125 * Cb * (1.0 + 4.0 * (uz - uEPz));
    		B6 = 0.125 * Cb * (1.0 - 4.0 * (uz - uEPz));

    		B0 = Cb - (B1 + B2 + B3 + B4 + B5 + B6);

    		B0 = Cb - (B1 + B2 + B3 + B4 + B5 + B6);

    		f0 = A0 - B0;                    
    		f1 = A1 - B1;
    		f2 = A2 - B2;
    		f3 = A3 - B3;
    		f4 = A4 - B4;
    		f5 = A5 - B5;
    		f6 = A6 - B6;     

    		dist[pH_ion*7*Np + n]   = f0;
    		dist[pH_ion*7*Np + nr2] = f1;       
    		dist[pH_ion*7*Np + nr1] = f2;
    		dist[pH_ion*7*Np + nr4] = f3;
    		dist[pH_ion*7*Np + nr3] = f4;
    		dist[pH_ion*7*Np + nr6] = f5;
    		dist[pH_ion*7*Np + nr5] = f6;

    	}      
    }
}

__global__  void dvc_ScaLBL_D3Q7_AAeven_pH_ionization( double *dist,
		double *Den, double *ElectricField, double * Velocity,
                double Di, double Vt,
		int pH_ion, int start, int finish, int Np) {
	
    int n;
    double Ex, Ey, Ez;       //electrical field
    double ux, uy, uz;
    double uEPx, uEPy, uEPz; //electrochemical induced velocity
    double Ca, Cb;
    double A0, A1, A2, A3, A4, A5, A6;
    double B0, B1, B2, B3, B4, B5, B6;
    double f0, f1, f2, f3, f4, f5, f6;
    double rhoe, tmp;

    int S = Np/NBLOCKS/NTHREADS + 1;
    for (int s=0; s<S; s++){
           //........Get 1-D index for this thread....................
           n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
           if (n<finish) {
		
        //Load data
        //Ci = Den[n];
        Ex = ElectricField[n + 0 * Np];
        Ey = ElectricField[n + 1 * Np];
        Ez = ElectricField[n + 2 * Np];
                
        ux = Velocity[n + 0 * Np];
        uy = Velocity[n + 1 * Np];
        uz = Velocity[n + 2 * Np];
        
        uEPx = Di / Vt * Ex;
        uEPy = Di / Vt * Ey;
        uEPz = Di / Vt * Ez;
        
        A0 = dist[pH_ion*7*Np + n];
        A1 = dist[pH_ion*7*Np +2 * Np + n];
        A2 = dist[pH_ion*7*Np +1 * Np + n];
        A3 = dist[pH_ion*7*Np +4 * Np + n];
        A4 = dist[pH_ion*7*Np +3 * Np + n];
        A5 = dist[pH_ion*7*Np +6 * Np + n];
        A6 = dist[pH_ion*7*Np +5 * Np + n];

        // charge density
        rhoe = A0 + A1 + A2 + A3 + A4 + A5 + A6;
        //rhoe = Ca - Cb;
        // new equilibrium
        tmp = sqrt(rhoe*rhoe + 4.04e-14);
        Ca = rhoe + tmp;
        Cb = Ca - rhoe;
        //if (Ca < 0.0) printf("Error in hydronium concentration, %f (charge density = %f) \n", Ca, rhoe);
        //if (Cb < 0.0) printf("Error in hydroxide concentration, %f \n", Cb);
        
        Den[pH_ion*Np + n] = Ca - Cb;

        // proton production
        A1 = 0.125 * Ca * (1.0 + 4.0 * (ux + uEPx));
        A2 = 0.125 * Ca * (1.0 - 4.0 * (ux + uEPx));
        A3 = 0.125 * Ca * (1.0 + 4.0 * (uy) + uEPy);
        A4 = 0.125 * Ca * (1.0 - 4.0 * (uy) + uEPy);
        A5 = 0.125 * Ca * (1.0 + 4.0 * (uz) + uEPz);
        A6 = 0.125 * Ca * (1.0 - 4.0 * (uz) + uEPz);  
        
        A0 = Ca - (A1+A2+A3+A4+A5+A6);
        
        // hydroxide ions created by water ionization (no net charge increase)
        //Cb += (f1 + f2 + f3 + f4 + f5 + f6);
        // use relative mass of hydroxide + momentum conservation
        B1 = 0.125 * Cb * (1.0 + 4.0 * (ux - uEPx));
        B2 = 0.125 * Cb * (1.0 - 4.0 * (ux - uEPx));
        B3 = 0.125 * Cb * (1.0 + 4.0 * (uy - uEPy));
        B4 = 0.125 * Cb * (1.0 - 4.0 * (uy - uEPy));
        B5 = 0.125 * Cb * (1.0 + 4.0 * (uz - uEPz));
        B6 = 0.125 * Cb * (1.0 - 4.0 * (uz - uEPz));
        
        B0 = Cb - (B1 + B2 + B3 + B4 + B5 + B6);
        
        f0 = A0 - B0;                    
        f1 = A1 - B1;
        f2 = A2 - B2;
        f3 = A3 - B3;
        f4 = A4 - B4;
        f5 = A5 - B5;
        f6 = A6 - B6;     

        dist[pH_ion*7*Np + n] = f0;
        dist[pH_ion*7*Np +1 * Np + n] = f1;
        dist[pH_ion*7*Np +2 * Np + n] = f2;
        dist[pH_ion*7*Np +3 * Np + n] = f3;
        dist[pH_ion*7*Np +4 * Np + n] = f4;
        dist[pH_ion*7*Np +5 * Np + n] = f5;
        dist[pH_ion*7*Np +6 * Np + n] = f6;

        }
    }
}
/**** end of pH equlibrium model ********/

extern "C" void Membrane_D3Q19_Unpack(int q, int *list, int *links, int start, int linkCount,
                                    double *recvbuf, double *dist, int N) {
    //....................................................................................
    // Unack distribution from the recv buffer
    // Distribution q matche Cqx, Cqy, Cqz
    // swap rule means that the distributions in recvbuf are OPPOSITE of q
    // dist may be even or odd distributions stored by stream layout
    //....................................................................................
    int n, idx, link;
    for (link=0; link<linkCount; link++){

    	idx = links[start+link];
        // Get the value from the list -- note that n is the index is from the send (non-local) process
        n = list[start + idx];
        // unpack the distribution to the proper location
        if (!(n < 0))
            dist[q * N + n] = recvbuf[start + idx];
    }
}

extern "C" void Membrane_D3Q19_Transport(int q, int *list, int *links, double *coef, int start, int offset, 
		int linkCount, double *recvbuf, double *dist, int N){
    //....................................................................................
    // Unack distribution from the recv buffer
    // Distribution q matche Cqx, Cqy, Cqz
    // swap rule means that the distributions in recvbuf are OPPOSITE of q
    // dist may be even or odd distributions stored by stream layout
    //....................................................................................
    int n, idx, link;
    double alpha;
    for (link=offset; link<linkCount; link++){

    	idx = list[start+link];
        // Get the value from the list -- note that n is the index is from the send (non-local) process
        n = list[start + idx];
        alpha = coef[start + idx];
        // unpack the distribution to the proper location
        if (!(n < 0))
            dist[q * N + n] = alpha*recvbuf[start + idx];
    }
}

__global__  void dvc_ScaLBL_D3Q7_Membrane_AssignLinkCoef(int *membrane, int *Map, double *Distance, double *Psi, double *coef,
		double Threshold, double MassFractionIn, double MassFractionOut, double ThresholdMassFractionIn, double ThresholdMassFractionOut,
		int memLinks, int Nx, int Ny, int Nz, int Np){

	int link,iq,ip,nq,np,nqm,npm;
	double aq, ap, membranePotential;
	/* Interior Links */

	int S = memLinks/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		link =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (link < memLinks) {

			// inside             	//outside
			aq = MassFractionIn;	ap = MassFractionOut;  
			iq = membrane[2*link]; 	ip = membrane[2*link+1];
			nq = iq%Np;				np = ip%Np;
			nqm = Map[nq];			npm = Map[np]; // strided layout

			/* membrane potential for this link */
			membranePotential = Psi[nqm] - Psi[npm];
			if (membranePotential > Threshold){
				aq = ThresholdMassFractionIn;	ap = ThresholdMassFractionOut;  
			}

			/* Save the mass transfer coefficients */
			coef[2*link] = aq;		coef[2*link+1] = ap;
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_Membrane_AssignLinkCoef_halo(
		const int Cqx, const int Cqy, int const Cqz, 
		int *Map, double *Distance, double *Psi, double Threshold, 
		double MassFractionIn, double MassFractionOut, double ThresholdMassFractionIn, double ThresholdMassFractionOut,
		int *d3q7_recvlist, int *d3q7_linkList, double *coef, int start, int nlinks, int count,
		const int N, const int Nx, const int Ny, const int Nz) {
	//....................................................................................
	// Unack distribution from the recv buffer
	// Distribution q matche Cqx, Cqy, Cqz
	// swap rule means that the distributions in recvbuf are OPPOSITE of q
	// dist may be even or odd distributions stored by stream layout
	//....................................................................................
	int n, idx, nqm, npm, label, i, j, k;
	double distanceLocal, distanceNonlocal;
	double psiLocal, psiNonlocal, membranePotential;
	double ap,aq; // coefficient

	/* second enforce custom rule for membrane links */
	int S = (count-nlinks)/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		idx =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;

		if (idx < count) {

			n = d3q7_recvlist[idx];
			label = d3q7_linkList[idx];
			ap = 1.0;  // regular streaming rule
			aq = 1.0;
			if (label > 0 && !(n < 0)){
				nqm = Map[n];
				distanceLocal = Distance[nqm];  
				psiLocal = Psi[nqm];

				// Get the 3-D indices from the send process
				k = nqm/(Nx*Ny); j = (nqm-Nx*Ny*k)/Nx; i = nqm-Nx*Ny*k-Nx*j;
				// Streaming link the non-local distribution
				i -= Cqx; j -= Cqy; k -= Cqz;
				npm = k*Nx*Ny + j*Nx + i;
				distanceNonlocal = Distance[npm];  
				psiNonlocal = Psi[npm];

				membranePotential = psiLocal - psiNonlocal;
				aq = MassFractionIn;
				ap = MassFractionOut;

				/* link is inside membrane */
				if (distanceLocal > 0.0){
					if (membranePotential < Threshold*(-1.0)){
						ap = MassFractionIn;
						aq = MassFractionOut;
					}
					else {
						ap = ThresholdMassFractionIn;
						aq = ThresholdMassFractionOut;
					}
				}
				else if (membranePotential > Threshold){
					aq = ThresholdMassFractionIn;
					ap = ThresholdMassFractionOut;
				}
			}
			coef[2*idx]=aq;
			coef[2*idx+1]=ap;
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_Membrane_Unpack(int q,  
		int *d3q7_recvlist, double *recvbuf, int count,
		double *dist, int N,  double *coef)  {
	//....................................................................................
	// Unack distribution from the recv buffer
	// Distribution q matche Cqx, Cqy, Cqz
	// swap rule means that the distributions in recvbuf are OPPOSITE of q
	// dist may be even or odd distributions stored by stream layout
	//....................................................................................
	int n, idx, link;
	double fq,fp,fqq,ap,aq; // coefficient

	/* second enforce custom rule for membrane links */
	int S = count/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		idx =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (idx < count){
	    	n = d3q7_recvlist[idx];
	        // update link based on mass transfer coefficients
	        if (!(n < 0)){
	        	aq = coef[2*idx];
	        	ap = coef[2*idx+1];
	        	fq = dist[q * N + n];
	        	fp = recvbuf[idx];
	        	fqq = (1-aq)*fq+ap*fp;
	            dist[q * N + n] = fqq;
	        }
		} 
	}
}

__global__  void dvc_ScaLBL_D3Q7_Membrane_IonTransport(int *membrane, double *coef, 
		double *dist, double *Den, int memLinks, int Np){	
	int link,iq,ip,nq,np;
	double aq, ap, fq, fp, fqq, fpp, Cq, Cp;

	int S = memLinks/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		link =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (link < memLinks){

			// inside             	//outside
			aq = coef[2*link];		ap = coef[2*link+1];
			iq = membrane[2*link]; 	ip = membrane[2*link+1];
			nq = iq%Np;				np = ip%Np;
			fq  = dist[iq];			fp = dist[ip];
			fqq = (1-aq)*fq+ap*fp;	fpp = (1-ap)*fp+aq*fq;
			Cq = Den[nq];			Cp = Den[np];
			Cq += fqq - fq;			Cp += fpp - fp;
			Den[nq] = Cq;			Den[np] = Cp;
			dist[iq] = fqq;			dist[ip] = fpp;
		}
	}
}


__global__  void dvc_ScaLBL_D3Q7_AAodd_IonConcentration(int *neighborList, double *dist, double *Den, int start, int finish, int Np){
    int n,nread;
    double fq,Ci;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {

            // q=0
            fq = dist[n];
            Ci = fq;

            // q=1
            nread = neighborList[n]; 
            fq = dist[nread]; 
            Ci += fq;
            
            // q=2
            nread = neighborList[n+Np]; 
            fq = dist[nread];  
            Ci += fq;
            
            // q=3
            nread = neighborList[n+2*Np]; 
            fq = dist[nread];
            Ci += fq;
            
            // q=4
            nread = neighborList[n+3*Np]; 
            fq = dist[nread];
            Ci += fq;
            
            // q=5
            nread = neighborList[n+4*Np];
            fq = dist[nread];
            Ci += fq;
            
            // q=6
            nread = neighborList[n+5*Np];
            fq = dist[nread];
            Ci += fq;

            Den[n]=Ci;
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_AAeven_IonConcentration(double *dist, double *Den, int start, int finish, int Np){
    int n;
    double fq,Ci;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {

            // q=0
            fq = dist[n];
            Ci = fq;
            
            // q=1
            fq = dist[2*Np+n];
            Ci += fq;

            // q=2
            fq = dist[1*Np+n];
            Ci += fq;

            // q=3
            fq = dist[4*Np+n];
            Ci += fq;

            // q=4
            fq = dist[3*Np+n];
            Ci += fq;

            // q=5
            fq = dist[6*Np+n];
            Ci += fq;

            // q=6
            fq = dist[5*Np+n];
            Ci += fq;

            Den[n]=Ci;
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_AAodd_Ion(int *neighborList, double *dist, double *Den, double *FluxDiffusive, double *FluxAdvective, double *FluxElectrical, double *Velocity, double *ElectricField, 
                                           double Di, int zi, double rlx, double Vt, int start, int finish, int Np){
    int n;
    double Ci;
    double ux, uy, uz;
    double uEPx, uEPy, uEPz; //electrochemical induced velocity
    double Ex, Ey, Ez;       //electrical field
    double flux_diffusive_x, flux_diffusive_y, flux_diffusive_z;
    double f0, f1, f2, f3, f4, f5, f6;
    //double X,Y,Z,factor_x, factor_y, factor_z;
    int nr1, nr2, nr3, nr4, nr5, nr6;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
	        //Load data

	        //Load data
	        //Ci = Den[n];
	        Ex = ElectricField[n + 0 * Np];
	        Ey = ElectricField[n + 1 * Np];
	        Ez = ElectricField[n + 2 * Np];
	        ux = Velocity[n + 0 * Np];
	        uy = Velocity[n + 1 * Np];
	        uz = Velocity[n + 2 * Np];
	        uEPx = zi * Di / Vt * Ex;
	        uEPy = zi * Di / Vt * Ey;
	        uEPz = zi * Di / Vt * Ez;

	        // q=0
	        f0 = dist[n];
	        // q=1
	        nr1 = neighborList[n]; // neighbor 2 ( > 10Np => odd part of dist)
	        f1 = dist[nr1];        // reading the f1 data into register fq
	        // q=2
	        nr2 = neighborList[n + Np]; // neighbor 1 ( < 10Np => even part of dist)
	        f2 = dist[nr2];             // reading the f2 data into register fq
	        // q=3
	        nr3 = neighborList[n + 2 * Np]; // neighbor 4
	        f3 = dist[nr3];
	        // q=4
	        nr4 = neighborList[n + 3 * Np]; // neighbor 3
	        f4 = dist[nr4];
	        // q=5
	        nr5 = neighborList[n + 4 * Np];
	        f5 = dist[nr5];
	        // q=6
	        nr6 = neighborList[n + 5 * Np];
	        f6 = dist[nr6];

	        // compute diffusive flux
	        Ci = f0 + f1 + f2 + f3 + f4 + f5 + f6;
	        flux_diffusive_x = (1.0 - 0.5 * rlx) * ((f1 - f2) - ux * Ci);
	        flux_diffusive_y = (1.0 - 0.5 * rlx) * ((f3 - f4) - uy * Ci);
	        flux_diffusive_z = (1.0 - 0.5 * rlx) * ((f5 - f6) - uz * Ci);
	        FluxDiffusive[n + 0 * Np] = flux_diffusive_x;
	        FluxDiffusive[n + 1 * Np] = flux_diffusive_y;
	        FluxDiffusive[n + 2 * Np] = flux_diffusive_z;
	        FluxAdvective[n + 0 * Np] = ux * Ci;
	        FluxAdvective[n + 1 * Np] = uy * Ci;
	        FluxAdvective[n + 2 * Np] = uz * Ci;
	        FluxElectrical[n + 0 * Np] = uEPx * Ci;
	        FluxElectrical[n + 1 * Np] = uEPy * Ci;
	        FluxElectrical[n + 2 * Np] = uEPz * Ci;
	        
	        Den[n] = Ci;

	        // q=0
	        dist[n] = f0 * (1.0 - rlx) + rlx * 0.25 * Ci;

	        // q = 1
	        dist[nr2] =
	        f1 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + 4.0 * (ux + uEPx));

	        // q=2
	        dist[nr1] =
	        f2 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - 4.0 * (ux + uEPx));

	        // q = 3
	        dist[nr4] =
	        f3 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + 4.0 * (uy + uEPy));

	        // q = 4
	        dist[nr3] =
	        f4 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - 4.0 * (uy + uEPy));

	        // q = 5
	        dist[nr6] =
	        f5 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + 4.0 * (uz + uEPz));

	        // q = 6
	        dist[nr5] =
	        f6 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - 4.0 * (uz + uEPz));

		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_AAeven_Ion(double *dist, double *Den, double *FluxDiffusive, double *FluxAdvective, double *FluxElectrical, double *Velocity, double *ElectricField, 
                                            double Di, int zi, double rlx, double Vt, int start, int finish, int Np){
    int n;
    double Ci;
    double ux, uy, uz;
    double uEPx, uEPy, uEPz; //electrochemical induced velocity
    double Ex, Ey, Ez;       //electrical field
    double flux_diffusive_x, flux_diffusive_y, flux_diffusive_z;
    double f0, f1, f2, f3, f4, f5, f6;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {

	        //Load data
	        //Ci = Den[n];
	        Ex = ElectricField[n + 0 * Np];
	        Ey = ElectricField[n + 1 * Np];
	        Ez = ElectricField[n + 2 * Np];
	        ux = Velocity[n + 0 * Np];
	        uy = Velocity[n + 1 * Np];
	        uz = Velocity[n + 2 * Np];
	        uEPx = zi * Di / Vt * Ex;
	        uEPy = zi * Di / Vt * Ey;
	        uEPz = zi * Di / Vt * Ez;

	        f0 = dist[n];
	        f1 = dist[2 * Np + n];
	        f2 = dist[1 * Np + n];
	        f3 = dist[4 * Np + n];
	        f4 = dist[3 * Np + n];
	        f5 = dist[6 * Np + n];
	        f6 = dist[5 * Np + n];

	        // compute diffusive flux
	        Ci = f0 + f1 + f2 + f3 + f4 + f5 + f6;
	        flux_diffusive_x = (1.0 - 0.5 * rlx) * ((f1 - f2) - ux * Ci);
	        flux_diffusive_y = (1.0 - 0.5 * rlx) * ((f3 - f4) - uy * Ci);
	        flux_diffusive_z = (1.0 - 0.5 * rlx) * ((f5 - f6) - uz * Ci);
	        FluxDiffusive[n + 0 * Np] = flux_diffusive_x;
	        FluxDiffusive[n + 1 * Np] = flux_diffusive_y;
	        FluxDiffusive[n + 2 * Np] = flux_diffusive_z;
	        FluxAdvective[n + 0 * Np] = ux * Ci;
	        FluxAdvective[n + 1 * Np] = uy * Ci;
	        FluxAdvective[n + 2 * Np] = uz * Ci;
	        FluxElectrical[n + 0 * Np] = uEPx * Ci;
	        FluxElectrical[n + 1 * Np] = uEPy * Ci;
	        FluxElectrical[n + 2 * Np] = uEPz * Ci;
	        
	        Den[n] = Ci;

	        // q=0
	        dist[n] = f0 * (1.0 - rlx) + rlx * 0.25 * Ci;

	        // q = 1
	        dist[1 * Np + n] =
	        f1 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + 4.0 * (ux + uEPx));

	        // q=2
	        dist[2 * Np + n] =
	        f2 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - 4.0 * (ux + uEPx));

	        // q = 3
	        dist[3 * Np + n] =
	        f3 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + 4.0 * (uy + uEPy));

	        // q = 4
	        dist[4 * Np + n] =
	        f4 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - 4.0 * (uy + uEPy));

	        // q = 5
	        dist[5 * Np + n] =
	        f5 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + 4.0 * (uz + uEPz));

	        // q = 6
	        dist[6 * Np + n] =
	        f6 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - 4.0 * (uz + uEPz));
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_Ion_Init(double *dist, double *Den, double DenInit, int Np){

	int n;
	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<Np) {
            dist[0*Np+n] = 0.25*DenInit;
            dist[1*Np+n] = 0.125*DenInit;		
            dist[2*Np+n] = 0.125*DenInit;	
            dist[3*Np+n] = 0.125*DenInit;	
            dist[4*Np+n] = 0.125*DenInit;	
            dist[5*Np+n] = 0.125*DenInit;	
            dist[6*Np+n] = 0.125*DenInit;	
            Den[n] = DenInit;
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_Ion_Init_FromFile(double *dist, double *Den, int Np){

	int n;
    double DenInit;
	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<Np) {
            DenInit = Den[n];
            dist[0*Np+n] = 0.25*DenInit;
            dist[1*Np+n] = 0.125*DenInit;		
            dist[2*Np+n] = 0.125*DenInit;	
            dist[3*Np+n] = 0.125*DenInit;	
            dist[4*Np+n] = 0.125*DenInit;	
            dist[5*Np+n] = 0.125*DenInit;	
            dist[6*Np+n] = 0.125*DenInit;	
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_Ion_ChargeDensity(double *Den, double *ChargeDensity, double IonValence, int ion_component, int start, int finish, int Np){

    int n;
    double Ci;//ion concentration of species i
    double CD;//charge density
    double CD_tmp;
    double F = 96485.0;//Faraday's constant; unit[C/mol]; F=e*Na, where Na is the Avogadro constant

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {

            Ci = Den[n+ion_component*Np];
            CD = ChargeDensity[n];
            if (ion_component == 0) CD=0.0;
            CD_tmp = F*IonValence*Ci;
            ChargeDensity[n] = CD + CD_tmp;

		}
	}
}
__global__  void dvc_ScaLBL_D3Q7_AAodd_Ion_v0(int *neighborList, double *dist,
                                      double *Den, double *FluxDiffusive,
                                      double *FluxAdvective,
                                      double *FluxElectrical, double *Velocity,
                                      double *ElectricField, double Di, int zi,
                                      double rlx, double Vt, int start,
                                      int finish, int Np) {
    int n;
    double Ci;
    double ux, uy, uz;
    double uEPx, uEPy, uEPz; //electrochemical induced velocity
    double Ex, Ey, Ez;       //electrical field
    double flux_diffusive_x, flux_diffusive_y, flux_diffusive_z;
    double f0, f1, f2, f3, f4, f5, f6;
    //double X,Y,Z,factor_x, factor_y, factor_z;
    int nr1, nr2, nr3, nr4, nr5, nr6;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {


        //Load data
        Ci = Den[n];
        Ex = ElectricField[n + 0 * Np];
        Ey = ElectricField[n + 1 * Np];
        Ez = ElectricField[n + 2 * Np];
        ux = Velocity[n + 0 * Np];
        uy = Velocity[n + 1 * Np];
        uz = Velocity[n + 2 * Np];
        uEPx = zi * Di / Vt * Ex;
        uEPy = zi * Di / Vt * Ey;
        uEPz = zi * Di / Vt * Ez;

        // q=0
        f0 = dist[n];
        // q=1
        nr1 = neighborList[n]; // neighbor 2 ( > 10Np => odd part of dist)
        f1 = dist[nr1];        // reading the f1 data into register fq
        // q=2
        nr2 = neighborList[n + Np]; // neighbor 1 ( < 10Np => even part of dist)
        f2 = dist[nr2];             // reading the f2 data into register fq
        // q=3
        nr3 = neighborList[n + 2 * Np]; // neighbor 4
        f3 = dist[nr3];
        // q=4
        nr4 = neighborList[n + 3 * Np]; // neighbor 3
        f4 = dist[nr4];
        // q=5
        nr5 = neighborList[n + 4 * Np];
        f5 = dist[nr5];
        // q=6
        nr6 = neighborList[n + 5 * Np];
        f6 = dist[nr6];

        // compute diffusive flux
        //Ci = f0 + f1 + f2 + f3 + f4 + f5 + f6;
        flux_diffusive_x = (1.0 - 0.5 * rlx) * ((f1 - f2) - ux * Ci);
        flux_diffusive_y = (1.0 - 0.5 * rlx) * ((f3 - f4) - uy * Ci);
        flux_diffusive_z = (1.0 - 0.5 * rlx) * ((f5 - f6) - uz * Ci);
        FluxDiffusive[n + 0 * Np] = flux_diffusive_x;
        FluxDiffusive[n + 1 * Np] = flux_diffusive_y;
        FluxDiffusive[n + 2 * Np] = flux_diffusive_z;
        FluxAdvective[n + 0 * Np] = ux * Ci;
        FluxAdvective[n + 1 * Np] = uy * Ci;
        FluxAdvective[n + 2 * Np] = uz * Ci;
        FluxElectrical[n + 0 * Np] = uEPx * Ci;
        FluxElectrical[n + 1 * Np] = uEPy * Ci;
        FluxElectrical[n + 2 * Np] = uEPz * Ci;
        
        //Den[n] = Ci;

        /* use logistic function to prevent negative distributions*/
        //X = 4.0 * (ux + uEPx);
        //Y = 4.0 * (uy + uEPy);
        //Z = 4.0 * (uz + uEPz);
        //factor_x = X / sqrt(1 + X*X);
        //factor_y = Y / sqrt(1 + Y*Y);
        //factor_z = Z / sqrt(1 + Z*Z);

        // q=0
        dist[n] = f0 * (1.0 - rlx) + rlx * 0.25 * Ci;

        // q = 1
        dist[nr2] =
        f1 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + 4.0 * (ux + uEPx));
        //    f1 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + factor_x);


        // q=2
        dist[nr1] =
        f2 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - 4.0 * (ux + uEPx));
        //        f2 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - factor_x);

        // q = 3
        dist[nr4] =
        f3 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + 4.0 * (uy + uEPy));
        //        f3 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + factor_y );

        // q = 4
        dist[nr3] =
        f4 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - 4.0 * (uy + uEPy));
        //        f4 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - factor_y);

        // q = 5
        dist[nr6] =
        f5 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + 4.0 * (uz + uEPz));
        //        f5 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 +  factor_z);

        // q = 6
        dist[nr5] =
        f6 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - 4.0 * (uz + uEPz));
        //    f6 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - factor_z);

		}
    }
}

__global__  void dvc_ScaLBL_D3Q7_AAeven_Ion_v0(
    double *dist, double *Den, double *FluxDiffusive, double *FluxAdvective,
    double *FluxElectrical, double *Velocity, double *ElectricField, double Di,
    int zi, double rlx, double Vt, int start, int finish, int Np) {
    int n;
    double Ci;
    double ux, uy, uz;
    double uEPx, uEPy, uEPz; //electrochemical induced velocity
    double Ex, Ey, Ez;       //electrical field
    double flux_diffusive_x, flux_diffusive_y, flux_diffusive_z;
    double f0, f1, f2, f3, f4, f5, f6;
    //double X,Y,Z, factor_x, factor_y, factor_z;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {

        //Load data
        Ci = Den[n];
        Ex = ElectricField[n + 0 * Np];
        Ey = ElectricField[n + 1 * Np];
        Ez = ElectricField[n + 2 * Np];
        ux = Velocity[n + 0 * Np];
        uy = Velocity[n + 1 * Np];
        uz = Velocity[n + 2 * Np];
        uEPx = zi * Di / Vt * Ex;
        uEPy = zi * Di / Vt * Ey;
        uEPz = zi * Di / Vt * Ez;

        f0 = dist[n];
        f1 = dist[2 * Np + n];
        f2 = dist[1 * Np + n];
        f3 = dist[4 * Np + n];
        f4 = dist[3 * Np + n];
        f5 = dist[6 * Np + n];
        f6 = dist[5 * Np + n];

        // compute diffusive flux
        //Ci = f0 + f1 + f2 + f3 + f4 + f5 + f6;
        flux_diffusive_x = (1.0 - 0.5 * rlx) * ((f1 - f2) - ux * Ci);
        flux_diffusive_y = (1.0 - 0.5 * rlx) * ((f3 - f4) - uy * Ci);
        flux_diffusive_z = (1.0 - 0.5 * rlx) * ((f5 - f6) - uz * Ci);
        FluxDiffusive[n + 0 * Np] = flux_diffusive_x;
        FluxDiffusive[n + 1 * Np] = flux_diffusive_y;
        FluxDiffusive[n + 2 * Np] = flux_diffusive_z;
        FluxAdvective[n + 0 * Np] = ux * Ci;
        FluxAdvective[n + 1 * Np] = uy * Ci;
        FluxAdvective[n + 2 * Np] = uz * Ci;
        FluxElectrical[n + 0 * Np] = uEPx * Ci;
        FluxElectrical[n + 1 * Np] = uEPy * Ci;
        FluxElectrical[n + 2 * Np] = uEPz * Ci;
        
        //Den[n] = Ci;
        
        /* use logistic function to prevent negative distributions*/
        //X = 4.0 * (ux + uEPx);
        //Y = 4.0 * (uy + uEPy);
        //Z = 4.0 * (uz + uEPz);
        //factor_x = X / sqrt(1 + X*X);
        //factor_y = Y / sqrt(1 + Y*Y);
        //factor_z = Z / sqrt(1 + Z*Z);

        // q=0
        dist[n] = f0 * (1.0 - rlx) + rlx * 0.25 * Ci;

        // q = 1
        dist[1 * Np + n] =
        f1 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + 4.0 * (ux + uEPx));
        //        f1 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + factor_x);

        // q=2
        dist[2 * Np + n] =
        f2 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - 4.0 * (ux + uEPx));
        //        f2 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - factor_x);

        // q = 3
        dist[3 * Np + n] =
        f3 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + 4.0 * (uy + uEPy));
        //        f3 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + factor_y);

        // q = 4
        dist[4 * Np + n] =
        f4 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - 4.0 * (uy + uEPy));
        //        f4 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - factor_y);

        // q = 5
        dist[5 * Np + n] =
        f5 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + 4.0 * (uz + uEPz));
        //        f5 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 + factor_z);

        // q = 6
        dist[6 * Np + n] =
        f6 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - 4.0 * (uz + uEPz));
        //        f6 * (1.0 - rlx) + rlx * 0.125 * Ci * (1.0 - factor_z);
        }
    }
}

extern "C" void ScaLBL_D3Q7_AAeven_Ion_v0(
    double *dist, double *Den, double *FluxDiffusive, double *FluxAdvective,
    double *FluxElectrical, double *Velocity, double *ElectricField, double Di,
    int zi, double rlx, double Vt, int start, int finish, int Np) {
    
    dvc_ScaLBL_D3Q7_AAeven_Ion_v0<<<NBLOCKS,NTHREADS >>>(dist,
                                      Den, FluxDiffusive, FluxAdvective,
                                      FluxElectrical, Velocity,
                                      ElectricField, Di, zi,
                                      rlx, Vt, start, finish,  Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("cuda error in dvc_ScaLBL_D3Q7_AAeven_Ion_v0: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q7_AAodd_Ion_v0(int *neighborList, double *dist,
                                      double *Den, double *FluxDiffusive,
                                      double *FluxAdvective,
                                      double *FluxElectrical, double *Velocity,
                                      double *ElectricField, double Di, int zi,
                                      double rlx, double Vt, int start,
                                      int finish, int Np) {
                                      
	dvc_ScaLBL_D3Q7_AAodd_Ion_v0<<<NBLOCKS,NTHREADS >>>(neighborList, dist,
                                      Den, FluxDiffusive, FluxAdvective,
                                      FluxElectrical, Velocity,
                                      ElectricField, Di, zi,
                                      rlx, Vt, start,
                                      finish,  Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("cuda error in dvc_ScaLBL_D3Q7_AAodd_Ion_v0: %s \n",hipGetErrorString(err));
	}
} 
                                 


extern "C" void ScaLBL_D3Q7_AAodd_IonConcentration(int *neighborList, double *dist, double *Den, int start, int finish, int Np){

	//cudaProfilerStart();
	dvc_ScaLBL_D3Q7_AAodd_IonConcentration<<<NBLOCKS,NTHREADS >>>(neighborList,dist,Den,start,finish,Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_AAodd_IonConcentration: %s \n",hipGetErrorString(err));
	}
	//cudaProfilerStop();
}

extern "C" void ScaLBL_D3Q7_AAeven_IonConcentration(double *dist, double *Den, int start, int finish, int Np){

	//cudaProfilerStart();
	dvc_ScaLBL_D3Q7_AAeven_IonConcentration<<<NBLOCKS,NTHREADS >>>(dist,Den,start,finish,Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_AAeven_IonConcentration: %s \n",hipGetErrorString(err));
	}
	//cudaProfilerStop();
}

extern "C" void ScaLBL_D3Q7_AAodd_Ion(int *neighborList, double *dist, double *Den, double *FluxDiffusive, double *FluxAdvective, double *FluxElectrical, double *Velocity, double *ElectricField,  
                                      double Di, int zi, double rlx, double Vt, int start, int finish, int Np){
	//cudaProfilerStart();
	dvc_ScaLBL_D3Q7_AAodd_Ion<<<NBLOCKS,NTHREADS >>>(neighborList,dist,Den,FluxDiffusive,FluxAdvective,FluxElectrical,Velocity,ElectricField,Di,zi,rlx,Vt,start,finish,Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_AAodd_Ion: %s \n",hipGetErrorString(err));
	}
	//cudaProfilerStop();
}

extern "C" void ScaLBL_D3Q7_AAeven_Ion(double *dist, double *Den, double *FluxDiffusive, double *FluxAdvective, double *FluxElectrical, double *Velocity, double *ElectricField, 
                                       double Di, int zi, double rlx, double Vt, int start, int finish, int Np){
	//cudaProfilerStart();
	dvc_ScaLBL_D3Q7_AAeven_Ion<<<NBLOCKS,NTHREADS >>>(dist,Den,FluxDiffusive,FluxAdvective,FluxElectrical,Velocity,ElectricField,Di,zi,rlx,Vt,start,finish,Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_AAeven_Ion: %s \n",hipGetErrorString(err));
	}
	//cudaProfilerStop();
}

extern "C" void ScaLBL_D3Q7_Ion_Init(double *dist, double *Den, double DenInit, int Np){

	//cudaProfilerStart();
	dvc_ScaLBL_D3Q7_Ion_Init<<<NBLOCKS,NTHREADS >>>(dist,Den,DenInit,Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_Ion_Init: %s \n",hipGetErrorString(err));
	}
	//cudaProfilerStop();
}

extern "C" void ScaLBL_D3Q7_Ion_Init_FromFile(double *dist, double *Den, int Np){

	//cudaProfilerStart();
	dvc_ScaLBL_D3Q7_Ion_Init_FromFile<<<NBLOCKS,NTHREADS >>>(dist,Den,Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_Ion_Init_FromFile: %s \n",hipGetErrorString(err));
	}
	//cudaProfilerStop();
}

extern "C" void ScaLBL_D3Q7_Ion_ChargeDensity(double *Den, double *ChargeDensity, double IonValence, int ion_component, int start, int finish, int Np){

	//cudaProfilerStart();
	dvc_ScaLBL_D3Q7_Ion_ChargeDensity<<<NBLOCKS,NTHREADS >>>(Den,ChargeDensity,IonValence,ion_component,start,finish,Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_Ion_ChargeDensity: %s \n",hipGetErrorString(err));
	}
	//cudaProfilerStop();
}

extern "C" void ScaLBL_D3Q7_Membrane_AssignLinkCoef(int *membrane, int *Map, double *Distance, double *Psi, double *coef,
		double Threshold, double MassFractionIn, double MassFractionOut, double ThresholdMassFractionIn, double ThresholdMassFractionOut,
		int memLinks, int Nx, int Ny, int Nz, int Np){
	
	dvc_ScaLBL_D3Q7_Membrane_AssignLinkCoef<<<NBLOCKS,NTHREADS >>>(membrane,  Map,  Distance,  Psi,  coef,
			 Threshold,  MassFractionIn,  MassFractionOut,  ThresholdMassFractionIn,  ThresholdMassFractionOut,
			 memLinks,  Nx,  Ny,  Nz,  Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in dvc_ScaLBL_D3Q7_Membrane_AssignLinkCoef: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q7_Membrane_AssignLinkCoef_halo(
		const int Cqx, const int Cqy, int const Cqz, 
		int *Map, double *Distance, double *Psi, double Threshold, 
		double MassFractionIn, double MassFractionOut, double ThresholdMassFractionIn, double ThresholdMassFractionOut,
		int *d3q7_recvlist, int *d3q7_linkList, double *coef, int start, int nlinks, int count,
		const int N, const int Nx, const int Ny, const int Nz) {
	
    int GRID = count / NTHREADS + 1;

	dvc_ScaLBL_D3Q7_Membrane_AssignLinkCoef_halo<<<GRID,NTHREADS >>>(
			 Cqx,  Cqy,  Cqz, Map, Distance, Psi,  Threshold, 
			 MassFractionIn,  MassFractionOut,  ThresholdMassFractionIn,  ThresholdMassFractionOut,
			d3q7_recvlist, d3q7_linkList, coef,  start,  nlinks,  count, N,  Nx,  Ny,  Nz);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in dvc_ScaLBL_D3Q7_Membrane_AssignLinkCoef_halo: %s \n",hipGetErrorString(err));
	}
}


extern "C" void ScaLBL_D3Q7_Membrane_Unpack(int q,  
		int *d3q7_recvlist, double *recvbuf, int count,
		double *dist, int N,  double *coef){
	
    int GRID = count / NTHREADS + 1;

	dvc_ScaLBL_D3Q7_Membrane_Unpack<<<GRID,NTHREADS >>>(q, d3q7_recvlist, recvbuf,count,
			 dist, N,  coef);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in dvc_ScaLBL_D3Q7_Membrane_Unpack: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q7_Membrane_IonTransport(int *membrane, double *coef, 
		double *dist, double *Den, int memLinks, int Np){
	
	dvc_ScaLBL_D3Q7_Membrane_IonTransport<<<NBLOCKS,NTHREADS >>>(membrane, coef, dist, Den, memLinks, Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in dvc_ScaLBL_D3Q7_Membrane_IonTransport: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q7_AAodd_pH_ionization(int *neighborList, double *dist,
                                      double *Den, double *ElectricField, double *Velocity,
                                      double Di, double Vt,
                                      int pH_ion, int start, int finish, int Np) {

        dvc_ScaLBL_D3Q7_AAodd_pH_ionization<<<NBLOCKS,NTHREADS >>>(neighborList,dist,Den,ElectricField,
	                                                           Velocity,Di,Vt,pH_ion,start,finish,Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in dvc_ScaLBL_D3Q7_AAodd_pH_ionization: %s \n",hipGetErrorString(err));
	}

}

extern "C" void ScaLBL_D3Q7_AAeven_pH_ionization( double *dist,
		double *Den, double *ElectricField, double * Velocity,
                double Di, double Vt,
		int pH_ion, int start, int finish, int Np) {

        dvc_ScaLBL_D3Q7_AAeven_pH_ionization<<<NBLOCKS,NTHREADS >>>(dist,Den,ElectricField,
	                                                            Velocity,Di,Vt,pH_ion,start,finish,Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in dvc_ScaLBL_D3Q7_AAeven_pH_ionization: %s \n",hipGetErrorString(err));
	}

}

