/* Implement Mixed Gradient (Lee et al. JCP 2016)*/
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#define NBLOCKS 560
#define NTHREADS 128

__global__ void dvc_ScaLBL_D3Q19_MixedGradient(int *Map, double *Phi, double *Gradient, int start, int finish, int Np, int Nx, int Ny, int Nz)
{
	static int D3Q19[18][3]={{1,0,0},{-1,0,0},{0,1,0},{0,-1,0},{0,0,1},{0,0,-1},
			{1,1,0},{-1,-1,0},{1,-1,0},{-1,1,0},
			{1,0,1},{-1,0,-1},{1,0,-1},{-1,0,1},
			{0,1,1},{0,-1,-1},{0,1,-1},{0,-1,1}};

	int i,j,k,n,N,idx;
	int np,np2,nm; // neighbors
	double v,vp,vp2,vm; // values at neighbors
	double grad;
	N = Nx*Ny*Nz;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){

		//........Get 1-D index for this thread....................
		idx = start + S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;

		if (idx<finish){
			n = Map[idx]; // layout in regular array
			//.......Back out the 3-D indices for node n..............
			k = n/(Nx*Ny);
			j = (n-Nx*Ny*k)/Nx;
			i = n-Nx*Ny*k-Nx*j;	
			v = Phi[n];
			grad = 0.0;
			for (int q=0; q<6; q++){
				int iqx = D3Q19[q][0];
				int iqy = D3Q19[q][1];
				int iqz = D3Q19[q][2];
				np = (k+iqz)*Nx*Ny + (j+iqy)*Nx + i + iqx;
				np2 = (k+2*iqz)*Nx*Ny + (j+2*iqy)*Nx + i + 2*iqx;
				nm = (k-iqz)*Nx*Ny + (j-iqy)*Nx + i - iqx;
				vp = Phi[np];
				vp2 = Phi[np2];
				vm = Phi[nm];
				grad += 0.25*(5.0*vp-vp2-3.0*v-vm);
			}
			for (int q=6; q<18; q++){
				int iqx = D3Q19[q][0];
				int iqy = D3Q19[q][1];
				int iqz = D3Q19[q][2];
				np = (k+iqz)*Nx*Ny + (j+iqy)*Nx + i + iqx;
				np2 = (k+2*iqz)*Nx*Ny + (j+2*iqy)*Nx + i + 2*iqx;
				nm = (k-iqz)*Nx*Ny + (j-iqy)*Nx + i - iqx;
				vp = Phi[np];
				vp2 = Phi[np2];
				vm = Phi[nm];
				grad += 0.125*(5.0*vp-vp2-3.0*v-vm);
			}
			Gradient[n] = grad;
		}
	}
}

extern "C" void ScaLBL_D3Q19_MixedGradient(int *Map, double *Phi, double *Gradient, int start, int finish, int Np, int Nx, int Ny, int Nz)
{
	hipProfilerStart();
	dvc_ScaLBL_D3Q19_MixedGradient<<<NBLOCKS,NTHREADS >>>(Map, Phi, Gradient, start, finish, Np, Nx, Ny, Nz);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_MixedGradient: %s \n",hipGetErrorString(err));
	}
	hipProfilerStop();
}

