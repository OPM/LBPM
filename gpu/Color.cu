
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#define NBLOCKS 1024
#define NTHREADS 256

__global__  void dvc_ScaLBL_Color_Init(char *ID, double *Den, double *Phi, double das, double dbs, int Nx, int Ny, int Nz)
{
	//int i,j,k;
	int n,N;
	char id;
	N = Nx*Ny*Nz;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N){
		
  		id=ID[n];	
 		//.......Back out the 3-D indices for node n..............
		//k = n/(Nx*Ny);
		//j = (n-Nx*Ny*k)/Nx;
		//i = n-Nx*Ny*k-Nx*j;

		if ( id == 1){
			Den[n] = 1.0;
			Den[N+n] = 0.0;
			Phi[n] = 1.0;
		}
		else if ( id == 2){
			Den[n] = 0.0;
			Den[N+n] = 1.0;
			Phi[n] = -1.0;
		}
		else{
			Den[n] = das;
			Den[N+n] = dbs;
			Phi[n] = (das-dbs)/(das+dbs);
		}
		}
	}
}
__global__  void dvc_ScaLBL_Color_InitDistancePacked(char *ID, double *Den, double *Phi, double *Distance,
								double das, double dbs, double beta, double xp, int Nx, int Ny, int Nz)
{
	int i,j,k,n,N;
	double d;

	N = Nx*Ny*Nz;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N){
			//.......Back out the 3-D indices for node n..............
			k = n/(Nx*Ny);
			j = (n-Nx*Ny*k)/Nx;
			i = n-Nx*Ny*k-Nx*j;

			if ( ID[n] == 1){
				Den[2*n] = 1.0;
				Den[2*n+1] = 0.0;
				Phi[n] = 1.0;
			}
			if (i == 0 || j == 0 || k == 0 || i == Nx-1 || j == Ny-1 || k == Nz-1){
				Den[2*n] = 0.0;
				Den[2*n+1] = 0.0;
			}
			else if ( ID[n] == 1){
				Den[2*n] = 1.0;
				Den[2*n+1] = 0.0;
				Phi[n] = 1.0;
			}
			else if ( ID[n] == 2){
				Den[2*n] = 0.0;
				Den[2*n+1] = 1.0;
				Phi[n] = -1.0;
			}
			else{
				Den[2*n] = das;
				Den[2*n+1] = dbs;
				Phi[n] = (das-dbs)/(das+dbs);
				d = fabs(Distance[n]);
				Phi[n] = (2.f*(exp(-2.f*beta*(d+xp)))/(1.f+exp(-2.f*beta*(d+xp))) - 1.f);
			}
		}
	}
}

__global__  void dvc_ScaLBL_Color_InitDistance(char *ID, double *Den, double *Phi, double *Distance,
		double das, double dbs, double beta, double xp, int Nx, int Ny, int Nz)
{
	int n,N;
	double d;

	N = Nx*Ny*Nz;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N){
			if ( ID[n] == 1){
				Den[n] = 1.0;
				Den[N+n] = 0.0;
				Phi[n] = 1.0;
			}
			else if ( ID[n] == 2){
				Den[n] = 0.0;
				Den[N+n] = 1.0;
				Phi[n] = -1.0;
			}
			else{
				Den[n] = das;
				Den[N+n] = dbs;
				Phi[n] = (das-dbs)/(das+dbs);
				d = fabs(Distance[n]);
				Phi[n] = (2.f*(exp(-2.f*beta*(d+xp)))/(1.f+exp(-2.f*beta*(d+xp))) - 1.f);
			}
		}
	}
}

//*************************************************************************

__global__  void dvc_ScaLBL_Color_BC(int *list, int *Map, double *Phi, double *Den, double vA, double vB, int count, int Np)
{
	int idx,n,nm;
	// Fill the outlet with component b
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		Den[n] = vA;
		Den[Np+n] = vB;
		
		//nm = Map[n];
		//Phi[nm] = (vA-vB)/(vA+vB);
		Phi[n] = (vA-vB)/(vA+vB);

	}
}
__global__  void dvc_ScaLBL_Color_BC_z(int *list, int *Map, double *Phi, double *Den, double vA, double vB, int count, int Np)
{
	int idx,n,nm;
	// Fill the outlet with component b
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		Den[n] = vA;
		Den[Np+n] = vB;
		//double valB = Den[Np+n]; // mass that reaches inlet is conserved

		nm = Map[n];
		Phi[nm] = (vA-vB)/(vA+vB);
		//Phi[n] = (vA-vB)/(vA+vB);
	}
}

__global__  void dvc_ScaLBL_Color_BC_Z(int *list, int *Map, double *Phi, double *Den, double vA, double vB, int count, int Np)
{
	int idx,n,nm;
	// Fill the outlet with component b
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < count){
		n = list[idx];
		//double valA = Den[n]; // mass that reaches outlet is conserved
		Den[n] = vA;
		Den[Np+n] = vB;
		
		nm = Map[n];
		Phi[nm] = (vA-vB)/(vA+vB);
		//Phi[n] = (vA-vB)/(vA+vB);

	}
}
//*************************************************************************

__global__  void dvc_ScaLBL_D3Q19_ColorGradient(char *ID, double *phi, double *ColorGrad, int Nx, int Ny, int Nz)
{
	int n,N,i,j,k,nn;
	// distributions
	double f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double nx,ny,nz;

	// non-conserved moments
	// additional variables needed for computations

	N = Nx*Ny*Nz;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N ){

			//.......Back out the 3-D indices for node n..............
			k = n/(Nx*Ny);
			j = (n-Nx*Ny*k)/Nx;
			i = n-Nx*Ny*k-Nx*j;
			//........................................................................
			//........Get 1-D index for this thread....................
			//		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
			//........................................................................
			//					COMPUTE THE COLOR GRADIENT
			//........................................................................
			//.................Read Phase Indicator Values............................
			//........................................................................
			nn = n-1;							// neighbor index (get convention)
			if (i-1<0)		nn += Nx;			// periodic BC along the x-boundary
			f1 = phi[nn];						// get neighbor for phi - 1
			//........................................................................
			nn = n+1;							// neighbor index (get convention)
			if (!(i+1<Nx))	nn -= Nx;			// periodic BC along the x-boundary
			f2 = phi[nn];						// get neighbor for phi - 2
			//........................................................................
			nn = n-Nx;							// neighbor index (get convention)
			if (j-1<0)		nn += Nx*Ny;		// Perioidic BC along the y-boundary
			f3 = phi[nn];					// get neighbor for phi - 3
			//........................................................................
			nn = n+Nx;							// neighbor index (get convention)
			if (!(j+1<Ny))	nn -= Nx*Ny;		// Perioidic BC along the y-boundary
			f4 = phi[nn];					// get neighbor for phi - 4
			//........................................................................
			nn = n-Nx*Ny;						// neighbor index (get convention)
			if (k-1<0)		nn += Nx*Ny*Nz;		// Perioidic BC along the z-boundary
			f5 = phi[nn];					// get neighbor for phi - 5
			//........................................................................
			nn = n+Nx*Ny;						// neighbor index (get convention)
			if (!(k+1<Nz))	nn -= Nx*Ny*Nz;		// Perioidic BC along the z-boundary
			f6 = phi[nn];					// get neighbor for phi - 6
			//........................................................................
			nn = n-Nx-1;						// neighbor index (get convention)
			if (i-1<0)			nn += Nx;		// periodic BC along the x-boundary
			if (j-1<0)			nn += Nx*Ny;	// Perioidic BC along the y-boundary
			f7 = phi[nn];					// get neighbor for phi - 7
			//........................................................................
			nn = n+Nx+1;						// neighbor index (get convention)
			if (!(i+1<Nx))		nn -= Nx;		// periodic BC along the x-boundary
			if (!(j+1<Ny))		nn -= Nx*Ny;	// Perioidic BC along the y-boundary
			f8 = phi[nn];					// get neighbor for phi - 8
			//........................................................................
			nn = n+Nx-1;						// neighbor index (get convention)
			if (i-1<0)			nn += Nx;		// periodic BC along the x-boundary
			if (!(j+1<Ny))		nn -= Nx*Ny;	// Perioidic BC along the y-boundary
			f9 = phi[nn];					// get neighbor for phi - 9
			//........................................................................
			nn = n-Nx+1;						// neighbor index (get convention)
			if (!(i+1<Nx))		nn -= Nx;		// periodic BC along the x-boundary
			if (j-1<0)			nn += Nx*Ny;	// Perioidic BC along the y-boundary
			f10 = phi[nn];					// get neighbor for phi - 10
			//........................................................................
			nn = n-Nx*Ny-1;						// neighbor index (get convention)
			if (i-1<0)			nn += Nx;		// periodic BC along the x-boundary
			if (k-1<0)			nn += Nx*Ny*Nz;	// Perioidic BC along the z-boundary
			f11 = phi[nn];					// get neighbor for phi - 11
			//........................................................................
			nn = n+Nx*Ny+1;						// neighbor index (get convention)
			if (!(i+1<Nx))		nn -= Nx;		// periodic BC along the x-boundary
			if (!(k+1<Nz))	nn -= Nx*Ny*Nz;		// Perioidic BC along the z-boundary
			f12 = phi[nn];					// get neighbor for phi - 12
			//........................................................................
			nn = n+Nx*Ny-1;						// neighbor index (get convention)
			if (i-1<0)			nn += Nx;		// periodic BC along the x-boundary
			if (!(k+1<Nz))		nn -= Nx*Ny*Nz;	// Perioidic BC along the z-boundary
			f13 = phi[nn];					// get neighbor for phi - 13
			//........................................................................
			nn = n-Nx*Ny+1;						// neighbor index (get convention)
			if (!(i+1<Nx))		nn -= Nx;		// periodic BC along the x-boundary
			if (k-1<0)			nn += Nx*Ny*Nz;	// Perioidic BC along the z-boundary
			f14 = phi[nn];					// get neighbor for phi - 14
			//........................................................................
			nn = n-Nx*Ny-Nx;					// neighbor index (get convention)
			if (j-1<0)		nn += Nx*Ny;		// Perioidic BC along the y-boundary
			if (k-1<0)		nn += Nx*Ny*Nz;		// Perioidic BC along the z-boundary
			f15 = phi[nn];					// get neighbor for phi - 15
			//........................................................................
			nn = n+Nx*Ny+Nx;					// neighbor index (get convention)
			if (!(j+1<Ny))	nn -= Nx*Ny;		// Perioidic BC along the y-boundary
			if (!(k+1<Nz))	nn -= Nx*Ny*Nz;		// Perioidic BC along the z-boundary
			f16 = phi[nn];					// get neighbor for phi - 16
			//........................................................................
			nn = n+Nx*Ny-Nx;					// neighbor index (get convention)
			if (j-1<0)		nn += Nx*Ny;		// Perioidic BC along the y-boundary
			if (!(k+1<Nz))	nn -= Nx*Ny*Nz;		// Perioidic BC along the z-boundary
			f17 = phi[nn];					// get neighbor for phi - 17
			//........................................................................
			nn = n-Nx*Ny+Nx;					// neighbor index (get convention)
			if (!(j+1<Ny))	nn -= Nx*Ny;		// Perioidic BC along the y-boundary
			if (k-1<0)		nn += Nx*Ny*Nz;		// Perioidic BC along the z-boundary
			f18 = phi[nn];					// get neighbor for phi - 18
			//............Compute the Color Gradient...................................
			nx = -(f1-f2+0.5*(f7-f8+f9-f10+f11-f12+f13-f14));
			ny = -(f3-f4+0.5*(f7-f8-f9+f10+f15-f16+f17-f18));
			nz = -(f5-f6+0.5*(f11-f12-f13+f14+f15-f16-f17+f18));
			//...........Normalize the Color Gradient.................................
			//	C = sqrt(nx*nx+ny*ny+nz*nz);
			//	nx = nx/C;
			//	ny = ny/C;
			//	nz = nz/C;
			//...Store the Color Gradient....................
			ColorGrad[n] = nx;
			ColorGrad[N+n] = ny;
			ColorGrad[2*N+n] = nz;
			//...............................................
		}
	}
}
//*************************************************************************
__global__  void dvc_ColorCollide( char *ID, double *disteven, double *distodd, double *ColorGrad,
								double *Velocity, int Nx, int Ny, int Nz, double rlx_setA, double rlx_setB,
								double alpha, double beta, double Fx, double Fy, double Fz, bool pBC)
{

	int n,N;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;

	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
	// additional variables needed for computations
	double rho,jx,jy,jz,C,nx,ny,nz;

	N = Nx*Ny*Nz;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N && ID[n] > 0){

			// Retrieve the color gradient
			nx = ColorGrad[n];
			ny = ColorGrad[N+n];
			nz = ColorGrad[2*N+n];
			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			nx = nx/C;
			ny = ny/C;
			nz = nz/C;
			//......No color gradient at z-boundary if pressure BC are set.............
			//	if (pBC && k==0) nx = ny = nz = 0.f;
			//	if (pBC && k==Nz-1) nx = ny = nz = 0.f;
			//........................................................................
			//					READ THE DISTRIBUTIONS
			//		(read from opposite array due to previous swap operation)
			//........................................................................
			f2 = distodd[n];
			f4 = distodd[N+n];
			f6 = distodd[2*N+n];
			f8 = distodd[3*N+n];
			f10 = distodd[4*N+n];
			f12 = distodd[5*N+n];
			f14 = distodd[6*N+n];
			f16 = distodd[7*N+n];
			f18 = distodd[8*N+n];
			//........................................................................
			f0 = disteven[n];
			f1 = disteven[N+n];
			f3 = disteven[2*N+n];
			f5 = disteven[3*N+n];
			f7 = disteven[4*N+n];
			f9 = disteven[5*N+n];
			f11 = disteven[6*N+n];
			f13 = disteven[7*N+n];
			f15 = disteven[8*N+n];
			f17 = disteven[9*N+n];
			//........................................................................
			//					PERFORM RELAXATION PROCESS
			//........................................................................
			//....................compute the moments...............................................
			rho = f0+f2+f1+f4+f3+f6+f5+f8+f7+f10+f9+f12+f11+f14+f13+f16+f15+f18+f17;
			m1 = -30*f0-11*(f2+f1+f4+f3+f6+f5)+8*(f8+f7+f10+f9+f12+f11+f14+f13+f16+f15+f18 +f17);
			m2 = 12*f0-4*(f2+f1 +f4+f3+f6 +f5)+f8+f7+f10+f9+f12+f11+f14+f13+f16+f15+f18+f17;
			jx = f1-f2+f7-f8+f9-f10+f11-f12+f13-f14;
			m4 = 4*(-f1+f2)+f7-f8+f9-f10+f11-f12+f13-f14;
			jy = f3-f4+f7-f8-f9+f10+f15-f16+f17-f18;
			m6 = -4*(f3-f4)+f7-f8-f9+f10+f15-f16+f17-f18;
			jz = f5-f6+f11-f12-f13+f14+f15-f16-f17+f18;
			m8 = -4*(f5-f6)+f11-f12-f13+f14+f15-f16-f17+f18;
			m9 = 2*(f1+f2)-f3-f4-f5-f6+f7+f8+f9+f10+f11+f12+f13+f14-2*(f15+f16+f17+f18);
			m10 = -4*(f1+f2)+2*(f4+f3+f6+f5)+f8+f7+f10+f9+f12+f11+f14+f13-2*(f16+f15+f18+f17);
			m11 = f4+f3-f6-f5+f8+f7+f10+f9-f12-f11-f14-f13;
			m12 = -2*(f4+f3-f6-f5)+f8+f7+f10+f9-f12-f11-f14-f13;
			m13 = f8+f7-f10-f9;
			m14 = f16+f15-f18-f17;
			m15 = f12+f11-f14-f13;
			m16 = f7-f8+f9-f10-f11+f12-f13+f14;
			m17 = -f7+f8+f9-f10+f15-f16+f17-f18;
			m18 = f11-f12-f13+f14-f15+f16+f17-f18;
			//..........Toelke, Fruediger et. al. 2006...............
			if (C == 0.0)	nx = ny = nz = 1.0;
			m1 = m1 + rlx_setA*((19*(jx*jx+jy*jy+jz*jz)/rho - 11*rho) -alpha*C - m1);
			m2 = m2 + rlx_setA*((3*rho - 5.5*(jx*jx+jy*jy+jz*jz)/rho)- m2);
			m4 = m4 + rlx_setB*((-0.6666666666666666*jx)- m4);
			m6 = m6 + rlx_setB*((-0.6666666666666666*jy)- m6);
			m8 = m8 + rlx_setB*((-0.6666666666666666*jz)- m8);
			m9 = m9 + rlx_setA*(((2*jx*jx-jy*jy-jz*jz)/rho) + 0.5*alpha*C*(2*nx*nx-ny*ny-nz*nz) - m9);
			m10 = m10 + rlx_setA*(-0.5*((2*jx*jx-jy*jy-jz*jz)/rho) - m10);
			m11 = m11 + rlx_setA*(((jy*jy-jz*jz)/rho) + 0.5*alpha*C*(ny*ny-nz*nz)- m11);
			m12 = m12 + rlx_setA*( -0.5*((jy*jy-jz*jz)/rho) - m12);
			m13 = m13 + rlx_setA*( (jx*jy/rho) + 0.5*alpha*C*nx*ny - m13);
			m14 = m14 + rlx_setA*( (jy*jz/rho) + 0.5*alpha*C*ny*nz - m14);
			m15 = m15 + rlx_setA*( (jx*jz/rho) + 0.5*alpha*C*nx*nz - m15);
			m16 = m16 + rlx_setB*( - m16);
			m17 = m17 + rlx_setB*( - m17);
			m18 = m18 + rlx_setB*( - m18);
			//.................inverse transformation......................................................
			f0 = 0.05263157894736842*rho-0.012531328320802*m1+0.04761904761904762*m2;
			f1 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
					+0.1*(jx-m4)+0.0555555555555555555555555*(m9-m10);
			f2 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
					+0.1*(m4-jx)+0.0555555555555555555555555*(m9-m10);
			f3 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
					+0.1*(jy-m6)+0.02777777777777778*(m10-m9)+0.08333333333333333*(m11-m12);
			f4 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
					+0.1*(m6-jy)+0.02777777777777778*(m10-m9)+0.08333333333333333*(m11-m12);
			f5 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
					+0.1*(jz-m8)+0.02777777777777778*(m10-m9)+0.08333333333333333*(m12-m11);
			f6 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
					+0.1*(m8-jz)+0.02777777777777778*(m10-m9)+0.08333333333333333*(m12-m11);
			f7 = 0.05263157894736842*rho+0.003341687552213868*m1+0.003968253968253968*m2+0.1*(jx+jy)+0.025*(m4+m6)
						+0.02777777777777778*m9+0.01388888888888889*m10+0.08333333333333333*m11
						+0.04166666666666666*m12+0.25*m13+0.125*(m16-m17);
			f8 = 0.05263157894736842*rho+0.003341687552213868*m1+0.003968253968253968*m2-0.1*(jx+jy)-0.025*(m4+m6)
						+0.02777777777777778*m9+0.01388888888888889*m10+0.08333333333333333*m11
						+0.04166666666666666*m12+0.25*m13+0.125*(m17-m16);
			f9 = 0.05263157894736842*rho+0.003341687552213868*m1+0.003968253968253968*m2+0.1*(jx-jy)+0.025*(m4-m6)
						+0.02777777777777778*m9+0.01388888888888889*m10+0.08333333333333333*m11
						+0.04166666666666666*m12-0.25*m13+0.125*(m16+m17);
			f10 = 0.05263157894736842*rho+0.003341687552213868*m1+0.003968253968253968*m2+0.1*(jy-jx)+0.025*(m6-m4)
						+0.02777777777777778*m9+0.01388888888888889*m10+0.08333333333333333*m11
						+0.04166666666666666*m12-0.25*m13-0.125*(m16+m17);
			f11 = 0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2+0.1*(jx+jz)+0.025*(m4+m8)
					+0.02777777777777778*m9+0.01388888888888889*m10-0.08333333333333333*m11
					-0.04166666666666666*m12+0.25*m15+0.125*(m18-m16);
			f12 = 0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2-0.1*(jx+jz)-0.025*(m4+m8)
					+0.02777777777777778*m9+0.01388888888888889*m10-0.08333333333333333*m11
					-0.04166666666666666*m12+0.25*m15+0.125*(m16-m18);
			f13 = 0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2+0.1*(jx-jz)+0.025*(m4-m8)
					+0.02777777777777778*m9+0.01388888888888889*m10-0.08333333333333333*m11
					-0.04166666666666666*m12-0.25*m15-0.125*(m16+m18);
			f14 = 0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2+0.1*(jz-jx)+0.025*(m8-m4)
					+0.02777777777777778*m9+0.01388888888888889*m10-0.08333333333333333*m11
					-0.04166666666666666*m12-0.25*m15+0.125*(m16+m18);
			f15 = 0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2+0.1*(jy+jz)+0.025*(m6+m8)
					-0.0555555555555555555555555*m9-0.02777777777777778*m10+0.25*m14+0.125*(m17-m18);
			f16 =  0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2-0.1*(jy+jz)-0.025*(m6+m8)
					-0.0555555555555555555555555*m9-0.02777777777777778*m10+0.25*m14+0.125*(m18-m17);
			f17 = 0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2+0.1*(jy-jz)+0.025*(m6-m8)
					-0.0555555555555555555555555*m9-0.02777777777777778*m10-0.25*m14+0.125*(m17+m18);
			f18 = 0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2+0.1*(jz-jy)+0.025*(m8-m6)
					-0.0555555555555555555555555*m9-0.02777777777777778*m10-0.25*m14-0.125*(m17+m18);
			//.......................................................................................................
			// incorporate external force
			f1 += 0.16666666*Fx;
			f2 -= 0.16666666*Fx;
			f3 += 0.16666666*Fy;
			f4 -= 0.16666666*Fy;
			f5 += 0.16666666*Fz;
			f6 -= 0.16666666*Fz;
			f7 += 0.08333333333*(Fx+Fy);
			f8 -= 0.08333333333*(Fx+Fy);
			f9 += 0.08333333333*(Fx-Fy);
			f10 -= 0.08333333333*(Fx-Fy);
			f11 += 0.08333333333*(Fx+Fz);
			f12 -= 0.08333333333*(Fx+Fz);
			f13 += 0.08333333333*(Fx-Fz);
			f14 -= 0.08333333333*(Fx-Fz);
			f15 += 0.08333333333*(Fy+Fz);
			f16 -= 0.08333333333*(Fy+Fz);
			f17 += 0.08333333333*(Fy-Fz);
			f18 -= 0.08333333333*(Fy-Fz);
			//*********** WRITE UPDATED VALUES TO MEMORY ******************
			// Write the updated distributions
			//....EVEN.....................................
			disteven[n] = f0;
			disteven[N+n] = f2;
			disteven[2*N+n] = f4;
			disteven[3*N+n] = f6;
			disteven[4*N+n] = f8;
			disteven[5*N+n] = f10;
			disteven[6*N+n] = f12;
			disteven[7*N+n] = f14;
			disteven[8*N+n] = f16;
			disteven[9*N+n] = f18;
			//....ODD......................................
			distodd[n] = f1;
			distodd[N+n] = f3;
			distodd[2*N+n] = f5;
			distodd[3*N+n] = f7;
			distodd[4*N+n] = f9;
			distodd[5*N+n] = f11;
			distodd[6*N+n] = f13;
			distodd[7*N+n] = f15;
			distodd[8*N+n] = f17;

			//...Store the Velocity..........................
			Velocity[n] = jx;
			Velocity[N+n] = jy;
			Velocity[2*N+n] = jz;
		/*	Velocity[3*n] = jx;
			Velocity[3*n+1] = jy;
			Velocity[3*n+2] = jz;
		*/	//...Store the Color Gradient....................
			//			ColorGrad[3*n] = nx*C;
			//			ColorGrad[3*n+1] = ny*C;
			//			ColorGrad[3*n+2] = nz*C;
			//...............................................
			//***************************************************************
		}	// check if n is in the solid
	} // loop over n
}

__global__  void 
__launch_bounds__(512,2)
dvc_ScaLBL_D3Q19_ColorCollide( char *ID, double *disteven, double *distodd, double *phi, double *ColorGrad,
								double *Velocity, int Nx, int Ny, int Nz, double rlx_setA, double rlx_setB, 
								double alpha, double beta, double Fx, double Fy, double Fz)
{
		
	int i,j,k,n,nn,N;
	// distributions
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;

	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
	// additional variables needed for computations
	double rho,jx,jy,jz,C,nx,ny,nz;
	char id;

	N = Nx*Ny*Nz;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N) {
			id = ID[n];
			if ( id > 0){

			//.......Back out the 3-D indices for node n..............
			k = n/(Nx*Ny);
			j = (n-Nx*Ny*k)/Nx;
			i = n-Nx*Ny*k-Nx*j;
			//........................................................................
			//........Get 1-D index for this thread....................
			//		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
			//........................................................................
			//					COMPUTE THE COLOR GRADIENT
			//........................................................................
			//.................Read Phase Indicator Values............................
			//........................................................................
			nn = n-1;							// neighbor index (get convention)
			if (i-1<0)		nn += Nx;			// periodic BC along the x-boundary
			f1 = phi[nn];						// get neighbor for phi - 1
			//........................................................................
			nn = n+1;							// neighbor index (get convention)
			if (!(i+1<Nx))	nn -= Nx;			// periodic BC along the x-boundary
			f2 = phi[nn];						// get neighbor for phi - 2
			//........................................................................
			nn = n-Nx;							// neighbor index (get convention)
			if (j-1<0)		nn += Nx*Ny;		// Perioidic BC along the y-boundary
			f3 = phi[nn];					// get neighbor for phi - 3
			//........................................................................
			nn = n+Nx;							// neighbor index (get convention)
			if (!(j+1<Ny))	nn -= Nx*Ny;		// Perioidic BC along the y-boundary
			f4 = phi[nn];					// get neighbor for phi - 4
			//........................................................................
			nn = n-Nx*Ny;						// neighbor index (get convention)
			if (k-1<0)		nn += Nx*Ny*Nz;		// Perioidic BC along the z-boundary
			f5 = phi[nn];					// get neighbor for phi - 5
			//........................................................................
			nn = n+Nx*Ny;						// neighbor index (get convention)
			if (!(k+1<Nz))	nn -= Nx*Ny*Nz;		// Perioidic BC along the z-boundary
			f6 = phi[nn];					// get neighbor for phi - 6
			//........................................................................
			nn = n-Nx-1;						// neighbor index (get convention)
			if (i-1<0)			nn += Nx;		// periodic BC along the x-boundary
			if (j-1<0)			nn += Nx*Ny;	// Perioidic BC along the y-boundary
			f7 = phi[nn];					// get neighbor for phi - 7
			//........................................................................
			nn = n+Nx+1;						// neighbor index (get convention)
			if (!(i+1<Nx))		nn -= Nx;		// periodic BC along the x-boundary
			if (!(j+1<Ny))		nn -= Nx*Ny;	// Perioidic BC along the y-boundary
			f8 = phi[nn];					// get neighbor for phi - 8
			//........................................................................
			nn = n+Nx-1;						// neighbor index (get convention)
			if (i-1<0)			nn += Nx;		// periodic BC along the x-boundary
			if (!(j+1<Ny))		nn -= Nx*Ny;	// Perioidic BC along the y-boundary
			f9 = phi[nn];					// get neighbor for phi - 9
			//........................................................................
			nn = n-Nx+1;						// neighbor index (get convention)
			if (!(i+1<Nx))		nn -= Nx;		// periodic BC along the x-boundary
			if (j-1<0)			nn += Nx*Ny;	// Perioidic BC along the y-boundary
			f10 = phi[nn];					// get neighbor for phi - 10
			//........................................................................
			nn = n-Nx*Ny-1;						// neighbor index (get convention)
			if (i-1<0)			nn += Nx;		// periodic BC along the x-boundary
			if (k-1<0)			nn += Nx*Ny*Nz;	// Perioidic BC along the z-boundary
			f11 = phi[nn];					// get neighbor for phi - 11
			//........................................................................
			nn = n+Nx*Ny+1;						// neighbor index (get convention)
			if (!(i+1<Nx))		nn -= Nx;		// periodic BC along the x-boundary
			if (!(k+1<Nz))		nn -= Nx*Ny*Nz;		// Perioidic BC along the z-boundary
			f12 = phi[nn];					// get neighbor for phi - 12
			//........................................................................
			nn = n+Nx*Ny-1;						// neighbor index (get convention)
			if (i-1<0)			nn += Nx;		// periodic BC along the x-boundary
			if (!(k+1<Nz))		nn -= Nx*Ny*Nz;	// Perioidic BC along the z-boundary
			f13 = phi[nn];					// get neighbor for phi - 13
			//........................................................................
			nn = n-Nx*Ny+1;						// neighbor index (get convention)
			if (!(i+1<Nx))		nn -= Nx;		// periodic BC along the x-boundary
			if (k-1<0)			nn += Nx*Ny*Nz;	// Perioidic BC along the z-boundary
			f14 = phi[nn];					// get neighbor for phi - 14
			//........................................................................
			nn = n-Nx*Ny-Nx;					// neighbor index (get convention)
			if (j-1<0)		nn += Nx*Ny;		// Perioidic BC along the y-boundary
			if (k-1<0)		nn += Nx*Ny*Nz;		// Perioidic BC along the z-boundary
			f15 = phi[nn];					// get neighbor for phi - 15
			//........................................................................
			nn = n+Nx*Ny+Nx;					// neighbor index (get convention)
			if (!(j+1<Ny))	nn -= Nx*Ny;		// Perioidic BC along the y-boundary
			if (!(k+1<Nz))	nn -= Nx*Ny*Nz;		// Perioidic BC along the z-boundary
			f16 = phi[nn];					// get neighbor for phi - 16
			//........................................................................
			nn = n+Nx*Ny-Nx;					// neighbor index (get convention)
			if (j-1<0)		nn += Nx*Ny;		// Perioidic BC along the y-boundary
			if (!(k+1<Nz))	nn -= Nx*Ny*Nz;		// Perioidic BC along the z-boundary
			f17 = phi[nn];					// get neighbor for phi - 17
			//........................................................................
			nn = n-Nx*Ny+Nx;					// neighbor index (get convention)
			if (!(j+1<Ny))	nn -= Nx*Ny;		// Perioidic BC along the y-boundary
			if (k-1<0)		nn += Nx*Ny*Nz;		// Perioidic BC along the z-boundary
			f18 = phi[nn];					// get neighbor for phi - 18
			//............Compute the Color Gradient...................................
			nx = -(f1-f2+0.5*(f7-f8+f9-f10+f11-f12+f13-f14));
			ny = -(f3-f4+0.5*(f7-f8-f9+f10+f15-f16+f17-f18));
			nz = -(f5-f6+0.5*(f11-f12-f13+f14+f15-f16-f17+f18));
			//...Store the Color Gradient....................
			ColorGrad[n] = nx;
			ColorGrad[N+n] = ny;
			ColorGrad[2*N+n] = nz;
			//...............................................
			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			if (C == 0.0) C=1.0;
			nx = nx/C;
			ny = ny/C;
			nz = nz/C;
			//......No color gradient at z-boundary if pressure BC are set.............
			//	if (pBC && k==0) nx = ny = nz = 0.f;
			//	if (pBC && k==Nz-1) nx = ny = nz = 0.f;
			//........................................................................
			//					READ THE DISTRIBUTIONS
			//		(read from opposite array due to previous swap operation)
			//........................................................................
			f2 = distodd[n];
			f4 = distodd[N+n];
			f6 = distodd[2*N+n];
			f0 = disteven[n];
			f1 = disteven[N+n];
			f3 = disteven[2*N+n];
			f5 = disteven[3*N+n];
			//........................................................................
			//....................compute the moments...............................................
			rho = f0+f2+f1+f4+f3+f6+f5;
			m1 = -30*f0-11*(f2+f1+f4+f3+f6+f5);
			m2 = 12*f0-4*(f2+f1 +f4+f3+f6 +f5);
			jx = f1-f2;
			m4 = 4*(-f1+f2);
			jy = f3-f4;
			m6 = -4*(f3-f4);
			jz = f5-f6;
			m8 = -4*(f5-f6);
			m9 = 2*(f1+f2)-f3-f4-f5-f6;
			m10 = -4*(f1+f2)+2*(f4+f3+f6+f5);
			m11 = f4+f3-f6-f5;
			m12 = -2*(f4+f3-f6-f5);
			//........................................................................
			f8 = distodd[3*N+n];
			f10 = distodd[4*N+n];
			f7 = disteven[4*N+n];
			f9 = disteven[5*N+n];
			//........................................................................
			rho += f8+f7+f10+f9;
			m1 += 8*(f8+f7+f10+f9);
			m2 += f8+f7+f10+f9;
			jx += f7-f8+f9-f10;
			m4 += f7-f8+f9-f10;
			jy += f7-f8-f9+f10;
			m6 += f7-f8-f9+f10;
			m9 += f7+f8+f9+f10;
			m10 += f8+f7+f10+f9;
			m11 += f8+f7+f10+f9;
			m12 += f8+f7+f10+f9;
			m13 = f8+f7-f10-f9;
			m16 = f7-f8+f9-f10;
			m17 = -f7+f8+f9-f10;
			//........................................................................
			f11 = disteven[6*N+n];
			f13 = disteven[7*N+n];
			f12 = distodd[5*N+n];
			f14 = distodd[6*N+n];
			//........................................................................
			//........................................................................
			f15 = disteven[8*N+n];
			f17 = disteven[9*N+n];
			f16 = distodd[7*N+n];
			f18 = distodd[8*N+n];
			//........................................................................
			//....................compute the moments...............................................
			rho += f12+f11+f14+f13+f16+f15+f18+f17;
			m1 += 8*(f12+f11+f14+f13+f16+f15+f18+f17);
			m2 += f12+f11+f14+f13+f16+f15+f18+f17;
			jx += f11-f12+f13-f14;
			m4 += f11-f12+f13-f14;
			jy += f15-f16+f17-f18;
			m6 += f15-f16+f17-f18;
			jz += f11-f12-f13+f14+f15-f16-f17+f18;
			m8 += f11-f12-f13+f14+f15-f16-f17+f18;
			m9 += f11+f12+f13+f14-2*(f15+f16+f17+f18);
			m10 += f12+f11+f14+f13-2*(f16+f15+f18+f17);
			m11 += -f12-f11-f14-f13;
			m12 += -f12-f11-f14-f13;
			m14 = f16+f15-f18-f17;
			m15 = f12+f11-f14-f13;
			m16 += -f11+f12-f13+f14;
			m17 += f15-f16+f17-f18;
			m18 = f11-f12-f13+f14-f15+f16+f17-f18;
			//........................................................................

			/*				f2 = distodd[n];
				f4 = distodd[N+n];
				f6 = distodd[2*N+n];
				f8 = distodd[3*N+n];
				//........................................................................
				f0 = disteven[n];
				f1 = disteven[N+n];
				f3 = disteven[2*N+n];
				f5 = disteven[3*N+n];
				f7 = disteven[4*N+n];
				//........................................................................
				//........................................................................
				//....................compute the moments...............................................
				rho = f0+f2+f1+f4+f3+f6+f5+f8+f7;
				m1 = -30*f0-11*(f2+f1+f4+f3+f6+f5)+8*(f8+f7);
				m2 = 12*f0-4*(f2+f1 +f4+f3+f6 +f5)+f8+f7;
				jx = f1-f2+f7-f8;
				m4 = 4*(-f1+f2)+f7-f8;
				jy = f3-f4+f7-f8;
				m6 = -4*(f3-f4)+f7-f8;
				jz = f5-f6;
				m8 = -4*(f5-f6);
				m9 = 2*(f1+f2)-f3-f4-f5-f6+f7+f8;
				m10 = -4*(f1+f2)+2*(f4+f3+f6+f5)+f8+f7;
				m11 = f4+f3-f6-f5+f8+f7;
				m12 = -2*(f4+f3-f6-f5)+f8+f7;
				m13 = f8+f7;
				m16 = f7-f8;
				m17 = -f7+f8;
				//........................................................................
				f9 = disteven[5*N+n];
				f11 = disteven[6*N+n];
				f13 = disteven[7*N+n];
				f15 = disteven[8*N+n];
				f17 = disteven[9*N+n];
				f10 = distodd[4*N+n];
				f12 = distodd[5*N+n];
				f14 = distodd[6*N+n];
				f16 = distodd[7*N+n];
				f18 = distodd[8*N+n];
				//........................................................................
				rho += f10+f9+f12+f11+f14+f13+f16+f15+f18+f17;
				m1 += 8*(f10+f9+f12+f11+f14+f13+f16+f15+f18 +f17);
				m2 += f10+f9+f12+f11+f14+f13+f16+f15+f18+f17;
				jx += f9-f10+f11-f12+f13-f14;
				m4 += f9-f10+f11-f12+f13-f14;
				jy += -f9+f10+f15-f16+f17-f18;
				m6 += -f9+f10+f15-f16+f17-f18;
				jz += f11-f12-f13+f14+f15-f16-f17+f18;
				m8 += f11-f12-f13+f14+f15-f16-f17+f18;
				m9 += f9+f10+f11+f12+f13+f14-2*(f15+f16+f17+f18);
				m10 += f10+f9+f12+f11+f14+f13-2*(f16+f15+f18+f17);
				m11 += f10+f9-f12-f11-f14-f13;
				m12 += f10+f9-f12-f11-f14-f13;
				m13 += -f10-f9;
				m14 = f16+f15-f18-f17;
				m15 = f12+f11-f14-f13;
				m16 += f9-f10-f11+f12-f13+f14;
				m17 += f9-f10+f15-f16+f17-f18;
				m18 = f11-f12-f13+f14-f15+f16+f17-f18;
			 */			//........................................................................
			//					PERFORM RELAXATION PROCESS
			//........................................................................
			//..........Toelke, Fruediger et. al. 2006...............
			if (C == 0.0)	nx = ny = nz = 0.0;
			m1 = m1 + rlx_setA*((19*(jx*jx+jy*jy+jz*jz)/rho - 11*rho) -alpha*C - m1);
			m2 = m2 + rlx_setA*((3*rho - 5.5*(jx*jx+jy*jy+jz*jz)/rho)- m2);
			m4 = m4 + rlx_setB*((-0.6666666666666666*jx)- m4);
			m6 = m6 + rlx_setB*((-0.6666666666666666*jy)- m6);
			m8 = m8 + rlx_setB*((-0.6666666666666666*jz)- m8);
			m9 = m9 + rlx_setA*(((2*jx*jx-jy*jy-jz*jz)/rho) + 0.5*alpha*C*(2*nx*nx-ny*ny-nz*nz) - m9);
			m10 = m10 + rlx_setA*( - m10);
			m11 = m11 + rlx_setA*(((jy*jy-jz*jz)/rho) + 0.5*alpha*C*(ny*ny-nz*nz)- m11);
			m12 = m12 + rlx_setA*( - m12);
			m13 = m13 + rlx_setA*( (jx*jy/rho) + 0.5*alpha*C*nx*ny - m13);
			m14 = m14 + rlx_setA*( (jy*jz/rho) + 0.5*alpha*C*ny*nz - m14);
			m15 = m15 + rlx_setA*( (jx*jz/rho) + 0.5*alpha*C*nx*nz - m15);
			m16 = m16 + rlx_setB*( - m16);
			m17 = m17 + rlx_setB*( - m17);
			m18 = m18 + rlx_setB*( - m18);
			//.................inverse transformation......................................................
			f0 = 0.05263157894736842*rho-0.012531328320802*m1+0.04761904761904762*m2;
			f1 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
					+0.1*(jx-m4)+0.0555555555555555555555555*(m9-m10);
			f2 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
					+0.1*(m4-jx)+0.0555555555555555555555555*(m9-m10);
			f3 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
					+0.1*(jy-m6)+0.02777777777777778*(m10-m9)+0.08333333333333333*(m11-m12);
			f4 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
					+0.1*(m6-jy)+0.02777777777777778*(m10-m9)+0.08333333333333333*(m11-m12);
			f5 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
					+0.1*(jz-m8)+0.02777777777777778*(m10-m9)+0.08333333333333333*(m12-m11);
			f6 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
					+0.1*(m8-jz)+0.02777777777777778*(m10-m9)+0.08333333333333333*(m12-m11);
			f7 = 0.05263157894736842*rho+0.003341687552213868*m1+0.003968253968253968*m2+0.1*(jx+jy)+0.025*(m4+m6)
						+0.02777777777777778*m9+0.01388888888888889*m10+0.08333333333333333*m11
						+0.04166666666666666*m12+0.25*m13+0.125*(m16-m17);
			f8 = 0.05263157894736842*rho+0.003341687552213868*m1+0.003968253968253968*m2-0.1*(jx+jy)-0.025*(m4+m6)
						+0.02777777777777778*m9+0.01388888888888889*m10+0.08333333333333333*m11
						+0.04166666666666666*m12+0.25*m13+0.125*(m17-m16);
			f9 = 0.05263157894736842*rho+0.003341687552213868*m1+0.003968253968253968*m2+0.1*(jx-jy)+0.025*(m4-m6)
						+0.02777777777777778*m9+0.01388888888888889*m10+0.08333333333333333*m11
						+0.04166666666666666*m12-0.25*m13+0.125*(m16+m17);
			f10 = 0.05263157894736842*rho+0.003341687552213868*m1+0.003968253968253968*m2+0.1*(jy-jx)+0.025*(m6-m4)
						+0.02777777777777778*m9+0.01388888888888889*m10+0.08333333333333333*m11
						+0.04166666666666666*m12-0.25*m13-0.125*(m16+m17);
			f11 = 0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2+0.1*(jx+jz)+0.025*(m4+m8)
					+0.02777777777777778*m9+0.01388888888888889*m10-0.08333333333333333*m11
					-0.04166666666666666*m12+0.25*m15+0.125*(m18-m16);
			f12 = 0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2-0.1*(jx+jz)-0.025*(m4+m8)
					+0.02777777777777778*m9+0.01388888888888889*m10-0.08333333333333333*m11
					-0.04166666666666666*m12+0.25*m15+0.125*(m16-m18);
			f13 = 0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2+0.1*(jx-jz)+0.025*(m4-m8)
					+0.02777777777777778*m9+0.01388888888888889*m10-0.08333333333333333*m11
					-0.04166666666666666*m12-0.25*m15-0.125*(m16+m18);
			f14 = 0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2+0.1*(jz-jx)+0.025*(m8-m4)
					+0.02777777777777778*m9+0.01388888888888889*m10-0.08333333333333333*m11
					-0.04166666666666666*m12-0.25*m15+0.125*(m16+m18);
			f15 = 0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2+0.1*(jy+jz)+0.025*(m6+m8)
					-0.0555555555555555555555555*m9-0.02777777777777778*m10+0.25*m14+0.125*(m17-m18);
			f16 =  0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2-0.1*(jy+jz)-0.025*(m6+m8)
					-0.0555555555555555555555555*m9-0.02777777777777778*m10+0.25*m14+0.125*(m18-m17);
			f17 = 0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2+0.1*(jy-jz)+0.025*(m6-m8)
					-0.0555555555555555555555555*m9-0.02777777777777778*m10-0.25*m14+0.125*(m17+m18);
			f18 = 0.05263157894736842*rho+0.003341687552213868*m1
					+0.003968253968253968*m2+0.1*(jz-jy)+0.025*(m8-m6)
					-0.0555555555555555555555555*m9-0.02777777777777778*m10-0.25*m14-0.125*(m17+m18);
			//.......................................................................................................
			// incorporate external force
			f1 += 0.16666666*Fx;
			f2 -= 0.16666666*Fx;
			f3 += 0.16666666*Fy;
			f4 -= 0.16666666*Fy;
			f5 += 0.16666666*Fz;
			f6 -= 0.16666666*Fz;
			f7 += 0.08333333333*(Fx+Fy);
			f8 -= 0.08333333333*(Fx+Fy);
			f9 += 0.08333333333*(Fx-Fy);
			f10 -= 0.08333333333*(Fx-Fy);
			f11 += 0.08333333333*(Fx+Fz);
			f12 -= 0.08333333333*(Fx+Fz);
			f13 += 0.08333333333*(Fx-Fz);
			f14 -= 0.08333333333*(Fx-Fz);
			f15 += 0.08333333333*(Fy+Fz);
			f16 -= 0.08333333333*(Fy+Fz);
			f17 += 0.08333333333*(Fy-Fz);
			f18 -= 0.08333333333*(Fy-Fz);
			//*********** WRITE UPDATED VALUES TO MEMORY ******************
			// Write the updated distributions
			//....EVEN.....................................
			disteven[n] = f0;
			disteven[N+n] = f2;
			disteven[2*N+n] = f4;
			disteven[3*N+n] = f6;
			disteven[4*N+n] = f8;
			disteven[5*N+n] = f10;
			disteven[6*N+n] = f12;
			disteven[7*N+n] = f14;
			disteven[8*N+n] = f16;
			disteven[9*N+n] = f18;
			//....ODD......................................
			distodd[n] = f1;
			distodd[N+n] = f3;
			distodd[2*N+n] = f5;
			distodd[3*N+n] = f7;
			distodd[4*N+n] = f9;
			distodd[5*N+n] = f11;
			distodd[6*N+n] = f13;
			distodd[7*N+n] = f15;
			distodd[8*N+n] = f17;
			//...Store the Velocity..........................
			Velocity[n] = jx;
			Velocity[N+n] = jy;
			Velocity[2*N+n] = jz;
			//***************************************************************

			}// check if n is in the solid
			}
	} // loop over n
}

__global__  void dvc_ScaLBL_D3Q7_ColorCollideMass(char *ID, double *A_even, double *A_odd, double *B_even, double *B_odd, 
		double *Den, double *Phi, double *ColorGrad, double *Velocity, double beta, int N, bool pBC)
{
	int n;
	double f0,f1,f2,f3,f4,f5,f6;
	double na,nb,nab;	// density values
	double ux,uy,uz;	// flow velocity
	double nx,ny,nz,C;	// color gradient components
	double a1,a2,b1,b2;
	double delta;
	char id;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;

		if (n<N)
			id = ID[n];
			if ( id != 0){

            //.....Load the Color gradient.........
            nx = ColorGrad[n];
            ny = ColorGrad[N+n];
            nz = ColorGrad[2*N+n];
            C = sqrt(nx*nx+ny*ny+nz*nz);
            if (C==0.0) C=1.0;
            nx = nx/C;
            ny = ny/C;
            nz = nz/C;
            //....Load the flow velocity...........
            ux = Velocity[n];
            uy = Velocity[N+n];
            uz = Velocity[2*N+n];
            //........................................................................
            //					READ THE DISTRIBUTIONS
            //		(read from opposite array due to previous swap operation)
            //........................................................................
            f2 = A_odd[n];
            f4 = A_odd[N+n];
            f6 = A_odd[2*N+n];
            f0 = A_even[n];
            f1 = A_even[N+n];
            f3 = A_even[2*N+n];
            f5 = A_even[3*N+n];
            na = f0+f1+f2+f3+f4+f5+f6;
            //........................................................................
            f2 = B_odd[n];
            f4 = B_odd[N+n];
            f6 = B_odd[2*N+n];
            f0 = B_even[n];
            f1 = B_even[N+n];
            f3 = B_even[2*N+n];
            f5 = B_even[3*N+n];
            nb = f0+f1+f2+f3+f4+f5+f6;
            nab = 1.0/(na+nb);
            //........................................................................
            //....Instantiate the density distributions
            // Generate Equilibrium Distributions and stream
            // Stationary value - distribution 0
            A_even[n] = 0.3333333333333333*na;
            B_even[n] = 0.3333333333333333*nb;
            // Non-Stationary equilibrium distributions
            //feq[0] = 0.1111111111111111*(1+4.5*ux);
            //feq[1] = 0.1111111111111111*(1-4.5*ux);
            //feq[2] = 0.1111111111111111*(1+4.5*uy);
            //feq[3] = 0.1111111111111111*(1-4.5*uy);
            //feq[4] = 0.1111111111111111*(1+4.5*uz);
            //feq[5] = 0.1111111111111111*(1-4.5*uz);

            //...............................................
            // q = 0,2,4
            // Cq = {1,0,0}, {0,1,0}, {0,0,1}
            delta = beta*na*nb*nab*0.1111111111111111*nx;
            if (!(na*nb*nab>0)) delta=0;
            a1 = na*(0.1111111111111111*(1+4.5*ux))+delta;
            b1 = nb*(0.1111111111111111*(1+4.5*ux))-delta;
            a2 = na*(0.1111111111111111*(1-4.5*ux))-delta;
            b2 = nb*(0.1111111111111111*(1-4.5*ux))+delta;

            A_odd[n] 	= a1;
            A_even[N+n] = a2;
            B_odd[n] 	= b1;
            B_even[N+n] = b2;
            //...............................................
            // q = 2
            // Cq = {0,1,0}
            delta = beta*na*nb*nab*0.1111111111111111*ny;
            if (!(na*nb*nab>0)) delta=0;
            a1 = na*(0.1111111111111111*(1+4.5*uy))+delta;
            b1 = nb*(0.1111111111111111*(1+4.5*uy))-delta;
            a2 = na*(0.1111111111111111*(1-4.5*uy))-delta;
            b2 = nb*(0.1111111111111111*(1-4.5*uy))+delta;

            A_odd[N+n] 	= a1;
            A_even[2*N+n] = a2;
            B_odd[N+n] 	= b1;
            B_even[2*N+n] = b2;
            //...............................................
            // q = 4
            // Cq = {0,0,1}
            delta = beta*na*nb*nab*0.1111111111111111*nz;
            if (!(na*nb*nab>0)) delta=0;
            a1 = na*(0.1111111111111111*(1+4.5*uz))+delta;
            b1 = nb*(0.1111111111111111*(1+4.5*uz))-delta;
            a2 = na*(0.1111111111111111*(1-4.5*uz))-delta;
            b2 = nb*(0.1111111111111111*(1-4.5*uz))+delta;

            A_odd[2*N+n] = a1;
            A_even[3*N+n] = a2;
            B_odd[2*N+n] = b1;
            B_even[3*N+n] = b2;

		}
	}
}

//*************************************************************************
__global__  void dvc_DensityStreamD3Q7(char *ID, double *Den, double *Copy, double *Phi, double *ColorGrad, double *Velocity,
		double beta, int Nx, int Ny, int Nz, bool pBC)
{
	char id;

	int idx;
	int in,jn,kn,n,nn,N;
	int q,Cqx,Cqy,Cqz;
	//	int sendLoc;

	double na,nb;		// density values
	double ux,uy,uz;	// flow velocity
	double nx,ny,nz,C;	// color gradient components
	double a1,a2,b1,b2;
	double sp,delta;
	double feq[6];		// equilibrium distributions
	// Set of Discrete velocities for the D3Q19 Model
	int D3Q7[3][3]={{1,0,0},{0,1,0},{0,0,1}};
	N = Nx*Ny*Nz;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N ){
			// Local Density Values
			na = Copy[2*n];
			nb = Copy[2*n+1];
			if (id > 0 && na+nb > 0.0){
				//.......Back out the 3-D indices for node n..............
				int	k = n/(Nx*Ny);
				int j = (n-Nx*Ny*k)/Nx;
				int i = n-Nx*Ny*k-Nx*j;
				//.....Load the Color gradient.........
				nx = ColorGrad[n];
				ny = ColorGrad[N+n];
				nz = ColorGrad[2*N+n];
				C = sqrt(nx*nx+ny*ny+nz*nz);
				if (C == 0.0) C=1.0;
				nx = nx/C;
				ny = ny/C;
				nz = nz/C;
				//....Load the flow velocity...........
				ux = Velocity[n];
				uy = Velocity[N+n];
				uz = Velocity[2*N+n];
				//....Instantiate the density distributions
				// Generate Equilibrium Distributions and stream
				// Stationary value - distribution 0
				//			Den[2*n] += 0.3333333333333333*na;
				//			Den[2*n+1] += 0.3333333333333333*nb;
				Den[2*n] += 0.3333333333333333*na;
				Den[2*n+1] += 0.3333333333333333*nb;
				// Non-Stationary equilibrium distributions
				feq[0] = 0.1111111111111111*(1+3*ux);
				feq[1] = 0.1111111111111111*(1-3*ux);
				feq[2] = 0.1111111111111111*(1+3*uy);
				feq[3] = 0.1111111111111111*(1-3*uy);
				feq[4] = 0.1111111111111111*(1+3*uz);
				feq[5] = 0.1111111111111111*(1-3*uz);
				// Construction and streaming for the components
				for (idx=0; idx<3; idx++){
					// Distribution index
					q = 2*idx;
					// Associated discrete velocity
					Cqx = D3Q7[idx][0];
					Cqy = D3Q7[idx][1];
					Cqz = D3Q7[idx][2];
					// Generate the Equilibrium Distribution
					a1 = na*feq[q];
					b1 = nb*feq[q];
					a2 = na*feq[q+1];
					b2 = nb*feq[q+1];
					// Recolor the distributions
					if (C > 0.0){
						sp = nx*double(Cqx)+ny*double(Cqy)+nz*double(Cqz);
						//if (idx > 2)	sp = 0.7071067811865475*sp;
						//delta = sp*min( min(a1,a2), min(b1,b2) );
						delta = na*nb/(na+nb)*0.1111111111111111*sp;
						//if (a1>0 && b1>0){
						a1 += beta*delta;
						a2 -= beta*delta;
						b1 -= beta*delta;
						b2 += beta*delta;
					}

					// .......Get the neighbor node..............
					//nn = n + Stride[idx];
					in = i+Cqx;
					jn = j+Cqy;
					kn = k+Cqz;

					// Adjust for periodic BC, if necessary
					//				if (in<0) in+= Nx;
					//				if (jn<0) jn+= Ny;
					//				if (kn<0) kn+= Nz;
					//				if (!(in<Nx)) in-= Nx;
					//				if (!(jn<Ny)) jn-= Ny;
					//				if (!(kn<Nz)) kn-= Nz;
					// Perform streaming or bounce-back as needed
					id = ID[kn*Nx*Ny+jn*Nx+in];
					if (id == 0){							//.....Bounce-back Rule...........
						//						Den[2*n] += a1;
						//						Den[2*n+1] += b1;
						Den[2*n] += a1;
						Den[2*n+1] += b1;
					}
					else{
						//......Push the "distribution" to neighboring node...........
						// Index of the neighbor in the local process
						//nn = (kn-zmin[rank]+1)*Nxp*Nyp + (jn-ymin[rank]+1)*Nxp + (in-xmin[rank]+1);
						nn = kn*Nx*Ny+jn*Nx+in;
						// Push to neighboring node
						//						Den[2*nn] += a1;
						//						Den[2*nn+1] += b1;
						Den[2*nn] += a1;
						Den[2*nn+1] += b1;
					}

					// .......Get the neighbor node..............
					q = 2*idx+1;
					in = i-Cqx;
					jn = j-Cqy;
					kn = k-Cqz;
					// Adjust for periodic BC, if necessary
					//				if (in<0) in+= Nx;
					//				if (jn<0) jn+= Ny;
					//				if (kn<0) kn+= Nz;
					//				if (!(in<Nx)) in-= Nx;
					//				if (!(jn<Ny)) jn-= Ny;
					//				if (!(kn<Nz)) kn-= Nz;
					// Perform streaming or bounce-back as needed
					id = ID[kn*Nx*Ny+jn*Nx+in];
					if (id == 0){
						//.....Bounce-back Rule...........
						//						Den[2*n] += a2;
						//					Den[2*n+1] += b2;
						Den[2*n] += a2;
						Den[2*n+1] += b2;
					}
					else{
						//......Push the "distribution" to neighboring node...........
						// Index of the neighbor in the local process
						//nn = (kn-zmin[rank]+1)*Nxp*Nyp + (jn-ymin[rank]+1)*Nxp + (in-xmin[rank]+1);
						nn = kn*Nx*Ny+jn*Nx+in;
						// Push to neighboring node
						//					Den[2*nn] += a2;
						//					Den[2*nn+1] += b2;
						Den[2*nn] += a2;
						Den[2*nn+1] += b2;
					}
				}
			}
		}
	}
}

__global__  void dvc_ScaLBL_ComputePhaseField(char *ID, double *Phi, double *Den, int N)
{
	int n;
	double Na,Nb;
	//...................................................................
	// Update Phi
	char id;
	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<N){
		   id=ID[n];
		   if (id > 0){
			// Get the density value (Streaming already performed)
			Na = Den[n];
			Nb = Den[N+n];
			Phi[n] = (Na-Nb)/(Na+Nb);
			}
		}
	}
	//...................................................................
}

__global__  void dvc_ScaLBL_SetSlice_z(double *Phi, double value, int Nx, int Ny, int Nz, int Slice)
{
	int n = Slice*Nx*Ny +  blockIdx.x*blockDim.x + threadIdx.x;
	if (n < (Slice+1)*Nx*Ny){
		Phi[n] = value;
	}
}



__global__  void dvc_ScaLBL_D3Q19_AAeven_Color(int *Map, double *dist, double *Aq, double *Bq, double *Den, double *Phi,
		double *Velocity, double rhoA, double rhoB, double tauA, double tauB, double alpha, double beta,
		double Fx, double Fy, double Fz, int strideY, int strideZ, int start, int finish, int Np){
	int ijk,nn,n;
	double fq;
	// conserved momemnts
	double rho,jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
	double m3,m5,m7;
	double nA,nB; // number density
	double a1,b1,a2,b2,nAB,delta;
	double C,nx,ny,nz; //color gradient magnitude and direction
	double ux,uy,uz;
	double phi,tau,rho0,rlx_setA,rlx_setB;

	const double mrt_V1=0.05263157894736842;
	const double mrt_V2=0.012531328320802;
	const double mrt_V3=0.04761904761904762;
	const double mrt_V4=0.004594820384294068;
	const double mrt_V5=0.01587301587301587;
	const double mrt_V6=0.0555555555555555555555555;
	const double mrt_V7=0.02777777777777778;
	const double mrt_V8=0.08333333333333333;
	const double mrt_V9=0.003341687552213868;
	const double mrt_V10=0.003968253968253968;
	const double mrt_V11=0.01388888888888889;
	const double mrt_V12=0.04166666666666666;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {

			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];

			// compute phase indicator field
			phi=(nA-nB)/(nA+nB);

			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
			// local relaxation time
			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
			rlx_setA = 1.f/tau;
			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);

			// Get the 1D index based on regular data layout
			ijk = Map[n];
			//					COMPUTE THE COLOR GRADIENT
			//........................................................................
			//.................Read Phase Indicator Values............................
			//........................................................................
			nn = ijk-1;							// neighbor index (get convention)
			m1 = Phi[nn];						// get neighbor for phi - 1
			//........................................................................
			nn = ijk+1;							// neighbor index (get convention)
			m2 = Phi[nn];						// get neighbor for phi - 2
			//........................................................................
			nn = ijk-strideY;							// neighbor index (get convention)
			m3 = Phi[nn];					// get neighbor for phi - 3
			//........................................................................
			nn = ijk+strideY;							// neighbor index (get convention)
			m4 = Phi[nn];					// get neighbor for phi - 4
			//........................................................................
			nn = ijk-strideZ;						// neighbor index (get convention)
			m5 = Phi[nn];					// get neighbor for phi - 5
			//........................................................................
			nn = ijk+strideZ;						// neighbor index (get convention)
			m6 = Phi[nn];					// get neighbor for phi - 6
			//........................................................................
			nn = ijk-strideY-1;						// neighbor index (get convention)
			m7 = Phi[nn];					// get neighbor for phi - 7
			//........................................................................
			nn = ijk+strideY+1;						// neighbor index (get convention)
			m8 = Phi[nn];					// get neighbor for phi - 8
			//........................................................................
			nn = ijk+strideY-1;						// neighbor index (get convention)
			m9 = Phi[nn];					// get neighbor for phi - 9
			//........................................................................
			nn = ijk-strideY+1;						// neighbor index (get convention)
			m10 = Phi[nn];					// get neighbor for phi - 10
			//........................................................................
			nn = ijk-strideZ-1;						// neighbor index (get convention)
			m11 = Phi[nn];					// get neighbor for phi - 11
			//........................................................................
			nn = ijk+strideZ+1;						// neighbor index (get convention)
			m12 = Phi[nn];					// get neighbor for phi - 12
			//........................................................................
			nn = ijk+strideZ-1;						// neighbor index (get convention)
			m13 = Phi[nn];					// get neighbor for phi - 13
			//........................................................................
			nn = ijk-strideZ+1;						// neighbor index (get convention)
			m14 = Phi[nn];					// get neighbor for phi - 14
			//........................................................................
			nn = ijk-strideZ-strideY;					// neighbor index (get convention)
			m15 = Phi[nn];					// get neighbor for phi - 15
			//........................................................................
			nn = ijk+strideZ+strideY;					// neighbor index (get convention)
			m16 = Phi[nn];					// get neighbor for phi - 16
			//........................................................................
			nn = ijk+strideZ-strideY;					// neighbor index (get convention)
			m17 = Phi[nn];					// get neighbor for phi - 17
			//........................................................................
			nn = ijk-strideZ+strideY;					// neighbor index (get convention)
			m18 = Phi[nn];					// get neighbor for phi - 18
			//............Compute the Color Gradient...................................
			nx = -(m1-m2+0.5*(m7-m8+m9-m10+m11-m12+m13-m14));
			ny = -(m3-m4+0.5*(m7-m8-m9+m10+m15-m16+m17-m18));
			nz = -(m5-m6+0.5*(m11-m12-m13+m14+m15-m16-m17+m18));

			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			double ColorMag = C;
			if (C==0.0) ColorMag=1.0;
			nx = nx/ColorMag;
			ny = ny/ColorMag;
			nz = nz/ColorMag;		

			// q=0
			fq = dist[n];
			rho = fq;
			m1  = -30.0*fq;
			m2  = 12.0*fq;

			// q=1
			fq = dist[2*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jx = fq;
			m4 = -4.0*fq;
			m9 = 2.0*fq;
			m10 = -4.0*fq;

			// f2 = dist[10*Np+n];
			fq = dist[1*Np+n];
			rho += fq;
			m1 -= 11.0*(fq);
			m2 -= 4.0*(fq);
			jx -= fq;
			m4 += 4.0*(fq);
			m9 += 2.0*(fq);
			m10 -= 4.0*(fq);

			// q=3
			fq = dist[4*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy = fq;
			m6 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 = fq;
			m12 = -2.0*fq;

			// q = 4
			fq = dist[3*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy -= fq;
			m6 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 += fq;
			m12 -= 2.0*fq;

			// q=5
			fq = dist[6*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz = fq;
			m8 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q = 6
			fq = dist[5*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz -= fq;
			m8 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q=7
			fq = dist[8*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy += fq;
			m6 += fq;
			m9  += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 = fq;
			m16 = fq;
			m17 = -fq;

			// q = 8
			fq = dist[7*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 += fq;
			m16 -= fq;
			m17 += fq;

			// q=9
			fq = dist[10*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 += fq;
			m17 += fq;

			// q = 10
			fq = dist[9*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy += fq;
			m6 += fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 -= fq;
			m17 -= fq;

			// q=11
			fq = dist[12*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 = fq;
			m16 -= fq;
			m18 = fq;

			// q=12
			fq = dist[11*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 += fq;
			m16 += fq;
			m18 -= fq;

			// q=13
			fq = dist[14*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 -= fq;
			m18 -= fq;

			// q=14
			fq = dist[13*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 += fq;
			m18 += fq;

			// q=15
			fq = dist[16*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 = fq;
			m17 += fq;
			m18 -= fq;

			// q=16
			fq = dist[15*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 += fq;
			m17 -= fq;
			m18 += fq;

			// q=17
			fq = dist[18*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 += fq;
			m18 += fq;

			// q=18
			fq = dist[17*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 -= fq;
			m18 -= fq;

			//........................................................................
			//..............carry out relaxation process..............................
			//..........Toelke, Fruediger et. al. 2006................................
			if (C == 0.0)	nx = ny = nz = 0.0;
			m1 = m1 + rlx_setA*((19*(jx*jx+jy*jy+jz*jz)/rho0 - 11*rho) -19*alpha*C - m1);
			m2 = m2 + rlx_setA*((3*rho - 5.5*(jx*jx+jy*jy+jz*jz)/rho0)- m2);
			m4 = m4 + rlx_setB*((-0.6666666666666666*jx)- m4);
			m6 = m6 + rlx_setB*((-0.6666666666666666*jy)- m6);
			m8 = m8 + rlx_setB*((-0.6666666666666666*jz)- m8);
			m9 = m9 + rlx_setA*(((2*jx*jx-jy*jy-jz*jz)/rho0) + 0.5*alpha*C*(2*nx*nx-ny*ny-nz*nz) - m9);
			m10 = m10 + rlx_setA*( - m10);
			m11 = m11 + rlx_setA*(((jy*jy-jz*jz)/rho0) + 0.5*alpha*C*(ny*ny-nz*nz)- m11);
			m12 = m12 + rlx_setA*( - m12);
			m13 = m13 + rlx_setA*( (jx*jy/rho0) + 0.5*alpha*C*nx*ny - m13);
			m14 = m14 + rlx_setA*( (jy*jz/rho0) + 0.5*alpha*C*ny*nz - m14);
			m15 = m15 + rlx_setA*( (jx*jz/rho0) + 0.5*alpha*C*nx*nz - m15);
			m16 = m16 + rlx_setB*( - m16);
			m17 = m17 + rlx_setB*( - m17);
			m18 = m18 + rlx_setB*( - m18);

			//.......................................................................................................
			//.................inverse transformation......................................................

			// q=0
			fq = mrt_V1*rho-mrt_V2*m1+mrt_V3*m2;
			dist[n] = fq;

			// q = 1
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10) + 0.16666666*Fx;
			dist[1*Np+n] = fq;

			// q=2
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10) -  0.16666666*Fx;
			dist[2*Np+n] = fq;

			// q = 3
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) + 0.16666666*Fy;
			dist[3*Np+n] = fq;

			// q = 4
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) - 0.16666666*Fy;
			dist[4*Np+n] = fq;

			// q = 5
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) + 0.16666666*Fz;
			dist[5*Np+n] = fq;

			// q = 6
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) - 0.16666666*Fz;
			dist[6*Np+n] = fq;

			// q = 7
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17) + 0.08333333333*(Fx+Fy);
			dist[7*Np+n] = fq;


			// q = 8
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11
					+mrt_V12*m12+0.25*m13+0.125*(m17-m16) - 0.08333333333*(Fx+Fy);
			dist[8*Np+n] = fq;

			// q = 9
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17) + 0.08333333333*(Fx-Fy);
			dist[9*Np+n] = fq;

			// q = 10
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17)- 0.08333333333*(Fx-Fy);
			dist[10*Np+n] = fq;


			// q = 11
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12+0.25*m15+0.125*(m18-m16) + 0.08333333333*(Fx+Fz);
			dist[11*Np+n] = fq;

			// q = 12
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+
					mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18)-0.08333333333*(Fx+Fz);
			dist[12*Np+n] = fq;

			// q = 13
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15-0.125*(m16+m18) + 0.08333333333*(Fx-Fz);
			dist[13*Np+n] = fq;

			// q= 14
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15+0.125*(m16+m18) - 0.08333333333*(Fx-Fz);

			dist[14*Np+n] = fq;

			// q = 15
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18) + 0.08333333333*(Fy+Fz);
			dist[15*Np+n] = fq;

			// q = 16
			fq =  mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17)- 0.08333333333*(Fy+Fz);
			dist[16*Np+n] = fq;


			// q = 17
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18) + 0.08333333333*(Fy-Fz);
			dist[17*Np+n] = fq;

			// q = 18
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18) - 0.08333333333*(Fy-Fz);
			dist[18*Np+n] = fq;

			//........................................................................

			// write the velocity 
			ux = jx / rho0;
			uy = jy / rho0;
			uz = jz / rho0;
			Velocity[n] = ux;
			Velocity[Np+n] = uy;
			Velocity[2*Np+n] = uz;

			// Instantiate mass transport distributions
			// Stationary value - distribution 0

			nAB = 1.0/(nA+nB);
			Aq[n] = 0.3333333333333333*nA;
			Bq[n] = 0.3333333333333333*nB;

			//...............................................
			// q = 0,2,4
			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nx;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*ux))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*ux))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*ux))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*ux))+delta;

			Aq[1*Np+n] = a1;
			Bq[1*Np+n] = b1;
			Aq[2*Np+n] = a2;
			Bq[2*Np+n] = b2;

			//...............................................
			// q = 2
			// Cq = {0,1,0}
			delta = beta*nA*nB*nAB*0.1111111111111111*ny;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uy))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uy))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uy))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uy))+delta;

			Aq[3*Np+n] = a1;
			Bq[3*Np+n] = b1;
			Aq[4*Np+n] = a2;
			Bq[4*Np+n] = b2;
			//...............................................
			// q = 4
			// Cq = {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nz;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uz))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uz))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uz))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uz))+delta;

			Aq[5*Np+n] = a1;
			Bq[5*Np+n] = b1;
			Aq[6*Np+n] = a2;
			Bq[6*Np+n] = b2;
			//...............................................

		}
	}
}


__global__ void dvc_ScaLBL_D3Q19_AAodd_Color(int *neighborList, int *Map, double *dist, double *Aq, double *Bq, double *Den,
		 double *Phi, double *Velocity, double rhoA, double rhoB, double tauA, double tauB, double alpha, double beta,
		double Fx, double Fy, double Fz, int strideY, int strideZ, int start, int finish, int Np){

	int n,nn,ijk,nread;
	int nr1,nr2,nr3,nr4,nr5,nr6;
	int nr7,nr8,nr9,nr10;
	int nr11,nr12,nr13,nr14;
	//int nr15,nr16,nr17,nr18;
	double fq;
	// conserved momemnts
	double rho,jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
	double m3,m5,m7;
	double nA,nB; // number density
	double a1,b1,a2,b2,nAB,delta;
	double C,nx,ny,nz; //color gradient magnitude and direction
	double ux,uy,uz;
	double phi,tau,rho0,rlx_setA,rlx_setB;

	const double mrt_V1=0.05263157894736842;
	const double mrt_V2=0.012531328320802;
	const double mrt_V3=0.04761904761904762;
	const double mrt_V4=0.004594820384294068;
	const double mrt_V5=0.01587301587301587;
	const double mrt_V6=0.0555555555555555555555555;
	const double mrt_V7=0.02777777777777778;
	const double mrt_V8=0.08333333333333333;
	const double mrt_V9=0.003341687552213868;
	const double mrt_V10=0.003968253968253968;
	const double mrt_V11=0.01388888888888889;
	const double mrt_V12=0.04166666666666666;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];

			// compute phase indicator field
			phi=(nA-nB)/(nA+nB);

			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
			// local relaxation time
			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
			rlx_setA = 1.f/tau;
			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);
			
			// Get the 1D index based on regular data layout
			ijk = Map[n];
			//					COMPUTE THE COLOR GRADIENT
			//........................................................................
			//.................Read Phase Indicator Values............................
			//........................................................................
			nn = ijk-1;							// neighbor index (get convention)
			m1 = Phi[nn];						// get neighbor for phi - 1
			//........................................................................
			nn = ijk+1;							// neighbor index (get convention)
			m2 = Phi[nn];						// get neighbor for phi - 2
			//........................................................................
			nn = ijk-strideY;							// neighbor index (get convention)
			m3 = Phi[nn];					// get neighbor for phi - 3
			//........................................................................
			nn = ijk+strideY;							// neighbor index (get convention)
			m4 = Phi[nn];					// get neighbor for phi - 4
			//........................................................................
			nn = ijk-strideZ;						// neighbor index (get convention)
			m5 = Phi[nn];					// get neighbor for phi - 5
			//........................................................................
			nn = ijk+strideZ;						// neighbor index (get convention)
			m6 = Phi[nn];					// get neighbor for phi - 6
			//........................................................................
			nn = ijk-strideY-1;						// neighbor index (get convention)
			m7 = Phi[nn];					// get neighbor for phi - 7
			//........................................................................
			nn = ijk+strideY+1;						// neighbor index (get convention)
			m8 = Phi[nn];					// get neighbor for phi - 8
			//........................................................................
			nn = ijk+strideY-1;						// neighbor index (get convention)
			m9 = Phi[nn];					// get neighbor for phi - 9
			//........................................................................
			nn = ijk-strideY+1;						// neighbor index (get convention)
			m10 = Phi[nn];					// get neighbor for phi - 10
			//........................................................................
			nn = ijk-strideZ-1;						// neighbor index (get convention)
			m11 = Phi[nn];					// get neighbor for phi - 11
			//........................................................................
			nn = ijk+strideZ+1;						// neighbor index (get convention)
			m12 = Phi[nn];					// get neighbor for phi - 12
			//........................................................................
			nn = ijk+strideZ-1;						// neighbor index (get convention)
			m13 = Phi[nn];					// get neighbor for phi - 13
			//........................................................................
			nn = ijk-strideZ+1;						// neighbor index (get convention)
			m14 = Phi[nn];					// get neighbor for phi - 14
			//........................................................................
			nn = ijk-strideZ-strideY;					// neighbor index (get convention)
			m15 = Phi[nn];					// get neighbor for phi - 15
			//........................................................................
			nn = ijk+strideZ+strideY;					// neighbor index (get convention)
			m16 = Phi[nn];					// get neighbor for phi - 16
			//........................................................................
			nn = ijk+strideZ-strideY;					// neighbor index (get convention)
			m17 = Phi[nn];					// get neighbor for phi - 17
			//........................................................................
			nn = ijk-strideZ+strideY;					// neighbor index (get convention)
			m18 = Phi[nn];					// get neighbor for phi - 18
			//............Compute the Color Gradient...................................
			nx = -(m1-m2+0.5*(m7-m8+m9-m10+m11-m12+m13-m14));
			ny = -(m3-m4+0.5*(m7-m8-m9+m10+m15-m16+m17-m18));
			nz = -(m5-m6+0.5*(m11-m12-m13+m14+m15-m16-m17+m18));

			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			double ColorMag = C;
			if (C==0.0) ColorMag=1.0;
			nx = nx/ColorMag;
			ny = ny/ColorMag;
			nz = nz/ColorMag;		

			// q=0
			fq = dist[n];
			rho = fq;
			m1  = -30.0*fq;
			m2  = 12.0*fq;

			// q=1
			//nread = neighborList[n]; // neighbor 2 
			//fq = dist[nread]; // reading the f1 data into register fq		
			nr1 = neighborList[n]; 
			fq = dist[nr1]; // reading the f1 data into register fq
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jx = fq;
			m4 = -4.0*fq;
			m9 = 2.0*fq;
			m10 = -4.0*fq;

			// f2 = dist[10*Np+n];
			//nread = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
			//fq = dist[nread];  // reading the f2 data into register fq
			nr2 = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
			fq = dist[nr2];  // reading the f2 data into register fq
			rho += fq;
			m1 -= 11.0*(fq);
			m2 -= 4.0*(fq);
			jx -= fq;
			m4 += 4.0*(fq);
			m9 += 2.0*(fq);
			m10 -= 4.0*(fq);

			// q=3
			//nread = neighborList[n+2*Np]; // neighbor 4
			//fq = dist[nread];
			nr3 = neighborList[n+2*Np]; // neighbor 4
			fq = dist[nr3];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy = fq;
			m6 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 = fq;
			m12 = -2.0*fq;

			// q = 4
			//nread = neighborList[n+3*Np]; // neighbor 3
			//fq = dist[nread];
			nr4 = neighborList[n+3*Np]; // neighbor 3
			fq = dist[nr4];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy -= fq;
			m6 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 += fq;
			m12 -= 2.0*fq;

			// q=5
			//nread = neighborList[n+4*Np];
			//fq = dist[nread];
			nr5 = neighborList[n+4*Np];
			fq = dist[nr5];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz = fq;
			m8 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;


			// q = 6
			//nread = neighborList[n+5*Np];
			//fq = dist[nread];
			nr6 = neighborList[n+5*Np];
			fq = dist[nr6];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz -= fq;
			m8 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q=7
			//nread = neighborList[n+6*Np];
			//fq = dist[nread];
			nr7 = neighborList[n+6*Np];
			fq = dist[nr7];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy += fq;
			m6 += fq;
			m9  += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 = fq;
			m16 = fq;
			m17 = -fq;

			// q = 8
			//nread = neighborList[n+7*Np];
			//fq = dist[nread];
			nr8 = neighborList[n+7*Np];
			fq = dist[nr8];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 += fq;
			m16 -= fq;
			m17 += fq;

			// q=9
			//nread = neighborList[n+8*Np];
			//fq = dist[nread];
			nr9 = neighborList[n+8*Np];
			fq = dist[nr9];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 += fq;
			m17 += fq;

			// q = 10
			//nread = neighborList[n+9*Np];
			//fq = dist[nread];
			nr10 = neighborList[n+9*Np];
			fq = dist[nr10];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy += fq;
			m6 += fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 -= fq;
			m17 -= fq;

			// q=11
			//nread = neighborList[n+10*Np];
			//fq = dist[nread];
			nr11 = neighborList[n+10*Np];
			fq = dist[nr11];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 = fq;
			m16 -= fq;
			m18 = fq;

			// q=12
			//nread = neighborList[n+11*Np];
			//fq = dist[nread];
			nr12 = neighborList[n+11*Np];
			fq = dist[nr12];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 += fq;
			m16 += fq;
			m18 -= fq;

			// q=13
			//nread = neighborList[n+12*Np];
			//fq = dist[nread];
			nr13 = neighborList[n+12*Np];
			fq = dist[nr13];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 -= fq;
			m18 -= fq;

			// q=14
			//nread = neighborList[n+13*Np];
			//fq = dist[nread];
			nr14 = neighborList[n+13*Np];
			fq = dist[nr14];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 += fq;
			m18 += fq;

			// q=15
			nread = neighborList[n+14*Np];
			fq = dist[nread];
			//fq = dist[17*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 = fq;
			m17 += fq;
			m18 -= fq;

			// q=16
			nread = neighborList[n+15*Np];
			fq = dist[nread];
			//fq = dist[8*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 += fq;
			m17 -= fq;
			m18 += fq;

			// q=17
			//fq = dist[18*Np+n];
			nread = neighborList[n+16*Np];
			fq = dist[nread];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 += fq;
			m18 += fq;

			// q=18
			nread = neighborList[n+17*Np];
			fq = dist[nread];
			//fq = dist[9*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 -= fq;
			m18 -= fq;
			
			//........................................................................
			//..............carry out relaxation process..............................
			//..........Toelke, Fruediger et. al. 2006................................
			if (C == 0.0)	nx = ny = nz = 0.0;
			m1 = m1 + rlx_setA*((19*(jx*jx+jy*jy+jz*jz)/rho0 - 11*rho) -19*alpha*C - m1);
			m2 = m2 + rlx_setA*((3*rho - 5.5*(jx*jx+jy*jy+jz*jz)/rho0)- m2);
			m4 = m4 + rlx_setB*((-0.6666666666666666*jx)- m4);
			m6 = m6 + rlx_setB*((-0.6666666666666666*jy)- m6);
			m8 = m8 + rlx_setB*((-0.6666666666666666*jz)- m8);
			m9 = m9 + rlx_setA*(((2*jx*jx-jy*jy-jz*jz)/rho0) + 0.5*alpha*C*(2*nx*nx-ny*ny-nz*nz) - m9);
			m10 = m10 + rlx_setA*( - m10);
			m11 = m11 + rlx_setA*(((jy*jy-jz*jz)/rho0) + 0.5*alpha*C*(ny*ny-nz*nz)- m11);
			m12 = m12 + rlx_setA*( - m12);
			m13 = m13 + rlx_setA*( (jx*jy/rho0) + 0.5*alpha*C*nx*ny - m13);
			m14 = m14 + rlx_setA*( (jy*jz/rho0) + 0.5*alpha*C*ny*nz - m14);
			m15 = m15 + rlx_setA*( (jx*jz/rho0) + 0.5*alpha*C*nx*nz - m15);
			m16 = m16 + rlx_setB*( - m16);
			m17 = m17 + rlx_setB*( - m17);
			m18 = m18 + rlx_setB*( - m18);
			//.................inverse transformation......................................................

			// q=0
			fq = mrt_V1*rho-mrt_V2*m1+mrt_V3*m2;
			dist[n] = fq;

			// q = 1
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10)+0.16666666*Fx;
			//nread = neighborList[n+Np];
			dist[nr2] = fq;

			// q=2
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10) -  0.16666666*Fx;
			//nread = neighborList[n];
			dist[nr1] = fq;

			// q = 3
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) + 0.16666666*Fy;
			//nread = neighborList[n+3*Np];
			dist[nr4] = fq;

			// q = 4
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) - 0.16666666*Fy;
			//nread = neighborList[n+2*Np];
			dist[nr3] = fq;

			// q = 5
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) + 0.16666666*Fz;
			//nread = neighborList[n+5*Np];
			dist[nr6] = fq;

			// q = 6
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) - 0.16666666*Fz;
			//nread = neighborList[n+4*Np];
			dist[nr5] = fq;

			// q = 7
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17) + 0.08333333333*(Fx+Fy);
			//nread = neighborList[n+7*Np];
			dist[nr8] = fq;

			// q = 8
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11
					+mrt_V12*m12+0.25*m13+0.125*(m17-m16) - 0.08333333333*(Fx+Fy);
			//nread = neighborList[n+6*Np];
			dist[nr7] = fq;

			// q = 9
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17) + 0.08333333333*(Fx-Fy);
			//nread = neighborList[n+9*Np];
			dist[nr10] = fq;

			// q = 10
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17)- 0.08333333333*(Fx-Fy);
			//nread = neighborList[n+8*Np];
			dist[nr9] = fq;

			// q = 11
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12+0.25*m15+0.125*(m18-m16) + 0.08333333333*(Fx+Fz);
			//nread = neighborList[n+11*Np];
			dist[nr12] = fq;

			// q = 12
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+
					mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18) - 0.08333333333*(Fx+Fz);
			//nread = neighborList[n+10*Np];
			dist[nr11]= fq;

			// q = 13
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15-0.125*(m16+m18) + 0.08333333333*(Fx-Fz);
			//nread = neighborList[n+13*Np];
			dist[nr14] = fq;

			// q= 14
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15+0.125*(m16+m18) - 0.08333333333*(Fx-Fz);
			//nread = neighborList[n+12*Np];
			dist[nr13] = fq;


			// q = 15
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18) + 0.08333333333*(Fy+Fz);
			nread = neighborList[n+15*Np];
			dist[nread] = fq;

			// q = 16
			fq =  mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17)- 0.08333333333*(Fy+Fz);
			nread = neighborList[n+14*Np];
			dist[nread] = fq;


			// q = 17
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18) + 0.08333333333*(Fy-Fz);
			nread = neighborList[n+17*Np];
			dist[nread] = fq;

			// q = 18
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18) - 0.08333333333*(Fy-Fz);
			nread = neighborList[n+16*Np];
			dist[nread] = fq;

			// write the velocity 
			ux = jx / rho0;
			uy = jy / rho0;
			uz = jz / rho0;
			Velocity[n] = ux;
			Velocity[Np+n] = uy;
			Velocity[2*Np+n] = uz;

			// Instantiate mass transport distributions
			// Stationary value - distribution 0
			nAB = 1.0/(nA+nB);
			Aq[n] = 0.3333333333333333*nA;
			Bq[n] = 0.3333333333333333*nB;

			//...............................................
			// q = 0,2,4
			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nx;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*ux))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*ux))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*ux))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*ux))+delta;

			// q = 1
			//nread = neighborList[n+Np];
			Aq[nr2] = a1;
			Bq[nr2] = b1;
			// q=2
			//nread = neighborList[n];
			Aq[nr1] = a2;
			Bq[nr1] = b2;

			//...............................................
			// Cq = {0,1,0}
			delta = beta*nA*nB*nAB*0.1111111111111111*ny;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uy))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uy))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uy))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uy))+delta;

			// q = 3
			//nread = neighborList[n+3*Np];
			Aq[nr4] = a1;
			Bq[nr4] = b1;
			// q = 4
			//nread = neighborList[n+2*Np];
			Aq[nr3] = a2;
			Bq[nr3] = b2;

			//...............................................
			// q = 4
			// Cq = {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nz;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uz))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uz))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uz))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uz))+delta;

			// q = 5
			//nread = neighborList[n+5*Np];
			Aq[nr6] = a1;
			Bq[nr6] = b1;
			// q = 6
			//nread = neighborList[n+4*Np];
			Aq[nr5] = a2;
			Bq[nr5] = b2;
			//...............................................
		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_AAodd_ColorMomentum(int *neighborList, double *dist, double *Den,
		double *Velocity, double *ColorGrad, double rhoA, double rhoB, double tauA, double tauB, double alpha, double beta,
		double Fx, double Fy, double Fz, int start, int finish, int Np){

	int n,nread;
	double fq;
	// conserved momemnts
	double rho,jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
	double nA,nB; // number density
	double C,nx,ny,nz; //color gradient magnitude and direction
	double ux,uy,uz;
	double phi,tau,rho0,rlx_setA,rlx_setB;

	const double mrt_V1=0.05263157894736842;
	const double mrt_V2=0.012531328320802;
	const double mrt_V3=0.04761904761904762;
	const double mrt_V4=0.004594820384294068;
	const double mrt_V5=0.01587301587301587;
	const double mrt_V6=0.0555555555555555555555555;
	const double mrt_V7=0.02777777777777778;
	const double mrt_V8=0.08333333333333333;
	const double mrt_V9=0.003341687552213868;
	const double mrt_V10=0.003968253968253968;
	const double mrt_V11=0.01388888888888889;
	const double mrt_V12=0.04166666666666666;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];

			// compute phase indicator field
			phi=(nA-nB)/(nA+nB);

			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
			// local relaxation time
			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
			rlx_setA = 1.f/tau;
			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);

			// read the color gradient
			nx = ColorGrad[n];
			ny = ColorGrad[Np+n];
			nz = ColorGrad[2*Np+n];

			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			if (C==0.0) C=1.0;
			nx = nx/C;
			ny = ny/C;
			nz = nz/C;		

			// q=0
			fq = dist[n];
			rho = fq;
			m1  = -30.0*fq;
			m2  = 12.0*fq;

			// q=1
			nread = neighborList[n]; // neighbor 2 ( > 10Np => odd part of dist)
			fq = dist[nread]; // reading the f1 data into register fq
			//fp = dist[10*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jx = fq;
			m4 = -4.0*fq;
			m9 = 2.0*fq;
			m10 = -4.0*fq;

			// f2 = dist[10*Np+n];
			nread = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
			fq = dist[nread];  // reading the f2 data into register fq
			//fq = dist[Np+n];
			rho += fq;
			m1 -= 11.0*(fq);
			m2 -= 4.0*(fq);
			jx -= fq;
			m4 += 4.0*(fq);
			m9 += 2.0*(fq);
			m10 -= 4.0*(fq);

			// q=3
			nread = neighborList[n+2*Np]; // neighbor 4
			fq = dist[nread];
			//fq = dist[11*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy = fq;
			m6 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 = fq;
			m12 = -2.0*fq;

			// q = 4
			nread = neighborList[n+3*Np]; // neighbor 3
			fq = dist[nread];
			//fq = dist[2*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy -= fq;
			m6 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 += fq;
			m12 -= 2.0*fq;

			// q=5
			nread = neighborList[n+4*Np];
			fq = dist[nread];
			//fq = dist[12*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz = fq;
			m8 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;


			// q = 6
			nread = neighborList[n+5*Np];
			fq = dist[nread];
			//fq = dist[3*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz -= fq;
			m8 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q=7
			nread = neighborList[n+6*Np];
			fq = dist[nread];
			//fq = dist[13*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy += fq;
			m6 += fq;
			m9  += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 = fq;
			m16 = fq;
			m17 = -fq;

			// q = 8
			nread = neighborList[n+7*Np];
			fq = dist[nread];
			//fq = dist[4*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 += fq;
			m16 -= fq;
			m17 += fq;

			// q=9
			nread = neighborList[n+8*Np];
			fq = dist[nread];
			//fq = dist[14*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 += fq;
			m17 += fq;

			// q = 10
			nread = neighborList[n+9*Np];
			fq = dist[nread];
			//fq = dist[5*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy += fq;
			m6 += fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 -= fq;
			m17 -= fq;

			// q=11
			nread = neighborList[n+10*Np];
			fq = dist[nread];
			//fq = dist[15*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 = fq;
			m16 -= fq;
			m18 = fq;

			// q=12
			nread = neighborList[n+11*Np];
			fq = dist[nread];
			//fq = dist[6*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 += fq;
			m16 += fq;
			m18 -= fq;

			// q=13
			nread = neighborList[n+12*Np];
			fq = dist[nread];
			//fq = dist[16*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 -= fq;
			m18 -= fq;

			// q=14
			nread = neighborList[n+13*Np];
			fq = dist[nread];
			//fq = dist[7*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 += fq;
			m18 += fq;

			// q=15
			nread = neighborList[n+14*Np];
			fq = dist[nread];
			//fq = dist[17*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 = fq;
			m17 += fq;
			m18 -= fq;

			// q=16
			nread = neighborList[n+15*Np];
			fq = dist[nread];
			//fq = dist[8*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 += fq;
			m17 -= fq;
			m18 += fq;

			// q=17
			//fq = dist[18*Np+n];
			nread = neighborList[n+16*Np];
			fq = dist[nread];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 += fq;
			m18 += fq;

			// q=18
			nread = neighborList[n+17*Np];
			fq = dist[nread];
			//fq = dist[9*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 -= fq;
			m18 -= fq;

			//........................................................................
			//..............carry out relaxation process..............................
			//..........Toelke, Fruediger et. al. 2006................................
			if (C == 0.0)	nx = ny = nz = 0.0;
			m1 = m1 + rlx_setA*((19*(jx*jx+jy*jy+jz*jz)/rho0 - 11*rho) -alpha*C - m1);
			m2 = m2 + rlx_setA*((3*rho - 5.5*(jx*jx+jy*jy+jz*jz)/rho0)- m2);
			m4 = m4 + rlx_setB*((-0.6666666666666666*jx)- m4);
			m6 = m6 + rlx_setB*((-0.6666666666666666*jy)- m6);
			m8 = m8 + rlx_setB*((-0.6666666666666666*jz)- m8);
			m9 = m9 + rlx_setA*(((2*jx*jx-jy*jy-jz*jz)/rho0) + 0.5*alpha*C*(2*nx*nx-ny*ny-nz*nz) - m9);
			m10 = m10 + rlx_setA*( - m10);
			m11 = m11 + rlx_setA*(((jy*jy-jz*jz)/rho0) + 0.5*alpha*C*(ny*ny-nz*nz)- m11);
			m12 = m12 + rlx_setA*( - m12);
			m13 = m13 + rlx_setA*( (jx*jy/rho0) + 0.5*alpha*C*nx*ny - m13);
			m14 = m14 + rlx_setA*( (jy*jz/rho0) + 0.5*alpha*C*ny*nz - m14);
			m15 = m15 + rlx_setA*( (jx*jz/rho0) + 0.5*alpha*C*nx*nz - m15);
			m16 = m16 + rlx_setB*( - m16);
			m17 = m17 + rlx_setB*( - m17);
			m18 = m18 + rlx_setB*( - m18);
			//.................inverse transformation......................................................

			// q=0
			fq = mrt_V1*rho-mrt_V2*m1+mrt_V3*m2;
			dist[n] = fq;

			// q = 1
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10)+0.16666666*Fx;
			nread = neighborList[n+Np];
			dist[nread] = fq;

			// q=2
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10) -  0.16666666*Fx;
			nread = neighborList[n];
			dist[nread] = fq;

			// q = 3
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) + 0.16666666*Fy;
			nread = neighborList[n+3*Np];
			dist[nread] = fq;

			// q = 4
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) - 0.16666666*Fy;
			nread = neighborList[n+2*Np];
			dist[nread] = fq;

			// q = 5
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) + 0.16666666*Fz;
			nread = neighborList[n+5*Np];
			dist[nread] = fq;

			// q = 6
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) - 0.16666666*Fz;
			nread = neighborList[n+4*Np];
			dist[nread] = fq;

			// q = 7
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17) + 0.08333333333*(Fx+Fy);
			nread = neighborList[n+7*Np];
			dist[nread] = fq;

			// q = 8
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11
					+mrt_V12*m12+0.25*m13+0.125*(m17-m16) - 0.08333333333*(Fx+Fy);
			nread = neighborList[n+6*Np];
			dist[nread] = fq;

			// q = 9
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17) + 0.08333333333*(Fx-Fy);
			nread = neighborList[n+9*Np];
			dist[nread] = fq;

			// q = 10
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17)- 0.08333333333*(Fx-Fy);
			nread = neighborList[n+8*Np];
			dist[nread] = fq;

			// q = 11
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12+0.25*m15+0.125*(m18-m16) + 0.08333333333*(Fx+Fz);
			nread = neighborList[n+11*Np];
			dist[nread] = fq;

			// q = 12
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+
					mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18) - 0.08333333333*(Fx+Fz);
			nread = neighborList[n+10*Np];
			dist[nread]= fq;

			// q = 13
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15-0.125*(m16+m18) + 0.08333333333*(Fx-Fz);
			nread = neighborList[n+13*Np];
			dist[nread] = fq;

			// q= 14
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15+0.125*(m16+m18) - 0.08333333333*(Fx-Fz);
			nread = neighborList[n+12*Np];
			dist[nread] = fq;


			// q = 15
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18) + 0.08333333333*(Fy+Fz);
			nread = neighborList[n+15*Np];
			dist[nread] = fq;

			// q = 16
			fq =  mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17)- 0.08333333333*(Fy+Fz);
			nread = neighborList[n+14*Np];
			dist[nread] = fq;


			// q = 17
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18) + 0.08333333333*(Fy-Fz);
			nread = neighborList[n+17*Np];
			dist[nread] = fq;

			// q = 18
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18) - 0.08333333333*(Fy-Fz);
			nread = neighborList[n+16*Np];
			dist[nread] = fq;

			// write the velocity 
			ux = jx / rho0;
			uy = jy / rho0;
			uz = jz / rho0;
			Velocity[n] = ux;
			Velocity[Np+n] = uy;
			Velocity[2*Np+n] = uz;
		}
	}
}

__global__  void dvc_ScaLBL_D3Q19_AAeven_ColorMomentum(double *dist, double *Den, double *Velocity,
		double *ColorGrad, double rhoA, double rhoB, double tauA, double tauB, double alpha, double beta,
		double Fx, double Fy, double Fz, int start, int finish, int Np){
	int n;
	double fq;
	// conserved momemnts
	double rho,jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
	double nA,nB; // number density
	double C,nx,ny,nz; //color gradient magnitude and direction
	double ux,uy,uz;
	double phi,tau,rho0,rlx_setA,rlx_setB;

	const double mrt_V1=0.05263157894736842;
	const double mrt_V2=0.012531328320802;
	const double mrt_V3=0.04761904761904762;
	const double mrt_V4=0.004594820384294068;
	const double mrt_V5=0.01587301587301587;
	const double mrt_V6=0.0555555555555555555555555;
	const double mrt_V7=0.02777777777777778;
	const double mrt_V8=0.08333333333333333;
	const double mrt_V9=0.003341687552213868;
	const double mrt_V10=0.003968253968253968;
	const double mrt_V11=0.01388888888888889;
	const double mrt_V12=0.04166666666666666;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {

			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];

			// compute phase indicator field
			phi=(nA-nB)/(nA+nB);

			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
			// local relaxation time
			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
			rlx_setA = 1.f/tau;
			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);

			// read the color gradient
			nx = ColorGrad[n];
			ny = ColorGrad[Np+n];
			nz = ColorGrad[2*Np+n];

			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			if (C==0.0) C=1.0;
			nx = nx/C;
			ny = ny/C;
			nz = nz/C;		

			// q=0
			fq = dist[n];
			rho = fq;
			m1  = -30.0*fq;
			m2  = 12.0*fq;

			// q=1
			fq = dist[2*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jx = fq;
			m4 = -4.0*fq;
			m9 = 2.0*fq;
			m10 = -4.0*fq;

			// f2 = dist[10*Np+n];
			fq = dist[1*Np+n];
			rho += fq;
			m1 -= 11.0*(fq);
			m2 -= 4.0*(fq);
			jx -= fq;
			m4 += 4.0*(fq);
			m9 += 2.0*(fq);
			m10 -= 4.0*(fq);

			// q=3
			fq = dist[4*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy = fq;
			m6 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 = fq;
			m12 = -2.0*fq;

			// q = 4
			fq = dist[3*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy -= fq;
			m6 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 += fq;
			m12 -= 2.0*fq;

			// q=5
			fq = dist[6*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz = fq;
			m8 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q = 6
			fq = dist[5*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz -= fq;
			m8 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q=7
			fq = dist[8*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy += fq;
			m6 += fq;
			m9  += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 = fq;
			m16 = fq;
			m17 = -fq;

			// q = 8
			fq = dist[7*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 += fq;
			m16 -= fq;
			m17 += fq;

			// q=9
			fq = dist[10*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 += fq;
			m17 += fq;

			// q = 10
			fq = dist[9*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy += fq;
			m6 += fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 -= fq;
			m17 -= fq;

			// q=11
			fq = dist[12*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 = fq;
			m16 -= fq;
			m18 = fq;

			// q=12
			fq = dist[11*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 += fq;
			m16 += fq;
			m18 -= fq;

			// q=13
			fq = dist[14*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 -= fq;
			m18 -= fq;

			// q=14
			fq = dist[13*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 += fq;
			m18 += fq;

			// q=15
			fq = dist[16*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 = fq;
			m17 += fq;
			m18 -= fq;

			// q=16
			fq = dist[15*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 += fq;
			m17 -= fq;
			m18 += fq;

			// q=17
			fq = dist[18*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 += fq;
			m18 += fq;

			// q=18
			fq = dist[17*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 -= fq;
			m18 -= fq;

			//........................................................................
			//..............carry out relaxation process..............................
			//..........Toelke, Fruediger et. al. 2006................................
			if (C == 0.0)	nx = ny = nz = 0.0;
			m1 = m1 + rlx_setA*((19*(jx*jx+jy*jy+jz*jz)/rho0 - 11*rho) -alpha*C - m1);
			m2 = m2 + rlx_setA*((3*rho - 5.5*(jx*jx+jy*jy+jz*jz)/rho0)- m2);
			m4 = m4 + rlx_setB*((-0.6666666666666666*jx)- m4);
			m6 = m6 + rlx_setB*((-0.6666666666666666*jy)- m6);
			m8 = m8 + rlx_setB*((-0.6666666666666666*jz)- m8);
			m9 = m9 + rlx_setA*(((2*jx*jx-jy*jy-jz*jz)/rho0) + 0.5*alpha*C*(2*nx*nx-ny*ny-nz*nz) - m9);
			m10 = m10 + rlx_setA*( - m10);
			m11 = m11 + rlx_setA*(((jy*jy-jz*jz)/rho0) + 0.5*alpha*C*(ny*ny-nz*nz)- m11);
			m12 = m12 + rlx_setA*( - m12);
			m13 = m13 + rlx_setA*( (jx*jy/rho0) + 0.5*alpha*C*nx*ny - m13);
			m14 = m14 + rlx_setA*( (jy*jz/rho0) + 0.5*alpha*C*ny*nz - m14);
			m15 = m15 + rlx_setA*( (jx*jz/rho0) + 0.5*alpha*C*nx*nz - m15);
			m16 = m16 + rlx_setB*( - m16);
			m17 = m17 + rlx_setB*( - m17);
			m18 = m18 + rlx_setB*( - m18);

			//.......................................................................................................
			//.................inverse transformation......................................................

			// q=0
			fq = mrt_V1*rho-mrt_V2*m1+mrt_V3*m2;
			dist[n] = fq;

			// q = 1
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10) + 0.16666666*Fx;
			dist[1*Np+n] = fq;

			// q=2
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10) -  0.16666666*Fx;
			dist[2*Np+n] = fq;

			// q = 3
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) + 0.16666666*Fy;
			dist[3*Np+n] = fq;

			// q = 4
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12) - 0.16666666*Fy;
			dist[4*Np+n] = fq;

			// q = 5
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) + 0.16666666*Fz;
			dist[5*Np+n] = fq;

			// q = 6
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11) - 0.16666666*Fz;
			dist[6*Np+n] = fq;

			// q = 7
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17) + 0.08333333333*(Fx+Fy);
			dist[7*Np+n] = fq;


			// q = 8
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11
					+mrt_V12*m12+0.25*m13+0.125*(m17-m16) - 0.08333333333*(Fx+Fy);
			dist[8*Np+n] = fq;

			// q = 9
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17) + 0.08333333333*(Fx-Fy);
			dist[9*Np+n] = fq;

			// q = 10
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17)- 0.08333333333*(Fx-Fy);
			dist[10*Np+n] = fq;


			// q = 11
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12+0.25*m15+0.125*(m18-m16) + 0.08333333333*(Fx+Fz);
			dist[11*Np+n] = fq;

			// q = 12
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+
					mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18)-0.08333333333*(Fx+Fz);
			dist[12*Np+n] = fq;

			// q = 13
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15-0.125*(m16+m18) + 0.08333333333*(Fx-Fz);
			dist[13*Np+n] = fq;

			// q= 14
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15+0.125*(m16+m18) - 0.08333333333*(Fx-Fz);

			dist[14*Np+n] = fq;

			// q = 15
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18) + 0.08333333333*(Fy+Fz);
			dist[15*Np+n] = fq;

			// q = 16
			fq =  mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17)- 0.08333333333*(Fy+Fz);
			dist[16*Np+n] = fq;


			// q = 17
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18) + 0.08333333333*(Fy-Fz);
			dist[17*Np+n] = fq;

			// q = 18
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18) - 0.08333333333*(Fy-Fz);
			dist[18*Np+n] = fq;

			//........................................................................

			// write the velocity 
			ux = jx / rho0;
			uy = jy / rho0;
			uz = jz / rho0;
			Velocity[n] = ux;
			Velocity[Np+n] = uy;
			Velocity[2*Np+n] = uz;
			
		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_AAeven_ColorMass(double *Aq, double *Bq, double *Den,
		double *Velocity, double *ColorGrad, double beta, int start, int finish, int Np){

	int n;
	double fq;
	// non-conserved moments
	double nA,nB; // number density
	double a1,b1,a2,b2,nAB,delta;
	double C,nx,ny,nz; //color gradient magnitude and direction
	double ux,uy,uz;
	double phi,tau,rho0,rlx_setA,rlx_setB;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];
			
			// read the color gradient
			ux = Velocity[n];
			uy = Velocity[Np+n];
			uz = Velocity[2*Np+n];

			// read the color gradient
			nx = ColorGrad[n];
			ny = ColorGrad[Np+n];
			nz = ColorGrad[2*Np+n];

			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			if (C==0.0) C=1.0;
			nx = nx/C;
			ny = ny/C;
			nz = nz/C;		

			// Instantiate mass transport distributions
			// Stationary value - distribution 0

			nAB = 1.0/(nA+nB);
			Aq[n] = 0.3333333333333333*nA;
			Bq[n] = 0.3333333333333333*nB;

			//...............................................
			// q = 0,2,4
			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nx;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*ux))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*ux))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*ux))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*ux))+delta;

			Aq[1*Np+n] = a1;
			Bq[1*Np+n] = b1;
			Aq[2*Np+n] = a2;
			Bq[2*Np+n] = b2;

			//...............................................
			// q = 2
			// Cq = {0,1,0}
			delta = beta*nA*nB*nAB*0.1111111111111111*ny;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uy))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uy))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uy))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uy))+delta;

			Aq[3*Np+n] = a1;
			Bq[3*Np+n] = b1;
			Aq[4*Np+n] = a2;
			Bq[4*Np+n] = b2;
			//...............................................
			// q = 4
			// Cq = {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nz;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uz))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uz))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uz))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uz))+delta;

			Aq[5*Np+n] = a1;
			Bq[5*Np+n] = b1;
			Aq[6*Np+n] = a2;
			Bq[6*Np+n] = b2;
			//...............................................

		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_AAodd_ColorMass(int *neighborList, double *Aq, double *Bq, double *Den,
		double *Velocity, double *ColorGrad, double beta, int start, int finish, int Np){

	int n,nread;
	double fq;
	// non-conserved moments
	double nA,nB; // number density
	double a1,b1,a2,b2,nAB,delta;
	double C,nx,ny,nz; //color gradient magnitude and direction
	double ux,uy,uz;
	double phi,tau,rho0,rlx_setA,rlx_setB;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];
			
			// read the color gradient
			ux = Velocity[n];
			uy = Velocity[Np+n];
			uz = Velocity[2*Np+n];

			// read the color gradient
			nx = ColorGrad[n];
			ny = ColorGrad[Np+n];
			nz = ColorGrad[2*Np+n];

			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			if (C==0.0) C=1.0;
			nx = nx/C;
			ny = ny/C;
			nz = nz/C;		

			// Instantiate mass transport distributions
			// Stationary value - distribution 0
			nAB = 1.0/(nA+nB);
			Aq[n] = 0.3333333333333333*nA;
			Bq[n] = 0.3333333333333333*nB;

			//...............................................
			// q = 0,2,4
			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nx;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*ux))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*ux))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*ux))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*ux))+delta;

			// q = 1
			nread = neighborList[n+Np];
			Aq[nread] = a1;
			Bq[nread] = b1;
			// q=2
			nread = neighborList[n];
			Aq[nread] = a2;
			Bq[nread] = b2;

			//...............................................
			// Cq = {0,1,0}
			delta = beta*nA*nB*nAB*0.1111111111111111*ny;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uy))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uy))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uy))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uy))+delta;

			// q = 3
			nread = neighborList[n+3*Np];
			Aq[nread] = a1;
			Bq[nread] = b1;
			// q = 4
			nread = neighborList[n+2*Np];
			Aq[nread] = a2;
			Bq[nread] = b2;

			//...............................................
			// q = 4
			// Cq = {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nz;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uz))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uz))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uz))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uz))+delta;

			// q = 5
			nread = neighborList[n+5*Np];
			Aq[nread] = a1;
			Bq[nread] = b1;
			// q = 6
			nread = neighborList[n+4*Np];
			Aq[nread] = a2;
			Bq[nread] = b2;
			//...............................................
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_AAodd_PhaseField(int *neighborList, int *Map, double *Aq, double *Bq, 
		double *Den, double *Phi, int start, int finish, int Np){
	int idx,n,nread;
	double fq,nA,nB;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			//..........Compute the number density for each component ............
			// q=0
			fq = Aq[n];
			nA = fq;
			fq = Bq[n];
			nB = fq;
			
			// q=1
			nread = neighborList[n]; 
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread]; 
			nB += fq;
			
			// q=2
			nread = neighborList[n+Np]; 
			fq = Aq[nread];  
			nA += fq;
			fq = Bq[nread]; 
			nB += fq;
			
			// q=3
			nread = neighborList[n+2*Np]; 
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;
			
			// q = 4
			nread = neighborList[n+3*Np]; 
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;

			// q=5
			nread = neighborList[n+4*Np];
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;
			
			// q = 6
			nread = neighborList[n+5*Np];
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;

			// save the number densities
			Den[n] = nA;
			Den[Np+n] = nB;

			// save the phase indicator field
			idx = Map[n];
			Phi[idx] = (nA-nB)/(nA+nB); 
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_AAeven_PhaseField(int *Map, double *Aq, double *Bq, double *Den, double *Phi, 
		int start, int finish, int Np){
	int idx,n;
	double fq,nA,nB;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			// compute number density for each component
			// q=0
			fq = Aq[n];
			nA = fq;
			fq = Bq[n];
			nB = fq;
			
			// q=1
			fq = Aq[2*Np+n];
			nA += fq;
			fq = Bq[2*Np+n];
			nB += fq;

			// q=2
			fq = Aq[1*Np+n];
			nA += fq;
			fq = Bq[1*Np+n];
			nB += fq;

			// q=3
			fq = Aq[4*Np+n];
			nA += fq;
			fq = Bq[4*Np+n];
			nB += fq;

			// q = 4
			fq = Aq[3*Np+n];
			nA += fq;
			fq = Bq[3*Np+n];
			nB += fq;
			
			// q=5
			fq = Aq[6*Np+n];
			nA += fq;
			fq = Bq[6*Np+n];
			nB += fq;
			
			// q = 6
			fq = Aq[5*Np+n];
			nA += fq;
			fq = Bq[5*Np+n];
			nB += fq;

			// save the number densities
			Den[n] = nA;
			Den[Np+n] = nB;

			// save the phase indicator field
			idx = Map[n];
			Phi[idx] = (nA-nB)/(nA+nB); 	
		}
	}
}
__global__ void dvc_ScaLBL_D3Q19_Gradient(int *Map, double *phi, double *ColorGrad, int start, int finish, int Np,
			int strideY, int strideZ){
	int idx,ijk,nn;
	// distributions
	double m1,m2,m3,m4,m5,m6,m7,m8,m9;
	double m10,m11,m12,m13,m14,m15,m16,m17,m18;
	double nx,ny,nz;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		idx =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (idx<finish) {
			// Get the 1D index based on regular data layout
			ijk = Map[idx];

			//.......Back out the 3D indices for node n..............
			//k = n/(Nx*Ny);
			//j = (n-Nx*Ny*k)/Nx;
			//i = n-Nx*Ny*k-Nx*j;
			//........................................................................
			//........Get 1-D index for this thread....................
			//		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
			//........................................................................
			//					COMPUTE THE COLOR GRADIENT
			//........................................................................
			//.................Read Phase Indicator Values............................
			//........................................................................
			nn = ijk-1;							// neighbor index (get convention)
			m1 = phi[nn];						// get neighbor for phi - 1
			//........................................................................
			nn = ijk+1;							// neighbor index (get convention)
			m2 = phi[nn];						// get neighbor for phi - 2
			//........................................................................
			nn = ijk-strideY;							// neighbor index (get convention)
			m3 = phi[nn];					// get neighbor for phi - 3
			//........................................................................
			nn = ijk+strideY;							// neighbor index (get convention)
			m4 = phi[nn];					// get neighbor for phi - 4
			//........................................................................
			nn = ijk-strideZ;						// neighbor index (get convention)
			m5 = phi[nn];					// get neighbor for phi - 5
			//........................................................................
			nn = ijk+strideZ;						// neighbor index (get convention)
			m6 = phi[nn];					// get neighbor for phi - 6
			//........................................................................
			nn = ijk-strideY-1;						// neighbor index (get convention)
			m7 = phi[nn];					// get neighbor for phi - 7
			//........................................................................
			nn = ijk+strideY+1;						// neighbor index (get convention)
			m8 = phi[nn];					// get neighbor for phi - 8
			//........................................................................
			nn = ijk+strideY-1;						// neighbor index (get convention)
			m9 = phi[nn];					// get neighbor for phi - 9
			//........................................................................
			nn = ijk-strideY+1;						// neighbor index (get convention)
			m10 = phi[nn];					// get neighbor for phi - 10
			//........................................................................
			nn = ijk-strideZ-1;						// neighbor index (get convention)
			m11 = phi[nn];					// get neighbor for phi - 11
			//........................................................................
			nn = ijk+strideZ+1;						// neighbor index (get convention)
			m12 = phi[nn];					// get neighbor for phi - 12
			//........................................................................
			nn = ijk+strideZ-1;						// neighbor index (get convention)
			m13 = phi[nn];					// get neighbor for phi - 13
			//........................................................................
			nn = ijk-strideZ+1;						// neighbor index (get convention)
			m14 = phi[nn];					// get neighbor for phi - 14
			//........................................................................
			nn = ijk-strideZ-strideY;					// neighbor index (get convention)
			m15 = phi[nn];					// get neighbor for phi - 15
			//........................................................................
			nn = ijk+strideZ+strideY;					// neighbor index (get convention)
			m16 = phi[nn];					// get neighbor for phi - 16
			//........................................................................
			nn = ijk+strideZ-strideY;					// neighbor index (get convention)
			m17 = phi[nn];					// get neighbor for phi - 17
			//........................................................................
			nn = ijk-strideZ+strideY;					// neighbor index (get convention)
			m18 = phi[nn];					// get neighbor for phi - 18
			//............Compute the Color Gradient...................................
			nx = -(m1-m2+0.5*(m7-m8+m9-m10+m11-m12+m13-m14));
			ny = -(m3-m4+0.5*(m7-m8-m9+m10+m15-m16+m17-m18));
			nz = -(m5-m6+0.5*(m11-m12-m13+m14+m15-m16-m17+m18));
			//...............................................
			//...Store the Color Gradient....................
			ColorGrad[idx] = nx;
			ColorGrad[Np+idx] = ny;
			ColorGrad[2*Np+idx] = nz;
			//...............................................
		}
	}
}
__global__ void dvc_ScaLBL_PhaseField_Init(int *Map, double *Phi, double *Den, double *Aq, double *Bq, int start, int finish, int Np){
	int idx,n;
	double phi,nA,nB;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		idx =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (idx<finish) {

			n = Map[idx];
			phi = Phi[n];
            if (phi > 1.f){
                    nA = 1.0; nB = 0.f;
            }
            else if (phi < -1.f){
                    nB = 1.0; nA = 0.f;
            }
            else{
                    nA=0.5*(phi+1.f);
                    nB=0.5*(1.f-phi);
            }
			Den[idx] = nA;
			Den[Np+idx] = nB;

			Aq[idx]=0.3333333333333333*nA;
			Aq[Np+idx]=0.1111111111111111*nA;
			Aq[2*Np+idx]=0.1111111111111111*nA;
			Aq[3*Np+idx]=0.1111111111111111*nA;
			Aq[4*Np+idx]=0.1111111111111111*nA;
			Aq[5*Np+idx]=0.1111111111111111*nA;
			Aq[6*Np+idx]=0.1111111111111111*nA;

			Bq[idx]=0.3333333333333333*nB;
			Bq[Np+idx]=0.1111111111111111*nB;
			Bq[2*Np+idx]=0.1111111111111111*nB;
			Bq[3*Np+idx]=0.1111111111111111*nB;
			Bq[4*Np+idx]=0.1111111111111111*nB;
			Bq[5*Np+idx]=0.1111111111111111*nB;
			Bq[6*Np+idx]=0.1111111111111111*nB;
		}
	}
}

extern "C" void ScaLBL_SetSlice_z(double *Phi, double value, int Nx, int Ny, int Nz, int Slice){
	int GRID = Nx*Ny / 512 + 1;
	dvc_ScaLBL_SetSlice_z<<<GRID,512>>>(Phi,value,Nx,Ny,Nz,Slice);
}

extern "C" void ScaLBL_Color_Init(char *ID, double *Den, double *Phi, double das, double dbs, int Nx, int Ny, int Nz){
	dvc_ScaLBL_Color_Init<<<NBLOCKS,NTHREADS >>>(ID, Den, Phi, das, dbs, Nx, Ny, Nz);
}

extern "C" void ScaLBL_Color_InitDistance(char *ID, double *Den, double *Phi, double *Distance,
								double das, double dbs, double beta, double xp, int Nx, int Ny, int Nz){

	dvc_ScaLBL_Color_InitDistance<<<NBLOCKS,NTHREADS >>>(ID, Den, Phi, Distance, das, dbs, beta, xp, Nx, Ny, Nz);
}

extern "C" void ScaLBL_D3Q19_ColorGradient(char *ID, double *phi, double *ColorGrad, int Nx, int Ny, int Nz){
	dvc_ScaLBL_D3Q19_ColorGradient<<<NBLOCKS,NTHREADS >>>(ID, phi, ColorGrad, Nx, Ny, Nz);
}

extern "C" void ColorCollide( char *ID, double *disteven, double *distodd, double *ColorGrad,
								double *Velocity, int Nx, int Ny, int Nz,double rlx_setA, double rlx_setB,
								double alpha, double beta, double Fx, double Fy, double Fz, bool pBC){
	dvc_ColorCollide<<<NBLOCKS,NTHREADS >>>( ID, disteven, distodd, ColorGrad,Velocity, Nx, Ny, Nz,rlx_setA, rlx_setB,
									alpha, beta, Fx, Fy, Fz, pBC);

}

extern "C" void ScaLBL_D3Q19_ColorCollide( char *ID, double *disteven, double *distodd, double *phi, double *ColorGrad,
								double *Velocity, int Nx, int Ny, int Nz,double rlx_setA, double rlx_setB,
								double alpha, double beta, double Fx, double Fy, double Fz){
	dvc_ScaLBL_D3Q19_ColorCollide<<<NBLOCKS,NTHREADS >>>(ID, disteven, distodd, phi, ColorGrad, Velocity, Nx, Ny, Nz, rlx_setA, rlx_setB,
									alpha, beta, Fx, Fy, Fz);

}

extern "C" void DensityStreamD3Q7(char *ID, double *Den, double *Copy, double *Phi, double *ColorGrad, double *Velocity,
		double beta, int Nx, int Ny, int Nz, bool pBC){

	dvc_DensityStreamD3Q7<<<NBLOCKS,NTHREADS >>>(ID, Den, Copy, Phi, ColorGrad, Velocity, beta, Nx, Ny, Nz, pBC);
}

extern "C" void ScaLBL_ComputePhaseField(char *ID, double *Phi, double *Den, int N){
	dvc_ScaLBL_ComputePhaseField<<<NBLOCKS,NTHREADS >>>(ID, Phi, Den, N);
}
extern "C" void ScaLBL_D3Q7_ColorCollideMass(char *ID, double *A_even, double *A_odd, double *B_even, double *B_odd,
		double *Den, double *Phi, double *ColorGrad, double *Velocity, double beta, int N, bool pBC){
	 dvc_ScaLBL_D3Q7_ColorCollideMass<<<NBLOCKS,NTHREADS >>>(ID, A_even, A_odd, B_even, B_odd, Den, Phi, ColorGrad, Velocity, beta, N, pBC);
}
// Pressure Boundary Conditions Functions

extern "C" void ScaLBL_D3Q19_AAeven_Color(int *Map, double *dist, double *Aq, double *Bq, double *Den, double *Phi,
		double *Vel, double rhoA, double rhoB, double tauA, double tauB, double alpha, double beta,
		double Fx, double Fy, double Fz, int strideY, int strideZ, int start, int finish, int Np){

	hipProfilerStart();
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(dvc_ScaLBL_D3Q19_AAeven_Color), hipFuncCachePreferL1);

	dvc_ScaLBL_D3Q19_AAeven_Color<<<NBLOCKS,NTHREADS >>>(Map, dist, Aq, Bq, Den, Phi, Vel, rhoA, rhoB, tauA, tauB, 
			alpha, beta, Fx, Fy, Fz, strideY, strideZ, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_Color: %s \n",hipGetErrorString(err));
	}
	hipProfilerStop();

}

extern "C" void ScaLBL_D3Q19_AAodd_Color(int *d_neighborList, int *Map, double *dist, double *Aq, double *Bq, double *Den, 
		double *Phi, double *Vel, double rhoA, double rhoB, double tauA, double tauB, double alpha, double beta,
		double Fx, double Fy, double Fz, int strideY, int strideZ, int start, int finish, int Np){

	hipProfilerStart();
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(dvc_ScaLBL_D3Q19_AAodd_Color), hipFuncCachePreferL1);
	
	dvc_ScaLBL_D3Q19_AAodd_Color<<<NBLOCKS,NTHREADS >>>(d_neighborList, Map, dist, Aq, Bq, Den, Phi, Vel, 
			rhoA, rhoB, tauA, tauB, alpha, beta, Fx, Fy, Fz, strideY, strideZ, start, finish, Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_Color: %s \n",hipGetErrorString(err));
	}
	hipProfilerStop();
}

extern "C" void ScaLBL_D3Q7_AAodd_PhaseField(int *NeighborList, int *Map, double *Aq, double *Bq, 
		double *Den, double *Phi, int start, int finish, int Np){

	hipProfilerStart();
	dvc_ScaLBL_D3Q7_AAodd_PhaseField<<<NBLOCKS,NTHREADS >>>(NeighborList, Map, Aq, Bq, Den, Phi, start, finish, Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_AAodd_PhaseField: %s \n",hipGetErrorString(err));
	}
	hipProfilerStop();
}

extern "C" void ScaLBL_D3Q7_AAeven_PhaseField(int *Map, double *Aq, double *Bq, double *Den, double *Phi, 
		int start, int finish, int Np){

	hipProfilerStart();
	dvc_ScaLBL_D3Q7_AAeven_PhaseField<<<NBLOCKS,NTHREADS >>>(Map, Aq, Bq, Den, Phi, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_AAeven_PhaseField: %s \n",hipGetErrorString(err));
	}
	hipProfilerStop();

}

extern "C" void ScaLBL_D3Q19_Gradient(int *Map, double *Phi, double *ColorGrad, int start, int finish, int Np,
		int Nx, int Ny, int Nz){

	int strideY=Nx;
	int strideZ=Nx*Ny;
	dvc_ScaLBL_D3Q19_Gradient<<<NBLOCKS,NTHREADS >>>(Map, Phi, ColorGrad, start, finish, Np, strideY, strideZ);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_ColorGrad: %s \n",hipGetErrorString(err));
	}

}

extern "C" void ScaLBL_PhaseField_Init(int *Map, double *Phi, double *Den, double *Aq, double *Bq, int start, int finish, int Np){
	dvc_ScaLBL_PhaseField_Init<<<NBLOCKS,NTHREADS >>>(Map, Phi, Den, Aq, Bq, start, finish, Np); 
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_PhaseField_Init: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAeven_ColorMomentum(double *dist, double *Den, double *Vel,
		double *ColorGrad, double rhoA, double rhoB, double tauA, double tauB, double alpha, double beta,
		double Fx, double Fy, double Fz, int start, int finish, int Np){

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(dvc_ScaLBL_D3Q19_AAeven_ColorMomentum), hipFuncCachePreferL1);

	dvc_ScaLBL_D3Q19_AAeven_ColorMomentum<<<NBLOCKS,NTHREADS >>>(dist, Den, Vel, ColorGrad, rhoA, rhoB, tauA, tauB, 
			alpha, beta, Fx, Fy, Fz, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_ColorMomentum: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAodd_ColorMomentum(int *d_neighborList, double *dist, double *Den, double *Vel,
		double *ColorGrad, double rhoA, double rhoB, double tauA, double tauB, double alpha, double beta,
		double Fx, double Fy, double Fz, int start, int finish, int Np){

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(dvc_ScaLBL_D3Q19_AAodd_ColorMomentum), hipFuncCachePreferL1);

	dvc_ScaLBL_D3Q19_AAodd_ColorMomentum<<<NBLOCKS,NTHREADS >>>(d_neighborList, dist, Den, Vel, ColorGrad, 
			rhoA, rhoB, tauA, tauB, alpha, beta, Fx, Fy, Fz, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_ColorMomentum: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAeven_ColorMass(double *Aq, double *Bq, double *Den, double *Vel,
		double *ColorGrad, double beta, int start, int finish, int Np){

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(dvc_ScaLBL_D3Q19_AAeven_ColorMass), hipFuncCachePreferL1);

	dvc_ScaLBL_D3Q19_AAeven_ColorMass<<<NBLOCKS,NTHREADS >>>(Aq, Bq, Den, Vel, ColorGrad, beta, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_Color: %s \n",hipGetErrorString(err));
	}

}

extern "C" void ScaLBL_D3Q19_AAodd_ColorMass(int *d_neighborList, double *Aq, double *Bq, double *Den, double *Vel,
		double *ColorGrad, double beta, int start, int finish, int Np){

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(dvc_ScaLBL_D3Q19_AAodd_ColorMass), hipFuncCachePreferL1);

	dvc_ScaLBL_D3Q19_AAodd_ColorMass<<<NBLOCKS,NTHREADS >>>(d_neighborList, Aq, Bq, Den, Vel, ColorGrad, beta, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_Color: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_Color_BC_z(int *list, int *Map, double *Phi, double *Den, double vA, double vB, int count, int Np){
    int GRID = count / 512 + 1;
    dvc_ScaLBL_Color_BC_z<<<GRID,512>>>(list, Map, Phi, Den, vA, vB, count, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_Color_BC_z: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_Color_BC_Z(int *list, int *Map, double *Phi, double *Den, double vA, double vB, int count, int Np){
    int GRID = count / 512 + 1;
    dvc_ScaLBL_Color_BC_Z<<<GRID,512>>>(list, Map, Phi, Den, vA, vB, count, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_Color_BC_Z: %s \n",hipGetErrorString(err));
	}
}



