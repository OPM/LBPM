#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <mpi.h>

inline void PackID(int *list, int count, char *sendbuf, char *ID){
	// Fill in the phase ID values from neighboring processors
	// This packs up the values that need to be sent from one processor to another
	int idx,n;

	for (idx=0; idx<count; idx++){
		n = list[idx];
		sendbuf[idx] = ID[n];
	}
}

inline void UnpackID(int *list, int count, char *recvbuf, char *ID){
	// Fill in the phase ID values from neighboring processors
	// This unpacks the values once they have been recieved from neighbors
	int idx,n;

	for (idx=0; idx<count; idx++){
		n = list[idx];
		ID[n] = recvbuf[idx];
	}
}


__global__ void PackDist(int q, int *list, int start, int count, double *sendbuf, double *dist, int N){
	//....................................................................................
	// Pack distribution q into the send buffer for the listed lattice sites
	// dist may be even or odd distributions stored by stream layout
	//....................................................................................
	int idx,n;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<count){
//	for (idx=0; idx<count; idx++){
		n = list[idx];
		sendbuf[start+idx] = dist[q*N+n];
	}
}


__global__ void MapRecvDist(int q, int Cqx, int Cqy, int Cqz, int *list,  int start, int count,
					   double *recvbuf, double *dist, int Nx, int Ny, int Nz){
	//....................................................................................
	// Unack distribution from the recv buffer
	// Distribution q matche Cqx, Cqy, Cqz
	// swap rule means that the distributions in recvbuf are OPPOSITE of q
	// dist may be even or odd distributions stored by stream layout
	//....................................................................................
	int i,j,k,n,nn,idx;
	int N = Nx*Ny*Nz;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<count){
//	for (idx=0; idx<count; idx++){
		// Get the value from the list -- note that n is the index is from the send (non-local) process
		n = list[idx];
		// Get the 3-D indices
		k = n/(Nx*Ny);
		j = (n-Nx*Ny*k)/Nx;
		i = n-Nx*Ny*k-Nz*j;
		// Streaming for the non-local distribution
		i += Cqx;
		j += Cqy;
		k += Cqz;
/*		if (i < 0) i += Nx;
		if (j < 0) j += Ny;
		if (k < 0) k += Nz;
		if (!(i<Nx)) i -= Nx;
		if (!(j<Ny)) j -= Ny;
		if (!(k<Nz)) k -= Nz;
*/
		nn = k*Nx*Ny+j*Nx+i;
		// unpack the distribution to the proper location
	//	if (recvbuf[start+idx] != dist[q*N+nn]){
	//		printf("Stopping to check error \n");
	//		printf("recvbuf[start+idx] = %f \n",recvbuf[start+idx]);
	//		printf("dist[q*N+nn] = %f \n",dist[q*N+nn]);
	//		printf("A bug! Again? \n");
	//		idx = count;
	//	}
//		list[idx] = nn;
		dist[q*N+nn] = recvbuf[start+idx];
	}
}


//************************************************************************* 
__global__ void INITIALIZE(char *ID, double *f_even, double *f_odd, int Nx, int Ny, int Nz, int S)
{
	int n,N;
	N = Nx*Ny*Nz;
	
	for (int s=0; s<S; s++){

		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		
		if (n<N){
			if (ID[n] > 0){
				f_even[n] = 0.3333333333333333;	
				f_odd[n] = 0.055555555555555555;		//double(100*n)+1.f;
				f_even[N+n] = 0.055555555555555555;	//double(100*n)+2.f;
				f_odd[N+n] = 0.055555555555555555;	//double(100*n)+3.f;
				f_even[2*N+n] = 0.055555555555555555;	//double(100*n)+4.f;
				f_odd[2*N+n] = 0.055555555555555555;	//double(100*n)+5.f;
				f_even[3*N+n] = 0.055555555555555555;	//double(100*n)+6.f;
				f_odd[3*N+n] = 0.0277777777777778;   //double(100*n)+7.f;
				f_even[4*N+n] = 0.0277777777777778;   //double(100*n)+8.f;
				f_odd[4*N+n] = 0.0277777777777778;   //double(100*n)+9.f;
				f_even[5*N+n] = 0.0277777777777778;  //double(100*n)+10.f;
				f_odd[5*N+n] = 0.0277777777777778;  //double(100*n)+11.f;
				f_even[6*N+n] = 0.0277777777777778;  //double(100*n)+12.f;
				f_odd[6*N+n] = 0.0277777777777778;  //double(100*n)+13.f;
				f_even[7*N+n] = 0.0277777777777778;  //double(100*n)+14.f;
				f_odd[7*N+n] = 0.0277777777777778;  //double(100*n)+15.f;
				f_even[8*N+n] = 0.0277777777777778;  //double(100*n)+16.f;
				f_odd[8*N+n] = 0.0277777777777778;  //double(100*n)+17.f;
				f_even[9*N+n] = 0.0277777777777778;  //double(100*n)+18.f;
			}
			else{
				for(int q=0; q<9; q++){
					f_even[q*N+n] = -1.0;
					f_odd[q*N+n] = -1.0;
				}
				f_even[9*N+n] = -1.0;
			}
		}
	}
}

__global__ void Compute_VELOCITY(char *ID, double *disteven, double *distodd, double *vel, int Nx, int Ny, int Nz, int S)
{
	int n,N;
	// distributions 
	double f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	double vx,vy,vz;
		
	N = Nx*Ny*Nz;

	// S - number of threadblocks per grid block
	for (int s=0; s<S; s++){

		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		
		if (n<N){
			if (ID[n] > 0){
				//........................................................................
				// Registers to store the distributions
				//........................................................................
				f2 = disteven[N+n];
				f4 = disteven[2*N+n];
				f6 = disteven[3*N+n];
				f8 = disteven[4*N+n];
				f10 = disteven[5*N+n];
				f12 = disteven[6*N+n];
				f14 = disteven[7*N+n];
				f16 = disteven[8*N+n];
				f18 = disteven[9*N+n];
				//........................................................................			
				f1 = distodd[n];
				f3 = distodd[1*N+n];
				f5 = distodd[2*N+n];
				f7 = distodd[3*N+n];
				f9 = distodd[4*N+n];
				f11 = distodd[5*N+n];
				f13 = distodd[6*N+n];
				f15 = distodd[7*N+n];
				f17 = distodd[8*N+n];
				//.................Compute the velocity...................................	
				vx = f1-f2+f7-f8+f9-f10+f11-f12+f13-f14;
				vy = f3-f4+f7-f8-f9+f10+f15-f16+f17-f18;
				vz = f5-f6+f11-f12-f13+f14+f15-f16-f17+f18;
				//..................Write the velocity.....................................	
				vel[n] = vx;
				vel[N+n] = vy;
				vel[2*N+n] = vz;
				//........................................................................			

			}
		}
	}
}


//************************************************************************* 
__global__ void SWAP(char *ID, double *disteven, double *distodd, int Nx, int Ny, int Nz, int S)
{
	int n,nn,N;
	// distributions 
	double f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	
	N = Nx*Ny*Nz;
	
	// S - number of threadblocks per grid block
	for (int s=0; s<S; s++){
		
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;

//	for (n=0; n<N; n++){
		//.......Back out the 3-D indices for node n..............
		int	k = n/(Nx*Ny);
		int j = (n-Nx*Ny*k)/Nx;
		int i = n-Nx*Ny*k-Nz*j;
		
		if (n<N){
			if (ID[n] > 0){				
				//........................................................................
				// Retrieve even distributions from the local node (swap convention)
				//		f0 = disteven[n];  // Does not particupate in streaming
				f1 = distodd[n];
				f3 = distodd[N+n];
				f5 = distodd[2*N+n];
				f7 = distodd[3*N+n];
				f9 = distodd[4*N+n];
				f11 = distodd[5*N+n];
				f13 = distodd[6*N+n];
				f15 = distodd[7*N+n];
				f17 = distodd[8*N+n];
				//........................................................................
				
				//........................................................................
				// Retrieve odd distributions from neighboring nodes (swap convention)
				//........................................................................	
				nn = n+1;							// neighbor index (pull convention)
					if (!(i+1<Nx))	nn -= Nx;			// periodic BC along the x-boundary
				//if (i+1<Nx){
					f2 = disteven[N+nn];					// pull neighbor for distribution 2
					if (f2 > 0){
						distodd[n] = f2;
						disteven[N+nn] = f1;
					}
				//}
				//........................................................................	
				nn = n+Nx;							// neighbor index (pull convention)
				if (!(j+1<Ny))	nn -= Nx*Ny;		// Perioidic BC along the y-boundary
				//if (j+1<Ny){
					f4 = disteven[2*N+nn];				// pull neighbor for distribution 4
					if (f4 > 0){
						distodd[N+n] = f4;
						disteven[2*N+nn] = f3;
				//	}
				}
				//........................................................................	
				nn = n+Nx*Ny;						// neighbor index (pull convention)
					if (!(k+1<Nz))	nn -= Nx*Ny*Nz;		// Perioidic BC along the z-boundary
				//if (k+1<Nz){
					f6 = disteven[3*N+nn];				// pull neighbor for distribution 6
					if (f6 > 0){
						distodd[2*N+n] = f6;
						disteven[3*N+nn] = f5;
				//	}
				}
				//........................................................................	
				nn = n+Nx+1;						// neighbor index (pull convention)
					if (!(i+1<Nx))		nn -= Nx;		// periodic BC along the x-boundary
					if (!(j+1<Ny))		nn -= Nx*Ny;	// Perioidic BC along the y-boundary
				//if ((i+1<Nx) && (j+1<Ny)){
					f8 = disteven[4*N+nn];				// pull neighbor for distribution 8
					if (f8 > 0){
						distodd[3*N+n] = f8;
						disteven[4*N+nn] = f7;
				//	}
				}
				//........................................................................			
				nn = n-Nx+1;						// neighbor index (pull convention)
					if (!(i+1<Nx))	nn -= Nx;		// periodic BC along the x-boundary
					if (j-1<0)		nn += Nx*Ny;	// Perioidic BC along the y-boundary
				//if (!(i-1<0) && (j+1<Ny)){
					f10 = disteven[5*N+nn];					// pull neighbor for distribution 9
					if (f10 > 0){
						distodd[4*N+n] = f10;
						disteven[5*N+nn] = f9;
				//	}
				}
				//........................................................................	
				nn = n+Nx*Ny+1;						// neighbor index (pull convention)
					if (!(i+1<Nx))	nn -= Nx;		// periodic BC along the x-boundary
					if (!(k+1<Nz))	nn -= Nx*Ny*Nz;	// Perioidic BC along the z-boundary
				//if ( !(i-1<0) && !(k-1<0)){
					f12 = disteven[6*N+nn];				// pull distribution 11
					if (f12 > 0){
						distodd[5*N+n] = f12;
						disteven[6*N+nn] = f11;
				//	}
				}
				//........................................................................			
				nn = n-Nx*Ny+1;						// neighbor index (pull convention)
					if (!(i+1<Nx))	nn -= Nx;		// periodic BC along the x-boundary
					if (k-1<0)		nn += Nx*Ny*Nz;	// Perioidic BC along the z-boundary
				//if (!(i-1<0) && (k+1<Nz)){
					f14 = disteven[7*N+nn];				// pull neighbor for distribution 13
					if (f14 > 0){
						distodd[6*N+n] = f14;
						disteven[7*N+nn] = f13;
				//	}
				}
				//........................................................................							
				nn = n+Nx*Ny+Nx;					// neighbor index (pull convention)
					if (!(j+1<Ny))	nn -= Nx*Ny;		// Perioidic BC along the y-boundary
					if (!(k+1<Nz))	nn -= Nx*Ny*Nz;		// Perioidic BC along the z-boundary	
				//if (!(j-1<0) && !(k-1<0)){
					f16 = disteven[8*N+nn];				// pull neighbor for distribution 15
					if (f16 > 0){
						distodd[7*N+n] = f16;
						disteven[8*N+nn] = f15;
				//	}
				}
				//........................................................................											
				nn = n-Nx*Ny+Nx;					// neighbor index (pull convention)
					if (!(j+1<Ny))	nn -= Nx*Ny;		// Perioidic BC along the y-boundary
					if (k-1<0)		nn += Nx*Ny*Nz;		// Perioidic BC along the z-boundary	
				//if (!(j-1<0) && (k+1<Nz)){
					f18 = disteven[9*N+nn];				// pull neighbor for distribution 17
					if (f18 > 0){
						distodd[8*N+n] = f18;
						disteven[9*N+nn] = f17;
				//	}
				}
				//........................................................................		
				
			}
		}
	}
}
//************************************************************************* 

//************************************************************************* 
__global__ void MRT(char *ID, double *disteven, double *distodd, int Nx, int Ny, int Nz, int S,
					double rlx_setA, double rlx_setB, double Fx, double Fy, double Fz)
{
		
	int n,N;
	// distributions 
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9;
	double f10,f11,f12,f13,f14,f15,f16,f17,f18;
	
	// conserved momemnts
	double rho,jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;	
	
	N = Nx*Ny*Nz;
	
	char id;
	
	// S - number of threadblocks per grid block
	for (int s=0; s<S; s++){
//	for (int n=0; n<N; n++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		
		id = ID[n];
		
		if (n<N){
			if (id > 0){				
				//........................................................................
				// Registers to store the distributions - read based on swap convention
				//........................................................................
				f2 = distodd[n];
				f4 = distodd[N+n];
				f6 = distodd[2*N+n];
				f8 = distodd[3*N+n];
				f10 = distodd[4*N+n];
				f12 = distodd[5*N+n];
				f14 = distodd[6*N+n];
				f16 = distodd[7*N+n];
				f18 = distodd[8*N+n];
				//........................................................................			
				f0 = disteven[n];
				f1 = disteven[N+n];
				f3 = disteven[2*N+n];
				f5 = disteven[3*N+n];
				f7 = disteven[4*N+n];
				f9 = disteven[5*N+n];
				f11 = disteven[6*N+n];
				f13 = disteven[7*N+n];
				f15 = disteven[8*N+n];
				f17 = disteven[9*N+n];
				//........................................................................		
				//....................compute the moments...............................................		
				rho = f0+f2+f1+f4+f3+f6+f5+f8+f7+f10+f9+f12+f11+f14+f13+f16+f15+f18+f17;
				m1 = -30*f0-11*(f2+f1+f4+f3+f6+f5)+8*(f8+f7+f10+f9+f12+f11+f14+f13+f16+f15+f18 +f17);
				m2 = 12*f0-4*(f2+f1 +f4+f3+f6 +f5)+f8+f7+f10+f9+f12+f11+f14+f13+f16+f15+f18+f17;
				jx = f1-f2+f7-f8+f9-f10+f11-f12+f13-f14;
				m4 = 4*(-f1+f2)+f7-f8+f9-f10+f11-f12+f13-f14;
				jy = f3-f4+f7-f8-f9+f10+f15-f16+f17-f18;
				m6 = -4*(f3-f4)+f7-f8-f9+f10+f15-f16+f17-f18;
				jz = f5-f6+f11-f12-f13+f14+f15-f16-f17+f18;
				m8 = -4*(f5-f6)+f11-f12-f13+f14+f15-f16-f17+f18;
				m9 = 2*(f1+f2)-f3-f4-f5-f6+f7+f8+f9+f10+f11+f12+f13+f14-2*(f15+f16+f17+f18);
				m10 = -4*(f1+f2)+2*(f4+f3+f6+f5)+f8+f7+f10+f9+f12+f11+f14+f13-2*(f16+f15+f18+f17);
				m11 = f4+f3-f6-f5+f8+f7+f10+f9-f12-f11-f14-f13;
				m12 = -2*(f4+f3-f6-f5)+f8+f7+f10+f9-f12-f11-f14-f13;
				m13 = f8+f7-f10-f9;
				m14 = f16+f15-f18-f17;
				m15 = f12+f11-f14-f13;
				m16 = f7-f8+f9-f10-f11+f12-f13+f14;
				m17 = -f7+f8+f9-f10+f15-f16+f17-f18;
				m18 = f11-f12-f13+f14-f15+f16+f17-f18;
				//..............incorporate external force................................................
				//jx += 0.5*Fx;
				//jy += 0.5*Fy;
				//jz += 0.5*Fz;
				//..............carry out relaxation process...............................................
				m1 = m1 + rlx_setA*((19*(jx*jx+jy*jy+jz*jz)/rho - 11*rho) - m1);	
				m2 = m2 + rlx_setA*((3*rho - 5.5*(jx*jx+jy*jy+jz*jz)/rho) - m2);
				m4 = m4 + rlx_setB*((-0.6666666666666666*jx) - m4);
				m6 = m6 + rlx_setB*((-0.6666666666666666*jy) - m6);
				m8 = m8 + rlx_setB*((-0.6666666666666666*jz) - m8);
				m9 = m9 + rlx_setA*(((2*jx*jx-jy*jy-jz*jz)/rho) - m9);
				m10 = m10 + rlx_setA*(-0.5*((2*jx*jx-jy*jy-jz*jz)/rho) - m10);
				m11 = m11 + rlx_setA*(((jy*jy-jz*jz)/rho) - m11);
				m12 = m12 + rlx_setA*(-0.5*((jy*jy-jz*jz)/rho) - m12);
				m13 = m13 + rlx_setA*((jx*jy/rho) - m13);
				m14 = m14 + rlx_setA*((jy*jz/rho) - m14);
				m15 = m15 + rlx_setA*((jx*jz/rho) - m15);
				m16 = m16 + rlx_setB*( - m16);
				m17 = m17 + rlx_setB*( - m17);
				m18 = m18 + rlx_setB*( - m18);
				//.................inverse transformation......................................................
				f0 = 0.05263157894736842*rho-0.012531328320802*m1+0.04761904761904762*m2;
				f1 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
				+0.1*(jx-m4)+0.05555555555555555*(m9-m10);		
				f2 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
				+0.1*(m4-jx)+0.05555555555555555*(m9-m10);	
				f3 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
				+0.1*(jy-m6)+0.02777777777777778*(m10-m9)+0.08333333333333333*(m11-m12);		
				f4 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
				+0.1*(m6-jy)+0.02777777777777778*(m10-m9)+0.08333333333333333*(m11-m12);
				f5 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
				+0.1*(jz-m8)+0.02777777777777778*(m10-m9)+0.08333333333333333*(m12-m11);		
				f6 = 0.05263157894736842*rho-0.004594820384294068*m1-0.01587301587301587*m2
				+0.1*(m8-jz)+0.02777777777777778*(m10-m9)+0.08333333333333333*(m12-m11);
				f7 = 0.05263157894736842*rho+0.003341687552213868*m1+0.003968253968253968*m2+0.1*(jx+jy)+0.025*(m4+m6)
				+0.02777777777777778*m9+0.01388888888888889*m10+0.08333333333333333*m11
				+0.04166666666666666*m12+0.25*m13+0.125*(m16-m17);
				f8 = 0.05263157894736842*rho+0.003341687552213868*m1+0.003968253968253968*m2-0.1*(jx+jy)-0.025*(m4+m6)
				+0.02777777777777778*m9+0.01388888888888889*m10+0.08333333333333333*m11
				+0.04166666666666666*m12+0.25*m13+0.125*(m17-m16);
				f9 = 0.05263157894736842*rho+0.003341687552213868*m1+0.003968253968253968*m2+0.1*(jx-jy)+0.025*(m4-m6)
				+0.02777777777777778*m9+0.01388888888888889*m10+0.08333333333333333*m11
				+0.04166666666666666*m12-0.25*m13+0.125*(m16+m17);
				f10 = 0.05263157894736842*rho+0.003341687552213868*m1+0.003968253968253968*m2+0.1*(jy-jx)+0.025*(m6-m4)
				+0.02777777777777778*m9+0.01388888888888889*m10+0.08333333333333333*m11
				+0.04166666666666666*m12-0.25*m13-0.125*(m16+m17);
				f11 = 0.05263157894736842*rho+0.003341687552213868*m1
				+0.003968253968253968*m2+0.1*(jx+jz)+0.025*(m4+m8)
				+0.02777777777777778*m9+0.01388888888888889*m10-0.08333333333333333*m11
				-0.04166666666666666*m12+0.25*m15+0.125*(m18-m16);
				f12 = 0.05263157894736842*rho+0.003341687552213868*m1
				+0.003968253968253968*m2-0.1*(jx+jz)-0.025*(m4+m8)
				+0.02777777777777778*m9+0.01388888888888889*m10-0.08333333333333333*m11
				-0.04166666666666666*m12+0.25*m15+0.125*(m16-m18);
				f13 = 0.05263157894736842*rho+0.003341687552213868*m1
				+0.003968253968253968*m2+0.1*(jx-jz)+0.025*(m4-m8)
				+0.02777777777777778*m9+0.01388888888888889*m10-0.08333333333333333*m11
				-0.04166666666666666*m12-0.25*m15-0.125*(m16+m18);
				f14 = 0.05263157894736842*rho+0.003341687552213868*m1
				+0.003968253968253968*m2+0.1*(jz-jx)+0.025*(m8-m4)
				+0.02777777777777778*m9+0.01388888888888889*m10-0.08333333333333333*m11
				-0.04166666666666666*m12-0.25*m15+0.125*(m16+m18);
				f15 = 0.05263157894736842*rho+0.003341687552213868*m1
				+0.003968253968253968*m2+0.1*(jy+jz)+0.025*(m6+m8)
				-0.05555555555555555*m9-0.02777777777777778*m10+0.25*m14+0.125*(m17-m18);
				f16 =  0.05263157894736842*rho+0.003341687552213868*m1
				+0.003968253968253968*m2-0.1*(jy+jz)-0.025*(m6+m8)
				-0.05555555555555555*m9-0.02777777777777778*m10+0.25*m14+0.125*(m18-m17);
				f17 = 0.05263157894736842*rho+0.003341687552213868*m1
				+0.003968253968253968*m2+0.1*(jy-jz)+0.025*(m6-m8)
				-0.05555555555555555*m9-0.02777777777777778*m10-0.25*m14+0.125*(m17+m18);
				f18 = 0.05263157894736842*rho+0.003341687552213868*m1
				+0.003968253968253968*m2+0.1*(jz-jy)+0.025*(m8-m6)
				-0.05555555555555555*m9-0.02777777777777778*m10-0.25*m14-0.125*(m17+m18);
				//.......................................................................................................
				// incorporate external force
				f1 += 0.16666666*Fx;		
				f2 -= 0.16666666*Fx;		
				f3 += 0.16666666*Fy;		
				f4 -= 0.16666666*Fy;		
				f5 += 0.16666666*Fz;		
				f6 -= 0.16666666*Fz;	
				f7 += 0.08333333333*(Fx+Fy);	
				f8 -= 0.08333333333*(Fx+Fy);	
				f9 += 0.08333333333*(Fx-Fy);	
				f10 -= 0.08333333333*(Fx-Fy);	
				f11 += 0.08333333333*(Fx+Fz);	
				f12 -= 0.08333333333*(Fx+Fz);	
				f13 += 0.08333333333*(Fx-Fz);	
				f14 -= 0.08333333333*(Fx-Fz);	
				f15 += 0.08333333333*(Fy+Fz);	
				f16 -= 0.08333333333*(Fy+Fz);	
				f17 += 0.08333333333*(Fy-Fz);	
				f18 -= 0.08333333333*(Fy-Fz);	
				//.......................................................................................................
				// Write data based on un-swapped convention				
				disteven[n] = f0;
				disteven[N+n] = f2;
				disteven[2*N+n] = f4;
				disteven[3*N+n] = f6;
				disteven[4*N+n] = f8;
				disteven[5*N+n] = f10;
				disteven[6*N+n] = f12;
				disteven[7*N+n] = f14;
				disteven[8*N+n] = f16;
				disteven[9*N+n] = f18;
				
				distodd[n] = f1;
				distodd[N+n] = f3;
				distodd[2*N+n] = f5;
				distodd[3*N+n] = f7;				
				distodd[4*N+n] = f9;				
				distodd[5*N+n] = f11;
				distodd[6*N+n] = f13;
				distodd[7*N+n] = f15;
				distodd[8*N+n] = f17;
				//.......................................................................................................
			}
		}
	}
}
//************************************************************************* 

using namespace std;

void Write_Out(double *array, int Nx, int Ny, int Nz){
	int value;
	FILE *output;
	output = fopen("dist.list","w");
	for (int k=0; k<Nz; k++){
		for (int j=0; j<Ny; j++){
			for (int i=0; i<Nx; i++){
				int index = k*Nx*Ny+j*Nx+i;
				value = int(array[index]);
				fprintf(output, "| %i",value);
			}
			fprintf(output, " | \n");
		}
		fprintf(output,"************************************** \n");	
	}
	fclose(output);
}

//************************************************************************* 
// MRT implementation of the LBM using CUDA
//************************************************************************* 

int main(int argc, char **argv)
{
	//*****************************************
	// ***** MPI STUFF ****************
	//*****************************************
	// Initialize MPI
	int rank,nprocs;
	MPI_Init(&argc,&argv);
    MPI_Comm comm = MPI_COMM_WORLD;
	MPI_Comm_rank(comm,&rank);
	MPI_Comm_size(comm,&nprocs);
	// parallel domain size (# of sub-domains)
	int nprocx,nprocy,nprocz;
	int iproc,jproc,kproc;
	int sendtag,recvtag;
	//*****************************************
	// MPI ranks for all 18 neighbors
	//**********************************
	int rank_x,rank_y,rank_z,rank_X,rank_Y,rank_Z;
	int rank_xy,rank_XY,rank_xY,rank_Xy;
	int rank_xz,rank_XZ,rank_xZ,rank_Xz;
	int rank_yz,rank_YZ,rank_yZ,rank_Yz;
	//**********************************
	MPI_Request req1[18],req2[18];
	MPI_Status stat1[18],stat2[18];
	//**********************************
	//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
	//!!!!!!!!!!! Random debugging communications!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
//	rank_X = rank+1;
//	if (!(rank_X < nprocs)) rank_X-=nprocs;
//	rank_x = rank-1;
//	if (rank_x < 0) rank_x +=nprocs;
//	rank_y = rank_z = rank_xy = rank_Xy = rank_xz = rank_Xz = rank_yz = rank_Yz = rank_x;
//	rank_Y = rank_Z = rank_XY = rank_xY = rank_XZ = rank_xZ = rank_YZ = rank_yZ = rank_X;
	//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	int device = 1;
	if (rank==0)	printf("Number of devices = %i \n", deviceCount);
	if (rank==0)	printf("Current device is = %i \n", device);
	hipSetDevice(device);
	
	// BGK Model parameters
	string FILENAME;	
	unsigned int nBlocks, nthreads;
	int iterMax, interval;
	double tau,Fx,Fy,Fz,tol;
	// Domain variables
	int Nx,Ny,Nz;
	int i,j,k,n;

	if (rank==0){
		ifstream input("MRT.in");
		input >> FILENAME;		// name of the input file
		input >> Nz;			// number of nodes (x,y,z)
		input >> nBlocks;
		input >> nthreads;
		input >> tau;				// relaxation time
		input >> Fx;			// External force components (x,y,z)
		input >> Fy;
		input >> Fz;
		input >> iterMax;			// max no. of iterations
		input >> interval;			// error interval
		input >> tol;				// error tolerance

		ifstream domain("Domain.in");
		domain >> nprocx;
		domain >> nprocy;
		domain >> nprocz;
	}

	// **************************************************************
	// Broadcast simulation parameters from rank 0 to all other procs
	MPI_Barrier(comm);
	//.................................................
	MPI_Bcast(&Nz,1,MPI_INT,0,comm);
	MPI_Bcast(&nBlocks,1,MPI_INT,0,comm);
	MPI_Bcast(&nthreads,1,MPI_INT,0,comm);
	MPI_Bcast(&tau,1,MPI_DOUBLE,0,comm);
	MPI_Bcast(&Fx,1,MPI_DOUBLE,0,comm);
	MPI_Bcast(&Fy,1,MPI_DOUBLE,0,comm);
	MPI_Bcast(&Fz,1,MPI_DOUBLE,0,comm);
	MPI_Bcast(&iterMax,1,MPI_INT,0,comm);
	MPI_Bcast(&interval,1,MPI_INT,0,comm);
	MPI_Bcast(&tol,1,MPI_DOUBLE,0,comm);

	MPI_Bcast(&nprocx,1,MPI_INT,0,comm);
	MPI_Bcast(&nprocy,1,MPI_INT,0,comm);
	MPI_Bcast(&nprocz,1,MPI_INT,0,comm);
	//.................................................
	MPI_Barrier(comm);
	// **************************************************************

	double rlx_setA = 1.f/tau;
	double rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);

	if (nprocs != nprocx*nprocy*nprocz){
		printf("Fatal error in processor number! \n");
		printf("nprocx =  %i \n",nprocx);
		printf("nprocy =  %i \n",nprocy);
		printf("nprocz =  %i \n",nprocz);
	}

	if (rank==0){
		printf("tau = %f \n", tau);
		printf("Set A = %f \n", rlx_setA);
		printf("Set B = %f \n", rlx_setB);
		printf("Force(x) = %f \n", Fx);
		printf("Force(y) = %f \n", Fy);
		printf("Force(z) = %f \n", Fz);
		printf("Sub-domain size = %i x %i x %i\n",Nz,Nz,Nz);
	}

	MPI_Barrier(comm);
	kproc = rank/(nprocx*nprocy);
	jproc = (rank-nprocx*nprocy*kproc)/nprocx;
	iproc = rank-nprocx*nprocy*kproc-nprocz*jproc;

	//..........................................
	// set up the neighbor ranks
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i+=1;
	j+=0;
	k+=0;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_X = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i-=1;
	j+=0;
	k+=0;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_x = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i+=0;
	j+=1;
	k+=0;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_Y = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i+=0;
	j-=1;
	k+=0;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_y = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i+=0;
	j+=0;
	k+=1;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-= nprocy;
	if (!(k<nprocz)) k-= nprocz;
	rank_Z = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i+=0;
	j+=0;
	k-=1;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-= nprocy;
	if (!(k<nprocz)) k-= nprocz;
	rank_z = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i+=1;
	j+=1;
	k+=0;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_XY = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i-=1;
	j-=1;
	k+=0;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_xy = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i+=1;
	j-=1;
	k+=0;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_Xy = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i-=1;
	j+=1;
	k+=0;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_xY = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i+=1;
	j+=0;
	k+=1;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_XZ = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i-=1;
	j+=0;
	k-=1;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_xz = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i-=1;
	j+=0;
	k+=1;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_xZ = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i+=1;
	j+=0;
	k-=1;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_Xz = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i+=0;
	j+=1;
	k+=1;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_YZ = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i+=0;
	j-=1;
	k-=1;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_yz = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k =kproc;
	i+=0;
	j-=1;
	k+=1;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_yZ = k*nprocx*nprocy+j*nprocx+i;
	//..........................................
	i=iproc; j=jproc; k=kproc;
	i+=0;
	j+=1;
	k-=1;
	if (i<0)	i+=nprocx;
	if (j<0)	j+=nprocy;
	if (k<0)	k+=nprocz;
	if (!(i<nprocx)) i-= nprocx;
	if (!(j<nprocy)) j-=nprocy;
	if (!(k<nprocz)) k-=nprocz;
	rank_Yz = k*nprocx*nprocy+j*nprocx+i;
	//..........................................

	Nz += 2;
	Nx = Ny = Nz;	// Cubic domain
	
	int N = Nx*Ny*Nz;
	int dist_mem_size = N*sizeof(double);
	
//	unsigned int nBlocks = 32;
//	int nthreads = 128;
	int S = N/nthreads/nBlocks;
	
//	unsigned int nBlocks = N/nthreads + (N%nthreads == 0?0:1);
	dim3 grid(nBlocks,1,1);
		
	if (rank==0) printf("Number of blocks = %i \n", nBlocks);
	if (rank==0) printf("Threads per block = %i \n", nthreads);
	if (rank==0) printf("Sweeps per thread = %i \n", S);
	if (rank==0) printf("Number of nodes per side = %i \n", Nx);
	if (rank==0) printf("Total Number of nodes = %i \n", N);
	
	//.......................................................................
	if (rank == 0)	printf("Read input media... \n");
	//.......................................................................
	char LocalRankString[8];
	char LocalRankFilename[40];
	sprintf(LocalRankString,"%05d",rank);
	sprintf(LocalRankFilename,"%s%s","ID.",LocalRankString);
	printf("Local File Name =  %s \n",LocalRankFilename);
	// .......... READ THE INPUT FILE .......................................
	char value;
	char *id;
	id = new char[N];	
	int sum = 0;
	double porosity;
	//.......................................................................
	ifstream PM(LocalRankFilename,ios::binary);
	for (k=0;k<Nz;k++){
		for (j=0;j<Ny;j++){
			for (i=0;i<Nx;i++){
				n = k*Nx*Ny+j*Nx+i;
				id[n] = 0;
			}
		}
	}
	for ( k=1;k<Nz-1;k++){
		for ( j=1;j<Ny-1;j++){
			for ( i=1;i<Nx-1;i++){
				PM.read((char *) (&value), sizeof(value));
				n = k*Nx*Ny+j*Nx+i;
				id[n] = value;
				if (value > 0) sum++;
			}
		}
	}
	PM.close();
//	printf("File porosity = %f\n", double(sum)/N);
	//...........................................................................
	MPI_Barrier(comm);
	if (rank == 0) cout << "Domain set." << endl;
	//...........................................................................
	// Write the communcation structure into a file for debugging
	char LocalCommFile[40];
	sprintf(LocalCommFile,"%s%s","Comm.",LocalRankString);
	FILE *CommFile;
	CommFile = fopen(LocalCommFile,"w");
	fprintf(CommFile,"rank=%d, ",rank);
	fprintf(CommFile,"i=%d,j=%d,k=%d :",iproc,jproc,kproc);
	fprintf(CommFile,"x=%d, ",rank_x);
	fprintf(CommFile,"X=%d, ",rank_X);
	fprintf(CommFile,"y=%d, ",rank_y);
	fprintf(CommFile,"Y=%d, ",rank_Y);
	fprintf(CommFile,"z=%d, ",rank_z);
	fprintf(CommFile,"Z=%d, ",rank_Z);
	fprintf(CommFile,"xy=%d, ",rank_xy);
	fprintf(CommFile,"XY=%d, ",rank_XY);
	fprintf(CommFile,"xY=%d, ",rank_xY);
	fprintf(CommFile,"Xy=%d, ",rank_Xy);
	fprintf(CommFile,"xz=%d, ",rank_xz);
	fprintf(CommFile,"XZ=%d, ",rank_XZ);
	fprintf(CommFile,"xZ=%d, ",rank_xZ);
	fprintf(CommFile,"Xz=%d, ",rank_Xz);
	fprintf(CommFile,"yz=%d, ",rank_yz);
	fprintf(CommFile,"YZ=%d, ",rank_YZ);
	fprintf(CommFile,"yZ=%d, ",rank_yZ);
	fprintf(CommFile,"Yz=%d, ",rank_Yz);
	fprintf(CommFile,"\n");
	fclose(CommFile);
	//...........................................................................

	// Set up MPI communication structures
	if (rank==0)	printf ("Setting up communication control structures \n");
	//......................................................................................
	// Get the actual D3Q19 communication counts (based on location of solid phase)
	// Discrete velocity set symmetry implies the sendcount = recvcount
	int sendCount_x, sendCount_y, sendCount_z, sendCount_X, sendCount_Y, sendCount_Z;
	int sendCount_xy, sendCount_yz, sendCount_xz, sendCount_Xy, sendCount_Yz, sendCount_xZ;
	int sendCount_xY, sendCount_yZ, sendCount_Xz, sendCount_XY, sendCount_YZ, sendCount_XZ;
	sendCount_x = sendCount_y = sendCount_z = sendCount_X = sendCount_Y = sendCount_Z = 0;
	sendCount_xy = sendCount_yz = sendCount_xz = sendCount_Xy = sendCount_Yz = sendCount_xZ = 0;
	sendCount_xY = sendCount_yZ = sendCount_Xz = sendCount_XY = sendCount_YZ = sendCount_XZ = 0;
	//......................................................................................
	for (k=0; k<Nz; k++){
		for (j=0; j<Ny; j++){
			for (i=0; i<Nx; i++){
				// Check the phase ID
				if (id[k*Nx*Ny+j*Nx+i] != 0){
					// Counts for the six faces
					if (i==1)	sendCount_x++;
					if (j==1)	sendCount_y++;
					if (k==1)	sendCount_z++;
					if (i==Nx-2)	sendCount_X++;
					if (j==Ny-2)	sendCount_Y++;
					if (k==Nz-2)	sendCount_Z++;
					// Counts for the twelve edges
					if (i==1 && j==1)	sendCount_xy++;
					if (i==1 && j==Ny-2)	sendCount_xY++;
					if (i==Nx-2 && j==1)	sendCount_Xy++;
					if (i==Nx-2 && j==Ny-2)	sendCount_XY++;

					if (i==1 && k==1)	sendCount_xz++;
					if (i==1 && k==Nz-2)	sendCount_xZ++;
					if (i==Nx-2 && k==1)	sendCount_Xz++;
					if (i==Nx-2 && k==Nz-2)	sendCount_XZ++;

					if (j==1 && k==1)	sendCount_yz++;
					if (j==1 && k==Nz-2)	sendCount_yZ++;
					if (j==Ny-2 && k==1)	sendCount_Yz++;
					if (j==Ny-2 && k==Nz-2)	sendCount_YZ++;
				}
			}
		}
	}
	//......................................................................................
	int *sendList_x, *sendList_y, *sendList_z, *sendList_X, *sendList_Y, *sendList_Z;
	int *sendList_xy, *sendList_yz, *sendList_xz, *sendList_Xy, *sendList_Yz, *sendList_xZ;
	int *sendList_xY, *sendList_yZ, *sendList_Xz, *sendList_XY, *sendList_YZ, *sendList_XZ;
	//......................................................................................
	// send buffers
	sendList_x = new int [sendCount_x];
	sendList_y = new int [sendCount_y];
	sendList_z = new int [sendCount_z];
	sendList_X = new int [sendCount_X];
	sendList_Y = new int [sendCount_Y];
	sendList_Z = new int [sendCount_Z];
	sendList_xy = new int [sendCount_xy];
	sendList_yz = new int [sendCount_yz];
	sendList_xz = new int [sendCount_xz];
	sendList_Xy = new int [sendCount_Xy];
	sendList_Yz = new int [sendCount_Yz];
	sendList_xZ = new int [sendCount_xZ];
	sendList_xY = new int [sendCount_xY];
	sendList_yZ = new int [sendCount_yZ];
	sendList_Xz = new int [sendCount_Xz];
	sendList_XY = new int [sendCount_XY];
	sendList_YZ = new int [sendCount_YZ];
	sendList_XZ = new int [sendCount_XZ];
	if (rank==0)	printf ("Preparing the sendlists \n");
	//......................................................................................
	// Populate the send list
	sendCount_x = sendCount_y = sendCount_z = sendCount_X = sendCount_Y = sendCount_Z = 0;
	sendCount_xy = sendCount_yz = sendCount_xz = sendCount_Xy = sendCount_Yz = sendCount_xZ = 0;
	sendCount_xY = sendCount_yZ = sendCount_Xz = sendCount_XY = sendCount_YZ = sendCount_XZ = 0;
	for (k=0; k<Nz; k++){
		for (j=0; j<Ny; j++){
			for (i=0; i<Nx; i++){
				// Local value to send
				n = k*Nx*Ny+j*Nx+i;
				if (id[n] != 0){
					// Counts for the six faces
					if (i==1)		sendList_x[sendCount_x++]=n;
					if (j==1)		sendList_y[sendCount_y++]=n;
					if (k==1)		sendList_z[sendCount_z++]=n;
					if (i==Nx-2)	sendList_X[sendCount_X++]=n;
					if (j==Ny-2)	sendList_Y[sendCount_Y++]=n;
					if (k==Nz-2)	sendList_Z[sendCount_Z++]=n;
					// Counts for the twelve edges
					if (i==1 && j==1)		sendList_xy[sendCount_xy++]=n;
					if (i==1 && j==Ny-2)	sendList_xY[sendCount_xY++]=n;
					if (i==Nx-2 && j==1)	sendList_Xy[sendCount_Xy++]=n;
					if (i==Nx-2 && j==Ny-2)	sendList_XY[sendCount_XY++]=n;

					if (i==1 && k==1)		sendList_xz[sendCount_xz++]=n;
					if (i==1 && k==Nz-2)	sendList_xZ[sendCount_xZ++]=n;
					if (i==Nx-2 && k==1)	sendList_Xz[sendCount_Xz++]=n;
					if (i==Nx-2 && k==Nz-2)	sendList_XZ[sendCount_XZ++]=n;

					if (j==1 && k==1)		sendList_yz[sendCount_yz++]=n;
					if (j==1 && k==Nz-2)	sendList_yZ[sendCount_yZ++]=n;
					if (j==Ny-2 && k==1)	sendList_Yz[sendCount_Yz++]=n;
					if (j==Ny-2 && k==Nz-2)	sendList_YZ[sendCount_YZ++]=n;
				}
			}
		}
	}
	MPI_Barrier(comm);
	if (rank==0)	printf ("SendLists are ready on host\n");
	//......................................................................................
	// Use MPI to fill in the recvCounts form the associated processes
	int recvCount_x, recvCount_y, recvCount_z, recvCount_X, recvCount_Y, recvCount_Z;
	int recvCount_xy, recvCount_yz, recvCount_xz, recvCount_Xy, recvCount_Yz, recvCount_xZ;
	int recvCount_xY, recvCount_yZ, recvCount_Xz, recvCount_XY, recvCount_YZ, recvCount_XZ;
	//......................................................................................
	//**********************************************************************************
	// Fill in the recieve counts using MPI
	sendtag = recvtag = 3;
	MPI_Send(&sendCount_x,1,MPI_INT,rank_X,sendtag,comm);
	MPI_Recv(&recvCount_X,1,MPI_INT,rank_x,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Send(&sendCount_X,1,MPI_INT,rank_x,sendtag,comm);
	MPI_Recv(&recvCount_x,1,MPI_INT,rank_X,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Send(&sendCount_y,1,MPI_INT,rank_Y,sendtag,comm);
	MPI_Recv(&recvCount_Y,1,MPI_INT,rank_y,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Send(&sendCount_Y,1,MPI_INT,rank_y,sendtag,comm);
	MPI_Recv(&recvCount_y,1,MPI_INT,rank_Y,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Send(&sendCount_z,1,MPI_INT,rank_Z,sendtag,comm);
	MPI_Recv(&recvCount_Z,1,MPI_INT,rank_z,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Send(&sendCount_Z,1,MPI_INT,rank_z,sendtag,comm);
	MPI_Recv(&recvCount_z,1,MPI_INT,rank_Z,recvtag,comm,MPI_STATUS_IGNORE);

	MPI_Send(&sendCount_xy,1,MPI_INT,rank_XY,sendtag,comm);
	MPI_Recv(&recvCount_XY,1,MPI_INT,rank_xy,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Send(&sendCount_XY,1,MPI_INT,rank_xy,sendtag,comm);
	MPI_Recv(&recvCount_xy,1,MPI_INT,rank_XY,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Send(&sendCount_Xy,1,MPI_INT,rank_xY,sendtag,comm);
	MPI_Recv(&recvCount_xY,1,MPI_INT,rank_Xy,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Send(&sendCount_xY,1,MPI_INT,rank_Xy,sendtag,comm);
	MPI_Recv(&recvCount_Xy,1,MPI_INT,rank_xY,recvtag,comm,MPI_STATUS_IGNORE);

	MPI_Send(&sendCount_xz,1,MPI_INT,rank_XZ,sendtag,comm);
	MPI_Recv(&recvCount_XZ,1,MPI_INT,rank_xz,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Send(&sendCount_XZ,1,MPI_INT,rank_xz,sendtag,comm);
	MPI_Recv(&recvCount_xz,1,MPI_INT,rank_XZ,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Send(&sendCount_Xz,1,MPI_INT,rank_xZ,sendtag,comm);
	MPI_Recv(&recvCount_xZ,1,MPI_INT,rank_Xz,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Send(&sendCount_xZ,1,MPI_INT,rank_Xz,sendtag,comm);
	MPI_Recv(&recvCount_Xz,1,MPI_INT,rank_xZ,recvtag,comm,MPI_STATUS_IGNORE);

	MPI_Send(&sendCount_yz,1,MPI_INT,rank_YZ,sendtag,comm);
	MPI_Recv(&recvCount_YZ,1,MPI_INT,rank_yz,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Send(&sendCount_YZ,1,MPI_INT,rank_yz,sendtag,comm);
	MPI_Recv(&recvCount_yz,1,MPI_INT,rank_YZ,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Send(&sendCount_Yz,1,MPI_INT,rank_yZ,sendtag,comm);
	MPI_Recv(&recvCount_yZ,1,MPI_INT,rank_Yz,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Send(&sendCount_yZ,1,MPI_INT,rank_Yz,sendtag,comm);
	MPI_Recv(&recvCount_Yz,1,MPI_INT,rank_yZ,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Barrier(comm);
	//**********************************************************************************
	//recvCount_x = sendCount_x;
	//recvCount_X = sendCount_X;
	//recvCount_y = sendCount_y;
	//recvCount_Y = sendCount_Y;
	//recvCount_z = sendCount_z;
	//recvCount_Z = sendCount_Z;
	//recvCount_xy = sendCount_xy;
	//recvCount_xY = sendCount_xY;
	//recvCount_Xy = sendCount_Xy;
	//recvCount_XY = sendCount_XY;
	//recvCount_xz = sendCount_xz;
	//recvCount_xZ = sendCount_xZ;
	//recvCount_Xz = sendCount_XZ;
	//recvCount_XZ = sendCount_XZ;
	//recvCount_yz = sendCount_yz;
	//recvCount_Yz = sendCount_Yz;
	//recvCount_yZ = sendCount_yZ;
	//recvCount_YZ = sendCount_YZ;
	//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
	//......................................................................................
	// Use MPI to fill in the appropriate values
	//	int tag = 5;
	//	MPI_Sendrecv(sendCount_x,1,MPI_INT,rank_x,tag,sendCount_X,1,MPI_INT,comm,req);
	//......................................................................................
	int *recvList_x, *recvList_y, *recvList_z, *recvList_X, *recvList_Y, *recvList_Z;
	int *recvList_xy, *recvList_yz, *recvList_xz, *recvList_Xy, *recvList_Yz, *recvList_xZ;
	int *recvList_xY, *recvList_yZ, *recvList_Xz, *recvList_XY, *recvList_YZ, *recvList_XZ;
	//......................................................................................
	// recv buffers
	recvList_x = new int [recvCount_x];
	recvList_y = new int [recvCount_y];
	recvList_z = new int [recvCount_z];
	recvList_X = new int [recvCount_X];
	recvList_Y = new int [recvCount_Y];
	recvList_Z = new int [recvCount_Z];
	recvList_xy = new int [recvCount_xy];
	recvList_yz = new int [recvCount_yz];
	recvList_xz = new int [recvCount_xz];
	recvList_Xy = new int [recvCount_Xy];
	recvList_Yz = new int [recvCount_Yz];
	recvList_xZ = new int [recvCount_xZ];
	recvList_xY = new int [recvCount_xY];
	recvList_yZ = new int [recvCount_yZ];
	recvList_Xz = new int [recvCount_Xz];
	recvList_XY = new int [recvCount_XY];
	recvList_YZ = new int [recvCount_YZ];
	recvList_XZ = new int [recvCount_XZ];
	//......................................................................................
	//......................................................................................
	// Use MPI to fill in the appropriate values for recvList
	// Fill in the recieve lists using MPI
	sendtag = recvtag = 4;
	MPI_Isend(sendList_x, sendCount_x,MPI_INT,rank_X,sendtag,comm,&req1[0]);
	MPI_Irecv(recvList_X, recvCount_X,MPI_INT,rank_x,recvtag,comm,&req2[0]);
	MPI_Isend(sendList_X, sendCount_X,MPI_INT,rank_x,sendtag,comm,&req1[1]);
	MPI_Irecv(recvList_x, recvCount_x,MPI_INT,rank_X,recvtag,comm,&req2[1]);
	MPI_Isend(sendList_y, sendCount_y,MPI_INT,rank_Y,sendtag,comm,&req1[2]);
	MPI_Irecv(recvList_Y, recvCount_Y,MPI_INT,rank_y,recvtag,comm,&req2[2]);
	MPI_Isend(sendList_Y, sendCount_Y,MPI_INT,rank_y,sendtag,comm,&req1[3]);
	MPI_Irecv(recvList_y, recvCount_y,MPI_INT,rank_Y,recvtag,comm,&req2[3]);
	MPI_Isend(sendList_z, sendCount_z,MPI_INT,rank_Z,sendtag,comm,&req1[4]);
	MPI_Irecv(recvList_Z, recvCount_Z,MPI_INT,rank_z,recvtag,comm,&req2[4]);
	MPI_Isend(sendList_Z, sendCount_Z,MPI_INT,rank_z,sendtag,comm,&req1[5]);
	MPI_Irecv(recvList_z, recvCount_z,MPI_INT,rank_Z,recvtag,comm,&req2[5]);

	MPI_Isend(sendList_xy, sendCount_xy,MPI_INT,rank_XY,sendtag,comm,&req1[6]);
	MPI_Irecv(recvList_XY, recvCount_XY,MPI_INT,rank_xy,recvtag,comm,&req2[6]);
	MPI_Isend(sendList_XY, sendCount_XY,MPI_INT,rank_xy,sendtag,comm,&req1[7]);
	MPI_Irecv(recvList_xy, recvCount_xy,MPI_INT,rank_XY,recvtag,comm,&req2[7]);
	MPI_Isend(sendList_Xy, sendCount_Xy,MPI_INT,rank_xY,sendtag,comm,&req1[8]);
	MPI_Irecv(recvList_xY, recvCount_xY,MPI_INT,rank_Xy,recvtag,comm,&req2[8]);
	MPI_Isend(sendList_xY, sendCount_xY,MPI_INT,rank_Xy,sendtag,comm,&req1[9]);
	MPI_Irecv(recvList_Xy, recvCount_Xy,MPI_INT,rank_xY,recvtag,comm,&req2[9]);

	MPI_Isend(sendList_xz, sendCount_xz,MPI_INT,rank_XZ,sendtag,comm,&req1[10]);
	MPI_Irecv(recvList_XZ, recvCount_XZ,MPI_INT,rank_xz,recvtag,comm,&req2[10]);
	MPI_Isend(sendList_XZ, sendCount_XZ,MPI_INT,rank_xz,sendtag,comm,&req1[11]);
	MPI_Irecv(recvList_xz, recvCount_xz,MPI_INT,rank_XZ,recvtag,comm,&req2[11]);
	MPI_Isend(sendList_Xz, sendCount_Xz,MPI_INT,rank_xZ,sendtag,comm,&req1[12]);
	MPI_Irecv(recvList_xZ, recvCount_xZ,MPI_INT,rank_Xz,recvtag,comm,&req2[12]);
	MPI_Isend(sendList_xZ, sendCount_xZ,MPI_INT,rank_Xz,sendtag,comm,&req1[13]);
	MPI_Irecv(recvList_Xz, recvCount_Xz,MPI_INT,rank_xZ,recvtag,comm,&req2[13]);

	MPI_Isend(sendList_yz, sendCount_yz,MPI_INT,rank_YZ,sendtag,comm,&req1[14]);
	MPI_Irecv(recvList_YZ, recvCount_YZ,MPI_INT,rank_yz,recvtag,comm,&req2[14]);
	MPI_Isend(sendList_YZ, sendCount_YZ,MPI_INT,rank_yz,sendtag,comm,&req1[15]);
	MPI_Irecv(recvList_yz, recvCount_yz,MPI_INT,rank_YZ,recvtag,comm,&req2[15]);
	MPI_Isend(sendList_Yz, sendCount_Yz,MPI_INT,rank_yZ,sendtag,comm,&req1[16]);
	MPI_Irecv(recvList_yZ, recvCount_yZ,MPI_INT,rank_Yz,recvtag,comm,&req2[16]);
	MPI_Isend(sendList_yZ, sendCount_yZ,MPI_INT,rank_Yz,sendtag,comm,&req1[17]);
	MPI_Irecv(recvList_Yz, recvCount_Yz,MPI_INT,rank_yZ,recvtag,comm,&req2[17]);
	MPI_Waitall(18,req1,stat1);
	MPI_Waitall(18,req2,stat2);
	MPI_Barrier(comm);
	//......................................................................................
	double *sendbuf_x, *sendbuf_y, *sendbuf_z, *sendbuf_X, *sendbuf_Y, *sendbuf_Z;
	double *sendbuf_xy, *sendbuf_yz, *sendbuf_xz, *sendbuf_Xy, *sendbuf_Yz, *sendbuf_xZ;
	double *sendbuf_xY, *sendbuf_yZ, *sendbuf_Xz, *sendbuf_XY, *sendbuf_YZ, *sendbuf_XZ;
	double *recvbuf_x, *recvbuf_y, *recvbuf_z, *recvbuf_X, *recvbuf_Y, *recvbuf_Z;
	double *recvbuf_xy, *recvbuf_yz, *recvbuf_xz, *recvbuf_Xy, *recvbuf_Yz, *recvbuf_xZ;
	double *recvbuf_xY, *recvbuf_yZ, *recvbuf_Xz, *recvbuf_XY, *recvbuf_YZ, *recvbuf_XZ;
	//......................................................................................
	hipMalloc((void **) &sendbuf_x, 5*sendCount_x*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_X, 5*sendCount_X*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_y, 5*sendCount_y*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_Y, 5*sendCount_Y*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_z, 5*sendCount_z*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_Z, 5*sendCount_Z*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_xy, sendCount_xy*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_xY, sendCount_xY*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_Xy, sendCount_Xy*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_XY, sendCount_XY*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_xz, sendCount_xz*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_xZ, sendCount_xZ*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_Xz, sendCount_Xz*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_XZ, sendCount_XZ*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_yz, sendCount_yz*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_yZ, sendCount_yZ*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_Yz, sendCount_Yz*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &sendbuf_YZ, sendCount_YZ*sizeof(double));	// Allocate device memory
	//......................................................................................
	hipMalloc((void **) &recvbuf_x, 5*recvCount_x*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_X, 5*recvCount_X*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_y, 5*recvCount_y*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_Y, 5*recvCount_Y*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_z, 5*recvCount_z*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_Z, 5*recvCount_Z*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_xy, recvCount_xy*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_xY, recvCount_xY*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_Xy, recvCount_Xy*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_XY, recvCount_XY*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_xz, recvCount_xz*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_xZ, recvCount_xZ*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_Xz, recvCount_Xz*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_XZ, recvCount_XZ*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_yz, recvCount_yz*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_yZ, recvCount_yZ*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_Yz, recvCount_Yz*sizeof(double));	// Allocate device memory
	hipMalloc((void **) &recvbuf_YZ, recvCount_YZ*sizeof(double));	// Allocate device memory
	//......................................................................................
	int *dvcSendList_x, *dvcSendList_y, *dvcSendList_z, *dvcSendList_X, *dvcSendList_Y, *dvcSendList_Z;
	int *dvcSendList_xy, *dvcSendList_yz, *dvcSendList_xz, *dvcSendList_Xy, *dvcSendList_Yz, *dvcSendList_xZ;
	int *dvcSendList_xY, *dvcSendList_yZ, *dvcSendList_Xz, *dvcSendList_XY, *dvcSendList_YZ, *dvcSendList_XZ;
	//......................................................................................
	int *dvcRecvList_x, *dvcRecvList_y, *dvcRecvList_z, *dvcRecvList_X, *dvcRecvList_Y, *dvcRecvList_Z;
	int *dvcRecvList_xy, *dvcRecvList_yz, *dvcRecvList_xz, *dvcRecvList_Xy, *dvcRecvList_Yz, *dvcRecvList_xZ;
	int *dvcRecvList_xY, *dvcRecvList_yZ, *dvcRecvList_Xz, *dvcRecvList_XY, *dvcRecvList_YZ, *dvcRecvList_XZ;
	//......................................................................................
	hipMalloc((void **) &dvcSendList_x, sendCount_x*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_X, sendCount_X*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_y, sendCount_y*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_Y, sendCount_Y*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_z, sendCount_z*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_Z, sendCount_Z*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_xy, sendCount_xy*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_xY, sendCount_xY*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_Xy, sendCount_Xy*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_XY, sendCount_XY*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_xz, sendCount_xz*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_xZ, sendCount_xZ*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_Xz, sendCount_Xz*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_XZ, sendCount_XZ*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_yz, sendCount_yz*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_yZ, sendCount_yZ*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_Yz, sendCount_Yz*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcSendList_YZ, sendCount_YZ*sizeof(int));	// Allocate device memory
	//......................................................................................
	hipMalloc((void **) &dvcRecvList_x, recvCount_x*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_X, recvCount_X*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_y, recvCount_y*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_Y, recvCount_Y*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_z, recvCount_z*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_Z, recvCount_Z*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_xy, recvCount_xy*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_xY, recvCount_xY*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_Xy, recvCount_Xy*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_XY, recvCount_XY*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_xz, recvCount_xz*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_xZ, recvCount_xZ*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_Xz, recvCount_Xz*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_XZ, recvCount_XZ*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_yz, recvCount_yz*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_yZ, recvCount_yZ*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_Yz, recvCount_Yz*sizeof(int));	// Allocate device memory
	hipMalloc((void **) &dvcRecvList_YZ, recvCount_YZ*sizeof(int));	// Allocate device memory
	//......................................................................................
	if (rank==0)	printf ("Prepare to copy send/recv Lists to device \n");
	hipMemcpy(dvcSendList_x,sendList_x,sendCount_x*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_X,sendList_X,sendCount_X*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_y,sendList_y,sendCount_y*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_Y,sendList_Y,sendCount_Y*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_z,sendList_z,sendCount_z*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_Z,sendList_Z,sendCount_Z*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_xy,sendList_xy,sendCount_xy*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_XY,sendList_XY,sendCount_XY*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_xY,sendList_xY,sendCount_xY*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_Xy,sendList_Xy,sendCount_Xy*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_xz,sendList_xz,sendCount_xz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_XZ,sendList_XZ,sendCount_XZ*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_xZ,sendList_xZ,sendCount_xZ*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_Xz,sendList_Xz,sendCount_Xz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_yz,sendList_yz,sendCount_yz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_YZ,sendList_YZ,sendCount_YZ*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_yZ,sendList_yZ,sendCount_yZ*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcSendList_Yz,sendList_Yz,sendCount_Yz*sizeof(int),hipMemcpyHostToDevice);
	//......................................................................................
	hipMemcpy(dvcRecvList_x,recvList_x,recvCount_x*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_X,recvList_X,recvCount_X*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_y,recvList_y,recvCount_y*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_Y,recvList_Y,recvCount_Y*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_z,recvList_z,recvCount_z*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_Z,recvList_Z,recvCount_Z*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_xy,recvList_xy,recvCount_xy*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_XY,recvList_XY,recvCount_XY*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_xY,recvList_xY,recvCount_xY*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_Xy,recvList_Xy,recvCount_Xy*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_xz,recvList_xz,recvCount_xz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_XZ,recvList_XZ,recvCount_XZ*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_xZ,recvList_xZ,recvCount_xZ*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_Xz,recvList_Xz,recvCount_Xz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_yz,recvList_yz,recvCount_yz*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_YZ,recvList_YZ,recvCount_YZ*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_yZ,recvList_yZ,recvCount_yZ*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dvcRecvList_Yz,recvList_Yz,recvCount_Yz*sizeof(int),hipMemcpyHostToDevice);
	//......................................................................................
	// Fill in the phase ID from neighboring processors
	char *sendID_x, *sendID_y, *sendID_z, *sendID_X, *sendID_Y, *sendID_Z;
	char *sendID_xy, *sendID_yz, *sendID_xz, *sendID_Xy, *sendID_Yz, *sendID_xZ;
	char *sendID_xY, *sendID_yZ, *sendID_Xz, *sendID_XY, *sendID_YZ, *sendID_XZ;
	char *recvID_x, *recvID_y, *recvID_z, *recvID_X, *recvID_Y, *recvID_Z;
	char *recvID_xy, *recvID_yz, *recvID_xz, *recvID_Xy, *recvID_Yz, *recvID_xZ;
	char *recvID_xY, *recvID_yZ, *recvID_Xz, *recvID_XY, *recvID_YZ, *recvID_XZ;
	// send buffers
	sendID_x = new char [sendCount_x];
	sendID_y = new char [sendCount_y];
	sendID_z = new char [sendCount_z];
	sendID_X = new char [sendCount_X];
	sendID_Y = new char [sendCount_Y];
	sendID_Z = new char [sendCount_Z];
	sendID_xy = new char [sendCount_xy];
	sendID_yz = new char [sendCount_yz];
	sendID_xz = new char [sendCount_xz];
	sendID_Xy = new char [sendCount_Xy];
	sendID_Yz = new char [sendCount_Yz];
	sendID_xZ = new char [sendCount_xZ];
	sendID_xY = new char [sendCount_xY];
	sendID_yZ = new char [sendCount_yZ];
	sendID_Xz = new char [sendCount_Xz];
	sendID_XY = new char [sendCount_XY];
	sendID_YZ = new char [sendCount_YZ];
	sendID_XZ = new char [sendCount_XZ];
	//......................................................................................
	// recv buffers
	recvID_x = new char [recvCount_x];
	recvID_y = new char [recvCount_y];
	recvID_z = new char [recvCount_z];
	recvID_X = new char [recvCount_X];
	recvID_Y = new char [recvCount_Y];
	recvID_Z = new char [recvCount_Z];
	recvID_xy = new char [recvCount_xy];
	recvID_yz = new char [recvCount_yz];
	recvID_xz = new char [recvCount_xz];
	recvID_Xy = new char [recvCount_Xy];
	recvID_xZ = new char [recvCount_xZ];
	recvID_xY = new char [recvCount_xY];
	recvID_yZ = new char [recvCount_yZ];
	recvID_Yz = new char [recvCount_Yz];
	recvID_Xz = new char [recvCount_Xz];
	recvID_XY = new char [recvCount_XY];
	recvID_YZ = new char [recvCount_YZ];
	recvID_XZ = new char [recvCount_XZ];
	//......................................................................................
	sendtag = recvtag = 7;
	PackID(sendList_x, sendCount_x ,sendID_x, id);
	PackID(sendList_X, sendCount_X ,sendID_X, id);
	PackID(sendList_y, sendCount_y ,sendID_y, id);
	PackID(sendList_Y, sendCount_Y ,sendID_Y, id);
	PackID(sendList_z, sendCount_z ,sendID_z, id);
	PackID(sendList_Z, sendCount_Z ,sendID_Z, id);
	PackID(sendList_xy, sendCount_xy ,sendID_xy, id);
	PackID(sendList_Xy, sendCount_Xy ,sendID_Xy, id);
	PackID(sendList_xY, sendCount_xY ,sendID_xY, id);
	PackID(sendList_XY, sendCount_XY ,sendID_XY, id);
	PackID(sendList_xz, sendCount_xz ,sendID_xz, id);
	PackID(sendList_Xz, sendCount_Xz ,sendID_Xz, id);
	PackID(sendList_xZ, sendCount_xZ ,sendID_xZ, id);
	PackID(sendList_XZ, sendCount_XZ ,sendID_XZ, id);
	PackID(sendList_yz, sendCount_yz ,sendID_yz, id);
	PackID(sendList_Yz, sendCount_Yz ,sendID_Yz, id);
	PackID(sendList_yZ, sendCount_yZ ,sendID_yZ, id);
	PackID(sendList_YZ, sendCount_YZ ,sendID_YZ, id);
	//......................................................................................
	MPI_Sendrecv(sendID_x,sendCount_x,MPI_CHAR,rank_X,sendtag,
			recvID_X,recvCount_X,MPI_CHAR,rank_x,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_X,sendCount_X,MPI_CHAR,rank_x,sendtag,
			recvID_x,recvCount_x,MPI_CHAR,rank_X,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_y,sendCount_y,MPI_CHAR,rank_Y,sendtag,
			recvID_Y,recvCount_Y,MPI_CHAR,rank_y,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_Y,sendCount_Y,MPI_CHAR,rank_y,sendtag,
			recvID_y,recvCount_y,MPI_CHAR,rank_Y,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_z,sendCount_z,MPI_CHAR,rank_Z,sendtag,
			recvID_Z,recvCount_Z,MPI_CHAR,rank_z,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_Z,sendCount_Z,MPI_CHAR,rank_z,sendtag,
			recvID_z,recvCount_z,MPI_CHAR,rank_Z,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_xy,sendCount_xy,MPI_CHAR,rank_XY,sendtag,
			recvID_XY,recvCount_XY,MPI_CHAR,rank_xy,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_XY,sendCount_XY,MPI_CHAR,rank_xy,sendtag,
			recvID_xy,recvCount_xy,MPI_CHAR,rank_XY,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_Xy,sendCount_Xy,MPI_CHAR,rank_xY,sendtag,
			recvID_xY,recvCount_xY,MPI_CHAR,rank_Xy,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_xY,sendCount_xY,MPI_CHAR,rank_Xy,sendtag,
			recvID_Xy,recvCount_Xy,MPI_CHAR,rank_xY,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_xz,sendCount_xz,MPI_CHAR,rank_XZ,sendtag,
			recvID_XZ,recvCount_XZ,MPI_CHAR,rank_xz,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_XZ,sendCount_XZ,MPI_CHAR,rank_xz,sendtag,
			recvID_xz,recvCount_xz,MPI_CHAR,rank_XZ,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_Xz,sendCount_Xz,MPI_CHAR,rank_xZ,sendtag,
			recvID_xZ,recvCount_xZ,MPI_CHAR,rank_Xz,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_xZ,sendCount_xZ,MPI_CHAR,rank_Xz,sendtag,
			recvID_Xz,recvCount_Xz,MPI_CHAR,rank_xZ,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_yz,sendCount_yz,MPI_CHAR,rank_YZ,sendtag,
			recvID_YZ,recvCount_YZ,MPI_CHAR,rank_yz,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_YZ,sendCount_YZ,MPI_CHAR,rank_yz,sendtag,
			recvID_yz,recvCount_yz,MPI_CHAR,rank_YZ,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_Yz,sendCount_Yz,MPI_CHAR,rank_yZ,sendtag,
			recvID_yZ,recvCount_yZ,MPI_CHAR,rank_Yz,recvtag,comm,MPI_STATUS_IGNORE);
	MPI_Sendrecv(sendID_yZ,sendCount_yZ,MPI_CHAR,rank_Yz,sendtag,
			recvID_Yz,recvCount_Yz,MPI_CHAR,rank_yZ,recvtag,comm,MPI_STATUS_IGNORE);
	//......................................................................................
	UnpackID(recvList_x, recvCount_x ,recvID_x, id);
	UnpackID(recvList_X, recvCount_X ,recvID_X, id);
	UnpackID(recvList_y, recvCount_y ,recvID_y, id);
	UnpackID(recvList_Y, recvCount_Y ,recvID_Y, id);
	UnpackID(recvList_z, recvCount_z ,recvID_z, id);
	UnpackID(recvList_Z, recvCount_Z ,recvID_Z, id);
	UnpackID(recvList_xy, recvCount_xy ,recvID_xy, id);
	UnpackID(recvList_Xy, recvCount_Xy ,recvID_Xy, id);
	UnpackID(recvList_xY, recvCount_xY ,recvID_xY, id);
	UnpackID(recvList_XY, recvCount_XY ,recvID_XY, id);
	UnpackID(recvList_xz, recvCount_xz ,recvID_xz, id);
	UnpackID(recvList_Xz, recvCount_Xz ,recvID_Xz, id);
	UnpackID(recvList_xZ, recvCount_xZ ,recvID_xZ, id);
	UnpackID(recvList_XZ, recvCount_XZ ,recvID_XZ, id);
	UnpackID(recvList_yz, recvCount_yz ,recvID_yz, id);
	UnpackID(recvList_Yz, recvCount_Yz ,recvID_Yz, id);
	UnpackID(recvList_yZ, recvCount_yZ ,recvID_yZ, id);
	UnpackID(recvList_YZ, recvCount_YZ ,recvID_YZ, id);
	//.....................................................................................
	// Once the ID is saved, free memory allocated to the buffers (no longer needed)
	//......................................................................................
	free(sendID_x); free(sendID_X); free(sendID_y); free(sendID_Y); free(sendID_z); free(sendID_Z);
	free(sendID_xy); free(sendID_XY); free(sendID_xY); free(sendID_Xy);
	free(sendID_xz); free(sendID_XZ); free(sendID_xZ); free(sendID_Xz);
	free(sendID_yz); free(sendID_YZ); free(sendID_yZ); free(sendID_Yz);
	free(recvID_x); free(recvID_X); free(recvID_y); free(recvID_Y); free(recvID_z); free(recvID_Z);
	free(recvID_xy); free(recvID_XY); free(recvID_xY); free(recvID_Xy);
	free(recvID_xz); free(recvID_XZ); free(recvID_xZ); free(recvID_Xz);
	free(recvID_yz); free(recvID_YZ); free(recvID_yZ); free(recvID_Yz);
	//......................................................................................
	if (rank==0)	printf ("Devices are ready to communicate. \n");
	MPI_Barrier(comm);

	//...........device phase ID.................................................
	if (rank==0)	printf ("Copying phase ID to device \n");
	char *ID;
	hipMalloc((void **) &ID, N);						// Allocate device memory
	// Copy to the device
	hipMemcpy(ID, id, N, hipMemcpyHostToDevice);
	//...........................................................................

	if (rank==0)	printf ("Allocating distributions \n");
	//......................device distributions.................................
	double *f_even,*f_odd;
	//...........................................................................
	hipMalloc((void **) &f_even, 10*dist_mem_size);	// Allocate device memory
	hipMalloc((void **) &f_odd, 9*dist_mem_size);		// Allocate device memory
	//...........................................................................

	if (rank==0)	printf("Setting the distributions, size = : %i\n", N);
	//...........................................................................
	INITIALIZE <<< grid, nthreads >>>  (ID, f_even, f_odd, Nx, Ny, Nz, S);
	//...........................................................................

	//...........................................................................
	// Grids used to pack faces on the GPU for MPI
	int faceGrid,edgeGrid,packThreads;
	packThreads=512;
	edgeGrid=1;
	faceGrid=Nx*Ny/packThreads;
	//...........................................................................

	int iter = 0;
	if (rank==0)	printf("No. of iterations: %i \n", iterMax);
	
	//.......create a stream for the LB calculation.......
	hipStream_t stream;
	hipStreamCreate(&stream);
	
	//.......create and start timer............
	double starttime,stoptime,cputime;
	MPI_Barrier(comm);
	starttime = MPI_Wtime();
	// Old cuda timer is below
//	hipEvent_t start, stop;
//	float time;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//	hipEventRecord( start, 0 );
	//.........................................

	sendtag = recvtag = 5;

	//************ MAIN ITERATION LOOP ***************************************/
	while (iter < iterMax){
		//...................................................................................
		PackDist<<<faceGrid,packThreads>>>(1,dvcSendList_x,0,sendCount_x,sendbuf_x,f_even,N);
		PackDist<<<faceGrid,packThreads>>>(4,dvcSendList_x,sendCount_x,sendCount_x,sendbuf_x,f_even,N);
		PackDist<<<faceGrid,packThreads>>>(5,dvcSendList_x,2*sendCount_x,sendCount_x,sendbuf_x,f_even,N);
		PackDist<<<faceGrid,packThreads>>>(6,dvcSendList_x,3*sendCount_x,sendCount_x,sendbuf_x,f_even,N);
		PackDist<<<faceGrid,packThreads>>>(7,dvcSendList_x,4*sendCount_x,sendCount_x,sendbuf_x,f_even,N);
		//...Packing for X face<<<faceGrid,packThreads>>>(1,7,9,11,13)................................
		PackDist<<<faceGrid,packThreads>>>(0,dvcSendList_X,0,sendCount_X,sendbuf_X,f_odd,N);
		PackDist<<<faceGrid,packThreads>>>(3,dvcSendList_X,sendCount_X,sendCount_X,sendbuf_X,f_odd,N);
		PackDist<<<faceGrid,packThreads>>>(4,dvcSendList_X,2*sendCount_X,sendCount_X,sendbuf_X,f_odd,N);
		PackDist<<<faceGrid,packThreads>>>(5,dvcSendList_X,3*sendCount_X,sendCount_X,sendbuf_X,f_odd,N);
		PackDist<<<faceGrid,packThreads>>>(6,dvcSendList_X,4*sendCount_X,sendCount_X,sendbuf_X,f_odd,N);
		//...Packing for y face<<<faceGrid,packThreads>>>(4,8,9,16,18).................................
		PackDist<<<faceGrid,packThreads>>>(2,dvcSendList_y,0,sendCount_y,sendbuf_y,f_even,N);
		PackDist<<<faceGrid,packThreads>>>(4,dvcSendList_y,sendCount_y,sendCount_y,sendbuf_y,f_even,N);
		PackDist<<<faceGrid,packThreads>>>(4,dvcSendList_y,2*sendCount_y,sendCount_y,sendbuf_y,f_odd,N);
		PackDist<<<faceGrid,packThreads>>>(8,dvcSendList_y,3*sendCount_y,sendCount_y,sendbuf_y,f_even,N);
		PackDist<<<faceGrid,packThreads>>>(9,dvcSendList_y,4*sendCount_y,sendCount_y,sendbuf_y,f_even,N);
		//...Packing for Y face<<<faceGrid,packThreads>>>(3,7,10,15,17).................................
		PackDist<<<faceGrid,packThreads>>>(1,dvcSendList_Y,0,sendCount_Y,sendbuf_Y,f_odd,N);
		PackDist<<<faceGrid,packThreads>>>(3,dvcSendList_Y,sendCount_Y,sendCount_Y,sendbuf_Y,f_odd,N);
		PackDist<<<faceGrid,packThreads>>>(5,dvcSendList_Y,2*sendCount_Y,sendCount_Y,sendbuf_Y,f_even,N);
		PackDist<<<faceGrid,packThreads>>>(7,dvcSendList_Y,3*sendCount_Y,sendCount_Y,sendbuf_Y,f_odd,N);
		PackDist<<<faceGrid,packThreads>>>(8,dvcSendList_Y,4*sendCount_Y,sendCount_Y,sendbuf_Y,f_odd,N);
		//...Packing for z face<<<faceGrid,packThreads>>>(6,12,13,16,17)................................
		PackDist<<<faceGrid,packThreads>>>(3,dvcSendList_z,0,sendCount_z,sendbuf_z,f_even,N);
		PackDist<<<faceGrid,packThreads>>>(6,dvcSendList_z,sendCount_z,sendCount_z,sendbuf_z,f_even,N);
		PackDist<<<faceGrid,packThreads>>>(6,dvcSendList_z,2*sendCount_z,sendCount_z,sendbuf_z,f_odd,N);
		PackDist<<<faceGrid,packThreads>>>(8,dvcSendList_z,3*sendCount_z,sendCount_z,sendbuf_z,f_even,N);
		PackDist<<<faceGrid,packThreads>>>(8,dvcSendList_z,4*sendCount_z,sendCount_z,sendbuf_z,f_odd,N);
		//...Packing for Z face<<<faceGrid,packThreads>>>(5,11,14,15,18)................................
		PackDist<<<faceGrid,packThreads>>>(2,dvcSendList_Z,0,sendCount_Z,sendbuf_Z,f_odd,N);
		PackDist<<<faceGrid,packThreads>>>(5,dvcSendList_Z,sendCount_Z,sendCount_Z,sendbuf_Z,f_odd,N);
		PackDist<<<faceGrid,packThreads>>>(7,dvcSendList_Z,2*sendCount_Z,sendCount_Z,sendbuf_Z,f_even,N);
		PackDist<<<faceGrid,packThreads>>>(7,dvcSendList_Z,3*sendCount_Z,sendCount_Z,sendbuf_Z,f_odd,N);
		PackDist<<<faceGrid,packThreads>>>(9,dvcSendList_Z,4*sendCount_Z,sendCount_Z,sendbuf_Z,f_even,N);
		//...Pack the xy edge <<<edgeGrid,packThreads>>>(8)................................
		PackDist<<<edgeGrid,packThreads>>>(4,dvcSendList_xy,0,sendCount_xy,sendbuf_xy,f_even,N);
		//...Pack the Xy edge <<<edgeGrid,packThreads>>>(9)................................
		PackDist<<<edgeGrid,packThreads>>>(4,dvcSendList_Xy,0,sendCount_Xy,sendbuf_Xy,f_odd,N);
		//...Pack the xY edge <<<edgeGrid,packThreads>>>(10)................................
		PackDist<<<edgeGrid,packThreads>>>(5,dvcSendList_xY,0,sendCount_xY,sendbuf_xY,f_even,N);
		//...Pack the XY edge <<<edgeGrid,packThreads>>>(7)................................
		PackDist<<<edgeGrid,packThreads>>>(3,dvcSendList_XY,0,sendCount_XY,sendbuf_XY,f_odd,N);
		//...Pack the xz edge <<<edgeGrid,packThreads>>>(12)................................
		PackDist<<<edgeGrid,packThreads>>>(6,dvcSendList_xz,0,sendCount_xz,sendbuf_xz,f_even,N);
		//...Pack the xZ edge <<<edgeGrid,packThreads>>>(14)................................
		PackDist<<<edgeGrid,packThreads>>>(7,dvcSendList_xZ,0,sendCount_xZ,sendbuf_xZ,f_even,N);
		//...Pack the Xz edge <<<edgeGrid,packThreads>>>(13)................................
		PackDist<<<edgeGrid,packThreads>>>(6,dvcSendList_Xz,0,sendCount_Xz,sendbuf_Xz,f_odd,N);
		//...Pack the XZ edge <<<edgeGrid,packThreads>>>(11)................................
		PackDist<<<edgeGrid,packThreads>>>(5,dvcSendList_XZ,0,sendCount_XZ,sendbuf_XZ,f_odd,N);
		//...Pack the xz edge <<<edgeGrid,packThreads>>>(12)................................
		PackDist<<<edgeGrid,packThreads>>>(6,dvcSendList_xz,0,sendCount_xz,sendbuf_xz,f_even,N);
		//...Pack the xZ edge <<<edgeGrid,packThreads>>>(14)................................
		PackDist<<<edgeGrid,packThreads>>>(7,dvcSendList_xZ,0,sendCount_xZ,sendbuf_xZ,f_even,N);
		//...Pack the Xz edge <<<edgeGrid,packThreads>>>(13)................................
		PackDist<<<edgeGrid,packThreads>>>(6,dvcSendList_Xz,0,sendCount_Xz,sendbuf_Xz,f_odd,N);
		//...Pack the XZ edge <<<edgeGrid,packThreads>>>(11)................................
		PackDist<<<edgeGrid,packThreads>>>(5,dvcSendList_XZ,0,sendCount_XZ,sendbuf_XZ,f_odd,N);
		//...Pack the yz edge <<<edgeGrid,packThreads>>>(16)................................
		PackDist<<<edgeGrid,packThreads>>>(8,dvcSendList_yz,0,sendCount_yz,sendbuf_yz,f_even,N);
		//...Pack the yZ edge <<<edgeGrid,packThreads>>>(18)................................
		PackDist<<<edgeGrid,packThreads>>>(9,dvcSendList_yZ,0,sendCount_yZ,sendbuf_yZ,f_even,N);
		//...Pack the Yz edge <<<edgeGrid,packThreads>>>(17)................................
		PackDist<<<edgeGrid,packThreads>>>(8,dvcSendList_Yz,0,sendCount_Yz,sendbuf_Yz,f_odd,N);
		//...Pack the YZ edge <<<edgeGrid,packThreads>>>(15)................................
		PackDist<<<edgeGrid,packThreads>>>(7,dvcSendList_YZ,0,sendCount_YZ,sendbuf_YZ,f_odd,N);
		//...................................................................................

		//*****************************************************************************
		//*****************************************************************************
		//*****************************************************************************
		//........ Execute the swap kernel (device) .........................
		//*****************************************************************************
		//*****************************************************************************
		SWAP <<< grid, nthreads >>> (ID, f_even, f_odd, Nx, Ny, Nz, S);
		//*****************************************************************************
		//*****************************************************************************
		//*****************************************************************************
		//*****************************************************************************

		//...................................................................................
		// Send all the distributions
		MPI_Isend(sendbuf_x, 5*sendCount_x,MPI_DOUBLE,rank_X,sendtag,comm,&req1[0]);
		MPI_Irecv(recvbuf_X, 5*recvCount_X,MPI_DOUBLE,rank_x,recvtag,comm,&req2[0]);
		MPI_Isend(sendbuf_X, 5*sendCount_X,MPI_DOUBLE,rank_x,sendtag,comm,&req1[1]);
		MPI_Irecv(recvbuf_x, 5*recvCount_x,MPI_DOUBLE,rank_X,recvtag,comm,&req2[1]);
		MPI_Isend(sendbuf_y, 5*sendCount_y,MPI_DOUBLE,rank_Y,sendtag,comm,&req1[2]);
		MPI_Irecv(recvbuf_Y, 5*recvCount_Y,MPI_DOUBLE,rank_y,recvtag,comm,&req2[2]);
		MPI_Isend(sendbuf_Y, 5*sendCount_Y,MPI_DOUBLE,rank_y,sendtag,comm,&req1[3]);
		MPI_Irecv(recvbuf_y, 5*recvCount_y,MPI_DOUBLE,rank_Y,recvtag,comm,&req2[3]);
		MPI_Isend(sendbuf_z, 5*sendCount_z,MPI_DOUBLE,rank_Z,sendtag,comm,&req1[4]);
		MPI_Irecv(recvbuf_Z, 5*recvCount_Z,MPI_DOUBLE,rank_z,recvtag,comm,&req2[4]);
		MPI_Isend(sendbuf_Z, 5*sendCount_Z,MPI_DOUBLE,rank_z,sendtag,comm,&req1[5]);
		MPI_Irecv(recvbuf_z, 5*recvCount_z,MPI_DOUBLE,rank_Z,recvtag,comm,&req2[5]);
		MPI_Isend(sendbuf_xy, sendCount_xy,MPI_DOUBLE,rank_XY,sendtag,comm,&req1[6]);
		MPI_Irecv(recvbuf_XY, recvCount_XY,MPI_DOUBLE,rank_xy,recvtag,comm,&req2[6]);
		MPI_Isend(sendbuf_XY, sendCount_XY,MPI_DOUBLE,rank_xy,sendtag,comm,&req1[7]);
		MPI_Irecv(recvbuf_xy, recvCount_xy,MPI_DOUBLE,rank_XY,recvtag,comm,&req2[7]);
		MPI_Isend(sendbuf_Xy, sendCount_Xy,MPI_DOUBLE,rank_xY,sendtag,comm,&req1[8]);
		MPI_Irecv(recvbuf_xY, recvCount_xY,MPI_DOUBLE,rank_Xy,recvtag,comm,&req2[8]);
		MPI_Isend(sendbuf_xY, sendCount_xY,MPI_DOUBLE,rank_Xy,sendtag,comm,&req1[9]);
		MPI_Irecv(recvbuf_Xy, recvCount_Xy,MPI_DOUBLE,rank_xY,recvtag,comm,&req2[9]);
		MPI_Isend(sendbuf_xz, sendCount_xz,MPI_DOUBLE,rank_XZ,sendtag,comm,&req1[10]);
		MPI_Irecv(recvbuf_XZ, recvCount_XZ,MPI_DOUBLE,rank_xz,recvtag,comm,&req2[10]);
		MPI_Isend(sendbuf_XZ, sendCount_XZ,MPI_DOUBLE,rank_xz,sendtag,comm,&req1[11]);
		MPI_Irecv(recvbuf_xz, recvCount_xz,MPI_DOUBLE,rank_XZ,recvtag,comm,&req2[11]);
		MPI_Isend(sendbuf_Xz, sendCount_Xz,MPI_DOUBLE,rank_xZ,sendtag,comm,&req1[12]);
		MPI_Irecv(recvbuf_xZ, recvCount_xZ,MPI_DOUBLE,rank_Xz,recvtag,comm,&req2[12]);
		MPI_Isend(sendbuf_xZ, sendCount_xZ,MPI_DOUBLE,rank_Xz,sendtag,comm,&req1[13]);
		MPI_Irecv(recvbuf_Xz, recvCount_Xz,MPI_DOUBLE,rank_xZ,recvtag,comm,&req2[13]);
		MPI_Isend(sendbuf_yz, sendCount_yz,MPI_DOUBLE,rank_YZ,sendtag,comm,&req1[14]);
		MPI_Irecv(recvbuf_YZ, recvCount_YZ,MPI_DOUBLE,rank_yz,recvtag,comm,&req2[14]);
		MPI_Isend(sendbuf_YZ, sendCount_YZ,MPI_DOUBLE,rank_yz,sendtag,comm,&req1[15]);
		MPI_Irecv(recvbuf_yz, recvCount_yz,MPI_DOUBLE,rank_YZ,recvtag,comm,&req2[15]);
		MPI_Isend(sendbuf_Yz, sendCount_Yz,MPI_DOUBLE,rank_yZ,sendtag,comm,&req1[16]);
		MPI_Irecv(recvbuf_yZ, recvCount_yZ,MPI_DOUBLE,rank_Yz,recvtag,comm,&req2[16]);
		MPI_Isend(sendbuf_yZ, sendCount_yZ,MPI_DOUBLE,rank_Yz,sendtag,comm,&req1[17]);
		MPI_Irecv(recvbuf_Yz, recvCount_Yz,MPI_DOUBLE,rank_yZ,recvtag,comm,&req2[17]);
		//...................................................................................

		//...................................................................................
		// Wait for completion of D3Q19 communication
		MPI_Waitall(18,req1,stat1);
		MPI_Waitall(18,req2,stat2);
		//...................................................................................
		// Unpack the distributions on the device
		//...................................................................................
		//...Map recieve list for the X face: q=2,8,10,12,13 .................................
		MapRecvDist<<<faceGrid,packThreads>>>(0,-1,0,0,dvcRecvList_X,0,recvCount_X,recvbuf_X,f_odd,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(3,-1,-1,0,dvcRecvList_X,recvCount_X,recvCount_X,recvbuf_X,f_odd,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(4,-1,1,0,dvcRecvList_X,2*recvCount_X,recvCount_X,recvbuf_X,f_odd,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(5,-1,0,-1,dvcRecvList_X,3*recvCount_X,recvCount_X,recvbuf_X,f_odd,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(6,-1,0,1,dvcRecvList_X,4*recvCount_X,recvCount_X,recvbuf_X,f_odd,Nx,Ny,Nz);
		//...................................................................................
		//...Map recieve list for the x face: q=1,7,9,11,13..................................
		MapRecvDist<<<faceGrid,packThreads>>>(1,1,0,0,dvcRecvList_x,0,recvCount_x,recvbuf_x,f_even,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(4,1,1,0,dvcRecvList_x,recvCount_x,recvCount_x,recvbuf_x,f_even,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(5,1,-1,0,dvcRecvList_x,2*recvCount_x,recvCount_x,recvbuf_x,f_even,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(6,1,0,1,dvcRecvList_x,3*recvCount_x,recvCount_x,recvbuf_x,f_even,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(7,1,0,-1,dvcRecvList_x,4*recvCount_x,recvCount_x,recvbuf_x,f_even,Nx,Ny,Nz);
		//...................................................................................
		//...Map recieve list for the y face: q=4,8,9,16,18 ...................................
		MapRecvDist<<<faceGrid,packThreads>>>(1,0,-1,0,dvcRecvList_Y,0,recvCount_Y,recvbuf_Y,f_odd,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(3,-1,-1,0,dvcRecvList_Y,recvCount_Y,recvCount_Y,recvbuf_Y,f_odd,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(5,1,-1,0,dvcRecvList_Y,2*recvCount_Y,recvCount_Y,recvbuf_Y,f_even,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(7,0,-1,-1,dvcRecvList_Y,3*recvCount_Y,recvCount_Y,recvbuf_Y,f_odd,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(8,0,-1,1,dvcRecvList_Y,4*recvCount_Y,recvCount_Y,recvbuf_Y,f_odd,Nx,Ny,Nz);
		//...................................................................................
		//...Map recieve list for the Y face: q=3,7,10,15,17 ..................................
		MapRecvDist<<<faceGrid,packThreads>>>(2,0,1,0,dvcRecvList_y,0,recvCount_y,recvbuf_y,f_even,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(4,1,1,0,dvcRecvList_y,recvCount_y,recvCount_y,recvbuf_y,f_even,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(4,-1,1,0,dvcRecvList_y,2*recvCount_y,recvCount_y,recvbuf_y,f_odd,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(8,0,1,1,dvcRecvList_y,3*recvCount_y,recvCount_y,recvbuf_y,f_even,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(9,0,1,-1,dvcRecvList_y,4*recvCount_y,recvCount_y,recvbuf_y,f_even,Nx,Ny,Nz);
		//...................................................................................
		//...Map recieve list for the z face<<<faceGrid,packThreads>>>(6,12,13,16,17)..............................................
		MapRecvDist<<<faceGrid,packThreads>>>(2,0,0,-1,dvcRecvList_Z,0,recvCount_Z,recvbuf_Z,f_odd,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(5,-1,0,-1,dvcRecvList_Z,recvCount_Z,recvCount_Z,recvbuf_Z,f_odd,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(7,1,0,-1,dvcRecvList_Z,2*recvCount_Z,recvCount_Z,recvbuf_Z,f_even,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(7,0,-1,-1,dvcRecvList_Z,3*recvCount_Z,recvCount_Z,recvbuf_Z,f_odd,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(9,0,1,-1,dvcRecvList_Z,4*recvCount_Z,recvCount_Z,recvbuf_Z,f_even,Nx,Ny,Nz);
		//...Map recieve list for the Z face<<<faceGrid,packThreads>>>(5,11,14,15,18)..............................................
		MapRecvDist<<<faceGrid,packThreads>>>(3,0,0,1,dvcRecvList_z,0,recvCount_z,recvbuf_z,f_even,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(6,1,0,1,dvcRecvList_z,recvCount_z,recvCount_z,recvbuf_z,f_even,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(6,-1,0,1,dvcRecvList_z,2*recvCount_z,recvCount_z,recvbuf_z,f_odd,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(8,0,1,1,dvcRecvList_z,3*recvCount_z,recvCount_z,recvbuf_z,f_even,Nx,Ny,Nz);
		MapRecvDist<<<faceGrid,packThreads>>>(8,0,-1,1,dvcRecvList_z,4*recvCount_z,recvCount_z,recvbuf_z,f_odd,Nx,Ny,Nz);
		//..................................................................................
		//...Map recieve list for the xy edge <<<edgeGrid,packThreads>>>(8)................................
		MapRecvDist<<<edgeGrid,packThreads>>>(3,-1,-1,0,dvcRecvList_XY,0,recvCount_XY,recvbuf_XY,f_odd,Nx,Ny,Nz);
		//...Map recieve list for the Xy edge <<<edgeGrid,packThreads>>>(9)................................
		MapRecvDist<<<edgeGrid,packThreads>>>(5,1,-1,0,dvcRecvList_xY,0,recvCount_xY,recvbuf_xY,f_even,Nx,Ny,Nz);
		//...Map recieve list for the xY edge <<<edgeGrid,packThreads>>>(10)................................
		MapRecvDist<<<edgeGrid,packThreads>>>(4,-1,1,0,dvcRecvList_Xy,0,recvCount_Xy,recvbuf_Xy,f_odd,Nx,Ny,Nz);
		//...Map recieve list for the XY edge <<<edgeGrid,packThreads>>>(7)................................
		MapRecvDist<<<edgeGrid,packThreads>>>(4,1,1,0,dvcRecvList_xy,0,recvCount_xy,recvbuf_xy,f_even,Nx,Ny,Nz);
		//...Map recieve list for the xz edge <<<edgeGrid,packThreads>>>(12)................................
		MapRecvDist<<<edgeGrid,packThreads>>>(5,-1,0,-1,dvcRecvList_XZ,0,recvCount_XZ,recvbuf_XZ,f_odd,Nx,Ny,Nz);
		//...Map recieve list for the xZ edge <<<edgeGrid,packThreads>>>(14)................................
		MapRecvDist<<<edgeGrid,packThreads>>>(6,-1,0,1,dvcRecvList_Xz,0,recvCount_Xz,recvbuf_Xz,f_odd,Nx,Ny,Nz);
		//...Map recieve list for the Xz edge <<<edgeGrid,packThreads>>>(13)................................
		MapRecvDist<<<edgeGrid,packThreads>>>(7,1,0,-1,dvcRecvList_xZ,0,recvCount_xZ,recvbuf_xZ,f_even,Nx,Ny,Nz);
		//...Map recieve list for the XZ edge <<<edgeGrid,packThreads>>>(11)................................
		MapRecvDist<<<edgeGrid,packThreads>>>(6,1,0,1,dvcRecvList_xz,0,recvCount_xz,recvbuf_xz,f_even,Nx,Ny,Nz);
		//...Map recieve list for the yz edge <<<edgeGrid,packThreads>>>(16)................................
		MapRecvDist<<<edgeGrid,packThreads>>>(7,0,-1,-1,dvcRecvList_YZ,0,recvCount_YZ,recvbuf_YZ,f_odd,Nx,Ny,Nz);
		//...Map recieve list for the yZ edge <<<edgeGrid,packThreads>>>(18)................................
		MapRecvDist<<<edgeGrid,packThreads>>>(8,0,-1,1,dvcRecvList_Yz,0,recvCount_Yz,recvbuf_Yz,f_odd,Nx,Ny,Nz);
		//...Map recieve list for the Yz edge <<<edgeGrid,packThreads>>>(17)................................
		MapRecvDist<<<edgeGrid,packThreads>>>(9,0,1,-1,dvcRecvList_yZ,0,recvCount_yZ,recvbuf_yZ,f_even,Nx,Ny,Nz);
		//...Map recieve list for the YZ edge <<<edgeGrid,packThreads>>>(15)................................
		MapRecvDist<<<edgeGrid,packThreads>>>(8,0,1,1,dvcRecvList_yz,0,recvCount_yz,recvbuf_yz,f_even,Nx,Ny,Nz);
		//...................................................................................

		//*****************************************************************************
		//*****************************************************************************
		//*****************************************************************************
		//........ Execute the collision kernel (device) ....................
		//*****************************************************************************
		//*****************************************************************************
		MRT <<< grid, nthreads >>> (ID, f_even, f_odd, Nx, Ny, Nz, S,
									rlx_setA, rlx_setB, Fx, Fy, Fz);
		//*****************************************************************************
		//*****************************************************************************
		//*****************************************************************************
		//*****************************************************************************

		MPI_Barrier(comm);
		// Iteration completed!
		iter++;
		//...................................................................
	}
	//************************************************************************/
	
	hipDeviceSynchronize();
	MPI_Barrier(comm);
	stoptime = MPI_Wtime();
//	cout << "CPU time: " << (stoptime - starttime) << " seconds" << endl;
	cputime = stoptime - starttime;
//	cout << "Lattice update rate: "<< double(Nx*Ny*Nz*iter)/cputime/1000000 <<  " MLUPS" << endl;
	double MLUPS = double(Nx*Ny*Nz*iter)/cputime/1000000;

	if (rank==0) printf("CPU time = %f \n", cputime);
	if (rank==0) printf("Lattice update rate = %f MLUPS \n", MLUPS);
	//.......... stop and destroy timer.............................
//	hipEventRecord( stop, stream);
//	hipEventSynchronize( stop );
//	hipEventElapsedTime( &time, start, stop );
//	printf("CPU time = %f \n", time);
//	float MLUPS = 0.001*float(Nx*Ny*Nz)*iter/time;
//	printf("MLUPS = %f \n", MLUPS);

	hipStreamDestroy(stream);
//	hipEventDestroy( start );
//	hipEventDestroy( stop );
	//..............................................................
	
	//..............................................................
	//.........Compute the velocity and copy result to host ........
	double *velocity;
	velocity = new double[3*N];
	//......................device distributions....................................
	double *vel;
	//..............................................................................
	hipMalloc((void **) &vel, 3*dist_mem_size);	// Allocate device memory
	//..............................................................................
	Compute_VELOCITY <<< grid, nthreads >>>  (ID, f_even, f_odd, vel, Nx, Ny, Nz, S);
	//..............................................................................
	hipMemcpy(velocity, vel, 3*dist_mem_size, hipMemcpyDeviceToHost);
	//..............................................................................
	hipDeviceSynchronize();
	MPI_Barrier(comm);
	//............................................................	
	//....Write the z-velocity to test poiseuille flow............
	double vz,vz_avg;	
	vz_avg = 0.0;

	FILE *output;
	output = fopen("velocity.out","w");
	for (int k=0; k<1; k++){
		for (int j=0; j<1; j++){
			for (int i=0; i<Nx; i++){
				int n = k*Nx*Ny+j*Nx+i;
				//.....print value........
				vz = velocity[2*N+n];
				vz_avg += vz;
				fprintf(output, " %e",vz);
			}
		}
	}
	fclose(output);
	
	vz = vz_avg/double(sum);
	printf("Average Velocity = %e\n", vz);

	// cleanup	
	hipFree(f_even);	hipFree(f_odd);	hipFree(vel);	hipFree(ID);
	free (velocity);	free(id);
	
	// ****************************************************
	MPI_Barrier(comm);
	MPI_Finalize();
	// ****************************************************
}
