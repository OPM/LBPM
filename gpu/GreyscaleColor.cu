
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define NBLOCKS 1024
#define NTHREADS 256

//Model-1 & 4
__global__ void dvc_ScaLBL_D3Q19_AAodd_GreyscaleColor(int *neighborList, int *Map, double *dist, double *Aq, double *Bq, double *Den,
		 double *Phi, double *GreySolidGrad, double *Poros,double *Perm, double *Velocity, 
         double rhoA, double rhoB, double tauA, double tauB,double tauA_eff,double tauB_eff,double alpha, double beta,
		double Gx, double Gy, double Gz, int strideY, int strideZ, int start, int finish, int Np){

	int n,nn,ijk,nread;
	int nr1,nr2,nr3,nr4,nr5,nr6;
	int nr7,nr8,nr9,nr10;
	int nr11,nr12,nr13,nr14;
	//int nr15,nr16,nr17,nr18;
	double fq;
	// conserved momemnts
	double rho,jx,jy,jz;
	double vx,vy,vz,v_mag;
    double ux,uy,uz,u_mag;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
	double m3,m5,m7;
	double nA,nB; // number density
	double a1,b1,a2,b2,nAB,delta;
	double C,nx,ny,nz; //color gradient magnitude and direction
	double phi,tau,rho0,rlx_setA,rlx_setB;

    double GeoFun=0.0;//geometric function from Guo's PRE 66, 036304 (2002)
    double porosity;
    double perm;//voxel permeability
    double c0, c1; //Guo's model parameters
    double tau_eff;
    double mu_eff;//kinematic viscosity
    double nx_gs,ny_gs,nz_gs;//grey-solid color gradient
    double nx_phase,ny_phase,nz_phase,C_phase;
    double Fx,Fy,Fz;

	const double mrt_V1=0.05263157894736842;
	const double mrt_V2=0.012531328320802;
	const double mrt_V3=0.04761904761904762;
	const double mrt_V4=0.004594820384294068;
	const double mrt_V5=0.01587301587301587;
	const double mrt_V6=0.0555555555555555555555555;
	const double mrt_V7=0.02777777777777778;
	const double mrt_V8=0.08333333333333333;
	const double mrt_V9=0.003341687552213868;
	const double mrt_V10=0.003968253968253968;
	const double mrt_V11=0.01388888888888889;
	const double mrt_V12=0.04166666666666666;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];

            porosity = Poros[n];
            perm = Perm[n];
            nx_gs = GreySolidGrad[n+0*Np];
            ny_gs = GreySolidGrad[n+1*Np];
            nz_gs = GreySolidGrad[n+2*Np];

			// compute phase indicator field
			phi=(nA-nB)/(nA+nB);

			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
			// local relaxation time
			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
			tau_eff=tauA_eff + 0.5*(1.0-phi)*(tauB_eff-tauA_eff);
			rlx_setA = 1.f/tau;
			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);
            mu_eff = (tau_eff-0.5)/3.0;//kinematic viscosity
			
			// Get the 1D index based on regular data layout
			ijk = Map[n];
			//					COMPUTE THE COLOR GRADIENT
			//........................................................................
			//.................Read Phase Indicator Values............................
			//........................................................................
			nn = ijk-1;							// neighbor index (get convention)
			m1 = Phi[nn];						// get neighbor for phi - 1
			//........................................................................
			nn = ijk+1;							// neighbor index (get convention)
			m2 = Phi[nn];						// get neighbor for phi - 2
			//........................................................................
			nn = ijk-strideY;							// neighbor index (get convention)
			m3 = Phi[nn];					// get neighbor for phi - 3
			//........................................................................
			nn = ijk+strideY;							// neighbor index (get convention)
			m4 = Phi[nn];					// get neighbor for phi - 4
			//........................................................................
			nn = ijk-strideZ;						// neighbor index (get convention)
			m5 = Phi[nn];					// get neighbor for phi - 5
			//........................................................................
			nn = ijk+strideZ;						// neighbor index (get convention)
			m6 = Phi[nn];					// get neighbor for phi - 6
			//........................................................................
			nn = ijk-strideY-1;						// neighbor index (get convention)
			m7 = Phi[nn];					// get neighbor for phi - 7
			//........................................................................
			nn = ijk+strideY+1;						// neighbor index (get convention)
			m8 = Phi[nn];					// get neighbor for phi - 8
			//........................................................................
			nn = ijk+strideY-1;						// neighbor index (get convention)
			m9 = Phi[nn];					// get neighbor for phi - 9
			//........................................................................
			nn = ijk-strideY+1;						// neighbor index (get convention)
			m10 = Phi[nn];					// get neighbor for phi - 10
			//........................................................................
			nn = ijk-strideZ-1;						// neighbor index (get convention)
			m11 = Phi[nn];					// get neighbor for phi - 11
			//........................................................................
			nn = ijk+strideZ+1;						// neighbor index (get convention)
			m12 = Phi[nn];					// get neighbor for phi - 12
			//........................................................................
			nn = ijk+strideZ-1;						// neighbor index (get convention)
			m13 = Phi[nn];					// get neighbor for phi - 13
			//........................................................................
			nn = ijk-strideZ+1;						// neighbor index (get convention)
			m14 = Phi[nn];					// get neighbor for phi - 14
			//........................................................................
			nn = ijk-strideZ-strideY;					// neighbor index (get convention)
			m15 = Phi[nn];					// get neighbor for phi - 15
			//........................................................................
			nn = ijk+strideZ+strideY;					// neighbor index (get convention)
			m16 = Phi[nn];					// get neighbor for phi - 16
			//........................................................................
			nn = ijk+strideZ-strideY;					// neighbor index (get convention)
			m17 = Phi[nn];					// get neighbor for phi - 17
			//........................................................................
			nn = ijk-strideZ+strideY;					// neighbor index (get convention)
			m18 = Phi[nn];					// get neighbor for phi - 18
			//............Compute the Color Gradient...................................
			nx_phase = -(m1-m2+0.5*(m7-m8+m9-m10+m11-m12+m13-m14));
			ny_phase = -(m3-m4+0.5*(m7-m8-m9+m10+m15-m16+m17-m18));
			nz_phase = -(m5-m6+0.5*(m11-m12-m13+m14+m15-m16-m17+m18));
			C_phase = sqrt(nx_phase*nx_phase+ny_phase*ny_phase+nz_phase*nz_phase);

            //correct the normal color gradient by considering the effect of grey solid
            nx = nx_phase + (1.0-porosity)*nx_gs; 
            ny = ny_phase + (1.0-porosity)*ny_gs; 
            nz = nz_phase + (1.0-porosity)*nz_gs; 
            if (C_phase==0.0){
                nx = nx_phase; 
                ny = ny_phase;
                nz = nz_phase;
            }

			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			double ColorMag = C;
			if (C==0.0) ColorMag=1.0;
			nx = nx/ColorMag;
			ny = ny/ColorMag;
			nz = nz/ColorMag;		

			// q=0
			fq = dist[n];
			rho = fq;
			m1  = -30.0*fq;
			m2  = 12.0*fq;

			// q=1
			//nread = neighborList[n]; // neighbor 2 
			//fq = dist[nread]; // reading the f1 data into register fq		
			nr1 = neighborList[n]; 
			fq = dist[nr1]; // reading the f1 data into register fq
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jx = fq;
			m4 = -4.0*fq;
			m9 = 2.0*fq;
			m10 = -4.0*fq;

			// f2 = dist[10*Np+n];
			//nread = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
			//fq = dist[nread];  // reading the f2 data into register fq
			nr2 = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
			fq = dist[nr2];  // reading the f2 data into register fq
			rho += fq;
			m1 -= 11.0*(fq);
			m2 -= 4.0*(fq);
			jx -= fq;
			m4 += 4.0*(fq);
			m9 += 2.0*(fq);
			m10 -= 4.0*(fq);

			// q=3
			//nread = neighborList[n+2*Np]; // neighbor 4
			//fq = dist[nread];
			nr3 = neighborList[n+2*Np]; // neighbor 4
			fq = dist[nr3];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy = fq;
			m6 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 = fq;
			m12 = -2.0*fq;

			// q = 4
			//nread = neighborList[n+3*Np]; // neighbor 3
			//fq = dist[nread];
			nr4 = neighborList[n+3*Np]; // neighbor 3
			fq = dist[nr4];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy -= fq;
			m6 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 += fq;
			m12 -= 2.0*fq;

			// q=5
			//nread = neighborList[n+4*Np];
			//fq = dist[nread];
			nr5 = neighborList[n+4*Np];
			fq = dist[nr5];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz = fq;
			m8 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;


			// q = 6
			//nread = neighborList[n+5*Np];
			//fq = dist[nread];
			nr6 = neighborList[n+5*Np];
			fq = dist[nr6];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz -= fq;
			m8 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q=7
			//nread = neighborList[n+6*Np];
			//fq = dist[nread];
			nr7 = neighborList[n+6*Np];
			fq = dist[nr7];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy += fq;
			m6 += fq;
			m9  += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 = fq;
			m16 = fq;
			m17 = -fq;

			// q = 8
			//nread = neighborList[n+7*Np];
			//fq = dist[nread];
			nr8 = neighborList[n+7*Np];
			fq = dist[nr8];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 += fq;
			m16 -= fq;
			m17 += fq;

			// q=9
			//nread = neighborList[n+8*Np];
			//fq = dist[nread];
			nr9 = neighborList[n+8*Np];
			fq = dist[nr9];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 += fq;
			m17 += fq;

			// q = 10
			//nread = neighborList[n+9*Np];
			//fq = dist[nread];
			nr10 = neighborList[n+9*Np];
			fq = dist[nr10];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy += fq;
			m6 += fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 -= fq;
			m17 -= fq;

			// q=11
			//nread = neighborList[n+10*Np];
			//fq = dist[nread];
			nr11 = neighborList[n+10*Np];
			fq = dist[nr11];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 = fq;
			m16 -= fq;
			m18 = fq;

			// q=12
			//nread = neighborList[n+11*Np];
			//fq = dist[nread];
			nr12 = neighborList[n+11*Np];
			fq = dist[nr12];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 += fq;
			m16 += fq;
			m18 -= fq;

			// q=13
			//nread = neighborList[n+12*Np];
			//fq = dist[nread];
			nr13 = neighborList[n+12*Np];
			fq = dist[nr13];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 -= fq;
			m18 -= fq;

			// q=14
			//nread = neighborList[n+13*Np];
			//fq = dist[nread];
			nr14 = neighborList[n+13*Np];
			fq = dist[nr14];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 += fq;
			m18 += fq;

			// q=15
			nread = neighborList[n+14*Np];
			fq = dist[nread];
			//fq = dist[17*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 = fq;
			m17 += fq;
			m18 -= fq;

			// q=16
			nread = neighborList[n+15*Np];
			fq = dist[nread];
			//fq = dist[8*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 += fq;
			m17 -= fq;
			m18 += fq;

			// q=17
			//fq = dist[18*Np+n];
			nread = neighborList[n+16*Np];
			fq = dist[nread];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 += fq;
			m18 += fq;

			// q=18
			nread = neighborList[n+17*Np];
			fq = dist[nread];
			//fq = dist[9*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 -= fq;
			m18 -= fq;
			
            // Compute greyscale related parameters
            c0 = 0.5*(1.0+porosity*0.5*mu_eff/perm);
            if (porosity==1.0) c0 = 0.5;//i.e. apparent pore nodes
            //GeoFun = 1.75/sqrt(150.0*porosity*porosity*porosity);
            c1 = porosity*0.5*GeoFun/sqrt(perm);
            if (porosity==1.0) c1 = 0.0;//i.e. apparent pore nodes

            vx = jx/rho0+0.5*(porosity*Gx);
            vy = jy/rho0+0.5*(porosity*Gy);
            vz = jz/rho0+0.5*(porosity*Gz);
            v_mag=sqrt(vx*vx+vy*vy+vz*vz);
            ux = vx/(c0+sqrt(c0*c0+c1*v_mag));
            uy = vy/(c0+sqrt(c0*c0+c1*v_mag));
            uz = vz/(c0+sqrt(c0*c0+c1*v_mag));
            u_mag=sqrt(ux*ux+uy*uy+uz*uz);

            //Update the total force to include linear (Darcy) and nonlinear (Forchheimer) drags due to the porous medium
            Fx = rho0*(-porosity*mu_eff/perm*ux - porosity*GeoFun/sqrt(perm)*u_mag*ux + porosity*Gx);
            Fy = rho0*(-porosity*mu_eff/perm*uy - porosity*GeoFun/sqrt(perm)*u_mag*uy + porosity*Gy);
            Fz = rho0*(-porosity*mu_eff/perm*uz - porosity*GeoFun/sqrt(perm)*u_mag*uz + porosity*Gz);
            if (porosity==1.0){
                Fx=rho0*(Gx);
                Fy=rho0*(Gy);
                Fz=rho0*(Gz);
            }

			// write the velocity 
			Velocity[n] = ux;
			Velocity[Np+n] = uy;
			Velocity[2*Np+n] = uz;

			//........................................................................
			//..............carry out relaxation process..............................
			//..........Toelke, Fruediger et. al. 2006................................
            //---------------- NO higher-order force -------------------------------//
			if (C == 0.0)	nx = ny = nz = 0.0;
			m1 = m1 + rlx_setA*((19*(ux*ux+uy*uy+uz*uz)*rho0/porosity - 11*rho) -19*alpha*C - m1);
			m2 = m2 + rlx_setA*((3*rho - 5.5*(ux*ux+uy*uy+uz*uz)*rho0/porosity)- m2);
            jx = jx + Fx;
			m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0)- m4)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
            jy = jy + Fy;
			m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0)- m6)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
            jz = jz + Fz;
			m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0)- m8)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
			m9 = m9 + rlx_setA*(((2*ux*ux-uy*uy-uz*uz)*rho0/porosity) + 0.5*alpha*C*(2*nx*nx-ny*ny-nz*nz) - m9);
			m10 = m10 + rlx_setA*( - m10);
            //m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10);
			m11 = m11 + rlx_setA*(((uy*uy-uz*uz)*rho0/porosity) + 0.5*alpha*C*(ny*ny-nz*nz)- m11);
			m12 = m12 + rlx_setA*( - m12);
            //m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12);
			m13 = m13 + rlx_setA*( (ux*uy*rho0/porosity) + 0.5*alpha*C*nx*ny - m13);
			m14 = m14 + rlx_setA*( (uy*uz*rho0/porosity) + 0.5*alpha*C*ny*nz - m14);
			m15 = m15 + rlx_setA*( (ux*uz*rho0/porosity) + 0.5*alpha*C*nx*nz - m15);
			m16 = m16 + rlx_setB*( - m16);
			m17 = m17 + rlx_setB*( - m17);
			m18 = m18 + rlx_setB*( - m18);
            //----------------------------------------------------------------------//

            //----------------With higher-order force ------------------------------//
			//if (C == 0.0)	nx = ny = nz = 0.0;
			//m1 = m1 + rlx_setA*((19*(ux*ux+uy*uy+uz*uz)*rho0/porosity - 11*rho) -19*alpha*C - m1)
            //        + (1-0.5*rlx_setA)*38*(Fx*ux+Fy*uy+Fz*uz)/porosity;
			//m2 = m2 + rlx_setA*((3*rho - 5.5*(ux*ux+uy*uy+uz*uz)*rho0/porosity)- m2)
            //        + (1-0.5*rlx_setA)*11*(-Fx*ux-Fy*uy-Fz*uz)/porosity;
            //jx = jx + Fx;
			//m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0)- m4)
            //        + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
            //jy = jy + Fy;
			//m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0)- m6)
            //        + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
            //jz = jz + Fz;
			//m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0)- m8)
            //        + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
			//m9 = m9 + rlx_setA*(((2*ux*ux-uy*uy-uz*uz)*rho0/porosity) + 0.5*alpha*C*(2*nx*nx-ny*ny-nz*nz) - m9)
            //        + (1-0.5*rlx_setA)*(4*Fx*ux-2*Fy*uy-2*Fz*uz)/porosity;
			////m10 = m10 + rlx_setA*( - m10);
            //m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10)
            //          + (1-0.5*rlx_setA)*(-2*Fx*ux+Fy*uy+Fz*uz)/porosity;
			//m11 = m11 + rlx_setA*(((uy*uy-uz*uz)*rho0/porosity) + 0.5*alpha*C*(ny*ny-nz*nz)- m11)
            //          + (1-0.5*rlx_setA)*(2*Fy*uy-2*Fz*uz)/porosity;
			////m12 = m12 + rlx_setA*( - m12);
            //m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12)
            //          + (1-0.5*rlx_setA)*(-Fy*uy+Fz*uz)/porosity;
			//m13 = m13 + rlx_setA*( (ux*uy*rho0/porosity) + 0.5*alpha*C*nx*ny - m13);
            //          + (1-0.5*rlx_setA)*(Fy*ux+Fx*uy)/porosity;
			//m14 = m14 + rlx_setA*( (uy*uz*rho0/porosity) + 0.5*alpha*C*ny*nz - m14);
            //          + (1-0.5*rlx_setA)*(Fz*uy+Fy*uz)/porosity;
			//m15 = m15 + rlx_setA*( (ux*uz*rho0/porosity) + 0.5*alpha*C*nx*nz - m15);
            //          + (1-0.5*rlx_setA)*(Fz*ux+Fx*uz)/porosity;
			//m16 = m16 + rlx_setB*( - m16);
			//m17 = m17 + rlx_setB*( - m17);
			//m18 = m18 + rlx_setB*( - m18);
            //----------------------------------------------------------------------//

			//.................inverse transformation......................................................
			// q=0
			fq = mrt_V1*rho-mrt_V2*m1+mrt_V3*m2;
			dist[n] = fq;

			// q = 1
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10);
			//nread = neighborList[n+Np];
			dist[nr2] = fq;

			// q=2
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10);
			//nread = neighborList[n];
			dist[nr1] = fq;

			// q = 3
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
			//nread = neighborList[n+3*Np];
			dist[nr4] = fq;

			// q = 4
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
			//nread = neighborList[n+2*Np];
			dist[nr3] = fq;

			// q = 5
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
			//nread = neighborList[n+5*Np];
			dist[nr6] = fq;

			// q = 6
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
			//nread = neighborList[n+4*Np];
			dist[nr5] = fq;

			// q = 7
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17);
			//nread = neighborList[n+7*Np];
			dist[nr8] = fq;

			// q = 8
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11
					+mrt_V12*m12+0.25*m13+0.125*(m17-m16);
			//nread = neighborList[n+6*Np];
			dist[nr7] = fq;

			// q = 9
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17);
			//nread = neighborList[n+9*Np];
			dist[nr10] = fq;

			// q = 10
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17);
			//nread = neighborList[n+8*Np];
			dist[nr9] = fq;

			// q = 11
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12+0.25*m15+0.125*(m18-m16);
			//nread = neighborList[n+11*Np];
			dist[nr12] = fq;

			// q = 12
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+
					mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18);
			//nread = neighborList[n+10*Np];
			dist[nr11]= fq;

			// q = 13
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15-0.125*(m16+m18);
			//nread = neighborList[n+13*Np];
			dist[nr14] = fq;

			// q= 14
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15+0.125*(m16+m18);
			//nread = neighborList[n+12*Np];
			dist[nr13] = fq;


			// q = 15
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18);
			nread = neighborList[n+15*Np];
			dist[nread] = fq;

			// q = 16
			fq =  mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17);
			nread = neighborList[n+14*Np];
			dist[nread] = fq;


			// q = 17
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18);
			nread = neighborList[n+17*Np];
			dist[nread] = fq;

			// q = 18
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18);
			nread = neighborList[n+16*Np];
			dist[nread] = fq;
			//........................................................................

			// Instantiate mass transport distributions
			// Stationary value - distribution 0
			nAB = 1.0/(nA+nB);
			Aq[n] = 0.3333333333333333*nA;
			Bq[n] = 0.3333333333333333*nB;

			//...............................................
			// q = 0,2,4
			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nx;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*ux))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*ux))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*ux))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*ux))+delta;

			// q = 1
			//nread = neighborList[n+Np];
			Aq[nr2] = a1;
			Bq[nr2] = b1;
			// q=2
			//nread = neighborList[n];
			Aq[nr1] = a2;
			Bq[nr1] = b2;

			//...............................................
			// Cq = {0,1,0}
			delta = beta*nA*nB*nAB*0.1111111111111111*ny;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uy))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uy))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uy))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uy))+delta;

			// q = 3
			//nread = neighborList[n+3*Np];
			Aq[nr4] = a1;
			Bq[nr4] = b1;
			// q = 4
			//nread = neighborList[n+2*Np];
			Aq[nr3] = a2;
			Bq[nr3] = b2;

			//...............................................
			// q = 4
			// Cq = {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nz;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uz))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uz))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uz))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uz))+delta;

			// q = 5
			//nread = neighborList[n+5*Np];
			Aq[nr6] = a1;
			Bq[nr6] = b1;
			// q = 6
			//nread = neighborList[n+4*Np];
			Aq[nr5] = a2;
			Bq[nr5] = b2;
			//...............................................
		}
	}
}

//Model-1 & 4
__global__  void dvc_ScaLBL_D3Q19_AAeven_GreyscaleColor(int *Map, double *dist, double *Aq, double *Bq, double *Den, 
        double *Phi, double *GreySolidGrad, double *Poros,double *Perm, double *Velocity, 
        double rhoA, double rhoB, double tauA, double tauB,double tauA_eff,double tauB_eff, double alpha, double beta,
		double Gx, double Gy, double Gz, int strideY, int strideZ, int start, int finish, int Np){
	int ijk,nn,n;
	double fq;
	// conserved momemnts
	double rho,jx,jy,jz;
	double vx,vy,vz,v_mag;
    double ux,uy,uz,u_mag;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
	double m3,m5,m7;
	double nA,nB; // number density
	double a1,b1,a2,b2,nAB,delta;
	double C,nx,ny,nz; //color gradient magnitude and direction
	double phi,tau,rho0,rlx_setA,rlx_setB;

    double GeoFun=0.0;//geometric function from Guo's PRE 66, 036304 (2002)
    double porosity;
    double perm;//voxel permeability
    double c0, c1; //Guo's model parameters
    double tau_eff;
    double mu_eff;//kinematic viscosity
    double nx_gs,ny_gs,nz_gs;//grey-solid color gradient
    double nx_phase,ny_phase,nz_phase,C_phase;
    double Fx,Fy,Fz;

	const double mrt_V1=0.05263157894736842;
	const double mrt_V2=0.012531328320802;
	const double mrt_V3=0.04761904761904762;
	const double mrt_V4=0.004594820384294068;
	const double mrt_V5=0.01587301587301587;
	const double mrt_V6=0.0555555555555555555555555;
	const double mrt_V7=0.02777777777777778;
	const double mrt_V8=0.08333333333333333;
	const double mrt_V9=0.003341687552213868;
	const double mrt_V10=0.003968253968253968;
	const double mrt_V11=0.01388888888888889;
	const double mrt_V12=0.04166666666666666;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {

			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];
            porosity = Poros[n];
            perm = Perm[n];
            nx_gs = GreySolidGrad[n+0*Np];
            ny_gs = GreySolidGrad[n+1*Np];
            nz_gs = GreySolidGrad[n+2*Np];

			// compute phase indicator field
			phi=(nA-nB)/(nA+nB);

			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
			// local relaxation time
			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
			tau_eff=tauA_eff + 0.5*(1.0-phi)*(tauB_eff-tauA_eff);
			rlx_setA = 1.f/tau;
			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);
            mu_eff = (tau_eff-0.5)/3.0;//kinematic viscosity

			// Get the 1D index based on regular data layout
			ijk = Map[n];
			//					COMPUTE THE COLOR GRADIENT
			//........................................................................
			//.................Read Phase Indicator Values............................
			//........................................................................
			nn = ijk-1;							// neighbor index (get convention)
			m1 = Phi[nn];						// get neighbor for phi - 1
			//........................................................................
			nn = ijk+1;							// neighbor index (get convention)
			m2 = Phi[nn];						// get neighbor for phi - 2
			//........................................................................
			nn = ijk-strideY;							// neighbor index (get convention)
			m3 = Phi[nn];					// get neighbor for phi - 3
			//........................................................................
			nn = ijk+strideY;							// neighbor index (get convention)
			m4 = Phi[nn];					// get neighbor for phi - 4
			//........................................................................
			nn = ijk-strideZ;						// neighbor index (get convention)
			m5 = Phi[nn];					// get neighbor for phi - 5
			//........................................................................
			nn = ijk+strideZ;						// neighbor index (get convention)
			m6 = Phi[nn];					// get neighbor for phi - 6
			//........................................................................
			nn = ijk-strideY-1;						// neighbor index (get convention)
			m7 = Phi[nn];					// get neighbor for phi - 7
			//........................................................................
			nn = ijk+strideY+1;						// neighbor index (get convention)
			m8 = Phi[nn];					// get neighbor for phi - 8
			//........................................................................
			nn = ijk+strideY-1;						// neighbor index (get convention)
			m9 = Phi[nn];					// get neighbor for phi - 9
			//........................................................................
			nn = ijk-strideY+1;						// neighbor index (get convention)
			m10 = Phi[nn];					// get neighbor for phi - 10
			//........................................................................
			nn = ijk-strideZ-1;						// neighbor index (get convention)
			m11 = Phi[nn];					// get neighbor for phi - 11
			//........................................................................
			nn = ijk+strideZ+1;						// neighbor index (get convention)
			m12 = Phi[nn];					// get neighbor for phi - 12
			//........................................................................
			nn = ijk+strideZ-1;						// neighbor index (get convention)
			m13 = Phi[nn];					// get neighbor for phi - 13
			//........................................................................
			nn = ijk-strideZ+1;						// neighbor index (get convention)
			m14 = Phi[nn];					// get neighbor for phi - 14
			//........................................................................
			nn = ijk-strideZ-strideY;					// neighbor index (get convention)
			m15 = Phi[nn];					// get neighbor for phi - 15
			//........................................................................
			nn = ijk+strideZ+strideY;					// neighbor index (get convention)
			m16 = Phi[nn];					// get neighbor for phi - 16
			//........................................................................
			nn = ijk+strideZ-strideY;					// neighbor index (get convention)
			m17 = Phi[nn];					// get neighbor for phi - 17
			//........................................................................
			nn = ijk-strideZ+strideY;					// neighbor index (get convention)
			m18 = Phi[nn];					// get neighbor for phi - 18
			//............Compute the Color Gradient...................................
			nx_phase = -(m1-m2+0.5*(m7-m8+m9-m10+m11-m12+m13-m14));
			ny_phase = -(m3-m4+0.5*(m7-m8-m9+m10+m15-m16+m17-m18));
			nz_phase = -(m5-m6+0.5*(m11-m12-m13+m14+m15-m16-m17+m18));
			C_phase = sqrt(nx_phase*nx_phase+ny_phase*ny_phase+nz_phase*nz_phase);

            //correct the normal color gradient by considering the effect of grey solid
            nx = nx_phase + (1.0-porosity)*nx_gs; 
            ny = ny_phase + (1.0-porosity)*ny_gs; 
            nz = nz_phase + (1.0-porosity)*nz_gs; 
            if (C_phase==0.0){
                nx = nx_phase; 
                ny = ny_phase;
                nz = nz_phase;
            }

			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			double ColorMag = C;
			if (C==0.0) ColorMag=1.0;
			nx = nx/ColorMag;
			ny = ny/ColorMag;
			nz = nz/ColorMag;		

			// q=0
			fq = dist[n];
			rho = fq;
			m1  = -30.0*fq;
			m2  = 12.0*fq;

			// q=1
			fq = dist[2*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jx = fq;
			m4 = -4.0*fq;
			m9 = 2.0*fq;
			m10 = -4.0*fq;

			// f2 = dist[10*Np+n];
			fq = dist[1*Np+n];
			rho += fq;
			m1 -= 11.0*(fq);
			m2 -= 4.0*(fq);
			jx -= fq;
			m4 += 4.0*(fq);
			m9 += 2.0*(fq);
			m10 -= 4.0*(fq);

			// q=3
			fq = dist[4*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy = fq;
			m6 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 = fq;
			m12 = -2.0*fq;

			// q = 4
			fq = dist[3*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jy -= fq;
			m6 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 += fq;
			m12 -= 2.0*fq;

			// q=5
			fq = dist[6*Np+n];
			rho += fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz = fq;
			m8 = -4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q = 6
			fq = dist[5*Np+n];
			rho+= fq;
			m1 -= 11.0*fq;
			m2 -= 4.0*fq;
			jz -= fq;
			m8 += 4.0*fq;
			m9 -= fq;
			m10 += 2.0*fq;
			m11 -= fq;
			m12 += 2.0*fq;

			// q=7
			fq = dist[8*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy += fq;
			m6 += fq;
			m9  += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 = fq;
			m16 = fq;
			m17 = -fq;

			// q = 8
			fq = dist[7*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 += fq;
			m16 -= fq;
			m17 += fq;

			// q=9
			fq = dist[10*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jy -= fq;
			m6 -= fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 += fq;
			m17 += fq;

			// q = 10
			fq = dist[9*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jy += fq;
			m6 += fq;
			m9 += fq;
			m10 += fq;
			m11 += fq;
			m12 += fq;
			m13 -= fq;
			m16 -= fq;
			m17 -= fq;

			// q=11
			fq = dist[12*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 = fq;
			m16 -= fq;
			m18 = fq;

			// q=12
			fq = dist[11*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 += fq;
			m16 += fq;
			m18 -= fq;

			// q=13
			fq = dist[14*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx += fq;
			m4 += fq;
			jz -= fq;
			m8 -= fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 -= fq;
			m18 -= fq;

			// q=14
			fq = dist[13*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jx -= fq;
			m4 -= fq;
			jz += fq;
			m8 += fq;
			m9 += fq;
			m10 += fq;
			m11 -= fq;
			m12 -= fq;
			m15 -= fq;
			m16 += fq;
			m18 += fq;

			// q=15
			fq = dist[16*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 = fq;
			m17 += fq;
			m18 -= fq;

			// q=16
			fq = dist[15*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 += fq;
			m17 -= fq;
			m18 += fq;

			// q=17
			fq = dist[18*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy += fq;
			m6 += fq;
			jz -= fq;
			m8 -= fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 += fq;
			m18 += fq;

			// q=18
			fq = dist[17*Np+n];
			rho += fq;
			m1 += 8.0*fq;
			m2 += fq;
			jy -= fq;
			m6 -= fq;
			jz += fq;
			m8 += fq;
			m9 -= 2.0*fq;
			m10 -= 2.0*fq;
			m14 -= fq;
			m17 -= fq;
			m18 -= fq;

            // Compute greyscale related parameters
            c0 = 0.5*(1.0+porosity*0.5*mu_eff/perm);
            if (porosity==1.0) c0 = 0.5;//i.e. apparent pore nodes
            //GeoFun = 1.75/sqrt(150.0*porosity*porosity*porosity);
            c1 = porosity*0.5*GeoFun/sqrt(perm);
            if (porosity==1.0) c1 = 0.0;//i.e. apparent pore nodes

            vx = jx/rho0+0.5*(porosity*Gx);
            vy = jy/rho0+0.5*(porosity*Gy);
            vz = jz/rho0+0.5*(porosity*Gz);
            v_mag=sqrt(vx*vx+vy*vy+vz*vz);
            ux = vx/(c0+sqrt(c0*c0+c1*v_mag));
            uy = vy/(c0+sqrt(c0*c0+c1*v_mag));
            uz = vz/(c0+sqrt(c0*c0+c1*v_mag));
            u_mag=sqrt(ux*ux+uy*uy+uz*uz);

            //Update the total force to include linear (Darcy) and nonlinear (Forchheimer) drags due to the porous medium
            Fx = rho0*(-porosity*mu_eff/perm*ux - porosity*GeoFun/sqrt(perm)*u_mag*ux + porosity*Gx);
            Fy = rho0*(-porosity*mu_eff/perm*uy - porosity*GeoFun/sqrt(perm)*u_mag*uy + porosity*Gy);
            Fz = rho0*(-porosity*mu_eff/perm*uz - porosity*GeoFun/sqrt(perm)*u_mag*uz + porosity*Gz);
            if (porosity==1.0){
                Fx=rho0*(Gx);
                Fy=rho0*(Gy);
                Fz=rho0*(Gz);
            }

			// write the velocity 
			Velocity[n] = ux;
			Velocity[Np+n] = uy;
			Velocity[2*Np+n] = uz;

			//........................................................................
			//..............carry out relaxation process..............................
			//..........Toelke, Fruediger et. al. 2006................................
            //---------------- NO higher-order force -------------------------------//
			if (C == 0.0)	nx = ny = nz = 0.0;
			m1 = m1 + rlx_setA*((19*(ux*ux+uy*uy+uz*uz)*rho0/porosity - 11*rho) -19*alpha*C - m1);
			m2 = m2 + rlx_setA*((3*rho - 5.5*(ux*ux+uy*uy+uz*uz)*rho0/porosity)- m2);
            jx = jx + Fx;
			m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0)- m4)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
            jy = jy + Fy;
			m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0)- m6)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
            jz = jz + Fz;
			m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0)- m8)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
			m9 = m9 + rlx_setA*(((2*ux*ux-uy*uy-uz*uz)*rho0/porosity) + 0.5*alpha*C*(2*nx*nx-ny*ny-nz*nz) - m9);
			m10 = m10 + rlx_setA*( - m10);
            //m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10);
			m11 = m11 + rlx_setA*(((uy*uy-uz*uz)*rho0/porosity) + 0.5*alpha*C*(ny*ny-nz*nz)- m11);
			m12 = m12 + rlx_setA*( - m12);
            //m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12);
			m13 = m13 + rlx_setA*( (ux*uy*rho0/porosity) + 0.5*alpha*C*nx*ny - m13);
			m14 = m14 + rlx_setA*( (uy*uz*rho0/porosity) + 0.5*alpha*C*ny*nz - m14);
			m15 = m15 + rlx_setA*( (ux*uz*rho0/porosity) + 0.5*alpha*C*nx*nz - m15);
			m16 = m16 + rlx_setB*( - m16);
			m17 = m17 + rlx_setB*( - m17);
			m18 = m18 + rlx_setB*( - m18);
            //----------------------------------------------------------------------//

            //----------------With higher-order force ------------------------------//
			//if (C == 0.0)	nx = ny = nz = 0.0;
			//m1 = m1 + rlx_setA*((19*(ux*ux+uy*uy+uz*uz)*rho0/porosity - 11*rho) -19*alpha*C - m1)
            //        + (1-0.5*rlx_setA)*38*(Fx*ux+Fy*uy+Fz*uz)/porosity;
			//m2 = m2 + rlx_setA*((3*rho - 5.5*(ux*ux+uy*uy+uz*uz)*rho0/porosity)- m2)
            //        + (1-0.5*rlx_setA)*11*(-Fx*ux-Fy*uy-Fz*uz)/porosity;
            //jx = jx + Fx;
			//m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0)- m4)
            //        + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
            //jy = jy + Fy;
			//m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0)- m6)
            //        + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
            //jz = jz + Fz;
			//m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0)- m8)
            //        + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
			//m9 = m9 + rlx_setA*(((2*ux*ux-uy*uy-uz*uz)*rho0/porosity) + 0.5*alpha*C*(2*nx*nx-ny*ny-nz*nz) - m9)
            //        + (1-0.5*rlx_setA)*(4*Fx*ux-2*Fy*uy-2*Fz*uz)/porosity;
			////m10 = m10 + rlx_setA*( - m10);
            //m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10)
            //          + (1-0.5*rlx_setA)*(-2*Fx*ux+Fy*uy+Fz*uz)/porosity;
			//m11 = m11 + rlx_setA*(((uy*uy-uz*uz)*rho0/porosity) + 0.5*alpha*C*(ny*ny-nz*nz)- m11)
            //          + (1-0.5*rlx_setA)*(2*Fy*uy-2*Fz*uz)/porosity;
			////m12 = m12 + rlx_setA*( - m12);
            //m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12)
            //          + (1-0.5*rlx_setA)*(-Fy*uy+Fz*uz)/porosity;
			//m13 = m13 + rlx_setA*( (ux*uy*rho0/porosity) + 0.5*alpha*C*nx*ny - m13);
            //          + (1-0.5*rlx_setA)*(Fy*ux+Fx*uy)/porosity;
			//m14 = m14 + rlx_setA*( (uy*uz*rho0/porosity) + 0.5*alpha*C*ny*nz - m14);
            //          + (1-0.5*rlx_setA)*(Fz*uy+Fy*uz)/porosity;
			//m15 = m15 + rlx_setA*( (ux*uz*rho0/porosity) + 0.5*alpha*C*nx*nz - m15);
            //          + (1-0.5*rlx_setA)*(Fz*ux+Fx*uz)/porosity;
			//m16 = m16 + rlx_setB*( - m16);
			//m17 = m17 + rlx_setB*( - m17);
			//m18 = m18 + rlx_setB*( - m18);
            //----------------------------------------------------------------------//

			//.................inverse transformation......................................................
			// q=0
			fq = mrt_V1*rho-mrt_V2*m1+mrt_V3*m2;
			dist[n] = fq;

			// q = 1
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10);
			dist[1*Np+n] = fq;

			// q=2
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10);
			dist[2*Np+n] = fq;

			// q = 3
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
			dist[3*Np+n] = fq;

			// q = 4
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
			dist[4*Np+n] = fq;

			// q = 5
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
			dist[5*Np+n] = fq;

			// q = 6
			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
			dist[6*Np+n] = fq;

			// q = 7
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17);
			dist[7*Np+n] = fq;


			// q = 8
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11
					+mrt_V12*m12+0.25*m13+0.125*(m17-m16);
			dist[8*Np+n] = fq;

			// q = 9
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17);
			dist[9*Np+n] = fq;

			// q = 10
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+
					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17);
			dist[10*Np+n] = fq;


			// q = 11
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12+0.25*m15+0.125*(m18-m16);
			dist[11*Np+n] = fq;

			// q = 12
			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+
					mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18);
			dist[12*Np+n] = fq;

			// q = 13
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15-0.125*(m16+m18);
			dist[13*Np+n] = fq;

			// q= 14
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)
					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
					-mrt_V12*m12-0.25*m15+0.125*(m16+m18);

			dist[14*Np+n] = fq;

			// q = 15
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18);
			dist[15*Np+n] = fq;

			// q = 16
			fq =  mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)
					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17);
			dist[16*Np+n] = fq;


			// q = 17
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18);
			dist[17*Np+n] = fq;

			// q = 18
			fq = mrt_V1*rho+mrt_V9*m1
					+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)
					-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18);
			dist[18*Np+n] = fq;
			//........................................................................

			// Instantiate mass transport distributions
			// Stationary value - distribution 0
			nAB = 1.0/(nA+nB);
			Aq[n] = 0.3333333333333333*nA;
			Bq[n] = 0.3333333333333333*nB;

			//...............................................
			// q = 0,2,4
			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nx;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*ux))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*ux))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*ux))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*ux))+delta;

			Aq[1*Np+n] = a1;
			Bq[1*Np+n] = b1;
			Aq[2*Np+n] = a2;
			Bq[2*Np+n] = b2;

			//...............................................
			// q = 2
			// Cq = {0,1,0}
			delta = beta*nA*nB*nAB*0.1111111111111111*ny;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uy))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uy))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uy))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uy))+delta;

			Aq[3*Np+n] = a1;
			Bq[3*Np+n] = b1;
			Aq[4*Np+n] = a2;
			Bq[4*Np+n] = b2;
			//...............................................
			// q = 4
			// Cq = {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nz;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uz))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uz))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uz))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uz))+delta;

			Aq[5*Np+n] = a1;
			Bq[5*Np+n] = b1;
			Aq[6*Np+n] = a2;
			Bq[6*Np+n] = b2;
			//...............................................

		}
	}
}

////Model-2&3
//__global__ void dvc_ScaLBL_D3Q19_AAodd_GreyscaleColor(int *neighborList, int *Map, double *dist, double *Aq, double *Bq, double *Den,
//		 double *Phi, double *GreySolidGrad, double *Poros,double *Perm, double *Velocity, 
//         double rhoA, double rhoB, double tauA, double tauB,double tauA_eff,double tauB_eff,double alpha, double beta,
//		 double Gx, double Gy, double Gz, int strideY, int strideZ, int start, int finish, int Np){
//
//	int n,nn,ijk,nread;
//	int nr1,nr2,nr3,nr4,nr5,nr6;
//	int nr7,nr8,nr9,nr10;
//	int nr11,nr12,nr13,nr14;
//	//int nr15,nr16,nr17,nr18;
//	double fq;
//	// conserved momemnts
//	double rho,jx,jy,jz;
//	double vx,vy,vz,v_mag;
//    double ux,uy,uz,u_mag;
//	// non-conserved moments
//	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
//	double m3,m5,m7;
//	double t1,t2,t4,t6,t8,t9,t10,t11,t12,t13,t14,t15,t16,t17,t18;
//	double t3,t5,t7;
//	double nA,nB; // number density
//	double a1,b1,a2,b2,nAB,delta;
//	double C,nx,ny,nz; //color gradient magnitude and direction
//	double phi,tau,rho0,rlx_setA,rlx_setB;
//
//    double GeoFun=0.0;//geometric function from Guo's PRE 66, 036304 (2002)
//    double porosity;
//    double perm;//voxel permeability
//    double c0, c1; //Guo's model parameters
//    double tau_eff;
//    double mu_eff;//kinematic viscosity
//    double nx_phase,ny_phase,nz_phase,C_phase;
//    double Fx,Fy,Fz;
//
//	const double mrt_V1=0.05263157894736842;
//	const double mrt_V2=0.012531328320802;
//	const double mrt_V3=0.04761904761904762;
//	const double mrt_V4=0.004594820384294068;
//	const double mrt_V5=0.01587301587301587;
//	const double mrt_V6=0.0555555555555555555555555;
//	const double mrt_V7=0.02777777777777778;
//	const double mrt_V8=0.08333333333333333;
//	const double mrt_V9=0.003341687552213868;
//	const double mrt_V10=0.003968253968253968;
//	const double mrt_V11=0.01388888888888889;
//	const double mrt_V12=0.04166666666666666;
//
//	int S = Np/NBLOCKS/NTHREADS + 1;
//	for (int s=0; s<S; s++){
//		//........Get 1-D index for this thread....................
//		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
//		if (n<finish) {
//			// read the component number densities
//			nA = Den[n];
//			nB = Den[Np + n];
//            porosity = Poros[n];
//            perm = Perm[n];
//
//			// compute phase indicator field
//			phi=(nA-nB)/(nA+nB);
//
//			// local density
//			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
//			// local relaxation time
//			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
//			tau_eff=tauA_eff + 0.5*(1.0-phi)*(tauB_eff-tauA_eff);
//			rlx_setA = 1.f/tau;
//			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);
//            mu_eff = (tau_eff-0.5)/3.0;//kinematic viscosity
//			
//			// Get the 1D index based on regular data layout
//			ijk = Map[n];
//			//					COMPUTE THE COLOR GRADIENT
//			//........................................................................
//			//.................Read Phase Indicator Values............................
//			//........................................................................
//			nn = ijk-1;							// neighbor index (get convention)
//			m1 = Phi[nn];						// get neighbor for phi - 1
//			t1 = m1+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t1)>1.0) t1 =((t1>0.0)-(t1<0.0))*(1.0-fabs(t1))+t1;
//			//........................................................................
//			nn = ijk+1;							// neighbor index (get convention)
//			m2 = Phi[nn];						// get neighbor for phi - 2
//			t2 = m2+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t2)>1.0) t2 =((t2>0.0)-(t2<0.0))*(1.0-fabs(t2))+t2;
//			//........................................................................
//			nn = ijk-strideY;							// neighbor index (get convention)
//			m3 = Phi[nn];					// get neighbor for phi - 3
//			t3 = m3+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t3)>1.0) t3 =((t3>0.0)-(t3<0.0))*(1.0-fabs(t3))+t3;
//			//........................................................................
//			nn = ijk+strideY;							// neighbor index (get convention)
//			m4 = Phi[nn];					// get neighbor for phi - 4
//			t4 = m4+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t4)>1.0) t4 =((t4>0.0)-(t4<0.0))*(1.0-fabs(t4))+t4;
//			//........................................................................
//			nn = ijk-strideZ;						// neighbor index (get convention)
//			m5 = Phi[nn];					// get neighbor for phi - 5
//			t5 = m5+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t5)>1.0) t5 =((t5>0.0)-(t5<0.0))*(1.0-fabs(t5))+t5;
//			//........................................................................
//			nn = ijk+strideZ;						// neighbor index (get convention)
//			m6 = Phi[nn];					// get neighbor for phi - 6
//			t6 = m6+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t6)>1.0) t6 =((t6>0.0)-(t6<0.0))*(1.0-fabs(t6))+t6;
//			//........................................................................
//			nn = ijk-strideY-1;						// neighbor index (get convention)
//			m7 = Phi[nn];					// get neighbor for phi - 7
//			t7 = m7+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t7)>1.0) t7 =((t7>0.0)-(t7<0.0))*(1.0-fabs(t7))+t7;
//			//........................................................................
//			nn = ijk+strideY+1;						// neighbor index (get convention)
//			m8 = Phi[nn];					// get neighbor for phi - 8
//			t8 = m8+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t8)>1.0) t8 =((t8>0.0)-(t8<0.0))*(1.0-fabs(t8))+t8;
//			//........................................................................
//			nn = ijk+strideY-1;						// neighbor index (get convention)
//			m9 = Phi[nn];					// get neighbor for phi - 9
//			t9 = m9+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t9)>1.0) t9 =((t9>0.0)-(t9<0.0))*(1.0-fabs(t9))+t9;
//			//........................................................................
//			nn = ijk-strideY+1;						// neighbor index (get convention)
//			m10 = Phi[nn];					// get neighbor for phi - 10
//			t10 = m10+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t10)>1.0) t10 =((t10>0.0)-(t10<0.0))*(1.0-fabs(t10))+t10;
//			//........................................................................
//			nn = ijk-strideZ-1;						// neighbor index (get convention)
//			m11 = Phi[nn];					// get neighbor for phi - 11
//			t11 = m11+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t11)>1.0) t11 =((t11>0.0)-(t11<0.0))*(1.0-fabs(t11))+t11;
//			//........................................................................
//			nn = ijk+strideZ+1;						// neighbor index (get convention)
//			m12 = Phi[nn];					// get neighbor for phi - 12
//			t12 = m12+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t12)>1.0) t12 =((t12>0.0)-(t12<0.0))*(1.0-fabs(t12))+t12;
//			//........................................................................
//			nn = ijk+strideZ-1;						// neighbor index (get convention)
//			m13 = Phi[nn];					// get neighbor for phi - 13
//			t13 = m13+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t13)>1.0) t13 =((t13>0.0)-(t13<0.0))*(1.0-fabs(t13))+t13;
//			//........................................................................
//			nn = ijk-strideZ+1;						// neighbor index (get convention)
//			m14 = Phi[nn];					// get neighbor for phi - 14
//			t14 = m14+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t14)>1.0) t14 =((t14>0.0)-(t14<0.0))*(1.0-fabs(t14))+t14;
//			//........................................................................
//			nn = ijk-strideZ-strideY;					// neighbor index (get convention)
//			m15 = Phi[nn];					// get neighbor for phi - 15
//			t15 = m15+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t15)>1.0) t15 =((t15>0.0)-(t15<0.0))*(1.0-fabs(t15))+t15;
//			//........................................................................
//			nn = ijk+strideZ+strideY;					// neighbor index (get convention)
//			m16 = Phi[nn];					// get neighbor for phi - 16
//			t16 = m16+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t16)>1.0) t16 =((t16>0.0)-(t16<0.0))*(1.0-fabs(t16))+t16;
//			//........................................................................
//			nn = ijk+strideZ-strideY;					// neighbor index (get convention)
//			m17 = Phi[nn];					// get neighbor for phi - 17
//			t17 = m17+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t17)>1.0) t17 =((t17>0.0)-(t17<0.0))*(1.0-fabs(t17))+t17;
//			//........................................................................
//			nn = ijk-strideZ+strideY;					// neighbor index (get convention)
//			m18 = Phi[nn];					// get neighbor for phi - 18
//			t18 = m18+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t18)>1.0) t18 =((t18>0.0)-(t18<0.0))*(1.0-fabs(t18))+t18;
//			//............Compute the Color Gradient...................................
//			nx_phase = -(m1-m2+0.5*(m7-m8+m9-m10+m11-m12+m13-m14));
//			ny_phase = -(m3-m4+0.5*(m7-m8-m9+m10+m15-m16+m17-m18));
//			nz_phase = -(m5-m6+0.5*(m11-m12-m13+m14+m15-m16-m17+m18));
//			C_phase = sqrt(nx_phase*nx_phase+ny_phase*ny_phase+nz_phase*nz_phase);
//            //correct the normal color gradient by considering the effect of grey solid
//			nx = -(t1-t2+0.5*(t7-t8+t9-t10+t11-t12+t13-t14));
//			ny = -(t3-t4+0.5*(t7-t8-t9+t10+t15-t16+t17-t18));
//			nz = -(t5-t6+0.5*(t11-t12-t13+t14+t15-t16-t17+t18));
//
//            if (C_phase==0.0){//i.e. if in a bulk phase, there is no need for grey-solid correction
//                nx = nx_phase; 
//                ny = ny_phase;
//                nz = nz_phase;
//            }
//
//			//...........Normalize the Color Gradient.................................
//			C = sqrt(nx*nx+ny*ny+nz*nz);
//			double ColorMag = C;
//			if (C==0.0) ColorMag=1.0;
//			nx = nx/ColorMag;
//			ny = ny/ColorMag;
//			nz = nz/ColorMag;		
//
//			// q=0
//			fq = dist[n];
//			rho = fq;
//			m1  = -30.0*fq;
//			m2  = 12.0*fq;
//
//			// q=1
//			//nread = neighborList[n]; // neighbor 2 
//			//fq = dist[nread]; // reading the f1 data into register fq		
//			nr1 = neighborList[n]; 
//			fq = dist[nr1]; // reading the f1 data into register fq
//			rho += fq;
//			m1 -= 11.0*fq;
//			m2 -= 4.0*fq;
//			jx = fq;
//			m4 = -4.0*fq;
//			m9 = 2.0*fq;
//			m10 = -4.0*fq;
//
//			// f2 = dist[10*Np+n];
//			//nread = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
//			//fq = dist[nread];  // reading the f2 data into register fq
//			nr2 = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
//			fq = dist[nr2];  // reading the f2 data into register fq
//			rho += fq;
//			m1 -= 11.0*(fq);
//			m2 -= 4.0*(fq);
//			jx -= fq;
//			m4 += 4.0*(fq);
//			m9 += 2.0*(fq);
//			m10 -= 4.0*(fq);
//
//			// q=3
//			//nread = neighborList[n+2*Np]; // neighbor 4
//			//fq = dist[nread];
//			nr3 = neighborList[n+2*Np]; // neighbor 4
//			fq = dist[nr3];
//			rho += fq;
//			m1 -= 11.0*fq;
//			m2 -= 4.0*fq;
//			jy = fq;
//			m6 = -4.0*fq;
//			m9 -= fq;
//			m10 += 2.0*fq;
//			m11 = fq;
//			m12 = -2.0*fq;
//
//			// q = 4
//			//nread = neighborList[n+3*Np]; // neighbor 3
//			//fq = dist[nread];
//			nr4 = neighborList[n+3*Np]; // neighbor 3
//			fq = dist[nr4];
//			rho+= fq;
//			m1 -= 11.0*fq;
//			m2 -= 4.0*fq;
//			jy -= fq;
//			m6 += 4.0*fq;
//			m9 -= fq;
//			m10 += 2.0*fq;
//			m11 += fq;
//			m12 -= 2.0*fq;
//
//			// q=5
//			//nread = neighborList[n+4*Np];
//			//fq = dist[nread];
//			nr5 = neighborList[n+4*Np];
//			fq = dist[nr5];
//			rho += fq;
//			m1 -= 11.0*fq;
//			m2 -= 4.0*fq;
//			jz = fq;
//			m8 = -4.0*fq;
//			m9 -= fq;
//			m10 += 2.0*fq;
//			m11 -= fq;
//			m12 += 2.0*fq;
//
//
//			// q = 6
//			//nread = neighborList[n+5*Np];
//			//fq = dist[nread];
//			nr6 = neighborList[n+5*Np];
//			fq = dist[nr6];
//			rho+= fq;
//			m1 -= 11.0*fq;
//			m2 -= 4.0*fq;
//			jz -= fq;
//			m8 += 4.0*fq;
//			m9 -= fq;
//			m10 += 2.0*fq;
//			m11 -= fq;
//			m12 += 2.0*fq;
//
//			// q=7
//			//nread = neighborList[n+6*Np];
//			//fq = dist[nread];
//			nr7 = neighborList[n+6*Np];
//			fq = dist[nr7];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx += fq;
//			m4 += fq;
//			jy += fq;
//			m6 += fq;
//			m9  += fq;
//			m10 += fq;
//			m11 += fq;
//			m12 += fq;
//			m13 = fq;
//			m16 = fq;
//			m17 = -fq;
//
//			// q = 8
//			//nread = neighborList[n+7*Np];
//			//fq = dist[nread];
//			nr8 = neighborList[n+7*Np];
//			fq = dist[nr8];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx -= fq;
//			m4 -= fq;
//			jy -= fq;
//			m6 -= fq;
//			m9 += fq;
//			m10 += fq;
//			m11 += fq;
//			m12 += fq;
//			m13 += fq;
//			m16 -= fq;
//			m17 += fq;
//
//			// q=9
//			//nread = neighborList[n+8*Np];
//			//fq = dist[nread];
//			nr9 = neighborList[n+8*Np];
//			fq = dist[nr9];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx += fq;
//			m4 += fq;
//			jy -= fq;
//			m6 -= fq;
//			m9 += fq;
//			m10 += fq;
//			m11 += fq;
//			m12 += fq;
//			m13 -= fq;
//			m16 += fq;
//			m17 += fq;
//
//			// q = 10
//			//nread = neighborList[n+9*Np];
//			//fq = dist[nread];
//			nr10 = neighborList[n+9*Np];
//			fq = dist[nr10];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx -= fq;
//			m4 -= fq;
//			jy += fq;
//			m6 += fq;
//			m9 += fq;
//			m10 += fq;
//			m11 += fq;
//			m12 += fq;
//			m13 -= fq;
//			m16 -= fq;
//			m17 -= fq;
//
//			// q=11
//			//nread = neighborList[n+10*Np];
//			//fq = dist[nread];
//			nr11 = neighborList[n+10*Np];
//			fq = dist[nr11];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx += fq;
//			m4 += fq;
//			jz += fq;
//			m8 += fq;
//			m9 += fq;
//			m10 += fq;
//			m11 -= fq;
//			m12 -= fq;
//			m15 = fq;
//			m16 -= fq;
//			m18 = fq;
//
//			// q=12
//			//nread = neighborList[n+11*Np];
//			//fq = dist[nread];
//			nr12 = neighborList[n+11*Np];
//			fq = dist[nr12];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx -= fq;
//			m4 -= fq;
//			jz -= fq;
//			m8 -= fq;
//			m9 += fq;
//			m10 += fq;
//			m11 -= fq;
//			m12 -= fq;
//			m15 += fq;
//			m16 += fq;
//			m18 -= fq;
//
//			// q=13
//			//nread = neighborList[n+12*Np];
//			//fq = dist[nread];
//			nr13 = neighborList[n+12*Np];
//			fq = dist[nr13];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx += fq;
//			m4 += fq;
//			jz -= fq;
//			m8 -= fq;
//			m9 += fq;
//			m10 += fq;
//			m11 -= fq;
//			m12 -= fq;
//			m15 -= fq;
//			m16 -= fq;
//			m18 -= fq;
//
//			// q=14
//			//nread = neighborList[n+13*Np];
//			//fq = dist[nread];
//			nr14 = neighborList[n+13*Np];
//			fq = dist[nr14];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx -= fq;
//			m4 -= fq;
//			jz += fq;
//			m8 += fq;
//			m9 += fq;
//			m10 += fq;
//			m11 -= fq;
//			m12 -= fq;
//			m15 -= fq;
//			m16 += fq;
//			m18 += fq;
//
//			// q=15
//			nread = neighborList[n+14*Np];
//			fq = dist[nread];
//			//fq = dist[17*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jy += fq;
//			m6 += fq;
//			jz += fq;
//			m8 += fq;
//			m9 -= 2.0*fq;
//			m10 -= 2.0*fq;
//			m14 = fq;
//			m17 += fq;
//			m18 -= fq;
//
//			// q=16
//			nread = neighborList[n+15*Np];
//			fq = dist[nread];
//			//fq = dist[8*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jy -= fq;
//			m6 -= fq;
//			jz -= fq;
//			m8 -= fq;
//			m9 -= 2.0*fq;
//			m10 -= 2.0*fq;
//			m14 += fq;
//			m17 -= fq;
//			m18 += fq;
//
//			// q=17
//			//fq = dist[18*Np+n];
//			nread = neighborList[n+16*Np];
//			fq = dist[nread];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jy += fq;
//			m6 += fq;
//			jz -= fq;
//			m8 -= fq;
//			m9 -= 2.0*fq;
//			m10 -= 2.0*fq;
//			m14 -= fq;
//			m17 += fq;
//			m18 += fq;
//
//			// q=18
//			nread = neighborList[n+17*Np];
//			fq = dist[nread];
//			//fq = dist[9*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jy -= fq;
//			m6 -= fq;
//			jz += fq;
//			m8 += fq;
//			m9 -= 2.0*fq;
//			m10 -= 2.0*fq;
//			m14 -= fq;
//			m17 -= fq;
//			m18 -= fq;
//			
//            // Compute greyscale related parameters
//            c0 = 0.5*(1.0+porosity*0.5*mu_eff/perm);
//            if (porosity==1.0) c0 = 0.5;//i.e. apparent pore nodes
//            //GeoFun = 1.75/sqrt(150.0*porosity*porosity*porosity);
//            c1 = porosity*0.5*GeoFun/sqrt(perm);
//            if (porosity==1.0) c1 = 0.0;//i.e. apparent pore nodes
//
//            vx = jx/rho0+0.5*(porosity*Gx);
//            vy = jy/rho0+0.5*(porosity*Gy);
//            vz = jz/rho0+0.5*(porosity*Gz);
//            v_mag=sqrt(vx*vx+vy*vy+vz*vz);
//            ux = vx/(c0+sqrt(c0*c0+c1*v_mag));
//            uy = vy/(c0+sqrt(c0*c0+c1*v_mag));
//            uz = vz/(c0+sqrt(c0*c0+c1*v_mag));
//            u_mag=sqrt(ux*ux+uy*uy+uz*uz);
//
//            //Update the total force to include linear (Darcy) and nonlinear (Forchheimer) drags due to the porous medium
//            Fx = rho0*(-porosity*mu_eff/perm*ux - porosity*GeoFun/sqrt(perm)*u_mag*ux + porosity*Gx);
//            Fy = rho0*(-porosity*mu_eff/perm*uy - porosity*GeoFun/sqrt(perm)*u_mag*uy + porosity*Gy);
//            Fz = rho0*(-porosity*mu_eff/perm*uz - porosity*GeoFun/sqrt(perm)*u_mag*uz + porosity*Gz);
//            if (porosity==1.0){
//                Fx=rho0*(Gx);
//                Fy=rho0*(Gy);
//                Fz=rho0*(Gz);
//            }
//
//			// write the velocity 
//			Velocity[n] = ux;
//			Velocity[Np+n] = uy;
//			Velocity[2*Np+n] = uz;
//
//			//........................................................................
//			//..............carry out relaxation process..............................
//			//..........Toelke, Fruediger et. al. 2006................................
//			if (C == 0.0)	nx = ny = nz = 0.0;
//			m1 = m1 + rlx_setA*((19*(ux*ux+uy*uy+uz*uz)*rho0/porosity - 11*rho) -19*alpha*C - m1);
//			m2 = m2 + rlx_setA*((3*rho - 5.5*(ux*ux+uy*uy+uz*uz)*rho0/porosity)- m2);
//            jx = jx + Fx;
//			m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0)- m4)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
//            jy = jy + Fy;
//			m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0)- m6)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
//            jz = jz + Fz;
//			m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0)- m8)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
//			m9 = m9 + rlx_setA*(((2*ux*ux-uy*uy-uz*uz)*rho0/porosity) + 0.5*alpha*C*(2*nx*nx-ny*ny-nz*nz) - m9);
//			m10 = m10 + rlx_setA*( - m10);
//            //m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10);
//			m11 = m11 + rlx_setA*(((uy*uy-uz*uz)*rho0/porosity) + 0.5*alpha*C*(ny*ny-nz*nz)- m11);
//			m12 = m12 + rlx_setA*( - m12);
//            //m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12);
//			m13 = m13 + rlx_setA*( (ux*uy*rho0/porosity) + 0.5*alpha*C*nx*ny - m13);
//			m14 = m14 + rlx_setA*( (uy*uz*rho0/porosity) + 0.5*alpha*C*ny*nz - m14);
//			m15 = m15 + rlx_setA*( (ux*uz*rho0/porosity) + 0.5*alpha*C*nx*nz - m15);
//			m16 = m16 + rlx_setB*( - m16);
//			m17 = m17 + rlx_setB*( - m17);
//			m18 = m18 + rlx_setB*( - m18);
//
//			//.................inverse transformation......................................................
//			// q=0
//			fq = mrt_V1*rho-mrt_V2*m1+mrt_V3*m2;
//			dist[n] = fq;
//
//			// q = 1
//			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10);
//			//nread = neighborList[n+Np];
//			dist[nr2] = fq;
//
//			// q=2
//			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10);
//			//nread = neighborList[n];
//			dist[nr1] = fq;
//
//			// q = 3
//			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
//			//nread = neighborList[n+3*Np];
//			dist[nr4] = fq;
//
//			// q = 4
//			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
//			//nread = neighborList[n+2*Np];
//			dist[nr3] = fq;
//
//			// q = 5
//			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
//			//nread = neighborList[n+5*Np];
//			dist[nr6] = fq;
//
//			// q = 6
//			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
//			//nread = neighborList[n+4*Np];
//			dist[nr5] = fq;
//
//			// q = 7
//			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+
//					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17);
//			//nread = neighborList[n+7*Np];
//			dist[nr8] = fq;
//
//			// q = 8
//			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11
//					+mrt_V12*m12+0.25*m13+0.125*(m17-m16);
//			//nread = neighborList[n+6*Np];
//			dist[nr7] = fq;
//
//			// q = 9
//			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+
//					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17);
//			//nread = neighborList[n+9*Np];
//			dist[nr10] = fq;
//
//			// q = 10
//			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+
//					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17);
//			//nread = neighborList[n+8*Np];
//			dist[nr9] = fq;
//
//			// q = 11
//			fq = mrt_V1*rho+mrt_V9*m1
//					+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)
//					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
//					-mrt_V12*m12+0.25*m15+0.125*(m18-m16);
//			//nread = neighborList[n+11*Np];
//			dist[nr12] = fq;
//
//			// q = 12
//			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+
//					mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18);
//			//nread = neighborList[n+10*Np];
//			dist[nr11]= fq;
//
//			// q = 13
//			fq = mrt_V1*rho+mrt_V9*m1
//					+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)
//					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
//					-mrt_V12*m12-0.25*m15-0.125*(m16+m18);
//			//nread = neighborList[n+13*Np];
//			dist[nr14] = fq;
//
//			// q= 14
//			fq = mrt_V1*rho+mrt_V9*m1
//					+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)
//					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
//					-mrt_V12*m12-0.25*m15+0.125*(m16+m18);
//			//nread = neighborList[n+12*Np];
//			dist[nr13] = fq;
//
//
//			// q = 15
//			fq = mrt_V1*rho+mrt_V9*m1
//					+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)
//					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18);
//			nread = neighborList[n+15*Np];
//			dist[nread] = fq;
//
//			// q = 16
//			fq =  mrt_V1*rho+mrt_V9*m1
//					+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)
//					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17);
//			nread = neighborList[n+14*Np];
//			dist[nread] = fq;
//
//
//			// q = 17
//			fq = mrt_V1*rho+mrt_V9*m1
//					+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)
//					-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18);
//			nread = neighborList[n+17*Np];
//			dist[nread] = fq;
//
//			// q = 18
//			fq = mrt_V1*rho+mrt_V9*m1
//					+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)
//					-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18);
//			nread = neighborList[n+16*Np];
//			dist[nread] = fq;
//			//........................................................................
//
//			// Instantiate mass transport distributions
//			// Stationary value - distribution 0
//			nAB = 1.0/(nA+nB);
//			Aq[n] = 0.3333333333333333*nA;
//			Bq[n] = 0.3333333333333333*nB;
//
//			//...............................................
//			// q = 0,2,4
//			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
//			delta = beta*nA*nB*nAB*0.1111111111111111*nx;
//			if (!(nA*nB*nAB>0)) delta=0;
//			a1 = nA*(0.1111111111111111*(1+4.5*ux))+delta;
//			b1 = nB*(0.1111111111111111*(1+4.5*ux))-delta;
//			a2 = nA*(0.1111111111111111*(1-4.5*ux))-delta;
//			b2 = nB*(0.1111111111111111*(1-4.5*ux))+delta;
//
//			// q = 1
//			//nread = neighborList[n+Np];
//			Aq[nr2] = a1;
//			Bq[nr2] = b1;
//			// q=2
//			//nread = neighborList[n];
//			Aq[nr1] = a2;
//			Bq[nr1] = b2;
//
//			//...............................................
//			// Cq = {0,1,0}
//			delta = beta*nA*nB*nAB*0.1111111111111111*ny;
//			if (!(nA*nB*nAB>0)) delta=0;
//			a1 = nA*(0.1111111111111111*(1+4.5*uy))+delta;
//			b1 = nB*(0.1111111111111111*(1+4.5*uy))-delta;
//			a2 = nA*(0.1111111111111111*(1-4.5*uy))-delta;
//			b2 = nB*(0.1111111111111111*(1-4.5*uy))+delta;
//
//			// q = 3
//			//nread = neighborList[n+3*Np];
//			Aq[nr4] = a1;
//			Bq[nr4] = b1;
//			// q = 4
//			//nread = neighborList[n+2*Np];
//			Aq[nr3] = a2;
//			Bq[nr3] = b2;
//
//			//...............................................
//			// q = 4
//			// Cq = {0,0,1}
//			delta = beta*nA*nB*nAB*0.1111111111111111*nz;
//			if (!(nA*nB*nAB>0)) delta=0;
//			a1 = nA*(0.1111111111111111*(1+4.5*uz))+delta;
//			b1 = nB*(0.1111111111111111*(1+4.5*uz))-delta;
//			a2 = nA*(0.1111111111111111*(1-4.5*uz))-delta;
//			b2 = nB*(0.1111111111111111*(1-4.5*uz))+delta;
//
//			// q = 5
//			//nread = neighborList[n+5*Np];
//			Aq[nr6] = a1;
//			Bq[nr6] = b1;
//			// q = 6
//			//nread = neighborList[n+4*Np];
//			Aq[nr5] = a2;
//			Bq[nr5] = b2;
//			//...............................................
//		}
//	}
//}
//
////Model-2&3
//__global__  void dvc_ScaLBL_D3Q19_AAeven_GreyscaleColor(int *Map, double *dist, double *Aq, double *Bq, double *Den, 
//        double *Phi, double *GreySolidGrad, double *Poros,double *Perm, double *Velocity, 
//        double rhoA, double rhoB, double tauA, double tauB,double tauA_eff,double tauB_eff, double alpha, double beta,
//		double Gx, double Gy, double Gz, int strideY, int strideZ, int start, int finish, int Np){
//	int ijk,nn,n;
//	double fq;
//	// conserved momemnts
//	double rho,jx,jy,jz;
//	double vx,vy,vz,v_mag;
//    double ux,uy,uz,u_mag;
//	// non-conserved moments
//	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
//	double m3,m5,m7;
//	double t1,t2,t4,t6,t8,t9,t10,t11,t12,t13,t14,t15,t16,t17,t18;
//	double t3,t5,t7;
//	double nA,nB; // number density
//	double a1,b1,a2,b2,nAB,delta;
//	double C,nx,ny,nz; //color gradient magnitude and direction
//	double phi,tau,rho0,rlx_setA,rlx_setB;
//
//    double GeoFun=0.0;//geometric function from Guo's PRE 66, 036304 (2002)
//    double porosity;
//    double perm;//voxel permeability
//    double c0, c1; //Guo's model parameters
//    double tau_eff;
//    double mu_eff;//kinematic viscosity
//    double nx_phase,ny_phase,nz_phase,C_phase;
//    double Fx,Fy,Fz;
//
//	const double mrt_V1=0.05263157894736842;
//	const double mrt_V2=0.012531328320802;
//	const double mrt_V3=0.04761904761904762;
//	const double mrt_V4=0.004594820384294068;
//	const double mrt_V5=0.01587301587301587;
//	const double mrt_V6=0.0555555555555555555555555;
//	const double mrt_V7=0.02777777777777778;
//	const double mrt_V8=0.08333333333333333;
//	const double mrt_V9=0.003341687552213868;
//	const double mrt_V10=0.003968253968253968;
//	const double mrt_V11=0.01388888888888889;
//	const double mrt_V12=0.04166666666666666;
//
//	int S = Np/NBLOCKS/NTHREADS + 1;
//	for (int s=0; s<S; s++){
//		//........Get 1-D index for this thread....................
//		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
//		if (n<finish) {
//
//			// read the component number densities
//			nA = Den[n];
//			nB = Den[Np + n];
//            porosity = Poros[n];
//            perm = Perm[n];
//
//			// compute phase indicator field
//			phi=(nA-nB)/(nA+nB);
//
//			// local density
//			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
//			// local relaxation time
//			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
//			tau_eff=tauA_eff + 0.5*(1.0-phi)*(tauB_eff-tauA_eff);
//			rlx_setA = 1.f/tau;
//			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);
//            mu_eff = (tau_eff-0.5)/3.0;//kinematic viscosity
//
//			// Get the 1D index based on regular data layout
//			ijk = Map[n];
//			//					COMPUTE THE COLOR GRADIENT
//			//........................................................................
//			//.................Read Phase Indicator Values............................
//			//........................................................................
//			nn = ijk-1;							// neighbor index (get convention)
//			m1 = Phi[nn];						// get neighbor for phi - 1
//			t1 = m1+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t1)>1.0) t1 =((t1>0.0)-(t1<0.0))*(1.0-fabs(t1))+t1;
//			//........................................................................
//			nn = ijk+1;							// neighbor index (get convention)
//			m2 = Phi[nn];						// get neighbor for phi - 2
//			t2 = m2+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t2)>1.0) t2 =((t2>0.0)-(t2<0.0))*(1.0-fabs(t2))+t2;
//			//........................................................................
//			nn = ijk-strideY;							// neighbor index (get convention)
//			m3 = Phi[nn];					// get neighbor for phi - 3
//			t3 = m3+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t3)>1.0) t3 =((t3>0.0)-(t3<0.0))*(1.0-fabs(t3))+t3;
//			//........................................................................
//			nn = ijk+strideY;							// neighbor index (get convention)
//			m4 = Phi[nn];					// get neighbor for phi - 4
//			t4 = m4+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t4)>1.0) t4 =((t4>0.0)-(t4<0.0))*(1.0-fabs(t4))+t4;
//			//........................................................................
//			nn = ijk-strideZ;						// neighbor index (get convention)
//			m5 = Phi[nn];					// get neighbor for phi - 5
//			t5 = m5+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t5)>1.0) t5 =((t5>0.0)-(t5<0.0))*(1.0-fabs(t5))+t5;
//			//........................................................................
//			nn = ijk+strideZ;						// neighbor index (get convention)
//			m6 = Phi[nn];					// get neighbor for phi - 6
//			t6 = m6+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t6)>1.0) t6 =((t6>0.0)-(t6<0.0))*(1.0-fabs(t6))+t6;
//			//........................................................................
//			nn = ijk-strideY-1;						// neighbor index (get convention)
//			m7 = Phi[nn];					// get neighbor for phi - 7
//			t7 = m7+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t7)>1.0) t7 =((t7>0.0)-(t7<0.0))*(1.0-fabs(t7))+t7;
//			//........................................................................
//			nn = ijk+strideY+1;						// neighbor index (get convention)
//			m8 = Phi[nn];					// get neighbor for phi - 8
//			t8 = m8+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t8)>1.0) t8 =((t8>0.0)-(t8<0.0))*(1.0-fabs(t8))+t8;
//			//........................................................................
//			nn = ijk+strideY-1;						// neighbor index (get convention)
//			m9 = Phi[nn];					// get neighbor for phi - 9
//			t9 = m9+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t9)>1.0) t9 =((t9>0.0)-(t9<0.0))*(1.0-fabs(t9))+t9;
//			//........................................................................
//			nn = ijk-strideY+1;						// neighbor index (get convention)
//			m10 = Phi[nn];					// get neighbor for phi - 10
//			t10 = m10+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t10)>1.0) t10 =((t10>0.0)-(t10<0.0))*(1.0-fabs(t10))+t10;
//			//........................................................................
//			nn = ijk-strideZ-1;						// neighbor index (get convention)
//			m11 = Phi[nn];					// get neighbor for phi - 11
//			t11 = m11+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t11)>1.0) t11 =((t11>0.0)-(t11<0.0))*(1.0-fabs(t11))+t11;
//			//........................................................................
//			nn = ijk+strideZ+1;						// neighbor index (get convention)
//			m12 = Phi[nn];					// get neighbor for phi - 12
//			t12 = m12+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t12)>1.0) t12 =((t12>0.0)-(t12<0.0))*(1.0-fabs(t12))+t12;
//			//........................................................................
//			nn = ijk+strideZ-1;						// neighbor index (get convention)
//			m13 = Phi[nn];					// get neighbor for phi - 13
//			t13 = m13+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t13)>1.0) t13 =((t13>0.0)-(t13<0.0))*(1.0-fabs(t13))+t13;
//			//........................................................................
//			nn = ijk-strideZ+1;						// neighbor index (get convention)
//			m14 = Phi[nn];					// get neighbor for phi - 14
//			t14 = m14+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t14)>1.0) t14 =((t14>0.0)-(t14<0.0))*(1.0-fabs(t14))+t14;
//			//........................................................................
//			nn = ijk-strideZ-strideY;					// neighbor index (get convention)
//			m15 = Phi[nn];					// get neighbor for phi - 15
//			t15 = m15+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t15)>1.0) t15 =((t15>0.0)-(t15<0.0))*(1.0-fabs(t15))+t15;
//			//........................................................................
//			nn = ijk+strideZ+strideY;					// neighbor index (get convention)
//			m16 = Phi[nn];					// get neighbor for phi - 16
//			t16 = m16+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t16)>1.0) t16 =((t16>0.0)-(t16<0.0))*(1.0-fabs(t16))+t16;
//			//........................................................................
//			nn = ijk+strideZ-strideY;					// neighbor index (get convention)
//			m17 = Phi[nn];					// get neighbor for phi - 17
//			t17 = m17+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t17)>1.0) t17 =((t17>0.0)-(t17<0.0))*(1.0-fabs(t17))+t17;
//			//........................................................................
//			nn = ijk-strideZ+strideY;					// neighbor index (get convention)
//			m18 = Phi[nn];					// get neighbor for phi - 18
//			t18 = m18+(1.0-porosity)*GreySolidGrad[nn];				
//            if (fabs(t18)>1.0) t18 =((t18>0.0)-(t18<0.0))*(1.0-fabs(t18))+t18;
//			//............Compute the Color Gradient...................................
//			nx_phase = -(m1-m2+0.5*(m7-m8+m9-m10+m11-m12+m13-m14));
//			ny_phase = -(m3-m4+0.5*(m7-m8-m9+m10+m15-m16+m17-m18));
//			nz_phase = -(m5-m6+0.5*(m11-m12-m13+m14+m15-m16-m17+m18));
//			C_phase = sqrt(nx_phase*nx_phase+ny_phase*ny_phase+nz_phase*nz_phase);
//            //correct the normal color gradient by considering the effect of grey solid
//			nx = -(t1-t2+0.5*(t7-t8+t9-t10+t11-t12+t13-t14));
//			ny = -(t3-t4+0.5*(t7-t8-t9+t10+t15-t16+t17-t18));
//			nz = -(t5-t6+0.5*(t11-t12-t13+t14+t15-t16-t17+t18));
//
//            if (C_phase==0.0){
//                nx = nx_phase; 
//                ny = ny_phase;
//                nz = nz_phase;
//            }
//
//			//...........Normalize the Color Gradient.................................
//			C = sqrt(nx*nx+ny*ny+nz*nz);
//			double ColorMag = C;
//			if (C==0.0) ColorMag=1.0;
//			nx = nx/ColorMag;
//			ny = ny/ColorMag;
//			nz = nz/ColorMag;		
//
//			// q=0
//			fq = dist[n];
//			rho = fq;
//			m1  = -30.0*fq;
//			m2  = 12.0*fq;
//
//			// q=1
//			fq = dist[2*Np+n];
//			rho += fq;
//			m1 -= 11.0*fq;
//			m2 -= 4.0*fq;
//			jx = fq;
//			m4 = -4.0*fq;
//			m9 = 2.0*fq;
//			m10 = -4.0*fq;
//
//			// f2 = dist[10*Np+n];
//			fq = dist[1*Np+n];
//			rho += fq;
//			m1 -= 11.0*(fq);
//			m2 -= 4.0*(fq);
//			jx -= fq;
//			m4 += 4.0*(fq);
//			m9 += 2.0*(fq);
//			m10 -= 4.0*(fq);
//
//			// q=3
//			fq = dist[4*Np+n];
//			rho += fq;
//			m1 -= 11.0*fq;
//			m2 -= 4.0*fq;
//			jy = fq;
//			m6 = -4.0*fq;
//			m9 -= fq;
//			m10 += 2.0*fq;
//			m11 = fq;
//			m12 = -2.0*fq;
//
//			// q = 4
//			fq = dist[3*Np+n];
//			rho+= fq;
//			m1 -= 11.0*fq;
//			m2 -= 4.0*fq;
//			jy -= fq;
//			m6 += 4.0*fq;
//			m9 -= fq;
//			m10 += 2.0*fq;
//			m11 += fq;
//			m12 -= 2.0*fq;
//
//			// q=5
//			fq = dist[6*Np+n];
//			rho += fq;
//			m1 -= 11.0*fq;
//			m2 -= 4.0*fq;
//			jz = fq;
//			m8 = -4.0*fq;
//			m9 -= fq;
//			m10 += 2.0*fq;
//			m11 -= fq;
//			m12 += 2.0*fq;
//
//			// q = 6
//			fq = dist[5*Np+n];
//			rho+= fq;
//			m1 -= 11.0*fq;
//			m2 -= 4.0*fq;
//			jz -= fq;
//			m8 += 4.0*fq;
//			m9 -= fq;
//			m10 += 2.0*fq;
//			m11 -= fq;
//			m12 += 2.0*fq;
//
//			// q=7
//			fq = dist[8*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx += fq;
//			m4 += fq;
//			jy += fq;
//			m6 += fq;
//			m9  += fq;
//			m10 += fq;
//			m11 += fq;
//			m12 += fq;
//			m13 = fq;
//			m16 = fq;
//			m17 = -fq;
//
//			// q = 8
//			fq = dist[7*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx -= fq;
//			m4 -= fq;
//			jy -= fq;
//			m6 -= fq;
//			m9 += fq;
//			m10 += fq;
//			m11 += fq;
//			m12 += fq;
//			m13 += fq;
//			m16 -= fq;
//			m17 += fq;
//
//			// q=9
//			fq = dist[10*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx += fq;
//			m4 += fq;
//			jy -= fq;
//			m6 -= fq;
//			m9 += fq;
//			m10 += fq;
//			m11 += fq;
//			m12 += fq;
//			m13 -= fq;
//			m16 += fq;
//			m17 += fq;
//
//			// q = 10
//			fq = dist[9*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx -= fq;
//			m4 -= fq;
//			jy += fq;
//			m6 += fq;
//			m9 += fq;
//			m10 += fq;
//			m11 += fq;
//			m12 += fq;
//			m13 -= fq;
//			m16 -= fq;
//			m17 -= fq;
//
//			// q=11
//			fq = dist[12*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx += fq;
//			m4 += fq;
//			jz += fq;
//			m8 += fq;
//			m9 += fq;
//			m10 += fq;
//			m11 -= fq;
//			m12 -= fq;
//			m15 = fq;
//			m16 -= fq;
//			m18 = fq;
//
//			// q=12
//			fq = dist[11*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx -= fq;
//			m4 -= fq;
//			jz -= fq;
//			m8 -= fq;
//			m9 += fq;
//			m10 += fq;
//			m11 -= fq;
//			m12 -= fq;
//			m15 += fq;
//			m16 += fq;
//			m18 -= fq;
//
//			// q=13
//			fq = dist[14*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx += fq;
//			m4 += fq;
//			jz -= fq;
//			m8 -= fq;
//			m9 += fq;
//			m10 += fq;
//			m11 -= fq;
//			m12 -= fq;
//			m15 -= fq;
//			m16 -= fq;
//			m18 -= fq;
//
//			// q=14
//			fq = dist[13*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jx -= fq;
//			m4 -= fq;
//			jz += fq;
//			m8 += fq;
//			m9 += fq;
//			m10 += fq;
//			m11 -= fq;
//			m12 -= fq;
//			m15 -= fq;
//			m16 += fq;
//			m18 += fq;
//
//			// q=15
//			fq = dist[16*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jy += fq;
//			m6 += fq;
//			jz += fq;
//			m8 += fq;
//			m9 -= 2.0*fq;
//			m10 -= 2.0*fq;
//			m14 = fq;
//			m17 += fq;
//			m18 -= fq;
//
//			// q=16
//			fq = dist[15*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jy -= fq;
//			m6 -= fq;
//			jz -= fq;
//			m8 -= fq;
//			m9 -= 2.0*fq;
//			m10 -= 2.0*fq;
//			m14 += fq;
//			m17 -= fq;
//			m18 += fq;
//
//			// q=17
//			fq = dist[18*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jy += fq;
//			m6 += fq;
//			jz -= fq;
//			m8 -= fq;
//			m9 -= 2.0*fq;
//			m10 -= 2.0*fq;
//			m14 -= fq;
//			m17 += fq;
//			m18 += fq;
//
//			// q=18
//			fq = dist[17*Np+n];
//			rho += fq;
//			m1 += 8.0*fq;
//			m2 += fq;
//			jy -= fq;
//			m6 -= fq;
//			jz += fq;
//			m8 += fq;
//			m9 -= 2.0*fq;
//			m10 -= 2.0*fq;
//			m14 -= fq;
//			m17 -= fq;
//			m18 -= fq;
//
//            // Compute greyscale related parameters
//            c0 = 0.5*(1.0+porosity*0.5*mu_eff/perm);
//            if (porosity==1.0) c0 = 0.5;//i.e. apparent pore nodes
//            //GeoFun = 1.75/sqrt(150.0*porosity*porosity*porosity);
//            c1 = porosity*0.5*GeoFun/sqrt(perm);
//            if (porosity==1.0) c1 = 0.0;//i.e. apparent pore nodes
//
//            vx = jx/rho0+0.5*(porosity*Gx);
//            vy = jy/rho0+0.5*(porosity*Gy);
//            vz = jz/rho0+0.5*(porosity*Gz);
//            v_mag=sqrt(vx*vx+vy*vy+vz*vz);
//            ux = vx/(c0+sqrt(c0*c0+c1*v_mag));
//            uy = vy/(c0+sqrt(c0*c0+c1*v_mag));
//            uz = vz/(c0+sqrt(c0*c0+c1*v_mag));
//            u_mag=sqrt(ux*ux+uy*uy+uz*uz);
//
//            //Update the total force to include linear (Darcy) and nonlinear (Forchheimer) drags due to the porous medium
//            Fx = rho0*(-porosity*mu_eff/perm*ux - porosity*GeoFun/sqrt(perm)*u_mag*ux + porosity*Gx);
//            Fy = rho0*(-porosity*mu_eff/perm*uy - porosity*GeoFun/sqrt(perm)*u_mag*uy + porosity*Gy);
//            Fz = rho0*(-porosity*mu_eff/perm*uz - porosity*GeoFun/sqrt(perm)*u_mag*uz + porosity*Gz);
//            if (porosity==1.0){
//                Fx=rho0*(Gx);
//                Fy=rho0*(Gy);
//                Fz=rho0*(Gz);
//            }
//
//			// write the velocity 
//			Velocity[n] = ux;
//			Velocity[Np+n] = uy;
//			Velocity[2*Np+n] = uz;
//
//			//........................................................................
//			//..............carry out relaxation process..............................
//			//..........Toelke, Fruediger et. al. 2006................................
//			if (C == 0.0)	nx = ny = nz = 0.0;
//			m1 = m1 + rlx_setA*((19*(ux*ux+uy*uy+uz*uz)*rho0/porosity - 11*rho) -19*alpha*C - m1);
//			m2 = m2 + rlx_setA*((3*rho - 5.5*(ux*ux+uy*uy+uz*uz)*rho0/porosity)- m2);
//            jx = jx + Fx;
//			m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0)- m4)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
//            jy = jy + Fy;
//			m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0)- m6)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
//            jz = jz + Fz;
//			m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0)- m8)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
//			m9 = m9 + rlx_setA*(((2*ux*ux-uy*uy-uz*uz)*rho0/porosity) + 0.5*alpha*C*(2*nx*nx-ny*ny-nz*nz) - m9);
//			m10 = m10 + rlx_setA*( - m10);
//            //m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10);
//			m11 = m11 + rlx_setA*(((uy*uy-uz*uz)*rho0/porosity) + 0.5*alpha*C*(ny*ny-nz*nz)- m11);
//			m12 = m12 + rlx_setA*( - m12);
//            //m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12);
//			m13 = m13 + rlx_setA*( (ux*uy*rho0/porosity) + 0.5*alpha*C*nx*ny - m13);
//			m14 = m14 + rlx_setA*( (uy*uz*rho0/porosity) + 0.5*alpha*C*ny*nz - m14);
//			m15 = m15 + rlx_setA*( (ux*uz*rho0/porosity) + 0.5*alpha*C*nx*nz - m15);
//			m16 = m16 + rlx_setB*( - m16);
//			m17 = m17 + rlx_setB*( - m17);
//			m18 = m18 + rlx_setB*( - m18);
//
//			//.................inverse transformation......................................................
//			// q=0
//			fq = mrt_V1*rho-mrt_V2*m1+mrt_V3*m2;
//			dist[n] = fq;
//
//			// q = 1
//			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10);
//			dist[1*Np+n] = fq;
//
//			// q=2
//			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10);
//			dist[2*Np+n] = fq;
//
//			// q = 3
//			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
//			dist[3*Np+n] = fq;
//
//			// q = 4
//			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
//			dist[4*Np+n] = fq;
//
//			// q = 5
//			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
//			dist[5*Np+n] = fq;
//
//			// q = 6
//			fq = mrt_V1*rho-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
//			dist[6*Np+n] = fq;
//
//			// q = 7
//			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+
//					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17);
//			dist[7*Np+n] = fq;
//
//
//			// q = 8
//			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11
//					+mrt_V12*m12+0.25*m13+0.125*(m17-m16);
//			dist[8*Np+n] = fq;
//
//			// q = 9
//			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+
//					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17);
//			dist[9*Np+n] = fq;
//
//			// q = 10
//			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+
//					mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17);
//			dist[10*Np+n] = fq;
//
//
//			// q = 11
//			fq = mrt_V1*rho+mrt_V9*m1
//					+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)
//					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
//					-mrt_V12*m12+0.25*m15+0.125*(m18-m16);
//			dist[11*Np+n] = fq;
//
//			// q = 12
//			fq = mrt_V1*rho+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+
//					mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18);
//			dist[12*Np+n] = fq;
//
//			// q = 13
//			fq = mrt_V1*rho+mrt_V9*m1
//					+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)
//					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
//					-mrt_V12*m12-0.25*m15-0.125*(m16+m18);
//			dist[13*Np+n] = fq;
//
//			// q= 14
//			fq = mrt_V1*rho+mrt_V9*m1
//					+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)
//					+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11
//					-mrt_V12*m12-0.25*m15+0.125*(m16+m18);
//
//			dist[14*Np+n] = fq;
//
//			// q = 15
//			fq = mrt_V1*rho+mrt_V9*m1
//					+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)
//					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18);
//			dist[15*Np+n] = fq;
//
//			// q = 16
//			fq =  mrt_V1*rho+mrt_V9*m1
//					+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)
//					-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17);
//			dist[16*Np+n] = fq;
//
//
//			// q = 17
//			fq = mrt_V1*rho+mrt_V9*m1
//					+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)
//					-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18);
//			dist[17*Np+n] = fq;
//
//			// q = 18
//			fq = mrt_V1*rho+mrt_V9*m1
//					+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)
//					-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18);
//			dist[18*Np+n] = fq;
//			//........................................................................
//
//			// Instantiate mass transport distributions
//			// Stationary value - distribution 0
//			nAB = 1.0/(nA+nB);
//			Aq[n] = 0.3333333333333333*nA;
//			Bq[n] = 0.3333333333333333*nB;
//
//			//...............................................
//			// q = 0,2,4
//			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
//			delta = beta*nA*nB*nAB*0.1111111111111111*nx;
//			if (!(nA*nB*nAB>0)) delta=0;
//			a1 = nA*(0.1111111111111111*(1+4.5*ux))+delta;
//			b1 = nB*(0.1111111111111111*(1+4.5*ux))-delta;
//			a2 = nA*(0.1111111111111111*(1-4.5*ux))-delta;
//			b2 = nB*(0.1111111111111111*(1-4.5*ux))+delta;
//
//			Aq[1*Np+n] = a1;
//			Bq[1*Np+n] = b1;
//			Aq[2*Np+n] = a2;
//			Bq[2*Np+n] = b2;
//
//			//...............................................
//			// q = 2
//			// Cq = {0,1,0}
//			delta = beta*nA*nB*nAB*0.1111111111111111*ny;
//			if (!(nA*nB*nAB>0)) delta=0;
//			a1 = nA*(0.1111111111111111*(1+4.5*uy))+delta;
//			b1 = nB*(0.1111111111111111*(1+4.5*uy))-delta;
//			a2 = nA*(0.1111111111111111*(1-4.5*uy))-delta;
//			b2 = nB*(0.1111111111111111*(1-4.5*uy))+delta;
//
//			Aq[3*Np+n] = a1;
//			Bq[3*Np+n] = b1;
//			Aq[4*Np+n] = a2;
//			Bq[4*Np+n] = b2;
//			//...............................................
//			// q = 4
//			// Cq = {0,0,1}
//			delta = beta*nA*nB*nAB*0.1111111111111111*nz;
//			if (!(nA*nB*nAB>0)) delta=0;
//			a1 = nA*(0.1111111111111111*(1+4.5*uz))+delta;
//			b1 = nB*(0.1111111111111111*(1+4.5*uz))-delta;
//			a2 = nA*(0.1111111111111111*(1-4.5*uz))-delta;
//			b2 = nB*(0.1111111111111111*(1-4.5*uz))+delta;
//
//			Aq[5*Np+n] = a1;
//			Bq[5*Np+n] = b1;
//			Aq[6*Np+n] = a2;
//			Bq[6*Np+n] = b2;
//			//...............................................
//
//		}
//	}
//}

//__global__ void dvc_ScaLBL_D3Q19_GreyscaleColor_Init(double *dist, double *Porosity, int Np)
//{
//	int n;
//	int S = Np/NBLOCKS/NTHREADS + 1;
//    double porosity;
//	for (int s=0; s<S; s++){
//		//........Get 1-D index for this thread....................
//		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
//		if (n<Np ){
//            porosity = Porosity[n];
//            if (porosity==0.0) porosity=1.f;
//			dist[n] = 0.3333333333333333/porosity;
//			dist[Np+n] = 0.055555555555555555/porosity;		//double(100*n)+1.f;
//			dist[2*Np+n] = 0.055555555555555555/porosity;	//double(100*n)+2.f;
//			dist[3*Np+n] = 0.055555555555555555/porosity;	//double(100*n)+3.f;
//			dist[4*Np+n] = 0.055555555555555555/porosity;	//double(100*n)+4.f;
//			dist[5*Np+n] = 0.055555555555555555/porosity;	//double(100*n)+5.f;
//			dist[6*Np+n] = 0.055555555555555555/porosity;	//double(100*n)+6.f;
//			dist[7*Np+n] = 0.0277777777777778/porosity;   //double(100*n)+7.f;
//			dist[8*Np+n] = 0.0277777777777778/porosity;   //double(100*n)+8.f;
//			dist[9*Np+n] = 0.0277777777777778/porosity;   //double(100*n)+9.f;
//			dist[10*Np+n] = 0.0277777777777778/porosity;  //double(100*n)+10.f;
//			dist[11*Np+n] = 0.0277777777777778/porosity;  //double(100*n)+11.f;
//			dist[12*Np+n] = 0.0277777777777778/porosity;  //double(100*n)+12.f;
//			dist[13*Np+n] = 0.0277777777777778/porosity;  //double(100*n)+13.f;
//			dist[14*Np+n] = 0.0277777777777778/porosity;  //double(100*n)+14.f;
//			dist[15*Np+n] = 0.0277777777777778/porosity;  //double(100*n)+15.f;
//			dist[16*Np+n] = 0.0277777777777778/porosity;  //double(100*n)+16.f;
//			dist[17*Np+n] = 0.0277777777777778/porosity;  //double(100*n)+17.f;
//			dist[18*Np+n] = 0.0277777777777778/porosity;  //double(100*n)+18.f;
//		}
//	}
//}


//extern "C" void ScaLBL_D3Q19_GreyscaleColor_Init(double *dist, double *Porosity, int Np){
//	dvc_ScaLBL_D3Q19_GreyscaleColor_Init<<<NBLOCKS,NTHREADS >>>(dist,Porosity,Np);
//	cudaError_t err = cudaGetLastError();
//	if (cudaSuccess != err){
//		printf("CUDA error in ScaLBL_D3Q19_GreyscaleColor_Init: %s \n",cudaGetErrorString(err));
//	}
//}

//Model-1 & 4
extern "C" void ScaLBL_D3Q19_AAeven_GreyscaleColor(int *Map, double *dist, double *Aq, double *Bq, double *Den, 
        double *Phi,double *GreySolidGrad, double *Poros,double *Perm,double *Vel, 
        double rhoA, double rhoB, double tauA, double tauB,double tauA_eff,double tauB_eff, double alpha, double beta,
		double Fx, double Fy, double Fz, int strideY, int strideZ, int start, int finish, int Np){

	//cudaProfilerStart();
	//cudaFuncSetCacheConfig(dvc_ScaLBL_D3Q19_AAeven_GreyscaleColor, cudaFuncCachePreferL1);

	dvc_ScaLBL_D3Q19_AAeven_GreyscaleColor<<<NBLOCKS,NTHREADS >>>(Map, dist, Aq, Bq, Den, Phi, GreySolidGrad, Poros, Perm, Vel, 
            rhoA, rhoB, tauA, tauB, tauA_eff, tauB_eff, alpha, beta, Fx, Fy, Fz, strideY, strideZ, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_GreyscaleColor: %s \n",hipGetErrorString(err));
	}
	//cudaProfilerStop();

}

//Model-1 & 4
extern "C" void ScaLBL_D3Q19_AAodd_GreyscaleColor(int *d_neighborList, int *Map, double *dist, double *Aq, double *Bq, double *Den, 
		double *Phi, double *GreySolidGrad, double *Poros,double *Perm,double *Vel, 
        double rhoA, double rhoB, double tauA, double tauB, double tauA_eff,double tauB_eff, double alpha, double beta,
		double Fx, double Fy, double Fz, int strideY, int strideZ, int start, int finish, int Np){

	//cudaProfilerStart();
	//cudaFuncSetCacheConfig(dvc_ScaLBL_D3Q19_AAodd_GreyscaleColor, cudaFuncCachePreferL1);
	
	dvc_ScaLBL_D3Q19_AAodd_GreyscaleColor<<<NBLOCKS,NTHREADS >>>(d_neighborList, Map, dist, Aq, Bq, Den, Phi,  GreySolidGrad, Poros, Perm,Vel, 
			rhoA, rhoB, tauA, tauB, tauA_eff, tauB_eff,alpha, beta, Fx, Fy, Fz, strideY, strideZ, start, finish, Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_GreyscaleColor: %s \n",hipGetErrorString(err));
	}
	//cudaProfilerStop();
}

////Model-2&3
//extern "C" void ScaLBL_D3Q19_AAeven_GreyscaleColor(int *Map, double *dist, double *Aq, double *Bq, double *Den, 
//        double *Phi,double *GreySolidGrad, double *Poros,double *Perm,double *Vel, 
//        double rhoA, double rhoB, double tauA, double tauB,double tauA_eff,double tauB_eff, double alpha, double beta,
//		double Fx, double Fy, double Fz, int strideY, int strideZ, int start, int finish, int Np){
//
//	//cudaProfilerStart();
//	//cudaFuncSetCacheConfig(dvc_ScaLBL_D3Q19_AAeven_GreyscaleColor, cudaFuncCachePreferL1);
//
//	dvc_ScaLBL_D3Q19_AAeven_GreyscaleColor<<<NBLOCKS,NTHREADS >>>(Map, dist, Aq, Bq, Den, Phi, GreySolidGrad, Poros, Perm, Vel, 
//            rhoA, rhoB, tauA, tauB, tauA_eff, tauB_eff, alpha, beta, Fx, Fy, Fz, strideY, strideZ, start, finish, Np);
//	cudaError_t err = cudaGetLastError();
//	if (cudaSuccess != err){
//		printf("CUDA error in ScaLBL_D3Q19_AAeven_GreyscaleColor: %s \n",cudaGetErrorString(err));
//	}
//	//cudaProfilerStop();
//
//}
//
////Model-2&3
//extern "C" void ScaLBL_D3Q19_AAodd_GreyscaleColor(int *d_neighborList, int *Map, double *dist, double *Aq, double *Bq, double *Den, 
//		double *Phi, double *GreySolidGrad, double *Poros,double *Perm,double *Vel, 
//        double rhoA, double rhoB, double tauA, double tauB, double tauA_eff,double tauB_eff, double alpha, double beta,
//		double Fx, double Fy, double Fz, int strideY, int strideZ, int start, int finish, int Np){
//
//	//cudaProfilerStart();
//	//cudaFuncSetCacheConfig(dvc_ScaLBL_D3Q19_AAodd_GreyscaleColor, cudaFuncCachePreferL1);
//	
//	dvc_ScaLBL_D3Q19_AAodd_GreyscaleColor<<<NBLOCKS,NTHREADS >>>(d_neighborList, Map, dist, Aq, Bq, Den, Phi,  GreySolidGrad, Poros, Perm,Vel, 
//			rhoA, rhoB, tauA, tauB, tauA_eff, tauB_eff,alpha, beta, Fx, Fy, Fz, strideY, strideZ, start, finish, Np);
//
//	cudaError_t err = cudaGetLastError();
//	if (cudaSuccess != err){
//		printf("CUDA error in ScaLBL_D3Q19_AAodd_GreyscaleColor: %s \n",cudaGetErrorString(err));
//	}
//	//cudaProfilerStop();
//}
