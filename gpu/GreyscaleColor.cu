
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS 1024
#define NTHREADS 256

__global__ void dvc_ScaLBL_D3Q19_AAeven_GreyscaleColor(double *dist, double *Aq, double *Bq, double *Den,
                double *DenGradA, double *DenGradB, double *SolidForce, int start, int finish, int Np,
                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double Gsc, double Gx, double Gy, double Gz,
                double *Poros,double *Perm, double *Velocity,double *Pressure){

	int n;
	double vx,vy,vz,v_mag;
    double ux,uy,uz,u_mag;
    double pressure;//defined for this incompressible model
	// conserved momemnts
	double jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
    double fq;
    // currently disable 'GeoFun'
    double GeoFun=0.0;//geometric function from Guo's PRE 66, 036304 (2002)
    double porosity;
    double perm;//voxel permeability
    double c0, c1; //Guo's model parameters
    double Fx, Fy, Fz;//The total body force including Brinkman force and user-specified (Gx,Gy,Gz)
	double tau,tau_eff,rlx_setA,rlx_setB;
    double mu_eff;//effective kinematic viscosity for Darcy term
    double rho0;
    double phi;
    double nx,ny,nz,C;
    double nA,nB;
	double a1,b1,a2,b2,nAB,delta;
    double beta=0.95;
    double nA_gradx,nA_grady,nA_gradz;
    double nB_gradx,nB_grady,nB_gradz;
    double Gff_x,Gff_y,Gff_z;
    double Gfs_x,Gfs_y,Gfs_z;


	const double mrt_V1=0.05263157894736842;
	const double mrt_V2=0.012531328320802;
	const double mrt_V3=0.04761904761904762;
	const double mrt_V4=0.004594820384294068;
	const double mrt_V5=0.01587301587301587;
	const double mrt_V6=0.0555555555555555555555555;
	const double mrt_V7=0.02777777777777778;
	const double mrt_V8=0.08333333333333333;
	const double mrt_V9=0.003341687552213868;
	const double mrt_V10=0.003968253968253968;
	const double mrt_V11=0.01388888888888889;
	const double mrt_V12=0.04166666666666666;


	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
	    //........Get 1-D index for this thread....................
	    n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;

		if ( n<finish ){

			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];
			nA_gradx = DenGradA[n+0*Np];
			nA_grady = DenGradA[n+1*Np];
			nA_gradz = DenGradA[n+2*Np];
			nB_gradx = DenGradB[n+0*Np];
			nB_grady = DenGradB[n+1*Np];
			nB_gradz = DenGradB[n+2*Np];

			// compute phase indicator field
			phi=(nA-nB)/(nA+nB);

			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
			// local relaxation time
			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
			rlx_setA = 1.f/tau;
			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);
			tau_eff=tauA_eff + 0.5*(1.0-phi)*(tauB_eff-tauA_eff);
            mu_eff = (tau_eff-0.5)/3.f;//kinematic viscosity


            //........................................................................
            //					READ THE DISTRIBUTIONS
            //		(read from opposite array due to previous swap operation)
            //........................................................................
            // q=0
            fq = dist[n];
            m1  = -30.0*fq;
            m2  = 12.0*fq;

            // q=1
            fq = dist[2*Np+n];
            pressure = fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jx = fq;
            m4 = -4.0*fq;
            m9 = 2.0*fq;
            m10 = -4.0*fq;

            // f2 = dist[10*Np+n];
            fq = dist[1*Np+n];
            pressure += fq;
            m1 -= 11.0*(fq);
            m2 -= 4.0*(fq);
            jx -= fq;
            m4 += 4.0*(fq);
            m9 += 2.0*(fq);
            m10 -= 4.0*(fq);

            // q=3
            fq = dist[4*Np+n];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jy = fq;
            m6 = -4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 = fq;
            m12 = -2.0*fq;

            // q = 4
            fq = dist[3*Np+n];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jy -= fq;
            m6 += 4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 += fq;
            m12 -= 2.0*fq;

            // q=5
            fq = dist[6*Np+n];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jz = fq;
            m8 = -4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 -= fq;
            m12 += 2.0*fq;

            // q = 6
            fq = dist[5*Np+n];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jz -= fq;
            m8 += 4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 -= fq;
            m12 += 2.0*fq;

            // q=7
            fq = dist[8*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jy += fq;
            m6 += fq;
            m9  += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 = fq;
            m16 = fq;
            m17 = -fq;

            // q = 8
            fq = dist[7*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jy -= fq;
            m6 -= fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 += fq;
            m16 -= fq;
            m17 += fq;

            // q=9
            fq = dist[10*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jy -= fq;
            m6 -= fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 -= fq;
            m16 += fq;
            m17 += fq;

            // q = 10
            fq = dist[9*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jy += fq;
            m6 += fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 -= fq;
            m16 -= fq;
            m17 -= fq;

            // q=11
            fq = dist[12*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jz += fq;
            m8 += fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 = fq;
            m16 -= fq;
            m18 = fq;

            // q=12
            fq = dist[11*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jz -= fq;
            m8 -= fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 += fq;
            m16 += fq;
            m18 -= fq;

            // q=13
            fq = dist[14*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jz -= fq;
            m8 -= fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 -= fq;
            m16 -= fq;
            m18 -= fq;

            // q=14
            fq = dist[13*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jz += fq;
            m8 += fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 -= fq;
            m16 += fq;
            m18 += fq;

            // q=15
            fq = dist[16*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy += fq;
            m6 += fq;
            jz += fq;
            m8 += fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 = fq;
            m17 += fq;
            m18 -= fq;

            // q=16
            fq = dist[15*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy -= fq;
            m6 -= fq;
            jz -= fq;
            m8 -= fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 += fq;
            m17 -= fq;
            m18 += fq;

            // q=17
            fq = dist[18*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy += fq;
            m6 += fq;
            jz -= fq;
            m8 -= fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 -= fq;
            m17 += fq;
            m18 += fq;

            // q=18
            fq = dist[17*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy -= fq;
            m6 -= fq;
            jz += fq;
            m8 += fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 -= fq;
            m17 -= fq;
            m18 -= fq;
            //---------------------------------------------------------------------//

            //---------------- Calculate SC fluid-fluid and fluid-solid forces ---------------//
            // fluid-fluid force
            Gff_x = -Gsc*nA*nB_gradx*int(phi>0.0)-Gsc*nB*nA_gradx*int(phi<0.0);
            Gff_y = -Gsc*nA*nB_grady*int(phi>0.0)-Gsc*nB*nA_grady*int(phi<0.0);
            Gff_z = -Gsc*nA*nB_gradz*int(phi>0.0)-Gsc*nB*nA_gradz*int(phi<0.0);
            // fluid-solid force
            Gfs_x = (nA-nB)*SolidForce[n+0*Np];    
            Gfs_y = (nA-nB)*SolidForce[n+1*Np];    
            Gfs_z = (nA-nB)*SolidForce[n+2*Np];    

            porosity = Poros[n];
            // use local saturation as an estimation of effective relperm values
            perm = Perm[n]*nA/(nA+nB)*int(phi>0.0)+Perm[n]*nB/(nA+nB)*int(phi<0.0);

            c0 = 0.5*(1.0+porosity*0.5*mu_eff/perm);
            if (porosity==1.0) c0 = 0.5;//i.e. apparent pore nodes
            //GeoFun = 1.75/sqrt(150.0*porosity*porosity*porosity);
            c1 = porosity*0.5*GeoFun/sqrt(perm);
            if (porosity==1.0) c1 = 0.0;//i.e. apparent pore nodes

            vx = jx/rho0+0.5*(porosity*Gx+Gff_x+Gfs_x);
            vy = jy/rho0+0.5*(porosity*Gy+Gff_y+Gfs_y);
            vz = jz/rho0+0.5*(porosity*Gz+Gff_z+Gfs_z);
            v_mag=sqrt(vx*vx+vy*vy+vz*vz);
            ux = vx/(c0+sqrt(c0*c0+c1*v_mag));
            uy = vy/(c0+sqrt(c0*c0+c1*v_mag));
            uz = vz/(c0+sqrt(c0*c0+c1*v_mag));
            u_mag=sqrt(ux*ux+uy*uy+uz*uz);

            //Update the total force to include linear (Darcy) and nonlinear (Forchheimer) drags due to the porous medium
            Fx = rho0*(-porosity*mu_eff/perm*ux - porosity*GeoFun/sqrt(perm)*u_mag*ux + porosity*Gx + Gff_x + Gfs_x);
            Fy = rho0*(-porosity*mu_eff/perm*uy - porosity*GeoFun/sqrt(perm)*u_mag*uy + porosity*Gy + Gff_y + Gfs_y);
            Fz = rho0*(-porosity*mu_eff/perm*uz - porosity*GeoFun/sqrt(perm)*u_mag*uz + porosity*Gz + Gff_z + Gfs_z);
            if (porosity==1.0){
                Fx=rho0*(Gx + Gff_x + Gfs_x);
                Fy=rho0*(Gy + Gff_y + Gfs_y);
                Fz=rho0*(Gz + Gff_z + Gfs_z);
            }

            //Calculate pressure for Incompressible-MRT model
            pressure=0.5/porosity*(pressure-0.5*rho0*u_mag*u_mag/porosity);

//            //..............carry out relaxation process...............................................
//            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1) 
//                    + (1-0.5*rlx_setA)*38*(Fx*ux+Fy*uy+Fz*uz)/porosity;
//            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2)
//                    + (1-0.5*rlx_setA)*11*(-Fx*ux-Fy*uy-Fz*uz)/porosity;
//            jx = jx + Fx;
//            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
//            jy = jy + Fy;
//            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
//            jz = jz + Fz;
//            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
//            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9)
//                    + (1-0.5*rlx_setA)*(4*Fx*ux-2*Fy*uy-2*Fz*uz)/porosity;
//            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10)
//                      + (1-0.5*rlx_setA)*(-2*Fx*ux+Fy*uy+Fz*uz)/porosity;
//            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11)
//                      + (1-0.5*rlx_setA)*(2*Fy*uy-2*Fz*uz)/porosity;
//            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12)
//                      + (1-0.5*rlx_setA)*(-Fy*uy+Fz*uz)/porosity;
//            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13)
//                      + (1-0.5*rlx_setA)*(Fy*ux+Fx*uy)/porosity;
//            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14)
//                      + (1-0.5*rlx_setA)*(Fz*uy+Fy*uz)/porosity;
//            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15)
//                      + (1-0.5*rlx_setA)*(Fz*ux+Fx*uz)/porosity;
//            m16 = m16 + rlx_setB*( - m16);
//            m17 = m17 + rlx_setB*( - m17);
//            m18 = m18 + rlx_setB*( - m18);
//            //.......................................................................................................

            //-------------------- IMRT collison where body force has NO higher-order terms -------------//
            //..............carry out relaxation process...............................................
            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1);
            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2);
            jx = jx + Fx;
            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
            jy = jy + Fy;
            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
            jz = jz + Fz;
            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9);
            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10);
            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11);
            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12);
            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13);
            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14);
            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15);
            m16 = m16 + rlx_setB*( - m16);
            m17 = m17 + rlx_setB*( - m17);
            m18 = m18 + rlx_setB*( - m18);
            //.......................................................................................................

            //.................inverse transformation......................................................
            // q=0
            fq = mrt_V1*rho0-mrt_V2*m1+mrt_V3*m2;
            dist[n] = fq;

            // q = 1
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10);
            dist[1*Np+n] = fq;

            // q=2
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10);
            dist[2*Np+n] = fq;

            // q = 3
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
            dist[3*Np+n] = fq;

            // q = 4
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
            dist[4*Np+n] = fq;

            // q = 5
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
            dist[5*Np+n] = fq;

            // q = 6
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
            dist[6*Np+n] = fq;

            // q = 7
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17);
            dist[7*Np+n] = fq;

            // q = 8
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m17-m16);
            dist[8*Np+n] = fq;

            // q = 9
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17);
            dist[9*Np+n] = fq;

            // q = 10
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17);
            dist[10*Np+n] = fq;

            // q = 11
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m18-m16);
            dist[11*Np+n] = fq;

            // q = 12
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18);
            dist[12*Np+n] = fq;

            // q = 13
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15-0.125*(m16+m18);
            dist[13*Np+n] = fq;

            // q= 14
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15+0.125*(m16+m18);
            dist[14*Np+n] = fq;

            // q = 15
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18);
            dist[15*Np+n] = fq;

            // q = 16
            fq =  mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17);
            dist[16*Np+n] = fq;

            // q = 17
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18);
            dist[17*Np+n] = fq;

            // q = 18
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18);
            dist[18*Np+n] = fq;
            //........................................................................

            //Update velocity on device
            Velocity[0*Np+n] = ux;
            Velocity[1*Np+n] = uy;
            Velocity[2*Np+n] = uz;
            //Update pressure on device
            Pressure[n] = pressure;

            //-----------------------Mass transport------------------------//
            // Calculate the color gradient
            nx = (2*nB*nA_gradx-2*nA*nB_gradx)/(nA+nB)/(nA+nB); 
            ny = (2*nB*nA_grady-2*nA*nB_grady)/(nA+nB)/(nA+nB); 
            nz = (2*nB*nA_gradz-2*nA*nB_gradz)/(nA+nB)/(nA+nB); 
			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			double ColorMag = C;
			if (C==0.0) ColorMag=1.0;
			nx = nx/ColorMag;
			ny = ny/ColorMag;
			nz = nz/ColorMag;		
			if (C == 0.0)	nx = ny = nz = 0.0;

			// Instantiate mass transport distributions
			// Stationary value - distribution 0
			nAB = 1.0/(nA+nB);
			Aq[n] = 0.3333333333333333*nA;
			Bq[n] = 0.3333333333333333*nB;

			//...............................................
			// q = 0,2,4
			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nx;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*ux))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*ux))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*ux))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*ux))+delta;

			Aq[1*Np+n] = a1;
			Bq[1*Np+n] = b1;
			Aq[2*Np+n] = a2;
			Bq[2*Np+n] = b2;

			//...............................................
			// q = 2
			// Cq = {0,1,0}
			delta = beta*nA*nB*nAB*0.1111111111111111*ny;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uy))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uy))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uy))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uy))+delta;

			Aq[3*Np+n] = a1;
			Bq[3*Np+n] = b1;
			Aq[4*Np+n] = a2;
			Bq[4*Np+n] = b2;
			//...............................................
			// q = 4
			// Cq = {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nz;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uz))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uz))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uz))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uz))+delta;

			Aq[5*Np+n] = a1;
			Bq[5*Np+n] = b1;
			Aq[6*Np+n] = a2;
			Bq[6*Np+n] = b2;
			//...............................................

		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_AAodd_GreyscaleColor(int *neighborList, double *dist, double *Aq, double *Bq, double *Den,
                double *DenGradA, double *DenGradB, double *SolidForce, int start, int finish, int Np,
                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double Gsc, double Gx, double Gy, double Gz,
                double *Poros,double *Perm, double *Velocity,double *Pressure){

	int n, nread, nr1,nr2,nr3,nr4,nr5,nr6;
	double vx,vy,vz,v_mag;
    double ux,uy,uz,u_mag;
    double pressure;//defined for this incompressible model
	// conserved momemnts
	double jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
    double fq;
    // currently disable 'GeoFun'
    double GeoFun=0.0;//geometric function from Guo's PRE 66, 036304 (2002)
    double porosity;
    double perm;//voxel permeability
    double c0, c1; //Guo's model parameters
    double Fx, Fy, Fz;//The total body force including Brinkman force and user-specified (Gx,Gy,Gz)
	double tau,tau_eff,rlx_setA,rlx_setB;
    double mu_eff;//effective kinematic viscosity for Darcy term
    double rho0;
    double phi;
    double nx,ny,nz,C;
    double nA,nB;
	double a1,b1,a2,b2,nAB,delta;
    double beta=0.95;
    double nA_gradx,nA_grady,nA_gradz;
    double nB_gradx,nB_grady,nB_gradz;
    double Gff_x,Gff_y,Gff_z;
    double Gfs_x,Gfs_y,Gfs_z;

	const double mrt_V1=0.05263157894736842;
	const double mrt_V2=0.012531328320802;
	const double mrt_V3=0.04761904761904762;
	const double mrt_V4=0.004594820384294068;
	const double mrt_V5=0.01587301587301587;
	const double mrt_V6=0.0555555555555555555555555;
	const double mrt_V7=0.02777777777777778;
	const double mrt_V8=0.08333333333333333;
	const double mrt_V9=0.003341687552213868;
	const double mrt_V10=0.003968253968253968;
	const double mrt_V11=0.01388888888888889;
	const double mrt_V12=0.04166666666666666;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
	    //........Get 1-D index for this thread....................
	    n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;

		if ( n<finish ){		

			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];
			nA_gradx = DenGradA[n+0*Np];
			nA_grady = DenGradA[n+1*Np];
			nA_gradz = DenGradA[n+2*Np];
			nB_gradx = DenGradB[n+0*Np];
			nB_grady = DenGradB[n+1*Np];
			nB_gradz = DenGradB[n+2*Np];

			// compute phase indicator field
			phi=(nA-nB)/(nA+nB);

			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
			// local relaxation time
			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
			rlx_setA = 1.f/tau;
			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);
			tau_eff=tauA_eff + 0.5*(1.0-phi)*(tauB_eff-tauA_eff);
            mu_eff = (tau_eff-0.5)/3.f;//kinematic viscosity

            //........................................................................
            //					READ THE DISTRIBUTIONS
            //		(read from opposite array due to previous swap operation)
            //........................................................................
            // q=0
            fq = dist[n];
            m1  = -30.0*fq;
            m2  = 12.0*fq;

            // q=1
            nr1 = neighborList[n]; // neighbor 2 ( > 10Np => odd part of dist)
            fq = dist[nr1]; // reading the f1 data into register fq
            pressure = fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jx = fq;
            m4 = -4.0*fq;
            m9 = 2.0*fq;
            m10 = -4.0*fq;

            // q=2
            nr2 = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
            fq = dist[nr2];  // reading the f2 data into register fq
            pressure += fq;
            m1 -= 11.0*(fq);
            m2 -= 4.0*(fq);
            jx -= fq;
            m4 += 4.0*(fq);
            m9 += 2.0*(fq);
            m10 -= 4.0*(fq);

            // q=3
            nr3 = neighborList[n+2*Np]; // neighbor 4
            fq = dist[nr3];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jy = fq;
            m6 = -4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 = fq;
            m12 = -2.0*fq;

            // q = 4
            nr4 = neighborList[n+3*Np]; // neighbor 3
            fq = dist[nr4];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jy -= fq;
            m6 += 4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 += fq;
            m12 -= 2.0*fq;

            // q=5
            nr5 = neighborList[n+4*Np];
            fq = dist[nr5];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jz = fq;
            m8 = -4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 -= fq;
            m12 += 2.0*fq;

            // q = 6
            nr6 = neighborList[n+5*Np];
            fq = dist[nr6];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jz -= fq;
            m8 += 4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 -= fq;
            m12 += 2.0*fq;

            // q=7
            nread = neighborList[n+6*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jy += fq;
            m6 += fq;
            m9  += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 = fq;
            m16 = fq;
            m17 = -fq;

            // q = 8
            nread = neighborList[n+7*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jy -= fq;
            m6 -= fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 += fq;
            m16 -= fq;
            m17 += fq;

            // q=9
            nread = neighborList[n+8*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jy -= fq;
            m6 -= fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 -= fq;
            m16 += fq;
            m17 += fq;

            // q = 10
            nread = neighborList[n+9*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jy += fq;
            m6 += fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 -= fq;
            m16 -= fq;
            m17 -= fq;

            // q=11
            nread = neighborList[n+10*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jz += fq;
            m8 += fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 = fq;
            m16 -= fq;
            m18 = fq;

            // q=12
            nread = neighborList[n+11*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jz -= fq;
            m8 -= fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 += fq;
            m16 += fq;
            m18 -= fq;

            // q=13
            nread = neighborList[n+12*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jz -= fq;
            m8 -= fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 -= fq;
            m16 -= fq;
            m18 -= fq;

            // q=14
            nread = neighborList[n+13*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jz += fq;
            m8 += fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 -= fq;
            m16 += fq;
            m18 += fq;

            // q=15
            nread = neighborList[n+14*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy += fq;
            m6 += fq;
            jz += fq;
            m8 += fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 = fq;
            m17 += fq;
            m18 -= fq;

            // q=16
            nread = neighborList[n+15*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy -= fq;
            m6 -= fq;
            jz -= fq;
            m8 -= fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 += fq;
            m17 -= fq;
            m18 += fq;

            // q=17
            nread = neighborList[n+16*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy += fq;
            m6 += fq;
            jz -= fq;
            m8 -= fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 -= fq;
            m17 += fq;
            m18 += fq;

            // q=18
            nread = neighborList[n+17*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy -= fq;
            m6 -= fq;
            jz += fq;
            m8 += fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 -= fq;
            m17 -= fq;
            m18 -= fq;
            //---------------------------------------------------------------------//

            //---------------- Calculate SC fluid-fluid and fluid-solid forces ---------------//
            // fluid-fluid force
            Gff_x = -Gsc*nA*nB_gradx*int(phi>0.0)-Gsc*nB*nA_gradx*int(phi<0.0);
            Gff_y = -Gsc*nA*nB_grady*int(phi>0.0)-Gsc*nB*nA_grady*int(phi<0.0);
            Gff_z = -Gsc*nA*nB_gradz*int(phi>0.0)-Gsc*nB*nA_gradz*int(phi<0.0);
            // fluid-solid force
            Gfs_x = (nA-nB)*SolidForce[n+0*Np];    
            Gfs_y = (nA-nB)*SolidForce[n+1*Np];    
            Gfs_z = (nA-nB)*SolidForce[n+2*Np];    

            porosity = Poros[n];
            // use local saturation as an estimation of effective relperm values
            perm = Perm[n]*nA/(nA+nB)*int(phi>0.0)+Perm[n]*nB/(nA+nB)*int(phi<0.0);

            c0 = 0.5*(1.0+porosity*0.5*mu_eff/perm);
            if (porosity==1.0) c0 = 0.5;//i.e. apparent pore nodes
            //GeoFun = 1.75/sqrt(150.0*porosity*porosity*porosity);
            c1 = porosity*0.5*GeoFun/sqrt(perm);
            if (porosity==1.0) c1 = 0.0;//i.e. apparent pore nodes

            vx = jx/rho0+0.5*(porosity*Gx+Gff_x+Gfs_x);
            vy = jy/rho0+0.5*(porosity*Gy+Gff_y+Gfs_y);
            vz = jz/rho0+0.5*(porosity*Gz+Gff_z+Gfs_z);
            v_mag=sqrt(vx*vx+vy*vy+vz*vz);
            ux = vx/(c0+sqrt(c0*c0+c1*v_mag));
            uy = vy/(c0+sqrt(c0*c0+c1*v_mag));
            uz = vz/(c0+sqrt(c0*c0+c1*v_mag));
            u_mag=sqrt(ux*ux+uy*uy+uz*uz);

            //Update the total force to include linear (Darcy) and nonlinear (Forchheimer) drags due to the porous medium
            Fx = rho0*(-porosity*mu_eff/perm*ux - porosity*GeoFun/sqrt(perm)*u_mag*ux + porosity*Gx + Gff_x + Gfs_x);
            Fy = rho0*(-porosity*mu_eff/perm*uy - porosity*GeoFun/sqrt(perm)*u_mag*uy + porosity*Gy + Gff_y + Gfs_y);
            Fz = rho0*(-porosity*mu_eff/perm*uz - porosity*GeoFun/sqrt(perm)*u_mag*uz + porosity*Gz + Gff_z + Gfs_z);
            if (porosity==1.0){
                Fx=rho0*(Gx + Gff_x + Gfs_x);
                Fy=rho0*(Gy + Gff_y + Gfs_y);
                Fz=rho0*(Gz + Gff_z + Gfs_z);
            }

            //Calculate pressure for Incompressible-MRT model
            pressure=0.5/porosity*(pressure-0.5*rho0*u_mag*u_mag/porosity);

//            //..............carry out relaxation process...............................................
//            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1) 
//                    + (1-0.5*rlx_setA)*38*(Fx*ux+Fy*uy+Fz*uz)/porosity;
//            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2)
//                    + (1-0.5*rlx_setA)*11*(-Fx*ux-Fy*uy-Fz*uz)/porosity;
//            jx = jx + Fx;
//            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
//            jy = jy + Fy;
//            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
//            jz = jz + Fz;
//            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
//            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9)
//                    + (1-0.5*rlx_setA)*(4*Fx*ux-2*Fy*uy-2*Fz*uz)/porosity;
//            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10)
//                      + (1-0.5*rlx_setA)*(-2*Fx*ux+Fy*uy+Fz*uz)/porosity;
//            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11)
//                      + (1-0.5*rlx_setA)*(2*Fy*uy-2*Fz*uz)/porosity;
//            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12)
//                      + (1-0.5*rlx_setA)*(-Fy*uy+Fz*uz)/porosity;
//            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13)
//                      + (1-0.5*rlx_setA)*(Fy*ux+Fx*uy)/porosity;
//            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14)
//                      + (1-0.5*rlx_setA)*(Fz*uy+Fy*uz)/porosity;
//            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15)
//                      + (1-0.5*rlx_setA)*(Fz*ux+Fx*uz)/porosity;
//            m16 = m16 + rlx_setB*( - m16);
//            m17 = m17 + rlx_setB*( - m17);
//            m18 = m18 + rlx_setB*( - m18);
//            //.......................................................................................................
           
            //-------------------- IMRT collison where body force has NO higher-order terms -------------//
            //..............carry out relaxation process...............................................
            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1);
            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2);
            jx = jx + Fx;
            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
            jy = jy + Fy;
            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
            jz = jz + Fz;
            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9);
            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10);
            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11);
            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12);
            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13);
            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14);
            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15);
            m16 = m16 + rlx_setB*( - m16);
            m17 = m17 + rlx_setB*( - m17);
            m18 = m18 + rlx_setB*( - m18);
            //.......................................................................................................


            //.................inverse transformation......................................................
            // q=0
            fq = mrt_V1*rho0-mrt_V2*m1+mrt_V3*m2;
            dist[n] = fq;

            // q = 1
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10);
            //nread = neighborList[n+Np];
            dist[nr2] = fq;

            // q=2
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10);
            //nread = neighborList[n];
            dist[nr1] = fq;

            // q = 3
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
            //nread = neighborList[n+3*Np];
            dist[nr4] = fq;

            // q = 4
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
            //nread = neighborList[n+2*Np];
            dist[nr3] = fq;

            // q = 5
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
            //nread = neighborList[n+5*Np];
            dist[nr6] = fq;

            // q = 6
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
            //nread = neighborList[n+4*Np];
            dist[nr5] = fq;

            // q = 7
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17);
            nread = neighborList[n+7*Np];
            dist[nread] = fq;

            // q = 8
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m17-m16);
            nread = neighborList[n+6*Np];
            dist[nread] = fq;

            // q = 9
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17);
            nread = neighborList[n+9*Np];
            dist[nread] = fq;

            // q = 10
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17);
            nread = neighborList[n+8*Np];
            dist[nread] = fq;

            // q = 11
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m18-m16);
            nread = neighborList[n+11*Np];
            dist[nread] = fq;

            // q = 12
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18);
            nread = neighborList[n+10*Np];
            dist[nread]= fq;

            // q = 13
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15-0.125*(m16+m18);
            nread = neighborList[n+13*Np];
            dist[nread] = fq;

            // q= 14
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15+0.125*(m16+m18);
            nread = neighborList[n+12*Np];
            dist[nread] = fq;

            // q = 15
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18);
            nread = neighborList[n+15*Np];
            dist[nread] = fq;

            // q = 16
            fq =  mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17);
            nread = neighborList[n+14*Np];
            dist[nread] = fq;

            // q = 17
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18);
            nread = neighborList[n+17*Np];
            dist[nread] = fq;

            // q = 18
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18);
            nread = neighborList[n+16*Np];
            dist[nread] = fq;
            //........................................................................

            //Update velocity on device
            Velocity[0*Np+n] = ux;
            Velocity[1*Np+n] = uy;
            Velocity[2*Np+n] = uz;
            //Update pressure on device
            Pressure[n] = pressure;

            //-----------------------Mass transport------------------------//
            // Calculate the color gradient
            nx = (2*nB*nA_gradx-2*nA*nB_gradx)/(nA+nB)/(nA+nB); 
            ny = (2*nB*nA_grady-2*nA*nB_grady)/(nA+nB)/(nA+nB); 
            nz = (2*nB*nA_gradz-2*nA*nB_gradz)/(nA+nB)/(nA+nB); 
			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			double ColorMag = C;
			if (C==0.0) ColorMag=1.0;
			nx = nx/ColorMag;
			ny = ny/ColorMag;
			nz = nz/ColorMag;		
			if (C == 0.0)	nx = ny = nz = 0.0;

			// Instantiate mass transport distributions
			// Stationary value - distribution 0
			nAB = 1.0/(nA+nB);
			Aq[n] = 0.3333333333333333*nA;
			Bq[n] = 0.3333333333333333*nB;

			//...............................................
			// q = 0,2,4
			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nx;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*ux))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*ux))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*ux))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*ux))+delta;

			// q = 1
			//nread = neighborList[n+Np];
			Aq[nr2] = a1;
			Bq[nr2] = b1;
			// q=2
			//nread = neighborList[n];
			Aq[nr1] = a2;
			Bq[nr1] = b2;

			//...............................................
			// Cq = {0,1,0}
			delta = beta*nA*nB*nAB*0.1111111111111111*ny;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uy))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uy))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uy))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uy))+delta;

			// q = 3
			//nread = neighborList[n+3*Np];
			Aq[nr4] = a1;
			Bq[nr4] = b1;
			// q = 4
			//nread = neighborList[n+2*Np];
			Aq[nr3] = a2;
			Bq[nr3] = b2;

			//...............................................
			// q = 4
			// Cq = {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nz;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uz))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uz))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uz))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uz))+delta;

			// q = 5
			//nread = neighborList[n+5*Np];
			Aq[nr6] = a1;
			Bq[nr6] = b1;
			// q = 6
			//nread = neighborList[n+4*Np];
			Aq[nr5] = a2;
			Bq[nr5] = b2;
			//...............................................
		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_GreyColorIMRT_Init(double *dist, double *Den, double rhoA, double rhoB, int Np){
	int n;
	int S = Np/NBLOCKS/NTHREADS + 1;
    double phi;
    double nA,nB;
    double Den0;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<Np ){
            nA = Den[n];
            nB = Den[n+Np];
            phi = (nA-nB)/(nA+nB);
            Den0 = 0.5*(1.f+phi)*rhoA + 0.5*(1.f-phi)*rhoB;

			dist[n] = Den0 - 0.6666666666666667;
			dist[Np+n] = 0.055555555555555555;		//double(100*n)+1.f;
			dist[2*Np+n] = 0.055555555555555555;	//double(100*n)+2.f;
			dist[3*Np+n] = 0.055555555555555555;	//double(100*n)+3.f;
			dist[4*Np+n] = 0.055555555555555555;	//double(100*n)+4.f;
			dist[5*Np+n] = 0.055555555555555555;	//double(100*n)+5.f;
			dist[6*Np+n] = 0.055555555555555555;	//double(100*n)+6.f;
			dist[7*Np+n] = 0.0277777777777778;   //double(100*n)+7.f;
			dist[8*Np+n] = 0.0277777777777778;   //double(100*n)+8.f;
			dist[9*Np+n] = 0.0277777777777778;   //double(100*n)+9.f;
			dist[10*Np+n] = 0.0277777777777778;  //double(100*n)+10.f;
			dist[11*Np+n] = 0.0277777777777778;  //double(100*n)+11.f;
			dist[12*Np+n] = 0.0277777777777778;  //double(100*n)+12.f;
			dist[13*Np+n] = 0.0277777777777778;  //double(100*n)+13.f;
			dist[14*Np+n] = 0.0277777777777778;  //double(100*n)+14.f;
			dist[15*Np+n] = 0.0277777777777778;  //double(100*n)+15.f;
			dist[16*Np+n] = 0.0277777777777778;  //double(100*n)+16.f;
			dist[17*Np+n] = 0.0277777777777778;  //double(100*n)+17.f;
			dist[18*Np+n] = 0.0277777777777778;  //double(100*n)+18.f;
		}
	}
}

__global__ void dvc_ScaLBL_D3Q7_GreyColorIMRT_Init(double *Den, double *Aq, double *Bq, int start, int finish, int Np){
	int idx;
    double nA,nB;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		idx =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (idx<finish) {
            nA = Den[idx];
            nB = Den[idx+Np];

			Aq[idx]=0.3333333333333333*nA;
			Aq[Np+idx]=0.1111111111111111*nA;
			Aq[2*Np+idx]=0.1111111111111111*nA;
			Aq[3*Np+idx]=0.1111111111111111*nA;
			Aq[4*Np+idx]=0.1111111111111111*nA;
			Aq[5*Np+idx]=0.1111111111111111*nA;
			Aq[6*Np+idx]=0.1111111111111111*nA;

			Bq[idx]=0.3333333333333333*nB;
			Bq[Np+idx]=0.1111111111111111*nB;
			Bq[2*Np+idx]=0.1111111111111111*nB;
			Bq[3*Np+idx]=0.1111111111111111*nB;
			Bq[4*Np+idx]=0.1111111111111111*nB;
			Bq[5*Np+idx]=0.1111111111111111*nB;
			Bq[6*Np+idx]=0.1111111111111111*nB;
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_AAodd_GreyscaleColorDensity(int *neighborList, double *Aq, double *Bq, double *Den, int start, int finish, int Np){
	int n,nread;
	double fq,nA,nB;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			//..........Compute the number density for each component ............
			// q=0
			fq = Aq[n];
			nA = fq;
			fq = Bq[n];
			nB = fq;
			
			// q=1
			nread = neighborList[n]; 
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread]; 
			nB += fq;
			
			// q=2
			nread = neighborList[n+Np]; 
			fq = Aq[nread];  
			nA += fq;
			fq = Bq[nread]; 
			nB += fq;
			
			// q=3
			nread = neighborList[n+2*Np]; 
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;
			
			// q = 4
			nread = neighborList[n+3*Np]; 
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;

			// q=5
			nread = neighborList[n+4*Np];
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;
			
			// q = 6
			nread = neighborList[n+5*Np];
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;

			// save the number densities
			Den[n] = nA;
			Den[Np+n] = nB;
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_AAeven_GreyscaleColorDensity(double *Aq, double *Bq, double *Den, int start, int finish, int Np){
	int n;
	double fq,nA,nB;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			// compute number density for each component
			// q=0
			fq = Aq[n];
			nA = fq;
			fq = Bq[n];
			nB = fq;
			
			// q=1
			fq = Aq[2*Np+n];
			nA += fq;
			fq = Bq[2*Np+n];
			nB += fq;

			// q=2
			fq = Aq[1*Np+n];
			nA += fq;
			fq = Bq[1*Np+n];
			nB += fq;

			// q=3
			fq = Aq[4*Np+n];
			nA += fq;
			fq = Bq[4*Np+n];
			nB += fq;

			// q = 4
			fq = Aq[3*Np+n];
			nA += fq;
			fq = Bq[3*Np+n];
			nB += fq;
			
			// q=5
			fq = Aq[6*Np+n];
			nA += fq;
			fq = Bq[6*Np+n];
			nB += fq;
			
			// q = 6
			fq = Aq[5*Np+n];
			nA += fq;
			fq = Bq[5*Np+n];
			nB += fq;

			// save the number densities
			Den[n] = nA;
			Den[Np+n] = nB;
		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_GreyscaleColor_Gradient(int *neighborList, double *Den, double *DenGrad, int start, int finish, int Np){

	int n,nn;
	// distributions
	double m1,m2,m3,m4,m5,m6,m7,m8,m9;
	double m10,m11,m12,m13,m14,m15,m16,m17,m18;
	double nx,ny,nz;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			nn = neighborList[n+Np]%Np;
			m1 = Den[nn]*int(n!=nn);
			nn = neighborList[n]%Np;
			m2 = Den[nn]*int(n!=nn);
			nn = neighborList[n+3*Np]%Np;
			m3 = Den[nn]*int(n!=nn);
			nn = neighborList[n+2*Np]%Np;
			m4 = Den[nn]*int(n!=nn);		
			nn = neighborList[n+5*Np]%Np;
			m5 = Den[nn]*int(n!=nn);
			nn = neighborList[n+4*Np]%Np;
			m6 = Den[nn]*int(n!=nn);		
			nn = neighborList[n+7*Np]%Np;
			m7 = Den[nn]*int(n!=nn);
			nn = neighborList[n+6*Np]%Np;
			m8 = Den[nn]*int(n!=nn);		
			nn = neighborList[n+9*Np]%Np;
			m9 = Den[nn]*int(n!=nn);
			nn = neighborList[n+8*Np]%Np;
			m10 = Den[nn]*int(n!=nn);		
			nn = neighborList[n+11*Np]%Np;
			m11 = Den[nn]*int(n!=nn);
			nn = neighborList[n+10*Np]%Np;
			m12 = Den[nn]*int(n!=nn);		
			nn = neighborList[n+13*Np]%Np;
			m13 = Den[nn]*int(n!=nn);
			nn = neighborList[n+12*Np]%Np;
			m14 = Den[nn]*int(n!=nn);		
			nn = neighborList[n+15*Np]%Np;
			m15 = Den[nn]*int(n!=nn);
			nn = neighborList[n+14*Np]%Np;
			m16 = Den[nn]*int(n!=nn);		
			nn = neighborList[n+17*Np]%Np;
			m17 = Den[nn]*int(n!=nn);
			nn = neighborList[n+16*Np]%Np;
			m18 = Den[nn]*int(n!=nn);					
			
			//............Compute the Color Gradient...................................
			nx = 1.f/18.f*(m1-m2+0.5*(m7-m8+m9-m10+m11-m12+m13-m14));
			ny = 1.f/18.f*(m3-m4+0.5*(m7-m8-m9+m10+m15-m16+m17-m18));
			nz = 1.f/18.f*(m5-m6+0.5*(m11-m12-m13+m14+m15-m16-m17+m18));
			
			DenGrad[n] = nx;
			DenGrad[Np+n] = ny;
			DenGrad[2*Np+n] = nz;
		}
	}
}


extern "C" void ScaLBL_D3Q19_AAeven_GreyscaleColor(double *dist, double *Aq, double *Bq, double *Den,
                double *DenGradA, double *DenGradB, double *SolidForce, int start, int finish, int Np,
                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double Gsc, double Gx, double Gy, double Gz,
                double *Poros,double *Perm, double *Velocity,double *Pressure){

    dvc_ScaLBL_D3Q19_AAeven_GreyscaleColor<<<NBLOCKS,NTHREADS >>>(dist, Aq, Bq, Den, DenGradA, DenGradB, SolidForce, start, finish, Np,
                                                                 tauA, tauB, tauA_eff, tauB_eff, rhoA, rhoB, Gsc, Gx, Gy, Gz, Poros, Perm, Velocity, Pressure);

    hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_GreyscaleColor: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAodd_GreyscaleColor(int *neighborList, double *dist, double *Aq, double *Bq, double *Den,
                double *DenGradA, double *DenGradB, double *SolidForce, int start, int finish, int Np,
                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double Gsc, double Gx, double Gy, double Gz,
                double *Poros,double *Perm, double *Velocity,double *Pressure){

    dvc_ScaLBL_D3Q19_AAodd_GreyscaleColor<<<NBLOCKS,NTHREADS >>>(neighborList, dist, Aq, Bq, Den, DenGradA, DenGradB, SolidForce, start, finish, Np,
                                                                 tauA, tauB, tauA_eff, tauB_eff, rhoA, rhoB, Gsc, Gx, Gy, Gz, Poros, Perm, Velocity, Pressure);

    hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_GreyscaleColor: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q7_GreyColorIMRT_Init(double *Den, double *Aq, double *Bq, int start, int finish, int Np){
	dvc_ScaLBL_D3Q7_GreyColorIMRT_Init<<<NBLOCKS,NTHREADS >>>(Den, Aq, Bq, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_GreyColorIMRT_Init: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_GreyColorIMRT_Init(double *dist, double *Den, double rhoA, double rhoB, int Np){
	dvc_ScaLBL_D3Q19_GreyColorIMRT_Init<<<NBLOCKS,NTHREADS >>>(dist,Den,rhoA,rhoB,Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_GreyColorIMRT_Init: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q7_AAodd_GreyscaleColorDensity(int *NeighborList, double *Aq, double *Bq, double *Den, int start, int finish, int Np){

	dvc_ScaLBL_D3Q7_AAodd_GreyscaleColorDensity<<<NBLOCKS,NTHREADS >>>(NeighborList, Aq, Bq, Den, start, finish, Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_AAodd_GreyscaleColorDensity: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q7_AAeven_GreyscaleColorDensity(double *Aq, double *Bq, double *Den, int start, int finish, int Np){

	dvc_ScaLBL_D3Q7_AAeven_GreyscaleColorDensity<<<NBLOCKS,NTHREADS >>>(Aq, Bq, Den, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_AAeven_GreyscaleColorDensity: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_GreyscaleColor_Gradient(int *neighborList, double *Den, double *DenGrad, int start, int finish, int Np){

	dvc_ScaLBL_D3Q19_GreyscaleColor_Gradient<<<NBLOCKS,NTHREADS >>>(neighborList, Den, DenGrad, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_GreyscaleColor_Gradient: %s \n",hipGetErrorString(err));
	}
}

