
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS 1024
#define NTHREADS 256

__global__ void dvc_ScaLBL_D3Q19_GreyscaleColor_Pressure(double *dist, double *Den, double *Poros,double *Velocity,
                double *Pressure, double rhoA,double rhoB, int N){

	int n;
    double ux,uy,uz,u_mag;
    double pressure;
    double porosity;
    double rho0;
    double phi;
    double nA,nB;

	int S = N/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
	    //........Get 1-D index for this thread....................
	    n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;

		if (n<N){		

            // initialize pressure value
            pressure = 0.0;
            pressure +=dist[1*N+n];
            pressure +=dist[2*N+n];
            pressure +=dist[3*N+n];
            pressure +=dist[4*N+n];
            pressure +=dist[5*N+n];
            pressure +=dist[6*N+n];
            pressure +=dist[7*N+n];
            pressure +=dist[8*N+n];
            pressure +=dist[9*N+n];
            pressure +=dist[10*N+n];
            pressure +=dist[11*N+n];
            pressure +=dist[12*N+n];
            pressure +=dist[13*N+n];
            pressure +=dist[14*N+n];
            pressure +=dist[15*N+n];
            pressure +=dist[16*N+n];
            pressure +=dist[17*N+n];
            pressure +=dist[18*N+n];

			// read the component number densities
			nA = Den[n];
			nB = Den[N + n];
			// compute phase indicator field
			phi=(nA-nB)/(nA+nB);
			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
            // read voxel porosity 
            porosity = Poros[n];
            // read velocity
            ux = Velocity[0*N+n]; 
            uy = Velocity[1*N+n];
            uz = Velocity[2*N+n];
            u_mag=sqrt(ux*ux+uy*uy+uz*uz);

            //Calculate pressure for Incompressible-MRT model
            pressure=0.5/porosity*(pressure-0.5*rho0*u_mag*u_mag/porosity);
            //Update pressure on device
            Pressure[n] = pressure;
		}
	}
}


__global__ void dvc_ScaLBL_D3Q19_AAeven_GreyscaleColor(double *dist, double *Aq, double *Bq, double *Den,
                double *DenGradA, double *DenGradB, double *SolidForce, int start, int finish, int Np,
                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double Gsc, double Gx, double Gy, double Gz,
                double *Poros,double *Perm, double *Velocity,double *Pressure){

	int n;
	double vx,vy,vz,v_mag;
    double ux,uy,uz,u_mag;
    double pressure;//defined for this incompressible model
	// conserved momemnts
	double jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
    double fq;
    // currently disable 'GeoFun'
    double GeoFun=0.0;//geometric function from Guo's PRE 66, 036304 (2002)
    double porosity;
    double perm;//voxel permeability
    double c0, c1; //Guo's model parameters
    double Fx, Fy, Fz;//The total body force including Brinkman force and user-specified (Gx,Gy,Gz)
	double tau,tau_eff,rlx_setA,rlx_setB;
    double mu_eff;//effective kinematic viscosity for Darcy term
    double rho0;
    double phi;
    double nx,ny,nz,C;
    double nA,nB;
	double a1,b1,a2,b2,nAB,delta;
    double beta=0.95;
    double nA_gradx,nA_grady,nA_gradz;
    double nB_gradx,nB_grady,nB_gradz;
    double Gff_x,Gff_y,Gff_z;
    double Gfs_x,Gfs_y,Gfs_z;


	const double mrt_V1=0.05263157894736842;
	const double mrt_V2=0.012531328320802;
	const double mrt_V3=0.04761904761904762;
	const double mrt_V4=0.004594820384294068;
	const double mrt_V5=0.01587301587301587;
	const double mrt_V6=0.0555555555555555555555555;
	const double mrt_V7=0.02777777777777778;
	const double mrt_V8=0.08333333333333333;
	const double mrt_V9=0.003341687552213868;
	const double mrt_V10=0.003968253968253968;
	const double mrt_V11=0.01388888888888889;
	const double mrt_V12=0.04166666666666666;


	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
	    //........Get 1-D index for this thread....................
	    n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;

		if ( n<finish ){

			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];
			nA_gradx = DenGradA[n+0*Np];
			nA_grady = DenGradA[n+1*Np];
			nA_gradz = DenGradA[n+2*Np];
			nB_gradx = DenGradB[n+0*Np];
			nB_grady = DenGradB[n+1*Np];
			nB_gradz = DenGradB[n+2*Np];

			// compute phase indicator field
			phi=(nA-nB)/(nA+nB);

			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
			// local relaxation time
			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
			rlx_setA = 1.f/tau;
			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);
			tau_eff=tauA_eff + 0.5*(1.0-phi)*(tauB_eff-tauA_eff);
            mu_eff = (tau_eff-0.5)/3.f;//kinematic viscosity


            //........................................................................
            //					READ THE DISTRIBUTIONS
            //		(read from opposite array due to previous swap operation)
            //........................................................................
            // q=0
            fq = dist[n];
            m1  = -30.0*fq;
            m2  = 12.0*fq;

            // q=1
            fq = dist[2*Np+n];
            pressure = fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jx = fq;
            m4 = -4.0*fq;
            m9 = 2.0*fq;
            m10 = -4.0*fq;

            // f2 = dist[10*Np+n];
            fq = dist[1*Np+n];
            pressure += fq;
            m1 -= 11.0*(fq);
            m2 -= 4.0*(fq);
            jx -= fq;
            m4 += 4.0*(fq);
            m9 += 2.0*(fq);
            m10 -= 4.0*(fq);

            // q=3
            fq = dist[4*Np+n];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jy = fq;
            m6 = -4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 = fq;
            m12 = -2.0*fq;

            // q = 4
            fq = dist[3*Np+n];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jy -= fq;
            m6 += 4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 += fq;
            m12 -= 2.0*fq;

            // q=5
            fq = dist[6*Np+n];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jz = fq;
            m8 = -4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 -= fq;
            m12 += 2.0*fq;

            // q = 6
            fq = dist[5*Np+n];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jz -= fq;
            m8 += 4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 -= fq;
            m12 += 2.0*fq;

            // q=7
            fq = dist[8*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jy += fq;
            m6 += fq;
            m9  += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 = fq;
            m16 = fq;
            m17 = -fq;

            // q = 8
            fq = dist[7*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jy -= fq;
            m6 -= fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 += fq;
            m16 -= fq;
            m17 += fq;

            // q=9
            fq = dist[10*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jy -= fq;
            m6 -= fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 -= fq;
            m16 += fq;
            m17 += fq;

            // q = 10
            fq = dist[9*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jy += fq;
            m6 += fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 -= fq;
            m16 -= fq;
            m17 -= fq;

            // q=11
            fq = dist[12*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jz += fq;
            m8 += fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 = fq;
            m16 -= fq;
            m18 = fq;

            // q=12
            fq = dist[11*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jz -= fq;
            m8 -= fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 += fq;
            m16 += fq;
            m18 -= fq;

            // q=13
            fq = dist[14*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jz -= fq;
            m8 -= fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 -= fq;
            m16 -= fq;
            m18 -= fq;

            // q=14
            fq = dist[13*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jz += fq;
            m8 += fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 -= fq;
            m16 += fq;
            m18 += fq;

            // q=15
            fq = dist[16*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy += fq;
            m6 += fq;
            jz += fq;
            m8 += fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 = fq;
            m17 += fq;
            m18 -= fq;

            // q=16
            fq = dist[15*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy -= fq;
            m6 -= fq;
            jz -= fq;
            m8 -= fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 += fq;
            m17 -= fq;
            m18 += fq;

            // q=17
            fq = dist[18*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy += fq;
            m6 += fq;
            jz -= fq;
            m8 -= fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 -= fq;
            m17 += fq;
            m18 += fq;

            // q=18
            fq = dist[17*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy -= fq;
            m6 -= fq;
            jz += fq;
            m8 += fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 -= fq;
            m17 -= fq;
            m18 -= fq;
            //---------------------------------------------------------------------//

            //---------------- Calculate SC fluid-fluid and fluid-solid forces ---------------//
            // fluid-fluid force
//            Gff_x = -Gsc*nA*nB_gradx*int(phi>0.0)-Gsc*nB*nA_gradx*int(phi<0.0);
//            Gff_y = -Gsc*nA*nB_grady*int(phi>0.0)-Gsc*nB*nA_grady*int(phi<0.0);
//            Gff_z = -Gsc*nA*nB_gradz*int(phi>0.0)-Gsc*nB*nA_gradz*int(phi<0.0);
            Gff_x = -Gsc*(nA*nB_gradx+nB*nA_gradx);
            Gff_y = -Gsc*(nA*nB_grady+nB*nA_grady);
            Gff_z = -Gsc*(nA*nB_gradz+nB*nA_gradz);
            // fluid-solid force
            Gfs_x = (nA-nB)*SolidForce[n+0*Np];    
            Gfs_y = (nA-nB)*SolidForce[n+1*Np];    
            Gfs_z = (nA-nB)*SolidForce[n+2*Np];    

            porosity = Poros[n];
            // use local saturation as an estimation of effective relperm values
            perm = Perm[n]*nA/(nA+nB)*int(phi>0.0)+Perm[n]*nB/(nA+nB)*int(phi<0.0);

            c0 = 0.5*(1.0+porosity*0.5*mu_eff/perm);
            if (porosity==1.0) c0 = 0.5;//i.e. apparent pore nodes
            //GeoFun = 1.75/sqrt(150.0*porosity*porosity*porosity);
            c1 = porosity*0.5*GeoFun/sqrt(perm);
            if (porosity==1.0) c1 = 0.0;//i.e. apparent pore nodes

            vx = jx/rho0+0.5*(porosity*Gx+Gff_x+Gfs_x);
            vy = jy/rho0+0.5*(porosity*Gy+Gff_y+Gfs_y);
            vz = jz/rho0+0.5*(porosity*Gz+Gff_z+Gfs_z);
            v_mag=sqrt(vx*vx+vy*vy+vz*vz);
            ux = vx/(c0+sqrt(c0*c0+c1*v_mag));
            uy = vy/(c0+sqrt(c0*c0+c1*v_mag));
            uz = vz/(c0+sqrt(c0*c0+c1*v_mag));
            u_mag=sqrt(ux*ux+uy*uy+uz*uz);

            //Update the total force to include linear (Darcy) and nonlinear (Forchheimer) drags due to the porous medium
            Fx = rho0*(-porosity*mu_eff/perm*ux - porosity*GeoFun/sqrt(perm)*u_mag*ux + porosity*Gx + Gff_x + Gfs_x);
            Fy = rho0*(-porosity*mu_eff/perm*uy - porosity*GeoFun/sqrt(perm)*u_mag*uy + porosity*Gy + Gff_y + Gfs_y);
            Fz = rho0*(-porosity*mu_eff/perm*uz - porosity*GeoFun/sqrt(perm)*u_mag*uz + porosity*Gz + Gff_z + Gfs_z);
            if (porosity==1.0){
                Fx=rho0*(Gx + Gff_x + Gfs_x);
                Fy=rho0*(Gy + Gff_y + Gfs_y);
                Fz=rho0*(Gz + Gff_z + Gfs_z);
            }

            //Calculate pressure for Incompressible-MRT model
            pressure=0.5/porosity*(pressure-0.5*rho0*u_mag*u_mag/porosity);

//            //..............carry out relaxation process...............................................
//            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1) 
//                    + (1-0.5*rlx_setA)*38*(Fx*ux+Fy*uy+Fz*uz)/porosity;
//            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2)
//                    + (1-0.5*rlx_setA)*11*(-Fx*ux-Fy*uy-Fz*uz)/porosity;
//            jx = jx + Fx;
//            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
//            jy = jy + Fy;
//            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
//            jz = jz + Fz;
//            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
//            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9)
//                    + (1-0.5*rlx_setA)*(4*Fx*ux-2*Fy*uy-2*Fz*uz)/porosity;
//            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10)
//                      + (1-0.5*rlx_setA)*(-2*Fx*ux+Fy*uy+Fz*uz)/porosity;
//            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11)
//                      + (1-0.5*rlx_setA)*(2*Fy*uy-2*Fz*uz)/porosity;
//            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12)
//                      + (1-0.5*rlx_setA)*(-Fy*uy+Fz*uz)/porosity;
//            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13)
//                      + (1-0.5*rlx_setA)*(Fy*ux+Fx*uy)/porosity;
//            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14)
//                      + (1-0.5*rlx_setA)*(Fz*uy+Fy*uz)/porosity;
//            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15)
//                      + (1-0.5*rlx_setA)*(Fz*ux+Fx*uz)/porosity;
//            m16 = m16 + rlx_setB*( - m16);
//            m17 = m17 + rlx_setB*( - m17);
//            m18 = m18 + rlx_setB*( - m18);
//            //.......................................................................................................

            //-------------------- IMRT collison where body force has NO higher-order terms -------------//
            //..............carry out relaxation process...............................................
            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1);
            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2);
            jx = jx + Fx;
            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
            jy = jy + Fy;
            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
            jz = jz + Fz;
            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9);
            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10);
            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11);
            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12);
            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13);
            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14);
            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15);
            m16 = m16 + rlx_setB*( - m16);
            m17 = m17 + rlx_setB*( - m17);
            m18 = m18 + rlx_setB*( - m18);
            //.......................................................................................................

            //.................inverse transformation......................................................
            // q=0
            fq = mrt_V1*rho0-mrt_V2*m1+mrt_V3*m2;
            dist[n] = fq;

            // q = 1
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10);
            dist[1*Np+n] = fq;

            // q=2
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10);
            dist[2*Np+n] = fq;

            // q = 3
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
            dist[3*Np+n] = fq;

            // q = 4
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
            dist[4*Np+n] = fq;

            // q = 5
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
            dist[5*Np+n] = fq;

            // q = 6
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
            dist[6*Np+n] = fq;

            // q = 7
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17);
            dist[7*Np+n] = fq;

            // q = 8
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m17-m16);
            dist[8*Np+n] = fq;

            // q = 9
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17);
            dist[9*Np+n] = fq;

            // q = 10
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17);
            dist[10*Np+n] = fq;

            // q = 11
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m18-m16);
            dist[11*Np+n] = fq;

            // q = 12
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18);
            dist[12*Np+n] = fq;

            // q = 13
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15-0.125*(m16+m18);
            dist[13*Np+n] = fq;

            // q= 14
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15+0.125*(m16+m18);
            dist[14*Np+n] = fq;

            // q = 15
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18);
            dist[15*Np+n] = fq;

            // q = 16
            fq =  mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17);
            dist[16*Np+n] = fq;

            // q = 17
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18);
            dist[17*Np+n] = fq;

            // q = 18
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18);
            dist[18*Np+n] = fq;
            //........................................................................

            //Update velocity on device
            Velocity[0*Np+n] = ux;
            Velocity[1*Np+n] = uy;
            Velocity[2*Np+n] = uz;
            //Update pressure on device
            Pressure[n] = pressure;

            //-----------------------Mass transport------------------------//
            // Calculate the color gradient
            nx = (2*nB*nA_gradx-2*nA*nB_gradx)/(nA+nB)/(nA+nB); 
            ny = (2*nB*nA_grady-2*nA*nB_grady)/(nA+nB)/(nA+nB); 
            nz = (2*nB*nA_gradz-2*nA*nB_gradz)/(nA+nB)/(nA+nB); 
			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			double ColorMag = C;
			if (C==0.0) ColorMag=1.0;
			nx = nx/ColorMag;
			ny = ny/ColorMag;
			nz = nz/ColorMag;		
			if (C == 0.0)	nx = ny = nz = 0.0;

			// Instantiate mass transport distributions
			// Stationary value - distribution 0
			nAB = 1.0/(nA+nB);
			Aq[n] = 0.3333333333333333*nA;
			Bq[n] = 0.3333333333333333*nB;

			//...............................................
			// q = 0,2,4
			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nx;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*ux))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*ux))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*ux))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*ux))+delta;

			Aq[1*Np+n] = a1;
			Bq[1*Np+n] = b1;
			Aq[2*Np+n] = a2;
			Bq[2*Np+n] = b2;

			//...............................................
			// q = 2
			// Cq = {0,1,0}
			delta = beta*nA*nB*nAB*0.1111111111111111*ny;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uy))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uy))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uy))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uy))+delta;

			Aq[3*Np+n] = a1;
			Bq[3*Np+n] = b1;
			Aq[4*Np+n] = a2;
			Bq[4*Np+n] = b2;
			//...............................................
			// q = 4
			// Cq = {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nz;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uz))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uz))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uz))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uz))+delta;

			Aq[5*Np+n] = a1;
			Bq[5*Np+n] = b1;
			Aq[6*Np+n] = a2;
			Bq[6*Np+n] = b2;
			//...............................................

		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_AAodd_GreyscaleColor(int *neighborList, double *dist, double *Aq, double *Bq, double *Den,
                double *DenGradA, double *DenGradB, double *SolidForce, int start, int finish, int Np,
                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double Gsc, double Gx, double Gy, double Gz,
                double *Poros,double *Perm, double *Velocity,double *Pressure){

	int n, nread, nr1,nr2,nr3,nr4,nr5,nr6;
	double vx,vy,vz,v_mag;
    double ux,uy,uz,u_mag;
    double pressure;//defined for this incompressible model
	// conserved momemnts
	double jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
    double fq;
    // currently disable 'GeoFun'
    double GeoFun=0.0;//geometric function from Guo's PRE 66, 036304 (2002)
    double porosity;
    double perm;//voxel permeability
    double c0, c1; //Guo's model parameters
    double Fx, Fy, Fz;//The total body force including Brinkman force and user-specified (Gx,Gy,Gz)
	double tau,tau_eff,rlx_setA,rlx_setB;
    double mu_eff;//effective kinematic viscosity for Darcy term
    double rho0;
    double phi;
    double nx,ny,nz,C;
    double nA,nB;
	double a1,b1,a2,b2,nAB,delta;
    double beta=0.95;
    double nA_gradx,nA_grady,nA_gradz;
    double nB_gradx,nB_grady,nB_gradz;
    double Gff_x,Gff_y,Gff_z;
    double Gfs_x,Gfs_y,Gfs_z;

	const double mrt_V1=0.05263157894736842;
	const double mrt_V2=0.012531328320802;
	const double mrt_V3=0.04761904761904762;
	const double mrt_V4=0.004594820384294068;
	const double mrt_V5=0.01587301587301587;
	const double mrt_V6=0.0555555555555555555555555;
	const double mrt_V7=0.02777777777777778;
	const double mrt_V8=0.08333333333333333;
	const double mrt_V9=0.003341687552213868;
	const double mrt_V10=0.003968253968253968;
	const double mrt_V11=0.01388888888888889;
	const double mrt_V12=0.04166666666666666;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
	    //........Get 1-D index for this thread....................
	    n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;

		if ( n<finish ){		

			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];
			nA_gradx = DenGradA[n+0*Np];
			nA_grady = DenGradA[n+1*Np];
			nA_gradz = DenGradA[n+2*Np];
			nB_gradx = DenGradB[n+0*Np];
			nB_grady = DenGradB[n+1*Np];
			nB_gradz = DenGradB[n+2*Np];

			// compute phase indicator field
			phi=(nA-nB)/(nA+nB);

			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
			// local relaxation time
			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
			rlx_setA = 1.f/tau;
			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);
			tau_eff=tauA_eff + 0.5*(1.0-phi)*(tauB_eff-tauA_eff);
            mu_eff = (tau_eff-0.5)/3.f;//kinematic viscosity

            //........................................................................
            //					READ THE DISTRIBUTIONS
            //		(read from opposite array due to previous swap operation)
            //........................................................................
            // q=0
            fq = dist[n];
            m1  = -30.0*fq;
            m2  = 12.0*fq;

            // q=1
            nr1 = neighborList[n]; // neighbor 2 ( > 10Np => odd part of dist)
            fq = dist[nr1]; // reading the f1 data into register fq
            pressure = fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jx = fq;
            m4 = -4.0*fq;
            m9 = 2.0*fq;
            m10 = -4.0*fq;

            // q=2
            nr2 = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
            fq = dist[nr2];  // reading the f2 data into register fq
            pressure += fq;
            m1 -= 11.0*(fq);
            m2 -= 4.0*(fq);
            jx -= fq;
            m4 += 4.0*(fq);
            m9 += 2.0*(fq);
            m10 -= 4.0*(fq);

            // q=3
            nr3 = neighborList[n+2*Np]; // neighbor 4
            fq = dist[nr3];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jy = fq;
            m6 = -4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 = fq;
            m12 = -2.0*fq;

            // q = 4
            nr4 = neighborList[n+3*Np]; // neighbor 3
            fq = dist[nr4];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jy -= fq;
            m6 += 4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 += fq;
            m12 -= 2.0*fq;

            // q=5
            nr5 = neighborList[n+4*Np];
            fq = dist[nr5];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jz = fq;
            m8 = -4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 -= fq;
            m12 += 2.0*fq;

            // q = 6
            nr6 = neighborList[n+5*Np];
            fq = dist[nr6];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jz -= fq;
            m8 += 4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 -= fq;
            m12 += 2.0*fq;

            // q=7
            nread = neighborList[n+6*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jy += fq;
            m6 += fq;
            m9  += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 = fq;
            m16 = fq;
            m17 = -fq;

            // q = 8
            nread = neighborList[n+7*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jy -= fq;
            m6 -= fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 += fq;
            m16 -= fq;
            m17 += fq;

            // q=9
            nread = neighborList[n+8*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jy -= fq;
            m6 -= fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 -= fq;
            m16 += fq;
            m17 += fq;

            // q = 10
            nread = neighborList[n+9*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jy += fq;
            m6 += fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 -= fq;
            m16 -= fq;
            m17 -= fq;

            // q=11
            nread = neighborList[n+10*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jz += fq;
            m8 += fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 = fq;
            m16 -= fq;
            m18 = fq;

            // q=12
            nread = neighborList[n+11*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jz -= fq;
            m8 -= fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 += fq;
            m16 += fq;
            m18 -= fq;

            // q=13
            nread = neighborList[n+12*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jz -= fq;
            m8 -= fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 -= fq;
            m16 -= fq;
            m18 -= fq;

            // q=14
            nread = neighborList[n+13*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jz += fq;
            m8 += fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 -= fq;
            m16 += fq;
            m18 += fq;

            // q=15
            nread = neighborList[n+14*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy += fq;
            m6 += fq;
            jz += fq;
            m8 += fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 = fq;
            m17 += fq;
            m18 -= fq;

            // q=16
            nread = neighborList[n+15*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy -= fq;
            m6 -= fq;
            jz -= fq;
            m8 -= fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 += fq;
            m17 -= fq;
            m18 += fq;

            // q=17
            nread = neighborList[n+16*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy += fq;
            m6 += fq;
            jz -= fq;
            m8 -= fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 -= fq;
            m17 += fq;
            m18 += fq;

            // q=18
            nread = neighborList[n+17*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy -= fq;
            m6 -= fq;
            jz += fq;
            m8 += fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 -= fq;
            m17 -= fq;
            m18 -= fq;
            //---------------------------------------------------------------------//

            //---------------- Calculate SC fluid-fluid and fluid-solid forces ---------------//
            // fluid-fluid force
//            Gff_x = -Gsc*nA*nB_gradx*int(phi>0.0)-Gsc*nB*nA_gradx*int(phi<0.0);
//            Gff_y = -Gsc*nA*nB_grady*int(phi>0.0)-Gsc*nB*nA_grady*int(phi<0.0);
//            Gff_z = -Gsc*nA*nB_gradz*int(phi>0.0)-Gsc*nB*nA_gradz*int(phi<0.0);
            Gff_x = -Gsc*(nA*nB_gradx+nB*nA_gradx);
            Gff_y = -Gsc*(nA*nB_grady+nB*nA_grady);
            Gff_z = -Gsc*(nA*nB_gradz+nB*nA_gradz);
            // fluid-solid force
            Gfs_x = (nA-nB)*SolidForce[n+0*Np];    
            Gfs_y = (nA-nB)*SolidForce[n+1*Np];    
            Gfs_z = (nA-nB)*SolidForce[n+2*Np];    

            porosity = Poros[n];
            // use local saturation as an estimation of effective relperm values
            perm = Perm[n]*nA/(nA+nB)*int(phi>0.0)+Perm[n]*nB/(nA+nB)*int(phi<0.0);

            c0 = 0.5*(1.0+porosity*0.5*mu_eff/perm);
            if (porosity==1.0) c0 = 0.5;//i.e. apparent pore nodes
            //GeoFun = 1.75/sqrt(150.0*porosity*porosity*porosity);
            c1 = porosity*0.5*GeoFun/sqrt(perm);
            if (porosity==1.0) c1 = 0.0;//i.e. apparent pore nodes

            vx = jx/rho0+0.5*(porosity*Gx+Gff_x+Gfs_x);
            vy = jy/rho0+0.5*(porosity*Gy+Gff_y+Gfs_y);
            vz = jz/rho0+0.5*(porosity*Gz+Gff_z+Gfs_z);
            v_mag=sqrt(vx*vx+vy*vy+vz*vz);
            ux = vx/(c0+sqrt(c0*c0+c1*v_mag));
            uy = vy/(c0+sqrt(c0*c0+c1*v_mag));
            uz = vz/(c0+sqrt(c0*c0+c1*v_mag));
            u_mag=sqrt(ux*ux+uy*uy+uz*uz);

            //Update the total force to include linear (Darcy) and nonlinear (Forchheimer) drags due to the porous medium
            Fx = rho0*(-porosity*mu_eff/perm*ux - porosity*GeoFun/sqrt(perm)*u_mag*ux + porosity*Gx + Gff_x + Gfs_x);
            Fy = rho0*(-porosity*mu_eff/perm*uy - porosity*GeoFun/sqrt(perm)*u_mag*uy + porosity*Gy + Gff_y + Gfs_y);
            Fz = rho0*(-porosity*mu_eff/perm*uz - porosity*GeoFun/sqrt(perm)*u_mag*uz + porosity*Gz + Gff_z + Gfs_z);
            if (porosity==1.0){
                Fx=rho0*(Gx + Gff_x + Gfs_x);
                Fy=rho0*(Gy + Gff_y + Gfs_y);
                Fz=rho0*(Gz + Gff_z + Gfs_z);
            }

            //Calculate pressure for Incompressible-MRT model
            pressure=0.5/porosity*(pressure-0.5*rho0*u_mag*u_mag/porosity);

//            //..............carry out relaxation process...............................................
//            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1) 
//                    + (1-0.5*rlx_setA)*38*(Fx*ux+Fy*uy+Fz*uz)/porosity;
//            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2)
//                    + (1-0.5*rlx_setA)*11*(-Fx*ux-Fy*uy-Fz*uz)/porosity;
//            jx = jx + Fx;
//            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
//            jy = jy + Fy;
//            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
//            jz = jz + Fz;
//            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
//            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9)
//                    + (1-0.5*rlx_setA)*(4*Fx*ux-2*Fy*uy-2*Fz*uz)/porosity;
//            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10)
//                      + (1-0.5*rlx_setA)*(-2*Fx*ux+Fy*uy+Fz*uz)/porosity;
//            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11)
//                      + (1-0.5*rlx_setA)*(2*Fy*uy-2*Fz*uz)/porosity;
//            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12)
//                      + (1-0.5*rlx_setA)*(-Fy*uy+Fz*uz)/porosity;
//            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13)
//                      + (1-0.5*rlx_setA)*(Fy*ux+Fx*uy)/porosity;
//            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14)
//                      + (1-0.5*rlx_setA)*(Fz*uy+Fy*uz)/porosity;
//            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15)
//                      + (1-0.5*rlx_setA)*(Fz*ux+Fx*uz)/porosity;
//            m16 = m16 + rlx_setB*( - m16);
//            m17 = m17 + rlx_setB*( - m17);
//            m18 = m18 + rlx_setB*( - m18);
//            //.......................................................................................................
           
            //-------------------- IMRT collison where body force has NO higher-order terms -------------//
            //..............carry out relaxation process...............................................
            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1);
            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2);
            jx = jx + Fx;
            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
            jy = jy + Fy;
            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
            jz = jz + Fz;
            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9);
            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10);
            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11);
            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12);
            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13);
            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14);
            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15);
            m16 = m16 + rlx_setB*( - m16);
            m17 = m17 + rlx_setB*( - m17);
            m18 = m18 + rlx_setB*( - m18);
            //.......................................................................................................


            //.................inverse transformation......................................................
            // q=0
            fq = mrt_V1*rho0-mrt_V2*m1+mrt_V3*m2;
            dist[n] = fq;

            // q = 1
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10);
            //nread = neighborList[n+Np];
            dist[nr2] = fq;

            // q=2
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10);
            //nread = neighborList[n];
            dist[nr1] = fq;

            // q = 3
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
            //nread = neighborList[n+3*Np];
            dist[nr4] = fq;

            // q = 4
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
            //nread = neighborList[n+2*Np];
            dist[nr3] = fq;

            // q = 5
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
            //nread = neighborList[n+5*Np];
            dist[nr6] = fq;

            // q = 6
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
            //nread = neighborList[n+4*Np];
            dist[nr5] = fq;

            // q = 7
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17);
            nread = neighborList[n+7*Np];
            dist[nread] = fq;

            // q = 8
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m17-m16);
            nread = neighborList[n+6*Np];
            dist[nread] = fq;

            // q = 9
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17);
            nread = neighborList[n+9*Np];
            dist[nread] = fq;

            // q = 10
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17);
            nread = neighborList[n+8*Np];
            dist[nread] = fq;

            // q = 11
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m18-m16);
            nread = neighborList[n+11*Np];
            dist[nread] = fq;

            // q = 12
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18);
            nread = neighborList[n+10*Np];
            dist[nread]= fq;

            // q = 13
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15-0.125*(m16+m18);
            nread = neighborList[n+13*Np];
            dist[nread] = fq;

            // q= 14
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15+0.125*(m16+m18);
            nread = neighborList[n+12*Np];
            dist[nread] = fq;

            // q = 15
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18);
            nread = neighborList[n+15*Np];
            dist[nread] = fq;

            // q = 16
            fq =  mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17);
            nread = neighborList[n+14*Np];
            dist[nread] = fq;

            // q = 17
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18);
            nread = neighborList[n+17*Np];
            dist[nread] = fq;

            // q = 18
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18);
            nread = neighborList[n+16*Np];
            dist[nread] = fq;
            //........................................................................

            //Update velocity on device
            Velocity[0*Np+n] = ux;
            Velocity[1*Np+n] = uy;
            Velocity[2*Np+n] = uz;
            //Update pressure on device
            Pressure[n] = pressure;

            //-----------------------Mass transport------------------------//
            // Calculate the color gradient
            nx = (2*nB*nA_gradx-2*nA*nB_gradx)/(nA+nB)/(nA+nB); 
            ny = (2*nB*nA_grady-2*nA*nB_grady)/(nA+nB)/(nA+nB); 
            nz = (2*nB*nA_gradz-2*nA*nB_gradz)/(nA+nB)/(nA+nB); 
			//...........Normalize the Color Gradient.................................
			C = sqrt(nx*nx+ny*ny+nz*nz);
			double ColorMag = C;
			if (C==0.0) ColorMag=1.0;
			nx = nx/ColorMag;
			ny = ny/ColorMag;
			nz = nz/ColorMag;		
			if (C == 0.0)	nx = ny = nz = 0.0;

			// Instantiate mass transport distributions
			// Stationary value - distribution 0
			nAB = 1.0/(nA+nB);
			Aq[n] = 0.3333333333333333*nA;
			Bq[n] = 0.3333333333333333*nB;

			//...............................................
			// q = 0,2,4
			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nx;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*ux))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*ux))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*ux))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*ux))+delta;

			// q = 1
			//nread = neighborList[n+Np];
			Aq[nr2] = a1;
			Bq[nr2] = b1;
			// q=2
			//nread = neighborList[n];
			Aq[nr1] = a2;
			Bq[nr1] = b2;

			//...............................................
			// Cq = {0,1,0}
			delta = beta*nA*nB*nAB*0.1111111111111111*ny;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uy))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uy))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uy))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uy))+delta;

			// q = 3
			//nread = neighborList[n+3*Np];
			Aq[nr4] = a1;
			Bq[nr4] = b1;
			// q = 4
			//nread = neighborList[n+2*Np];
			Aq[nr3] = a2;
			Bq[nr3] = b2;

			//...............................................
			// q = 4
			// Cq = {0,0,1}
			delta = beta*nA*nB*nAB*0.1111111111111111*nz;
			if (!(nA*nB*nAB>0)) delta=0;
			a1 = nA*(0.1111111111111111*(1+4.5*uz))+delta;
			b1 = nB*(0.1111111111111111*(1+4.5*uz))-delta;
			a2 = nA*(0.1111111111111111*(1-4.5*uz))-delta;
			b2 = nB*(0.1111111111111111*(1-4.5*uz))+delta;

			// q = 5
			//nread = neighborList[n+5*Np];
			Aq[nr6] = a1;
			Bq[nr6] = b1;
			// q = 6
			//nread = neighborList[n+4*Np];
			Aq[nr5] = a2;
			Bq[nr5] = b2;
			//...............................................
		}
	}
}

//__global__ void dvc_ScaLBL_D3Q19_AAeven_GreyscaleColorChem(double *dist, double *Aq, double *Bq, double *Den,double *SolidForce, int start, int finish, int Np,
//                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double gamma,double kappaA,double kappaB,double lambdaA,double lambdaB,
//                double Gx, double Gy, double Gz,
//                double *Poros,double *Perm, double *Velocity,double *Pressure,double *PressureGrad,double *PressTensorGrad,double *PhiLap){
//	int n;
//	double vx,vy,vz,v_mag;
//    double ux,uy,uz,u_mag;
//    double pressure;//defined for this incompressible model
//	// conserved momemnts
//	double jx,jy,jz;
//	// non-conserved moments
//	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
//    double fq;
//    // currently disable 'GeoFun'
//    double GeoFun=0.0;//geometric function from Guo's PRE 66, 036304 (2002)
//    double porosity;
//    double perm;//voxel permeability
//    double c0, c1; //Guo's model parameters
//    double Fx, Fy, Fz;//The total body force including Brinkman force and user-specified (Gx,Gy,Gz)
//	double tau,tau_eff,rlx_setA,rlx_setB;
//    double mu_eff;//effective kinematic viscosity for Darcy term
//    double rho0;
//    double phi;
//    double phi_lap;//laplacian of phase field
//    double nA,nB;
//	double a1,b1,a2,b2;
//    double Gfs_x,Gfs_y,Gfs_z;
//    double Gff_x,Gff_y,Gff_z;
//    double chem_a,chem_b;
//    double rlx_massA,rlx_massB;
//    // *---------------------------------Pressure Tensor Gradient------------------------------------*//
//    double Pxx_x,Pyy_y,Pzz_z;
//    double Pxy_x,Pxy_y;
//    double Pyz_y,Pyz_z;
//    double Pxz_x,Pxz_z;
//    double px,py,pz; //pressure gradient
//
//
//	const double mrt_V1=0.05263157894736842;
//	const double mrt_V2=0.012531328320802;
//	const double mrt_V3=0.04761904761904762;
//	const double mrt_V4=0.004594820384294068;
//	const double mrt_V5=0.01587301587301587;
//	const double mrt_V6=0.0555555555555555555555555;
//	const double mrt_V7=0.02777777777777778;
//	const double mrt_V8=0.08333333333333333;
//	const double mrt_V9=0.003341687552213868;
//	const double mrt_V10=0.003968253968253968;
//	const double mrt_V11=0.01388888888888889;
//	const double mrt_V12=0.04166666666666666;
//
//
//	int S = Np/NBLOCKS/NTHREADS + 1;
//	for (int s=0; s<S; s++){
//	    //........Get 1-D index for this thread....................
//	    n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
//
//		if ( n<finish ){
//
//			// read the component number densities
//			nA = Den[n];
//			nB = Den[Np + n];
//			// compute phase indicator field
//			phi=(nA-nB)/(nA+nB);
//            // load laplacian of phase field
//            phi_lap = PhiLap[n];
//            // Load voxel porosity and perm
//            porosity = Poros[n];
//            // use local saturation as an estimation of effective relperm values
//            perm = Perm[n]*nA/(nA+nB)*int(phi>0.0)+Perm[n]*nB/(nA+nB)*int(phi<0.0);
//
//            //Load pressure gradient
//            px=PressureGrad[0*Np+n];
//            py=PressureGrad[1*Np+n];
//            pz=PressureGrad[2*Np+n];
//
//            //Load pressure tensor gradient
//            //For reference full list of PressTensorGrad
//            //PressTensorGrad[n+0*Np]  = Pxx_x
//            //PressTensorGrad[n+1*Np]  = Pxx_y
//            //PressTensorGrad[n+2*Np]  = Pxx_z
//            //PressTensorGrad[n+3*Np]  = Pyy_x
//            //PressTensorGrad[n+4*Np]  = Pyy_y
//            //PressTensorGrad[n+5*Np]  = Pyy_z
//            //PressTensorGrad[n+6*Np]  = Pzz_x
//            //PressTensorGrad[n+7*Np]  = Pzz_y
//            //PressTensorGrad[n+8*Np]  = Pzz_z
//            //PressTensorGrad[n+9*Np]  = Pxy_x
//            //PressTensorGrad[n+10*Np] = Pxy_y
//            //PressTensorGrad[n+11*Np] = Pxy_z
//            //PressTensorGrad[n+12*Np] = Pyz_x
//            //PressTensorGrad[n+13*Np] = Pyz_y
//            //PressTensorGrad[n+14*Np] = Pyz_z
//            //PressTensorGrad[n+15*Np] = Pxz_x
//            //PressTensorGrad[n+16*Np] = Pxz_y
//            //PressTensorGrad[n+17*Np] = Pxz_z
//            Pxx_x = PressTensorGrad[0*Np+n];
//            Pyy_y = PressTensorGrad[4*Np+n];
//            Pzz_z = PressTensorGrad[8*Np+n];
//            Pxy_x = PressTensorGrad[9*Np+n];
//            Pxz_x = PressTensorGrad[15*Np+n];
//		    Pxy_y = PressTensorGrad[10*Np+n];
//		    Pyz_y = PressTensorGrad[13*Np+n];
//		    Pyz_z = PressTensorGrad[14*Np+n];
//		    Pxz_z = PressTensorGrad[17*Np+n];
//		    //............Compute the fluid-fluid force (gfx,gfy,gfz)...................................
//            //TODO double check if you need porosity as a fre-factor
//            Gff_x = porosity*px-(Pxx_x+Pxy_y+Pxz_z);
//            Gff_y = porosity*py-(Pxy_x+Pyy_y+Pyz_z);
//            Gff_z = porosity*pz-(Pxz_x+Pyz_y+Pzz_z);
//            // fluid-solid force
//            Gfs_x = (nA-nB)*SolidForce[n+0*Np];    
//            Gfs_y = (nA-nB)*SolidForce[n+1*Np];    
//            Gfs_z = (nA-nB)*SolidForce[n+2*Np];    
//
//			// local density
//			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
//			// local relaxation time
//			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
//			rlx_setA = 1.f/tau;
//			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);
//			tau_eff=tauA_eff + 0.5*(1.0-phi)*(tauB_eff-tauA_eff);
//            mu_eff = (tau_eff-0.5)/3.f;//kinematic viscosity
//
//
//            //........................................................................
//            //					READ THE DISTRIBUTIONS
//            //		(read from opposite array due to previous swap operation)
//            //........................................................................
//            // q=0
//            fq = dist[n];
//            m1  = -30.0*fq;
//            m2  = 12.0*fq;
//
//            // q=1
//            fq = dist[2*Np+n];
//            pressure = fq;
//            m1 -= 11.0*fq;
//            m2 -= 4.0*fq;
//            jx = fq;
//            m4 = -4.0*fq;
//            m9 = 2.0*fq;
//            m10 = -4.0*fq;
//
//            // f2 = dist[10*Np+n];
//            fq = dist[1*Np+n];
//            pressure += fq;
//            m1 -= 11.0*(fq);
//            m2 -= 4.0*(fq);
//            jx -= fq;
//            m4 += 4.0*(fq);
//            m9 += 2.0*(fq);
//            m10 -= 4.0*(fq);
//
//            // q=3
//            fq = dist[4*Np+n];
//            pressure += fq;
//            m1 -= 11.0*fq;
//            m2 -= 4.0*fq;
//            jy = fq;
//            m6 = -4.0*fq;
//            m9 -= fq;
//            m10 += 2.0*fq;
//            m11 = fq;
//            m12 = -2.0*fq;
//
//            // q = 4
//            fq = dist[3*Np+n];
//            pressure += fq;
//            m1 -= 11.0*fq;
//            m2 -= 4.0*fq;
//            jy -= fq;
//            m6 += 4.0*fq;
//            m9 -= fq;
//            m10 += 2.0*fq;
//            m11 += fq;
//            m12 -= 2.0*fq;
//
//            // q=5
//            fq = dist[6*Np+n];
//            pressure += fq;
//            m1 -= 11.0*fq;
//            m2 -= 4.0*fq;
//            jz = fq;
//            m8 = -4.0*fq;
//            m9 -= fq;
//            m10 += 2.0*fq;
//            m11 -= fq;
//            m12 += 2.0*fq;
//
//            // q = 6
//            fq = dist[5*Np+n];
//            pressure += fq;
//            m1 -= 11.0*fq;
//            m2 -= 4.0*fq;
//            jz -= fq;
//            m8 += 4.0*fq;
//            m9 -= fq;
//            m10 += 2.0*fq;
//            m11 -= fq;
//            m12 += 2.0*fq;
//
//            // q=7
//            fq = dist[8*Np+n];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx += fq;
//            m4 += fq;
//            jy += fq;
//            m6 += fq;
//            m9  += fq;
//            m10 += fq;
//            m11 += fq;
//            m12 += fq;
//            m13 = fq;
//            m16 = fq;
//            m17 = -fq;
//
//            // q = 8
//            fq = dist[7*Np+n];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx -= fq;
//            m4 -= fq;
//            jy -= fq;
//            m6 -= fq;
//            m9 += fq;
//            m10 += fq;
//            m11 += fq;
//            m12 += fq;
//            m13 += fq;
//            m16 -= fq;
//            m17 += fq;
//
//            // q=9
//            fq = dist[10*Np+n];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx += fq;
//            m4 += fq;
//            jy -= fq;
//            m6 -= fq;
//            m9 += fq;
//            m10 += fq;
//            m11 += fq;
//            m12 += fq;
//            m13 -= fq;
//            m16 += fq;
//            m17 += fq;
//
//            // q = 10
//            fq = dist[9*Np+n];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx -= fq;
//            m4 -= fq;
//            jy += fq;
//            m6 += fq;
//            m9 += fq;
//            m10 += fq;
//            m11 += fq;
//            m12 += fq;
//            m13 -= fq;
//            m16 -= fq;
//            m17 -= fq;
//
//            // q=11
//            fq = dist[12*Np+n];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx += fq;
//            m4 += fq;
//            jz += fq;
//            m8 += fq;
//            m9 += fq;
//            m10 += fq;
//            m11 -= fq;
//            m12 -= fq;
//            m15 = fq;
//            m16 -= fq;
//            m18 = fq;
//
//            // q=12
//            fq = dist[11*Np+n];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx -= fq;
//            m4 -= fq;
//            jz -= fq;
//            m8 -= fq;
//            m9 += fq;
//            m10 += fq;
//            m11 -= fq;
//            m12 -= fq;
//            m15 += fq;
//            m16 += fq;
//            m18 -= fq;
//
//            // q=13
//            fq = dist[14*Np+n];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx += fq;
//            m4 += fq;
//            jz -= fq;
//            m8 -= fq;
//            m9 += fq;
//            m10 += fq;
//            m11 -= fq;
//            m12 -= fq;
//            m15 -= fq;
//            m16 -= fq;
//            m18 -= fq;
//
//            // q=14
//            fq = dist[13*Np+n];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx -= fq;
//            m4 -= fq;
//            jz += fq;
//            m8 += fq;
//            m9 += fq;
//            m10 += fq;
//            m11 -= fq;
//            m12 -= fq;
//            m15 -= fq;
//            m16 += fq;
//            m18 += fq;
//
//            // q=15
//            fq = dist[16*Np+n];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jy += fq;
//            m6 += fq;
//            jz += fq;
//            m8 += fq;
//            m9 -= 2.0*fq;
//            m10 -= 2.0*fq;
//            m14 = fq;
//            m17 += fq;
//            m18 -= fq;
//
//            // q=16
//            fq = dist[15*Np+n];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jy -= fq;
//            m6 -= fq;
//            jz -= fq;
//            m8 -= fq;
//            m9 -= 2.0*fq;
//            m10 -= 2.0*fq;
//            m14 += fq;
//            m17 -= fq;
//            m18 += fq;
//
//            // q=17
//            fq = dist[18*Np+n];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jy += fq;
//            m6 += fq;
//            jz -= fq;
//            m8 -= fq;
//            m9 -= 2.0*fq;
//            m10 -= 2.0*fq;
//            m14 -= fq;
//            m17 += fq;
//            m18 += fq;
//
//            // q=18
//            fq = dist[17*Np+n];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jy -= fq;
//            m6 -= fq;
//            jz += fq;
//            m8 += fq;
//            m9 -= 2.0*fq;
//            m10 -= 2.0*fq;
//            m14 -= fq;
//            m17 -= fq;
//            m18 -= fq;
//            //---------------------------------------------------------------------//
//
//            c0 = 0.5*(1.0+porosity*0.5*mu_eff/perm);
//            if (porosity==1.0) c0 = 0.5;//i.e. apparent pore nodes
//            //GeoFun = 1.75/sqrt(150.0*porosity*porosity*porosity);
//            c1 = porosity*0.5*GeoFun/sqrt(perm);
//            if (porosity==1.0) c1 = 0.0;//i.e. apparent pore nodes
//
//            vx = jx/rho0+0.5*(porosity*Gx+Gff_x+Gfs_x);
//            vy = jy/rho0+0.5*(porosity*Gy+Gff_y+Gfs_y);
//            vz = jz/rho0+0.5*(porosity*Gz+Gff_z+Gfs_z);
//            v_mag=sqrt(vx*vx+vy*vy+vz*vz);
//            ux = vx/(c0+sqrt(c0*c0+c1*v_mag));
//            uy = vy/(c0+sqrt(c0*c0+c1*v_mag));
//            uz = vz/(c0+sqrt(c0*c0+c1*v_mag));
//            u_mag=sqrt(ux*ux+uy*uy+uz*uz);
//
//            //Update the total force to include linear (Darcy) and nonlinear (Forchheimer) drags due to the porous medium
//            Fx = rho0*(-porosity*mu_eff/perm*ux - porosity*GeoFun/sqrt(perm)*u_mag*ux + porosity*Gx + Gff_x + Gfs_x);
//            Fy = rho0*(-porosity*mu_eff/perm*uy - porosity*GeoFun/sqrt(perm)*u_mag*uy + porosity*Gy + Gff_y + Gfs_y);
//            Fz = rho0*(-porosity*mu_eff/perm*uz - porosity*GeoFun/sqrt(perm)*u_mag*uz + porosity*Gz + Gff_z + Gfs_z);
//            if (porosity==1.0){
//                Fx=rho0*(Gx + Gff_x + Gfs_x);
//                Fy=rho0*(Gy + Gff_y + Gfs_y);
//                Fz=rho0*(Gz + Gff_z + Gfs_z);
//            }
//
//            //Calculate pressure for Incompressible-MRT model
//            pressure=0.5/porosity*(pressure-0.5*rho0*u_mag*u_mag/porosity);
//
////            //..............carry out relaxation process...............................................
////            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1) 
////                    + (1-0.5*rlx_setA)*38*(Fx*ux+Fy*uy+Fz*uz)/porosity;
////            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2)
////                    + (1-0.5*rlx_setA)*11*(-Fx*ux-Fy*uy-Fz*uz)/porosity;
////            jx = jx + Fx;
////            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
////                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
////            jy = jy + Fy;
////            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
////                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
////            jz = jz + Fz;
////            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
////                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
////            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9)
////                    + (1-0.5*rlx_setA)*(4*Fx*ux-2*Fy*uy-2*Fz*uz)/porosity;
////            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10)
////                      + (1-0.5*rlx_setA)*(-2*Fx*ux+Fy*uy+Fz*uz)/porosity;
////            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11)
////                      + (1-0.5*rlx_setA)*(2*Fy*uy-2*Fz*uz)/porosity;
////            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12)
////                      + (1-0.5*rlx_setA)*(-Fy*uy+Fz*uz)/porosity;
////            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13)
////                      + (1-0.5*rlx_setA)*(Fy*ux+Fx*uy)/porosity;
////            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14)
////                      + (1-0.5*rlx_setA)*(Fz*uy+Fy*uz)/porosity;
////            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15)
////                      + (1-0.5*rlx_setA)*(Fz*ux+Fx*uz)/porosity;
////            m16 = m16 + rlx_setB*( - m16);
////            m17 = m17 + rlx_setB*( - m17);
////            m18 = m18 + rlx_setB*( - m18);
////            //.......................................................................................................
//
//            //-------------------- IMRT collison where body force has NO higher-order terms -------------//
//            //..............carry out relaxation process...............................................
//            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1);
//            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2);
//            jx = jx + Fx;
//            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
//            jy = jy + Fy;
//            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
//            jz = jz + Fz;
//            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
//            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9);
//            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10);
//            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11);
//            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12);
//            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13);
//            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14);
//            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15);
//            m16 = m16 + rlx_setB*( - m16);
//            m17 = m17 + rlx_setB*( - m17);
//            m18 = m18 + rlx_setB*( - m18);
//            //.......................................................................................................
//
//            //.................inverse transformation......................................................
//            // q=0
//            fq = mrt_V1*rho0-mrt_V2*m1+mrt_V3*m2;
//            dist[n] = fq;
//
//            // q = 1
//            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10);
//            dist[1*Np+n] = fq;
//
//            // q=2
//            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10);
//            dist[2*Np+n] = fq;
//
//            // q = 3
//            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
//            dist[3*Np+n] = fq;
//
//            // q = 4
//            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
//            dist[4*Np+n] = fq;
//
//            // q = 5
//            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
//            dist[5*Np+n] = fq;
//
//            // q = 6
//            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
//            dist[6*Np+n] = fq;
//
//            // q = 7
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17);
//            dist[7*Np+n] = fq;
//
//            // q = 8
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m17-m16);
//            dist[8*Np+n] = fq;
//
//            // q = 9
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17);
//            dist[9*Np+n] = fq;
//
//            // q = 10
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17);
//            dist[10*Np+n] = fq;
//
//            // q = 11
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m18-m16);
//            dist[11*Np+n] = fq;
//
//            // q = 12
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18);
//            dist[12*Np+n] = fq;
//
//            // q = 13
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15-0.125*(m16+m18);
//            dist[13*Np+n] = fq;
//
//            // q= 14
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15+0.125*(m16+m18);
//            dist[14*Np+n] = fq;
//
//            // q = 15
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18);
//            dist[15*Np+n] = fq;
//
//            // q = 16
//            fq =  mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17);
//            dist[16*Np+n] = fq;
//
//            // q = 17
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18);
//            dist[17*Np+n] = fq;
//
//            // q = 18
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18);
//            dist[18*Np+n] = fq;
//            //........................................................................
//
//            //Update velocity on device
//            Velocity[0*Np+n] = ux;
//            Velocity[1*Np+n] = uy;
//            Velocity[2*Np+n] = uz;
//            //Update pressure on device
//            Pressure[n] = pressure;
//
//            //-----------------------Mass transport------------------------//
//            // calcuale chemical potential
//            chem_a = lambdaA*(nA*nA*nA-1.5*nA*nA+0.5*nA)-0.25*kappaA*phi_lap;
//            chem_b = -lambdaB*(nB*nB*nB-1.5*nB*nB+0.5*nB)-0.25*kappaB*phi_lap;
//            rlx_massA = 3.f-sqrt(3.f);
//            rlx_massB = 3.f-sqrt(3.f);
//
//			//...............................................
//			// q = 0,2,4
//			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
//			a1 = Aq[1*Np+n];
//			b1 = Bq[1*Np+n];
//			a2 = Aq[2*Np+n];
//			b2 = Bq[2*Np+n];
//			a1 = (1.0-rlx_massA)*a1+rlx_massA*(0.1111111111111111*4.5*(gamma*chem_a+nA*ux));
//			b1 = (1.0-rlx_massB)*b1+rlx_massB*(0.1111111111111111*4.5*(gamma*chem_b+nB*ux));
//			a2 = (1.0-rlx_massA)*a2+rlx_massA*(0.1111111111111111*4.5*(gamma*chem_a-nA*ux));
//			b2 = (1.0-rlx_massB)*b2+rlx_massB*(0.1111111111111111*4.5*(gamma*chem_b-nB*ux));
//
//			Aq[1*Np+n] = a1;
//			Bq[1*Np+n] = b1;
//			Aq[2*Np+n] = a2;
//			Bq[2*Np+n] = b2;
//
//			//...............................................
//			// q = 2
//			// Cq = {0,1,0}
//			a1 = Aq[3*Np+n];
//			b1 = Bq[3*Np+n];
//			a2 = Aq[4*Np+n];
//			b2 = Bq[4*Np+n];
//			a1 = (1.0-rlx_massA)*a1+rlx_massA*(0.1111111111111111*4.5*(gamma*chem_a+nA*uy));
//			b1 = (1.0-rlx_massB)*b1+rlx_massB*(0.1111111111111111*4.5*(gamma*chem_b+nB*uy));
//			a2 = (1.0-rlx_massA)*a2+rlx_massA*(0.1111111111111111*4.5*(gamma*chem_a-nA*uy));
//			b2 = (1.0-rlx_massB)*b2+rlx_massB*(0.1111111111111111*4.5*(gamma*chem_b-nB*uy));
//
//			Aq[3*Np+n] = a1;
//			Bq[3*Np+n] = b1;
//			Aq[4*Np+n] = a2;
//			Bq[4*Np+n] = b2;
//			//...............................................
//			// q = 4
//			// Cq = {0,0,1}
//			a1 = Aq[5*Np+n];
//			b1 = Bq[5*Np+n];
//			a2 = Aq[6*Np+n];
//			b2 = Bq[6*Np+n];
//			a1 = (1.0-rlx_massA)*a1+rlx_massA*(0.1111111111111111*4.5*(gamma*chem_a+nA*uz));
//			b1 = (1.0-rlx_massB)*b1+rlx_massB*(0.1111111111111111*4.5*(gamma*chem_b+nB*uz));
//			a2 = (1.0-rlx_massA)*a2+rlx_massA*(0.1111111111111111*4.5*(gamma*chem_a-nA*uz));
//			b2 = (1.0-rlx_massB)*b2+rlx_massB*(0.1111111111111111*4.5*(gamma*chem_b-nB*uz));
//
//			Aq[5*Np+n] = a1;
//			Bq[5*Np+n] = b1;
//			Aq[6*Np+n] = a2;
//			Bq[6*Np+n] = b2;
//			//...............................................
//
//			// Instantiate mass transport distributions
//			// Stationary value - distribution 0
//            a1=Aq[n];
//            b1=Bq[n];
//			Aq[n] = (1.0-rlx_massA)*a1+rlx_massA*(nA-3.0*gamma*chem_a);
//			Bq[n] = (1.0-rlx_massB)*b1+rlx_massB*(nB-3.0*gamma*chem_b);
//
//
//		}
//	}
//}

//__global__ void dvc_ScaLBL_D3Q19_AAodd_GreyscaleColorChem(int *neighborList, double *dist, double *Aq, double *Bq, double *Den,double *SolidForce, int start, int finish, int Np,
//                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double gamma,double kappaA,double kappaB,double lambdaA,double lambdaB,
//                double Gx, double Gy, double Gz,
//                double *Poros,double *Perm, double *Velocity,double *Pressure,double *PressureGrad,double *PressTensorGrad,double *PhiLap){
//
//	int n, nread, nr1,nr2,nr3,nr4,nr5,nr6;
//	double vx,vy,vz,v_mag;
//    double ux,uy,uz,u_mag;
//    double pressure;//defined for this incompressible model
//	// conserved momemnts
//	double jx,jy,jz;
//	// non-conserved moments
//	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
//    double fq;
//    // currently disable 'GeoFun'
//    double GeoFun=0.0;//geometric function from Guo's PRE 66, 036304 (2002)
//    double porosity;
//    double perm;//voxel permeability
//    double c0, c1; //Guo's model parameters
//    double Fx, Fy, Fz;//The total body force including Brinkman force and user-specified (Gx,Gy,Gz)
//	double tau,tau_eff,rlx_setA,rlx_setB;
//    double mu_eff;//effective kinematic viscosity for Darcy term
//    double rho0;
//    double phi;
//    double phi_lap;//laplacian of phase field
//    double nA,nB;
//	double a1,b1,a2,b2;
//    double Gfs_x,Gfs_y,Gfs_z;
//    double Gff_x,Gff_y,Gff_z;
//    double chem_a,chem_b;
//    double rlx_massA,rlx_massB;
//    // *---------------------------------Pressure Tensor Gradient------------------------------------*//
//    double Pxx_x,Pyy_y,Pzz_z;
//    double Pxy_x,Pxy_y;
//    double Pyz_y,Pyz_z;
//    double Pxz_x,Pxz_z;
//    double px,py,pz; //pressure gradient
//
//	const double mrt_V1=0.05263157894736842;
//	const double mrt_V2=0.012531328320802;
//	const double mrt_V3=0.04761904761904762;
//	const double mrt_V4=0.004594820384294068;
//	const double mrt_V5=0.01587301587301587;
//	const double mrt_V6=0.0555555555555555555555555;
//	const double mrt_V7=0.02777777777777778;
//	const double mrt_V8=0.08333333333333333;
//	const double mrt_V9=0.003341687552213868;
//	const double mrt_V10=0.003968253968253968;
//	const double mrt_V11=0.01388888888888889;
//	const double mrt_V12=0.04166666666666666;
//
//	int S = Np/NBLOCKS/NTHREADS + 1;
//	for (int s=0; s<S; s++){
//	    //........Get 1-D index for this thread....................
//	    n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
//
//		if ( n<finish ){		
//
//			// read the component number densities
//			nA = Den[n];
//			nB = Den[Np + n];
//			// compute phase indicator field
//			phi=(nA-nB)/(nA+nB);
//            // load laplacian of phase field
//            phi_lap = PhiLap[n];
//            // Load voxel porosity and perm
//            porosity = Poros[n];
//            // use local saturation as an estimation of effective relperm values
//            perm = Perm[n]*nA/(nA+nB)*int(phi>0.0)+Perm[n]*nB/(nA+nB)*int(phi<0.0);
//
//            //Load pressure gradient
//            px=PressureGrad[0*Np+n];
//            py=PressureGrad[1*Np+n];
//            pz=PressureGrad[2*Np+n];
//
//            //Load pressure tensor gradient
//            //For reference full list of PressTensorGrad
//            //PressTensorGrad[n+0*Np]  = Pxx_x
//            //PressTensorGrad[n+1*Np]  = Pxx_y
//            //PressTensorGrad[n+2*Np]  = Pxx_z
//            //PressTensorGrad[n+3*Np]  = Pyy_x
//            //PressTensorGrad[n+4*Np]  = Pyy_y
//            //PressTensorGrad[n+5*Np]  = Pyy_z
//            //PressTensorGrad[n+6*Np]  = Pzz_x
//            //PressTensorGrad[n+7*Np]  = Pzz_y
//            //PressTensorGrad[n+8*Np]  = Pzz_z
//            //PressTensorGrad[n+9*Np]  = Pxy_x
//            //PressTensorGrad[n+10*Np] = Pxy_y
//            //PressTensorGrad[n+11*Np] = Pxy_z
//            //PressTensorGrad[n+12*Np] = Pyz_x
//            //PressTensorGrad[n+13*Np] = Pyz_y
//            //PressTensorGrad[n+14*Np] = Pyz_z
//            //PressTensorGrad[n+15*Np] = Pxz_x
//            //PressTensorGrad[n+16*Np] = Pxz_y
//            //PressTensorGrad[n+17*Np] = Pxz_z
//            Pxx_x = PressTensorGrad[0*Np+n];
//            Pyy_y = PressTensorGrad[4*Np+n];
//            Pzz_z = PressTensorGrad[8*Np+n];
//            Pxy_x = PressTensorGrad[9*Np+n];
//            Pxz_x = PressTensorGrad[15*Np+n];
//		    Pxy_y = PressTensorGrad[10*Np+n];
//		    Pyz_y = PressTensorGrad[13*Np+n];
//		    Pyz_z = PressTensorGrad[14*Np+n];
//		    Pxz_z = PressTensorGrad[17*Np+n];
//		    //............Compute the fluid-fluid force (gfx,gfy,gfz)...................................
//            //TODO double check if you need porosity as a fre-factor
//            Gff_x = porosity*px-(Pxx_x+Pxy_y+Pxz_z);
//            Gff_y = porosity*py-(Pxy_x+Pyy_y+Pyz_z);
//            Gff_z = porosity*pz-(Pxz_x+Pyz_y+Pzz_z);
//            // fluid-solid force
//            Gfs_x = (nA-nB)*SolidForce[n+0*Np];    
//            Gfs_y = (nA-nB)*SolidForce[n+1*Np];    
//            Gfs_z = (nA-nB)*SolidForce[n+2*Np];    
//
//			// local density
//			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
//			// local relaxation time
//			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
//			rlx_setA = 1.f/tau;
//			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);
//			tau_eff=tauA_eff + 0.5*(1.0-phi)*(tauB_eff-tauA_eff);
//            mu_eff = (tau_eff-0.5)/3.f;//kinematic viscosity
//
//            //........................................................................
//            //					READ THE DISTRIBUTIONS
//            //		(read from opposite array due to previous swap operation)
//            //........................................................................
//            // q=0
//            fq = dist[n];
//            m1  = -30.0*fq;
//            m2  = 12.0*fq;
//
//            // q=1
//            nr1 = neighborList[n]; // neighbor 2 ( > 10Np => odd part of dist)
//            fq = dist[nr1]; // reading the f1 data into register fq
//            pressure = fq;
//            m1 -= 11.0*fq;
//            m2 -= 4.0*fq;
//            jx = fq;
//            m4 = -4.0*fq;
//            m9 = 2.0*fq;
//            m10 = -4.0*fq;
//
//            // q=2
//            nr2 = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
//            fq = dist[nr2];  // reading the f2 data into register fq
//            pressure += fq;
//            m1 -= 11.0*(fq);
//            m2 -= 4.0*(fq);
//            jx -= fq;
//            m4 += 4.0*(fq);
//            m9 += 2.0*(fq);
//            m10 -= 4.0*(fq);
//
//            // q=3
//            nr3 = neighborList[n+2*Np]; // neighbor 4
//            fq = dist[nr3];
//            pressure += fq;
//            m1 -= 11.0*fq;
//            m2 -= 4.0*fq;
//            jy = fq;
//            m6 = -4.0*fq;
//            m9 -= fq;
//            m10 += 2.0*fq;
//            m11 = fq;
//            m12 = -2.0*fq;
//
//            // q = 4
//            nr4 = neighborList[n+3*Np]; // neighbor 3
//            fq = dist[nr4];
//            pressure += fq;
//            m1 -= 11.0*fq;
//            m2 -= 4.0*fq;
//            jy -= fq;
//            m6 += 4.0*fq;
//            m9 -= fq;
//            m10 += 2.0*fq;
//            m11 += fq;
//            m12 -= 2.0*fq;
//
//            // q=5
//            nr5 = neighborList[n+4*Np];
//            fq = dist[nr5];
//            pressure += fq;
//            m1 -= 11.0*fq;
//            m2 -= 4.0*fq;
//            jz = fq;
//            m8 = -4.0*fq;
//            m9 -= fq;
//            m10 += 2.0*fq;
//            m11 -= fq;
//            m12 += 2.0*fq;
//
//            // q = 6
//            nr6 = neighborList[n+5*Np];
//            fq = dist[nr6];
//            pressure += fq;
//            m1 -= 11.0*fq;
//            m2 -= 4.0*fq;
//            jz -= fq;
//            m8 += 4.0*fq;
//            m9 -= fq;
//            m10 += 2.0*fq;
//            m11 -= fq;
//            m12 += 2.0*fq;
//
//            // q=7
//            nread = neighborList[n+6*Np];
//            fq = dist[nread];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx += fq;
//            m4 += fq;
//            jy += fq;
//            m6 += fq;
//            m9  += fq;
//            m10 += fq;
//            m11 += fq;
//            m12 += fq;
//            m13 = fq;
//            m16 = fq;
//            m17 = -fq;
//
//            // q = 8
//            nread = neighborList[n+7*Np];
//            fq = dist[nread];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx -= fq;
//            m4 -= fq;
//            jy -= fq;
//            m6 -= fq;
//            m9 += fq;
//            m10 += fq;
//            m11 += fq;
//            m12 += fq;
//            m13 += fq;
//            m16 -= fq;
//            m17 += fq;
//
//            // q=9
//            nread = neighborList[n+8*Np];
//            fq = dist[nread];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx += fq;
//            m4 += fq;
//            jy -= fq;
//            m6 -= fq;
//            m9 += fq;
//            m10 += fq;
//            m11 += fq;
//            m12 += fq;
//            m13 -= fq;
//            m16 += fq;
//            m17 += fq;
//
//            // q = 10
//            nread = neighborList[n+9*Np];
//            fq = dist[nread];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx -= fq;
//            m4 -= fq;
//            jy += fq;
//            m6 += fq;
//            m9 += fq;
//            m10 += fq;
//            m11 += fq;
//            m12 += fq;
//            m13 -= fq;
//            m16 -= fq;
//            m17 -= fq;
//
//            // q=11
//            nread = neighborList[n+10*Np];
//            fq = dist[nread];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx += fq;
//            m4 += fq;
//            jz += fq;
//            m8 += fq;
//            m9 += fq;
//            m10 += fq;
//            m11 -= fq;
//            m12 -= fq;
//            m15 = fq;
//            m16 -= fq;
//            m18 = fq;
//
//            // q=12
//            nread = neighborList[n+11*Np];
//            fq = dist[nread];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx -= fq;
//            m4 -= fq;
//            jz -= fq;
//            m8 -= fq;
//            m9 += fq;
//            m10 += fq;
//            m11 -= fq;
//            m12 -= fq;
//            m15 += fq;
//            m16 += fq;
//            m18 -= fq;
//
//            // q=13
//            nread = neighborList[n+12*Np];
//            fq = dist[nread];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx += fq;
//            m4 += fq;
//            jz -= fq;
//            m8 -= fq;
//            m9 += fq;
//            m10 += fq;
//            m11 -= fq;
//            m12 -= fq;
//            m15 -= fq;
//            m16 -= fq;
//            m18 -= fq;
//
//            // q=14
//            nread = neighborList[n+13*Np];
//            fq = dist[nread];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jx -= fq;
//            m4 -= fq;
//            jz += fq;
//            m8 += fq;
//            m9 += fq;
//            m10 += fq;
//            m11 -= fq;
//            m12 -= fq;
//            m15 -= fq;
//            m16 += fq;
//            m18 += fq;
//
//            // q=15
//            nread = neighborList[n+14*Np];
//            fq = dist[nread];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jy += fq;
//            m6 += fq;
//            jz += fq;
//            m8 += fq;
//            m9 -= 2.0*fq;
//            m10 -= 2.0*fq;
//            m14 = fq;
//            m17 += fq;
//            m18 -= fq;
//
//            // q=16
//            nread = neighborList[n+15*Np];
//            fq = dist[nread];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jy -= fq;
//            m6 -= fq;
//            jz -= fq;
//            m8 -= fq;
//            m9 -= 2.0*fq;
//            m10 -= 2.0*fq;
//            m14 += fq;
//            m17 -= fq;
//            m18 += fq;
//
//            // q=17
//            nread = neighborList[n+16*Np];
//            fq = dist[nread];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jy += fq;
//            m6 += fq;
//            jz -= fq;
//            m8 -= fq;
//            m9 -= 2.0*fq;
//            m10 -= 2.0*fq;
//            m14 -= fq;
//            m17 += fq;
//            m18 += fq;
//
//            // q=18
//            nread = neighborList[n+17*Np];
//            fq = dist[nread];
//            pressure += fq;
//            m1 += 8.0*fq;
//            m2 += fq;
//            jy -= fq;
//            m6 -= fq;
//            jz += fq;
//            m8 += fq;
//            m9 -= 2.0*fq;
//            m10 -= 2.0*fq;
//            m14 -= fq;
//            m17 -= fq;
//            m18 -= fq;
//            //---------------------------------------------------------------------//
//
//            c0 = 0.5*(1.0+porosity*0.5*mu_eff/perm);
//            if (porosity==1.0) c0 = 0.5;//i.e. apparent pore nodes
//            //GeoFun = 1.75/sqrt(150.0*porosity*porosity*porosity);
//            c1 = porosity*0.5*GeoFun/sqrt(perm);
//            if (porosity==1.0) c1 = 0.0;//i.e. apparent pore nodes
//
//            vx = jx/rho0+0.5*(porosity*Gx+Gff_x+Gfs_x);
//            vy = jy/rho0+0.5*(porosity*Gy+Gff_y+Gfs_y);
//            vz = jz/rho0+0.5*(porosity*Gz+Gff_z+Gfs_z);
//            v_mag=sqrt(vx*vx+vy*vy+vz*vz);
//            ux = vx/(c0+sqrt(c0*c0+c1*v_mag));
//            uy = vy/(c0+sqrt(c0*c0+c1*v_mag));
//            uz = vz/(c0+sqrt(c0*c0+c1*v_mag));
//            u_mag=sqrt(ux*ux+uy*uy+uz*uz);
//
//            //Update the total force to include linear (Darcy) and nonlinear (Forchheimer) drags due to the porous medium
//            Fx = rho0*(-porosity*mu_eff/perm*ux - porosity*GeoFun/sqrt(perm)*u_mag*ux + porosity*Gx + Gff_x + Gfs_x);
//            Fy = rho0*(-porosity*mu_eff/perm*uy - porosity*GeoFun/sqrt(perm)*u_mag*uy + porosity*Gy + Gff_y + Gfs_y);
//            Fz = rho0*(-porosity*mu_eff/perm*uz - porosity*GeoFun/sqrt(perm)*u_mag*uz + porosity*Gz + Gff_z + Gfs_z);
//            if (porosity==1.0){
//                Fx=rho0*(Gx + Gff_x + Gfs_x);
//                Fy=rho0*(Gy + Gff_y + Gfs_y);
//                Fz=rho0*(Gz + Gff_z + Gfs_z);
//            }
//
//            //Calculate pressure for Incompressible-MRT model
//            pressure=0.5/porosity*(pressure-0.5*rho0*u_mag*u_mag/porosity);
//
////            //..............carry out relaxation process...............................................
////            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1) 
////                    + (1-0.5*rlx_setA)*38*(Fx*ux+Fy*uy+Fz*uz)/porosity;
////            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2)
////                    + (1-0.5*rlx_setA)*11*(-Fx*ux-Fy*uy-Fz*uz)/porosity;
////            jx = jx + Fx;
////            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
////                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
////            jy = jy + Fy;
////            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
////                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
////            jz = jz + Fz;
////            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
////                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
////            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9)
////                    + (1-0.5*rlx_setA)*(4*Fx*ux-2*Fy*uy-2*Fz*uz)/porosity;
////            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10)
////                      + (1-0.5*rlx_setA)*(-2*Fx*ux+Fy*uy+Fz*uz)/porosity;
////            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11)
////                      + (1-0.5*rlx_setA)*(2*Fy*uy-2*Fz*uz)/porosity;
////            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12)
////                      + (1-0.5*rlx_setA)*(-Fy*uy+Fz*uz)/porosity;
////            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13)
////                      + (1-0.5*rlx_setA)*(Fy*ux+Fx*uy)/porosity;
////            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14)
////                      + (1-0.5*rlx_setA)*(Fz*uy+Fy*uz)/porosity;
////            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15)
////                      + (1-0.5*rlx_setA)*(Fz*ux+Fx*uz)/porosity;
////            m16 = m16 + rlx_setB*( - m16);
////            m17 = m17 + rlx_setB*( - m17);
////            m18 = m18 + rlx_setB*( - m18);
////            //.......................................................................................................
//           
//            //-------------------- IMRT collison where body force has NO higher-order terms -------------//
//            //..............carry out relaxation process...............................................
//            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1);
//            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2);
//            jx = jx + Fx;
//            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
//            jy = jy + Fy;
//            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
//            jz = jz + Fz;
//            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
//            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9);
//            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10);
//            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11);
//            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12);
//            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13);
//            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14);
//            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15);
//            m16 = m16 + rlx_setB*( - m16);
//            m17 = m17 + rlx_setB*( - m17);
//            m18 = m18 + rlx_setB*( - m18);
//            //.......................................................................................................
//
//
//            //.................inverse transformation......................................................
//            // q=0
//            fq = mrt_V1*rho0-mrt_V2*m1+mrt_V3*m2;
//            dist[n] = fq;
//
//            // q = 1
//            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10);
//            //nread = neighborList[n+Np];
//            dist[nr2] = fq;
//
//            // q=2
//            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10);
//            //nread = neighborList[n];
//            dist[nr1] = fq;
//
//            // q = 3
//            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
//            //nread = neighborList[n+3*Np];
//            dist[nr4] = fq;
//
//            // q = 4
//            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
//            //nread = neighborList[n+2*Np];
//            dist[nr3] = fq;
//
//            // q = 5
//            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
//            //nread = neighborList[n+5*Np];
//            dist[nr6] = fq;
//
//            // q = 6
//            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
//            //nread = neighborList[n+4*Np];
//            dist[nr5] = fq;
//
//            // q = 7
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17);
//            nread = neighborList[n+7*Np];
//            dist[nread] = fq;
//
//            // q = 8
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m17-m16);
//            nread = neighborList[n+6*Np];
//            dist[nread] = fq;
//
//            // q = 9
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17);
//            nread = neighborList[n+9*Np];
//            dist[nread] = fq;
//
//            // q = 10
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17);
//            nread = neighborList[n+8*Np];
//            dist[nread] = fq;
//
//            // q = 11
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m18-m16);
//            nread = neighborList[n+11*Np];
//            dist[nread] = fq;
//
//            // q = 12
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18);
//            nread = neighborList[n+10*Np];
//            dist[nread]= fq;
//
//            // q = 13
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15-0.125*(m16+m18);
//            nread = neighborList[n+13*Np];
//            dist[nread] = fq;
//
//            // q= 14
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15+0.125*(m16+m18);
//            nread = neighborList[n+12*Np];
//            dist[nread] = fq;
//
//            // q = 15
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18);
//            nread = neighborList[n+15*Np];
//            dist[nread] = fq;
//
//            // q = 16
//            fq =  mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17);
//            nread = neighborList[n+14*Np];
//            dist[nread] = fq;
//
//            // q = 17
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18);
//            nread = neighborList[n+17*Np];
//            dist[nread] = fq;
//
//            // q = 18
//            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18);
//            nread = neighborList[n+16*Np];
//            dist[nread] = fq;
//            //........................................................................
//
//            //Update velocity on device
//            Velocity[0*Np+n] = ux;
//            Velocity[1*Np+n] = uy;
//            Velocity[2*Np+n] = uz;
//            //Update pressure on device
//            Pressure[n] = pressure;
//
//            //-----------------------Mass transport------------------------//
//            // calcuale chemical potential
//            chem_a = lambdaA*(nA*nA*nA-1.5*nA*nA+0.5*nA)-0.25*kappaA*phi_lap;
//            chem_b = -lambdaB*(nB*nB*nB-1.5*nB*nB+0.5*nB)-0.25*kappaB*phi_lap;
//            rlx_massA = 3.f-sqrt(3.f);
//            rlx_massB = 3.f-sqrt(3.f);
//
//			//...............................................
//			// q = 0,2,4
//			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
//			a1 = Aq[nr2];
//			b1 = Bq[nr2];
//			a2 = Aq[nr1];
//			b2 = Bq[nr1];
//			a1 = (1.0-rlx_massA)*a1+rlx_massA*(0.1111111111111111*4.5*(gamma*chem_a+nA*ux));
//			b1 = (1.0-rlx_massB)*b1+rlx_massB*(0.1111111111111111*4.5*(gamma*chem_b+nB*ux));
//			a2 = (1.0-rlx_massA)*a2+rlx_massA*(0.1111111111111111*4.5*(gamma*chem_a-nA*ux));
//			b2 = (1.0-rlx_massB)*b2+rlx_massB*(0.1111111111111111*4.5*(gamma*chem_b-nB*ux));
//
//			// q = 1
//			//nread = neighborList[n+Np];
//			Aq[nr2] = a1;
//			Bq[nr2] = b1;
//			// q=2
//			//nread = neighborList[n];
//			Aq[nr1] = a2;
//			Bq[nr1] = b2;
//
//			//...............................................
//			// Cq = {0,1,0}
//			a1 = Aq[nr4];
//			b1 = Bq[nr4];
//			a2 = Aq[nr3];
//			b2 = Bq[nr3];
//			a1 = (1.0-rlx_massA)*a1+rlx_massA*(0.1111111111111111*4.5*(gamma*chem_a+nA*uy));
//			b1 = (1.0-rlx_massB)*b1+rlx_massB*(0.1111111111111111*4.5*(gamma*chem_b+nB*uy));
//			a2 = (1.0-rlx_massA)*a2+rlx_massA*(0.1111111111111111*4.5*(gamma*chem_a-nA*uy));
//			b2 = (1.0-rlx_massB)*b2+rlx_massB*(0.1111111111111111*4.5*(gamma*chem_b-nB*uy));
//
//			// q = 3
//			//nread = neighborList[n+3*Np];
//			Aq[nr4] = a1;
//			Bq[nr4] = b1;
//			// q = 4
//			//nread = neighborList[n+2*Np];
//			Aq[nr3] = a2;
//			Bq[nr3] = b2;
//
//			//...............................................
//			// q = 4
//			// Cq = {0,0,1}
//			a1 = Aq[nr6];
//			b1 = Bq[nr6];
//			a2 = Aq[nr5];
//			b2 = Bq[nr5];
//			a1 = (1.0-rlx_massA)*a1+rlx_massA*(0.1111111111111111*4.5*(gamma*chem_a+nA*uz));
//			b1 = (1.0-rlx_massB)*b1+rlx_massB*(0.1111111111111111*4.5*(gamma*chem_b+nB*uz));
//			a2 = (1.0-rlx_massA)*a2+rlx_massA*(0.1111111111111111*4.5*(gamma*chem_a-nA*uz));
//			b2 = (1.0-rlx_massB)*b2+rlx_massB*(0.1111111111111111*4.5*(gamma*chem_b-nB*uz));
//
//			// q = 5
//			//nread = neighborList[n+5*Np];
//			Aq[nr6] = a1;
//			Bq[nr6] = b1;
//			// q = 6
//			//nread = neighborList[n+4*Np];
//			Aq[nr5] = a2;
//			Bq[nr5] = b2;
//			//...............................................
//
//			// Instantiate mass transport distributions
//			// Stationary value - distribution 0
//            a1=Aq[n];
//            b1=Bq[n];
//			Aq[n] = (1.0-rlx_massA)*a1+rlx_massA*(nA-3.0*gamma*chem_a);
//			Bq[n] = (1.0-rlx_massB)*b1+rlx_massB*(nB-3.0*gamma*chem_b);
//
//
//		}
//	}
//}

__global__ void dvc_ScaLBL_D3Q19_AAodd_GreyscaleColorChem(int *neighborList, double *dist, double *Cq, double *Phi, double *Den,double *SolidForce, int start, int finish, int Np,
                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double gamma,double kappaA,double kappaB,double lambdaA,double lambdaB,
                double Gx, double Gy, double Gz,
                double *Poros,double *Perm, double *Velocity,double *Pressure,double *PressureGrad,double *PressTensorGrad,double *PhiLap){

	int n, nread, nr1,nr2,nr3,nr4,nr5,nr6;
	double vx,vy,vz,v_mag;
    double ux,uy,uz,u_mag;
    double pressure;//defined for this incompressible model
	// conserved momemnts
	double jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
    double fq;
    // currently disable 'GeoFun'
    double GeoFun=0.0;//geometric function from Guo's PRE 66, 036304 (2002)
    double porosity;
    double perm;//voxel permeability
    double c0, c1; //Guo's model parameters
    double Fx, Fy, Fz;//The total body force including Brinkman force and user-specified (Gx,Gy,Gz)
	double tau,tau_eff,rlx_setA,rlx_setB;
    double mu_eff;//effective kinematic viscosity for Darcy term
    double rho0;
    double phi;
    double phi_lap;//laplacian of phase field
    double nA,nB;
	//double a1,b1,a2,b2;
    double Gfs_x,Gfs_y,Gfs_z;
    double Gff_x,Gff_y,Gff_z;
    double chem;
    //double rlx_massA,rlx_massB;
    double rlx_phi;
    double a1,a2;//PDF of phase field
    // *---------------------------------Pressure Tensor Gradient------------------------------------*//
    double Pxx_x,Pyy_y,Pzz_z;
    double Pxy_x,Pxy_y;
    double Pyz_y,Pyz_z;
    double Pxz_x,Pxz_z;
    double px,py,pz; //pressure gradient

	const double mrt_V1=0.05263157894736842;
	const double mrt_V2=0.012531328320802;
	const double mrt_V3=0.04761904761904762;
	const double mrt_V4=0.004594820384294068;
	const double mrt_V5=0.01587301587301587;
	const double mrt_V6=0.0555555555555555555555555;
	const double mrt_V7=0.02777777777777778;
	const double mrt_V8=0.08333333333333333;
	const double mrt_V9=0.003341687552213868;
	const double mrt_V10=0.003968253968253968;
	const double mrt_V11=0.01388888888888889;
	const double mrt_V12=0.04166666666666666;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
	    //........Get 1-D index for this thread....................
	    n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;

		if ( n<finish ){		

			// read the component number densities
			nA = Den[n];
			nB = Den[Np + n];
            // read phase field
            phi = Phi[n];
            // load laplacian of phase field
            phi_lap = PhiLap[n];
            // Load voxel porosity and perm
            porosity = Poros[n];
            // use local saturation as an estimation of effective relperm values
            perm = Perm[n]*nA/(nA+nB)*int(phi>0.0)+Perm[n]*nB/(nA+nB)*int(phi<0.0);

            //Load pressure gradient
            px=PressureGrad[0*Np+n];
            py=PressureGrad[1*Np+n];
            pz=PressureGrad[2*Np+n];

            //Load pressure tensor gradient
            //For reference full list of PressTensorGrad
            //PressTensorGrad[n+0*Np]  = Pxx_x
            //PressTensorGrad[n+1*Np]  = Pxx_y
            //PressTensorGrad[n+2*Np]  = Pxx_z
            //PressTensorGrad[n+3*Np]  = Pyy_x
            //PressTensorGrad[n+4*Np]  = Pyy_y
            //PressTensorGrad[n+5*Np]  = Pyy_z
            //PressTensorGrad[n+6*Np]  = Pzz_x
            //PressTensorGrad[n+7*Np]  = Pzz_y
            //PressTensorGrad[n+8*Np]  = Pzz_z
            //PressTensorGrad[n+9*Np]  = Pxy_x
            //PressTensorGrad[n+10*Np] = Pxy_y
            //PressTensorGrad[n+11*Np] = Pxy_z
            //PressTensorGrad[n+12*Np] = Pyz_x
            //PressTensorGrad[n+13*Np] = Pyz_y
            //PressTensorGrad[n+14*Np] = Pyz_z
            //PressTensorGrad[n+15*Np] = Pxz_x
            //PressTensorGrad[n+16*Np] = Pxz_y
            //PressTensorGrad[n+17*Np] = Pxz_z
            Pxx_x = PressTensorGrad[0*Np+n];
            Pyy_y = PressTensorGrad[4*Np+n];
            Pzz_z = PressTensorGrad[8*Np+n];
            Pxy_x = PressTensorGrad[9*Np+n];
            Pxz_x = PressTensorGrad[15*Np+n];
		    Pxy_y = PressTensorGrad[10*Np+n];
		    Pyz_y = PressTensorGrad[13*Np+n];
		    Pyz_z = PressTensorGrad[14*Np+n];
		    Pxz_z = PressTensorGrad[17*Np+n];
		    //............Compute the fluid-fluid force (gfx,gfy,gfz)...................................
            //TODO double check if you need porosity as a fre-factor
            Gff_x = porosity*px-(Pxx_x+Pxy_y+Pxz_z);
            Gff_y = porosity*py-(Pxy_x+Pyy_y+Pyz_z);
            Gff_z = porosity*pz-(Pxz_x+Pyz_y+Pzz_z);
            // fluid-solid force
            Gfs_x = (nA-nB)*SolidForce[n+0*Np];    
            Gfs_y = (nA-nB)*SolidForce[n+1*Np];    
            Gfs_z = (nA-nB)*SolidForce[n+2*Np];    

			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
			// local relaxation time
			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
			rlx_setA = 1.f/tau;
			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);
			tau_eff=tauA_eff + 0.5*(1.0-phi)*(tauB_eff-tauA_eff);
            mu_eff = (tau_eff-0.5)/3.f;//kinematic viscosity

            //........................................................................
            //					READ THE DISTRIBUTIONS
            //		(read from opposite array due to previous swap operation)
            //........................................................................
            // q=0
            fq = dist[n];
            m1  = -30.0*fq;
            m2  = 12.0*fq;

            // q=1
            nr1 = neighborList[n]; // neighbor 2 ( > 10Np => odd part of dist)
            fq = dist[nr1]; // reading the f1 data into register fq
            pressure = fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jx = fq;
            m4 = -4.0*fq;
            m9 = 2.0*fq;
            m10 = -4.0*fq;

            // q=2
            nr2 = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
            fq = dist[nr2];  // reading the f2 data into register fq
            pressure += fq;
            m1 -= 11.0*(fq);
            m2 -= 4.0*(fq);
            jx -= fq;
            m4 += 4.0*(fq);
            m9 += 2.0*(fq);
            m10 -= 4.0*(fq);

            // q=3
            nr3 = neighborList[n+2*Np]; // neighbor 4
            fq = dist[nr3];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jy = fq;
            m6 = -4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 = fq;
            m12 = -2.0*fq;

            // q = 4
            nr4 = neighborList[n+3*Np]; // neighbor 3
            fq = dist[nr4];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jy -= fq;
            m6 += 4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 += fq;
            m12 -= 2.0*fq;

            // q=5
            nr5 = neighborList[n+4*Np];
            fq = dist[nr5];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jz = fq;
            m8 = -4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 -= fq;
            m12 += 2.0*fq;

            // q = 6
            nr6 = neighborList[n+5*Np];
            fq = dist[nr6];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jz -= fq;
            m8 += 4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 -= fq;
            m12 += 2.0*fq;

            // q=7
            nread = neighborList[n+6*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jy += fq;
            m6 += fq;
            m9  += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 = fq;
            m16 = fq;
            m17 = -fq;

            // q = 8
            nread = neighborList[n+7*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jy -= fq;
            m6 -= fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 += fq;
            m16 -= fq;
            m17 += fq;

            // q=9
            nread = neighborList[n+8*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jy -= fq;
            m6 -= fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 -= fq;
            m16 += fq;
            m17 += fq;

            // q = 10
            nread = neighborList[n+9*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jy += fq;
            m6 += fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 -= fq;
            m16 -= fq;
            m17 -= fq;

            // q=11
            nread = neighborList[n+10*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jz += fq;
            m8 += fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 = fq;
            m16 -= fq;
            m18 = fq;

            // q=12
            nread = neighborList[n+11*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jz -= fq;
            m8 -= fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 += fq;
            m16 += fq;
            m18 -= fq;

            // q=13
            nread = neighborList[n+12*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jz -= fq;
            m8 -= fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 -= fq;
            m16 -= fq;
            m18 -= fq;

            // q=14
            nread = neighborList[n+13*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jz += fq;
            m8 += fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 -= fq;
            m16 += fq;
            m18 += fq;

            // q=15
            nread = neighborList[n+14*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy += fq;
            m6 += fq;
            jz += fq;
            m8 += fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 = fq;
            m17 += fq;
            m18 -= fq;

            // q=16
            nread = neighborList[n+15*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy -= fq;
            m6 -= fq;
            jz -= fq;
            m8 -= fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 += fq;
            m17 -= fq;
            m18 += fq;

            // q=17
            nread = neighborList[n+16*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy += fq;
            m6 += fq;
            jz -= fq;
            m8 -= fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 -= fq;
            m17 += fq;
            m18 += fq;

            // q=18
            nread = neighborList[n+17*Np];
            fq = dist[nread];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy -= fq;
            m6 -= fq;
            jz += fq;
            m8 += fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 -= fq;
            m17 -= fq;
            m18 -= fq;
            //---------------------------------------------------------------------//

            c0 = 0.5*(1.0+porosity*0.5*mu_eff/perm);
            if (porosity==1.0) c0 = 0.5;//i.e. apparent pore nodes
            //GeoFun = 1.75/sqrt(150.0*porosity*porosity*porosity);
            c1 = porosity*0.5*GeoFun/sqrt(perm);
            if (porosity==1.0) c1 = 0.0;//i.e. apparent pore nodes

            vx = jx/rho0+0.5*(porosity*Gx+Gff_x+Gfs_x);
            vy = jy/rho0+0.5*(porosity*Gy+Gff_y+Gfs_y);
            vz = jz/rho0+0.5*(porosity*Gz+Gff_z+Gfs_z);
            v_mag=sqrt(vx*vx+vy*vy+vz*vz);
            ux = vx/(c0+sqrt(c0*c0+c1*v_mag));
            uy = vy/(c0+sqrt(c0*c0+c1*v_mag));
            uz = vz/(c0+sqrt(c0*c0+c1*v_mag));
            u_mag=sqrt(ux*ux+uy*uy+uz*uz);

            //Update the total force to include linear (Darcy) and nonlinear (Forchheimer) drags due to the porous medium
            Fx = rho0*(-porosity*mu_eff/perm*ux - porosity*GeoFun/sqrt(perm)*u_mag*ux + porosity*Gx + Gff_x + Gfs_x);
            Fy = rho0*(-porosity*mu_eff/perm*uy - porosity*GeoFun/sqrt(perm)*u_mag*uy + porosity*Gy + Gff_y + Gfs_y);
            Fz = rho0*(-porosity*mu_eff/perm*uz - porosity*GeoFun/sqrt(perm)*u_mag*uz + porosity*Gz + Gff_z + Gfs_z);
            if (porosity==1.0){
                Fx=rho0*(Gx + Gff_x + Gfs_x);
                Fy=rho0*(Gy + Gff_y + Gfs_y);
                Fz=rho0*(Gz + Gff_z + Gfs_z);
            }

            //Calculate pressure for Incompressible-MRT model
            pressure=0.5/porosity*(pressure-0.5*rho0*u_mag*u_mag/porosity);

//            //..............carry out relaxation process...............................................
//            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1) 
//                    + (1-0.5*rlx_setA)*38*(Fx*ux+Fy*uy+Fz*uz)/porosity;
//            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2)
//                    + (1-0.5*rlx_setA)*11*(-Fx*ux-Fy*uy-Fz*uz)/porosity;
//            jx = jx + Fx;
//            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
//            jy = jy + Fy;
//            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
//            jz = jz + Fz;
//            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
//            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9)
//                    + (1-0.5*rlx_setA)*(4*Fx*ux-2*Fy*uy-2*Fz*uz)/porosity;
//            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10)
//                      + (1-0.5*rlx_setA)*(-2*Fx*ux+Fy*uy+Fz*uz)/porosity;
//            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11)
//                      + (1-0.5*rlx_setA)*(2*Fy*uy-2*Fz*uz)/porosity;
//            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12)
//                      + (1-0.5*rlx_setA)*(-Fy*uy+Fz*uz)/porosity;
//            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13)
//                      + (1-0.5*rlx_setA)*(Fy*ux+Fx*uy)/porosity;
//            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14)
//                      + (1-0.5*rlx_setA)*(Fz*uy+Fy*uz)/porosity;
//            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15)
//                      + (1-0.5*rlx_setA)*(Fz*ux+Fx*uz)/porosity;
//            m16 = m16 + rlx_setB*( - m16);
//            m17 = m17 + rlx_setB*( - m17);
//            m18 = m18 + rlx_setB*( - m18);
//            //.......................................................................................................
           
            //-------------------- IMRT collison where body force has NO higher-order terms -------------//
            //..............carry out relaxation process...............................................
            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1);
            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2);
            jx = jx + Fx;
            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
            jy = jy + Fy;
            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
            jz = jz + Fz;
            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9);
            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10);
            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11);
            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12);
            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13);
            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14);
            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15);
            m16 = m16 + rlx_setB*( - m16);
            m17 = m17 + rlx_setB*( - m17);
            m18 = m18 + rlx_setB*( - m18);
            //.......................................................................................................


            //.................inverse transformation......................................................
            // q=0
            fq = mrt_V1*rho0-mrt_V2*m1+mrt_V3*m2;
            dist[n] = fq;

            // q = 1
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10);
            //nread = neighborList[n+Np];
            dist[nr2] = fq;

            // q=2
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10);
            //nread = neighborList[n];
            dist[nr1] = fq;

            // q = 3
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
            //nread = neighborList[n+3*Np];
            dist[nr4] = fq;

            // q = 4
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
            //nread = neighborList[n+2*Np];
            dist[nr3] = fq;

            // q = 5
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
            //nread = neighborList[n+5*Np];
            dist[nr6] = fq;

            // q = 6
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
            //nread = neighborList[n+4*Np];
            dist[nr5] = fq;

            // q = 7
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17);
            nread = neighborList[n+7*Np];
            dist[nread] = fq;

            // q = 8
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m17-m16);
            nread = neighborList[n+6*Np];
            dist[nread] = fq;

            // q = 9
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17);
            nread = neighborList[n+9*Np];
            dist[nread] = fq;

            // q = 10
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17);
            nread = neighborList[n+8*Np];
            dist[nread] = fq;

            // q = 11
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m18-m16);
            nread = neighborList[n+11*Np];
            dist[nread] = fq;

            // q = 12
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18);
            nread = neighborList[n+10*Np];
            dist[nread]= fq;

            // q = 13
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15-0.125*(m16+m18);
            nread = neighborList[n+13*Np];
            dist[nread] = fq;

            // q= 14
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15+0.125*(m16+m18);
            nread = neighborList[n+12*Np];
            dist[nread] = fq;

            // q = 15
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18);
            nread = neighborList[n+15*Np];
            dist[nread] = fq;

            // q = 16
            fq =  mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17);
            nread = neighborList[n+14*Np];
            dist[nread] = fq;

            // q = 17
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18);
            nread = neighborList[n+17*Np];
            dist[nread] = fq;

            // q = 18
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18);
            nread = neighborList[n+16*Np];
            dist[nread] = fq;
            //........................................................................

            //Update velocity on device
            Velocity[0*Np+n] = ux;
            Velocity[1*Np+n] = uy;
            Velocity[2*Np+n] = uz;
            //Update pressure on device
            Pressure[n] = pressure;

            //-----------------------Mass transport------------------------//
            // calcuale chemical potential
            chem = lambdaA*(nA*nA*nA-1.5*nA*nA+0.5*nA)-lambdaB*(nB*nB*nB-1.5*nB*nB+0.5*nB)-0.25*(kappaA+kappaB)*phi_lap;
            //rlx_phi = 3.f-sqrt(3.f);
            rlx_phi = 1.0;

			//...............................................
			// q = 0,2,4
			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
			a1 = Cq[nr2];
			a2 = Cq[nr1];
			a1 = (1.0-rlx_phi)*a1+rlx_phi*(0.1111111111111111*4.5*(gamma*chem+phi*ux));
			a2 = (1.0-rlx_phi)*a2+rlx_phi*(0.1111111111111111*4.5*(gamma*chem-phi*ux));

			// q = 1
			//nread = neighborList[n+Np];
			Cq[nr2] = a1;
			// q=2
			//nread = neighborList[n];
			Cq[nr1] = a2;

			//...............................................
			// Cq = {0,1,0}
			a1 = Cq[nr4];
			a2 = Cq[nr3];
			a1 = (1.0-rlx_phi)*a1+rlx_phi*(0.1111111111111111*4.5*(gamma*chem+phi*uy));
			a2 = (1.0-rlx_phi)*a2+rlx_phi*(0.1111111111111111*4.5*(gamma*chem-phi*uy));

			// q = 3
			//nread = neighborList[n+3*Np];
			Cq[nr4] = a1;
			// q = 4
			//nread = neighborList[n+2*Np];
			Cq[nr3] = a2;

			//...............................................
			// q = 4
			// Cq = {0,0,1}
			a1 = Cq[nr6];
			a2 = Cq[nr5];
			a1 = (1.0-rlx_phi)*a1+rlx_phi*(0.1111111111111111*4.5*(gamma*chem+phi*uz));
			a2 = (1.0-rlx_phi)*a2+rlx_phi*(0.1111111111111111*4.5*(gamma*chem-phi*uz));

			// q = 5
			//nread = neighborList[n+5*Np];
			Cq[nr6] = a1;
			// q = 6
			//nread = neighborList[n+4*Np];
			Cq[nr5] = a2;
			//...............................................

			// Instantiate mass transport distributions
			// Stationary value - distribution 0
            a1=Cq[n];
			Cq[n] = (1.0-rlx_phi)*a1+rlx_phi*(a1-3.0*gamma*chem);

		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_AAeven_GreyscaleColorChem(double *dist, double *Cq, double *Phi, double *Den,double *SolidForce, int start, int finish, int Np,
                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double gamma,double kappaA,double kappaB,double lambdaA,double lambdaB,
                double Gx, double Gy, double Gz,
                double *Poros,double *Perm, double *Velocity,double *Pressure,double *PressureGrad,double *PressTensorGrad,double *PhiLap){
	int n;
	double vx,vy,vz,v_mag;
    double ux,uy,uz,u_mag;
    double pressure;//defined for this incompressible model
	// conserved momemnts
	double jx,jy,jz;
	// non-conserved moments
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
    double fq;
    // currently disable 'GeoFun'
    double GeoFun=0.0;//geometric function from Guo's PRE 66, 036304 (2002)
    double porosity;
    double perm;//voxel permeability
    double c0, c1; //Guo's model parameters
    double Fx, Fy, Fz;//The total body force including Brinkman force and user-specified (Gx,Gy,Gz)
	double tau,tau_eff,rlx_setA,rlx_setB;
    double mu_eff;//effective kinematic viscosity for Darcy term
    double rho0;
    double phi;
    double phi_lap;//laplacian of phase field
    double nA,nB;
	//double a1,b1,a2,b2;
    double Gfs_x,Gfs_y,Gfs_z;
    double Gff_x,Gff_y,Gff_z;
    double chem;
    //double rlx_massA,rlx_massB;
    double rlx_phi;
    double a1,a2;//PDF of phase field
    // *---------------------------------Pressure Tensor Gradient------------------------------------*//
    double Pxx_x,Pyy_y,Pzz_z;
    double Pxy_x,Pxy_y;
    double Pyz_y,Pyz_z;
    double Pxz_x,Pxz_z;
    double px,py,pz; //pressure gradient


	const double mrt_V1=0.05263157894736842;
	const double mrt_V2=0.012531328320802;
	const double mrt_V3=0.04761904761904762;
	const double mrt_V4=0.004594820384294068;
	const double mrt_V5=0.01587301587301587;
	const double mrt_V6=0.0555555555555555555555555;
	const double mrt_V7=0.02777777777777778;
	const double mrt_V8=0.08333333333333333;
	const double mrt_V9=0.003341687552213868;
	const double mrt_V10=0.003968253968253968;
	const double mrt_V11=0.01388888888888889;
	const double mrt_V12=0.04166666666666666;


	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
	    //........Get 1-D index for this thread....................
	    n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;

		if ( n<finish ){

			// read the component number densities
            // TODO you can eliminate this, get nA and nB from phi
			nA = Den[n];
			nB = Den[Np + n];
            // read phase field
            phi = Phi[n];
            // load laplacian of phase field
            phi_lap = PhiLap[n];
            // Load voxel porosity and perm
            porosity = Poros[n];
            // use local saturation as an estimation of effective relperm values
            perm = Perm[n]*nA/(nA+nB)*int(phi>0.0)+Perm[n]*nB/(nA+nB)*int(phi<0.0);

            //Load pressure gradient
            px=PressureGrad[0*Np+n];
            py=PressureGrad[1*Np+n];
            pz=PressureGrad[2*Np+n];

            //Load pressure tensor gradient
            //For reference full list of PressTensorGrad
            //PressTensorGrad[n+0*Np]  = Pxx_x
            //PressTensorGrad[n+1*Np]  = Pxx_y
            //PressTensorGrad[n+2*Np]  = Pxx_z
            //PressTensorGrad[n+3*Np]  = Pyy_x
            //PressTensorGrad[n+4*Np]  = Pyy_y
            //PressTensorGrad[n+5*Np]  = Pyy_z
            //PressTensorGrad[n+6*Np]  = Pzz_x
            //PressTensorGrad[n+7*Np]  = Pzz_y
            //PressTensorGrad[n+8*Np]  = Pzz_z
            //PressTensorGrad[n+9*Np]  = Pxy_x
            //PressTensorGrad[n+10*Np] = Pxy_y
            //PressTensorGrad[n+11*Np] = Pxy_z
            //PressTensorGrad[n+12*Np] = Pyz_x
            //PressTensorGrad[n+13*Np] = Pyz_y
            //PressTensorGrad[n+14*Np] = Pyz_z
            //PressTensorGrad[n+15*Np] = Pxz_x
            //PressTensorGrad[n+16*Np] = Pxz_y
            //PressTensorGrad[n+17*Np] = Pxz_z
            Pxx_x = PressTensorGrad[0*Np+n];
            Pyy_y = PressTensorGrad[4*Np+n];
            Pzz_z = PressTensorGrad[8*Np+n];
            Pxy_x = PressTensorGrad[9*Np+n];
            Pxz_x = PressTensorGrad[15*Np+n];
		    Pxy_y = PressTensorGrad[10*Np+n];
		    Pyz_y = PressTensorGrad[13*Np+n];
		    Pyz_z = PressTensorGrad[14*Np+n];
		    Pxz_z = PressTensorGrad[17*Np+n];
		    //............Compute the fluid-fluid force (gfx,gfy,gfz)...................................
            //TODO double check if you need porosity as a fre-factor
            Gff_x = porosity*px-(Pxx_x+Pxy_y+Pxz_z);
            Gff_y = porosity*py-(Pxy_x+Pyy_y+Pyz_z);
            Gff_z = porosity*pz-(Pxz_x+Pyz_y+Pzz_z);
            // fluid-solid force
            Gfs_x = (nA-nB)*SolidForce[n+0*Np];    
            Gfs_y = (nA-nB)*SolidForce[n+1*Np];    
            Gfs_z = (nA-nB)*SolidForce[n+2*Np];    

			// local density
			rho0=rhoA + 0.5*(1.0-phi)*(rhoB-rhoA);
			// local relaxation time
			tau=tauA + 0.5*(1.0-phi)*(tauB-tauA);
			rlx_setA = 1.f/tau;
			rlx_setB = 8.f*(2.f-rlx_setA)/(8.f-rlx_setA);
			tau_eff=tauA_eff + 0.5*(1.0-phi)*(tauB_eff-tauA_eff);
            mu_eff = (tau_eff-0.5)/3.f;//kinematic viscosity


            //........................................................................
            //					READ THE DISTRIBUTIONS
            //		(read from opposite array due to previous swap operation)
            //........................................................................
            // q=0
            fq = dist[n];
            m1  = -30.0*fq;
            m2  = 12.0*fq;

            // q=1
            fq = dist[2*Np+n];
            pressure = fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jx = fq;
            m4 = -4.0*fq;
            m9 = 2.0*fq;
            m10 = -4.0*fq;

            // f2 = dist[10*Np+n];
            fq = dist[1*Np+n];
            pressure += fq;
            m1 -= 11.0*(fq);
            m2 -= 4.0*(fq);
            jx -= fq;
            m4 += 4.0*(fq);
            m9 += 2.0*(fq);
            m10 -= 4.0*(fq);

            // q=3
            fq = dist[4*Np+n];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jy = fq;
            m6 = -4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 = fq;
            m12 = -2.0*fq;

            // q = 4
            fq = dist[3*Np+n];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jy -= fq;
            m6 += 4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 += fq;
            m12 -= 2.0*fq;

            // q=5
            fq = dist[6*Np+n];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jz = fq;
            m8 = -4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 -= fq;
            m12 += 2.0*fq;

            // q = 6
            fq = dist[5*Np+n];
            pressure += fq;
            m1 -= 11.0*fq;
            m2 -= 4.0*fq;
            jz -= fq;
            m8 += 4.0*fq;
            m9 -= fq;
            m10 += 2.0*fq;
            m11 -= fq;
            m12 += 2.0*fq;

            // q=7
            fq = dist[8*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jy += fq;
            m6 += fq;
            m9  += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 = fq;
            m16 = fq;
            m17 = -fq;

            // q = 8
            fq = dist[7*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jy -= fq;
            m6 -= fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 += fq;
            m16 -= fq;
            m17 += fq;

            // q=9
            fq = dist[10*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jy -= fq;
            m6 -= fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 -= fq;
            m16 += fq;
            m17 += fq;

            // q = 10
            fq = dist[9*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jy += fq;
            m6 += fq;
            m9 += fq;
            m10 += fq;
            m11 += fq;
            m12 += fq;
            m13 -= fq;
            m16 -= fq;
            m17 -= fq;

            // q=11
            fq = dist[12*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jz += fq;
            m8 += fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 = fq;
            m16 -= fq;
            m18 = fq;

            // q=12
            fq = dist[11*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jz -= fq;
            m8 -= fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 += fq;
            m16 += fq;
            m18 -= fq;

            // q=13
            fq = dist[14*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx += fq;
            m4 += fq;
            jz -= fq;
            m8 -= fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 -= fq;
            m16 -= fq;
            m18 -= fq;

            // q=14
            fq = dist[13*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jx -= fq;
            m4 -= fq;
            jz += fq;
            m8 += fq;
            m9 += fq;
            m10 += fq;
            m11 -= fq;
            m12 -= fq;
            m15 -= fq;
            m16 += fq;
            m18 += fq;

            // q=15
            fq = dist[16*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy += fq;
            m6 += fq;
            jz += fq;
            m8 += fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 = fq;
            m17 += fq;
            m18 -= fq;

            // q=16
            fq = dist[15*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy -= fq;
            m6 -= fq;
            jz -= fq;
            m8 -= fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 += fq;
            m17 -= fq;
            m18 += fq;

            // q=17
            fq = dist[18*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy += fq;
            m6 += fq;
            jz -= fq;
            m8 -= fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 -= fq;
            m17 += fq;
            m18 += fq;

            // q=18
            fq = dist[17*Np+n];
            pressure += fq;
            m1 += 8.0*fq;
            m2 += fq;
            jy -= fq;
            m6 -= fq;
            jz += fq;
            m8 += fq;
            m9 -= 2.0*fq;
            m10 -= 2.0*fq;
            m14 -= fq;
            m17 -= fq;
            m18 -= fq;
            //---------------------------------------------------------------------//

            c0 = 0.5*(1.0+porosity*0.5*mu_eff/perm);
            if (porosity==1.0) c0 = 0.5;//i.e. apparent pore nodes
            //GeoFun = 1.75/sqrt(150.0*porosity*porosity*porosity);
            c1 = porosity*0.5*GeoFun/sqrt(perm);
            if (porosity==1.0) c1 = 0.0;//i.e. apparent pore nodes

            vx = jx/rho0+0.5*(porosity*Gx+Gff_x+Gfs_x);
            vy = jy/rho0+0.5*(porosity*Gy+Gff_y+Gfs_y);
            vz = jz/rho0+0.5*(porosity*Gz+Gff_z+Gfs_z);
            v_mag=sqrt(vx*vx+vy*vy+vz*vz);
            ux = vx/(c0+sqrt(c0*c0+c1*v_mag));
            uy = vy/(c0+sqrt(c0*c0+c1*v_mag));
            uz = vz/(c0+sqrt(c0*c0+c1*v_mag));
            u_mag=sqrt(ux*ux+uy*uy+uz*uz);

            //Update the total force to include linear (Darcy) and nonlinear (Forchheimer) drags due to the porous medium
            Fx = rho0*(-porosity*mu_eff/perm*ux - porosity*GeoFun/sqrt(perm)*u_mag*ux + porosity*Gx + Gff_x + Gfs_x);
            Fy = rho0*(-porosity*mu_eff/perm*uy - porosity*GeoFun/sqrt(perm)*u_mag*uy + porosity*Gy + Gff_y + Gfs_y);
            Fz = rho0*(-porosity*mu_eff/perm*uz - porosity*GeoFun/sqrt(perm)*u_mag*uz + porosity*Gz + Gff_z + Gfs_z);
            if (porosity==1.0){
                Fx=rho0*(Gx + Gff_x + Gfs_x);
                Fy=rho0*(Gy + Gff_y + Gfs_y);
                Fz=rho0*(Gz + Gff_z + Gfs_z);
            }

            //Calculate pressure for Incompressible-MRT model
            pressure=0.5/porosity*(pressure-0.5*rho0*u_mag*u_mag/porosity);

//            //..............carry out relaxation process...............................................
//            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1) 
//                    + (1-0.5*rlx_setA)*38*(Fx*ux+Fy*uy+Fz*uz)/porosity;
//            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2)
//                    + (1-0.5*rlx_setA)*11*(-Fx*ux-Fy*uy-Fz*uz)/porosity;
//            jx = jx + Fx;
//            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
//            jy = jy + Fy;
//            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
//            jz = jz + Fz;
//            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
//                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
//            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9)
//                    + (1-0.5*rlx_setA)*(4*Fx*ux-2*Fy*uy-2*Fz*uz)/porosity;
//            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10)
//                      + (1-0.5*rlx_setA)*(-2*Fx*ux+Fy*uy+Fz*uz)/porosity;
//            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11)
//                      + (1-0.5*rlx_setA)*(2*Fy*uy-2*Fz*uz)/porosity;
//            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12)
//                      + (1-0.5*rlx_setA)*(-Fy*uy+Fz*uz)/porosity;
//            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13)
//                      + (1-0.5*rlx_setA)*(Fy*ux+Fx*uy)/porosity;
//            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14)
//                      + (1-0.5*rlx_setA)*(Fz*uy+Fy*uz)/porosity;
//            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15)
//                      + (1-0.5*rlx_setA)*(Fz*ux+Fx*uz)/porosity;
//            m16 = m16 + rlx_setB*( - m16);
//            m17 = m17 + rlx_setB*( - m17);
//            m18 = m18 + rlx_setB*( - m18);
//            //.......................................................................................................

            //-------------------- IMRT collison where body force has NO higher-order terms -------------//
            //..............carry out relaxation process...............................................
            m1 = m1 + rlx_setA*((-30*rho0+19*(ux*ux+uy*uy+uz*uz)/porosity + 57*pressure*porosity) - m1);
            m2 = m2 + rlx_setA*((12*rho0 - 5.5*(ux*ux+uy*uy+uz*uz)/porosity-27*pressure*porosity) - m2);
            jx = jx + Fx;
            m4 = m4 + rlx_setB*((-0.6666666666666666*ux*rho0) - m4)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fx);
            jy = jy + Fy;
            m6 = m6 + rlx_setB*((-0.6666666666666666*uy*rho0) - m6)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fy);
            jz = jz + Fz;
            m8 = m8 + rlx_setB*((-0.6666666666666666*uz*rho0) - m8)
                    + (1-0.5*rlx_setB)*(-0.6666666666666666*Fz);
            m9 = m9 + rlx_setA*((rho0*(2*ux*ux-uy*uy-uz*uz)/porosity) - m9);
            m10 = m10 + rlx_setA*(-0.5*rho0*((2*ux*ux-uy*uy-uz*uz)/porosity)- m10);
            m11 = m11 + rlx_setA*((rho0*(uy*uy-uz*uz)/porosity) - m11);
            m12 = m12 + rlx_setA*(-0.5*(rho0*(uy*uy-uz*uz)/porosity)- m12);
            m13 = m13 + rlx_setA*((rho0*ux*uy/porosity) - m13);
            m14 = m14 + rlx_setA*((rho0*uy*uz/porosity) - m14);
            m15 = m15 + rlx_setA*((rho0*ux*uz/porosity) - m15);
            m16 = m16 + rlx_setB*( - m16);
            m17 = m17 + rlx_setB*( - m17);
            m18 = m18 + rlx_setB*( - m18);
            //.......................................................................................................

            //.................inverse transformation......................................................
            // q=0
            fq = mrt_V1*rho0-mrt_V2*m1+mrt_V3*m2;
            dist[n] = fq;

            // q = 1
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jx-m4)+mrt_V6*(m9-m10);
            dist[1*Np+n] = fq;

            // q=2
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m4-jx)+mrt_V6*(m9-m10);
            dist[2*Np+n] = fq;

            // q = 3
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jy-m6)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
            dist[3*Np+n] = fq;

            // q = 4
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m6-jy)+mrt_V7*(m10-m9)+mrt_V8*(m11-m12);
            dist[4*Np+n] = fq;

            // q = 5
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(jz-m8)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
            dist[5*Np+n] = fq;

            // q = 6
            fq = mrt_V1*rho0-mrt_V4*m1-mrt_V5*m2+0.1*(m8-jz)+mrt_V7*(m10-m9)+mrt_V8*(m12-m11);
            dist[6*Np+n] = fq;

            // q = 7
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jy)+0.025*(m4+m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m16-m17);
            dist[7*Np+n] = fq;

            // q = 8
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jy)-0.025*(m4+m6) +mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12+0.25*m13+0.125*(m17-m16);
            dist[8*Np+n] = fq;

            // q = 9
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jy)+0.025*(m4-m6)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13+0.125*(m16+m17);
            dist[9*Np+n] = fq;

            // q = 10
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jx)+0.025*(m6-m4)+mrt_V7*m9+mrt_V11*m10+mrt_V8*m11+mrt_V12*m12-0.25*m13-0.125*(m16+m17);
            dist[10*Np+n] = fq;

            // q = 11
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx+jz)+0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m18-m16);
            dist[11*Np+n] = fq;

            // q = 12
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jx+jz)-0.025*(m4+m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12+0.25*m15+0.125*(m16-m18);
            dist[12*Np+n] = fq;

            // q = 13
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jx-jz)+0.025*(m4-m8)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15-0.125*(m16+m18);
            dist[13*Np+n] = fq;

            // q= 14
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jx)+0.025*(m8-m4)+mrt_V7*m9+mrt_V11*m10-mrt_V8*m11-mrt_V12*m12-0.25*m15+0.125*(m16+m18);
            dist[14*Np+n] = fq;

            // q = 15
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy+jz)+0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m17-m18);
            dist[15*Np+n] = fq;

            // q = 16
            fq =  mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2-0.1*(jy+jz)-0.025*(m6+m8)-mrt_V6*m9-mrt_V7*m10+0.25*m14+0.125*(m18-m17);
            dist[16*Np+n] = fq;

            // q = 17
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jy-jz)+0.025*(m6-m8)-mrt_V6*m9-mrt_V7*m10-0.25*m14+0.125*(m17+m18);
            dist[17*Np+n] = fq;

            // q = 18
            fq = mrt_V1*rho0+mrt_V9*m1+mrt_V10*m2+0.1*(jz-jy)+0.025*(m8-m6)-mrt_V6*m9-mrt_V7*m10-0.25*m14-0.125*(m17+m18);
            dist[18*Np+n] = fq;
            //........................................................................

            //Update velocity on device
            Velocity[0*Np+n] = ux;
            Velocity[1*Np+n] = uy;
            Velocity[2*Np+n] = uz;
            //Update pressure on device
            Pressure[n] = pressure;

            //-----------------------Mass transport------------------------//
            // calcuale chemical potential
            chem = lambdaA*(nA*nA*nA-1.5*nA*nA+0.5*nA)-lambdaB*(nB*nB*nB-1.5*nB*nB+0.5*nB)-0.25*(kappaA+kappaB)*phi_lap;
            //rlx_phi = 3.f-sqrt(3.f);
            rlx_phi = 1.0;

			//...............................................
			// q = 0,2,4
			// Cq = {1,0,0}, {0,1,0}, {0,0,1}
			a1 = Cq[1*Np+n];
			a2 = Cq[2*Np+n];
			a1 = (1.0-rlx_phi)*a1+rlx_phi*(0.1111111111111111*4.5*(gamma*chem+phi*ux));
			a2 = (1.0-rlx_phi)*a2+rlx_phi*(0.1111111111111111*4.5*(gamma*chem-phi*ux));

			Cq[1*Np+n] = a1;
			Cq[2*Np+n] = a2;

			//...............................................
			// q = 2
			// Cq = {0,1,0}
			a1 = Cq[3*Np+n];
			a2 = Cq[4*Np+n];
			a1 = (1.0-rlx_phi)*a1+rlx_phi*(0.1111111111111111*4.5*(gamma*chem+phi*uy));
			a2 = (1.0-rlx_phi)*a2+rlx_phi*(0.1111111111111111*4.5*(gamma*chem-phi*uy));

			Cq[3*Np+n] = a1;
			Cq[4*Np+n] = a2;
			//...............................................
			// q = 4
			// Cq = {0,0,1}
			a1 = Cq[5*Np+n];
			a2 = Cq[6*Np+n];
			a1 = (1.0-rlx_phi)*a1+rlx_phi*(0.1111111111111111*4.5*(gamma*chem+phi*uz));
			a2 = (1.0-rlx_phi)*a2+rlx_phi*(0.1111111111111111*4.5*(gamma*chem-phi*uz));

			Cq[5*Np+n] = a1;
			Cq[6*Np+n] = a2;
			//...............................................

			// Instantiate mass transport distributions
			// Stationary value - distribution 0
            a1=Cq[n];
			Cq[n] = (1.0-rlx_phi)*a1+rlx_phi*(a1-3.0*gamma*chem);
		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_GreyColorIMRT_Init(double *dist, double *Den, double rhoA, double rhoB, int Np){
	int n;
	int S = Np/NBLOCKS/NTHREADS + 1;
    double phi;
    double nA,nB;
    double Den0;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
		if (n<Np ){
            nA = Den[n];
            nB = Den[n+Np];
            phi = (nA-nB)/(nA+nB);
            Den0 = 0.5*(1.f+phi)*rhoA + 0.5*(1.f-phi)*rhoB;

			dist[n] = Den0 - 0.6666666666666667;
			dist[Np+n] = 0.055555555555555555;		//double(100*n)+1.f;
			dist[2*Np+n] = 0.055555555555555555;	//double(100*n)+2.f;
			dist[3*Np+n] = 0.055555555555555555;	//double(100*n)+3.f;
			dist[4*Np+n] = 0.055555555555555555;	//double(100*n)+4.f;
			dist[5*Np+n] = 0.055555555555555555;	//double(100*n)+5.f;
			dist[6*Np+n] = 0.055555555555555555;	//double(100*n)+6.f;
			dist[7*Np+n] = 0.0277777777777778;   //double(100*n)+7.f;
			dist[8*Np+n] = 0.0277777777777778;   //double(100*n)+8.f;
			dist[9*Np+n] = 0.0277777777777778;   //double(100*n)+9.f;
			dist[10*Np+n] = 0.0277777777777778;  //double(100*n)+10.f;
			dist[11*Np+n] = 0.0277777777777778;  //double(100*n)+11.f;
			dist[12*Np+n] = 0.0277777777777778;  //double(100*n)+12.f;
			dist[13*Np+n] = 0.0277777777777778;  //double(100*n)+13.f;
			dist[14*Np+n] = 0.0277777777777778;  //double(100*n)+14.f;
			dist[15*Np+n] = 0.0277777777777778;  //double(100*n)+15.f;
			dist[16*Np+n] = 0.0277777777777778;  //double(100*n)+16.f;
			dist[17*Np+n] = 0.0277777777777778;  //double(100*n)+17.f;
			dist[18*Np+n] = 0.0277777777777778;  //double(100*n)+18.f;
		}
	}
}

//__global__ void dvc_ScaLBL_D3Q7_GreyColorIMRT_Init(double *Den, double *Aq, double *Bq, double *Phi, int start, int finish, int Np){
//	int idx;
//    double nA,nB;
//
//	int S = Np/NBLOCKS/NTHREADS + 1;
//	for (int s=0; s<S; s++){
//		//........Get 1-D index for this thread....................
//		idx =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
//		if (idx<finish) {
//            nA = Den[idx];
//            nB = Den[idx+Np];
//
//			Aq[idx]=0.3333333333333333*nA;
//			Aq[Np+idx]=0.1111111111111111*nA;
//			Aq[2*Np+idx]=0.1111111111111111*nA;
//			Aq[3*Np+idx]=0.1111111111111111*nA;
//			Aq[4*Np+idx]=0.1111111111111111*nA;
//			Aq[5*Np+idx]=0.1111111111111111*nA;
//			Aq[6*Np+idx]=0.1111111111111111*nA;
//
//			Bq[idx]=0.3333333333333333*nB;
//			Bq[Np+idx]=0.1111111111111111*nB;
//			Bq[2*Np+idx]=0.1111111111111111*nB;
//			Bq[3*Np+idx]=0.1111111111111111*nB;
//			Bq[4*Np+idx]=0.1111111111111111*nB;
//			Bq[5*Np+idx]=0.1111111111111111*nB;
//			Bq[6*Np+idx]=0.1111111111111111*nB;
//
//            Phi[idx] = nA-nB;
//		}
//	}
//}

__global__ void dvc_ScaLBL_D3Q7_GreyColorIMRT_Init(double *Den, double *Cq, double *PhiLap, double gamma, double kappaA, double kappaB, double lambdaA, double lambdaB,
                int start, int finish, int Np){
	int idx;
    double nA,nB;
    double phi;
    double phi_lap;//laplacian of the phase field
    double chem;//chemical potential
	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		idx =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (idx<finish) {
            nA = Den[idx];
            nB = Den[idx+Np];
            phi = nA-nB;
            phi_lap = PhiLap[idx];
            chem = lambdaA*(nA*nA*nA-1.5*nA*nA+0.5*nA)-lambdaB*(nB*nB*nB-1.5*nB*nB+0.5*nB)-0.25*(kappaA+kappaB)*phi_lap;

			Cq[1*Np+idx]=0.5*gamma*chem;
			Cq[2*Np+idx]=0.5*gamma*chem;
			Cq[3*Np+idx]=0.5*gamma*chem;
			Cq[4*Np+idx]=0.5*gamma*chem;
			Cq[5*Np+idx]=0.5*gamma*chem;
			Cq[6*Np+idx]=0.5*gamma*chem;

			Cq[0*Np+idx]= phi - 3.0*gamma*chem;
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_AAodd_GreyscaleColorDensity(int *neighborList, double *Aq, double *Bq, double *Den, double *Phi, int start, int finish, int Np){
	int n,nread;
	double fq,nA,nB;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			//..........Compute the number density for each component ............
			// q=0
			fq = Aq[n];
			nA = fq;
			fq = Bq[n];
			nB = fq;
			
			// q=1
			nread = neighborList[n]; 
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread]; 
			nB += fq;
			
			// q=2
			nread = neighborList[n+Np]; 
			fq = Aq[nread];  
			nA += fq;
			fq = Bq[nread]; 
			nB += fq;
			
			// q=3
			nread = neighborList[n+2*Np]; 
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;
			
			// q = 4
			nread = neighborList[n+3*Np]; 
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;

			// q=5
			nread = neighborList[n+4*Np];
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;
			
			// q = 6
			nread = neighborList[n+5*Np];
			fq = Aq[nread];
			nA += fq;
			fq = Bq[nread];
			nB += fq;

			// save the number densities
			Den[n] = nA;
			Den[Np+n] = nB;
            // save the phase field
			Phi[n] = nA-nB; 	
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_AAeven_GreyscaleColorDensity(double *Aq, double *Bq, double *Den, double *Phi, int start, int finish, int Np){
	int n;
	double fq,nA,nB;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			// compute number density for each component
			// q=0
			fq = Aq[n];
			nA = fq;
			fq = Bq[n];
			nB = fq;
			
			// q=1
			fq = Aq[2*Np+n];
			nA += fq;
			fq = Bq[2*Np+n];
			nB += fq;

			// q=2
			fq = Aq[1*Np+n];
			nA += fq;
			fq = Bq[1*Np+n];
			nB += fq;

			// q=3
			fq = Aq[4*Np+n];
			nA += fq;
			fq = Bq[4*Np+n];
			nB += fq;

			// q = 4
			fq = Aq[3*Np+n];
			nA += fq;
			fq = Bq[3*Np+n];
			nB += fq;
			
			// q=5
			fq = Aq[6*Np+n];
			nA += fq;
			fq = Bq[6*Np+n];
			nB += fq;
			
			// q = 6
			fq = Aq[5*Np+n];
			nA += fq;
			fq = Bq[5*Np+n];
			nB += fq;

			// save the number densities
			Den[n] = nA;
			Den[Np+n] = nB;
            // save the phase field
			Phi[n] = nA-nB; 	
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_AAodd_GreyscaleColorPhi(int *neighborList, double *Cq, double *Den, double *Phi, int start, int finish, int Np){
	int n,nread;
	double fq,phi;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			//..........Compute the number density for each component ............
			// q=0
			fq = Cq[n];
			phi = fq;
			
			// q=1
			nread = neighborList[n]; 
			fq = Cq[nread];
			phi += fq;
			
			// q=2
			nread = neighborList[n+Np]; 
			fq = Cq[nread];  
			phi += fq;
			
			// q=3
			nread = neighborList[n+2*Np]; 
			fq = Cq[nread];
			phi += fq;
			
			// q = 4
			nread = neighborList[n+3*Np]; 
			fq = Cq[nread];
			phi += fq;

			// q=5
			nread = neighborList[n+4*Np];
			fq = Cq[nread];
			phi += fq;
			
			// q = 6
			nread = neighborList[n+5*Np];
			fq = Cq[nread];
			phi += fq;

			// save the number densities
			Den[0*Np+n] = 0.5*(1.0+phi);
			Den[1*Np+n] = 0.5*(1.0-phi);
            // save the phase field
			Phi[n] = phi; 	
		}
	}
}

__global__  void dvc_ScaLBL_D3Q7_AAeven_GreyscaleColorPhi(double *Cq, double *Den, double *Phi, int start, int finish, int Np){
	int n;
	double fq,phi;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
			// compute number density for each component
			// q=0
			fq = Cq[n];
			phi = fq;
			
			// q=1
			fq = Cq[2*Np+n];
			phi += fq;

			// q=2
			fq = Cq[1*Np+n];
			phi += fq;

			// q=3
			fq = Cq[4*Np+n];
			phi += fq;

			// q = 4
			fq = Cq[3*Np+n];
			phi += fq;
			
			// q=5
			fq = Cq[6*Np+n];
			phi += fq;
			
			// q = 6
			fq = Cq[5*Np+n];
			phi += fq;

			// save the number densities
			Den[0*Np+n] = 0.5*(1.0+phi);
			Den[1*Np+n] = 0.5*(1.0-phi);
            // save the phase field
			Phi[n] = phi; 	
		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_GreyscaleColor_Gradient(int *neighborList, double *Den, double *DenGrad, int start, int finish, int Np){

	int n,nn;
	// distributions
	double m1,m2,m3,m4,m5,m6,m7,m8,m9;
	double m10,m11,m12,m13,m14,m15,m16,m17,m18;
	double nx,ny,nz;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
//			nn = neighborList[n+Np]%Np;
//			m1 = Den[nn]*int(n!=nn);
//			nn = neighborList[n]%Np;
//			m2 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+3*Np]%Np;
//			m3 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+2*Np]%Np;
//			m4 = Den[nn]*int(n!=nn);		
//			nn = neighborList[n+5*Np]%Np;
//			m5 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+4*Np]%Np;
//			m6 = Den[nn]*int(n!=nn);		
//			nn = neighborList[n+7*Np]%Np;
//			m7 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+6*Np]%Np;
//			m8 = Den[nn]*int(n!=nn);		
//			nn = neighborList[n+9*Np]%Np;
//			m9 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+8*Np]%Np;
//			m10 = Den[nn]*int(n!=nn);		
//			nn = neighborList[n+11*Np]%Np;
//			m11 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+10*Np]%Np;
//			m12 = Den[nn]*int(n!=nn);		
//			nn = neighborList[n+13*Np]%Np;
//			m13 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+12*Np]%Np;
//			m14 = Den[nn]*int(n!=nn);		
//			nn = neighborList[n+15*Np]%Np;
//			m15 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+14*Np]%Np;
//			m16 = Den[nn]*int(n!=nn);		
//			nn = neighborList[n+17*Np]%Np;
//			m17 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+16*Np]%Np;
//			m18 = Den[nn]*int(n!=nn);					

			nn = neighborList[n+Np]%Np;
			m1 = Den[nn];
			nn = neighborList[n]%Np;
			m2 = Den[nn];
			nn = neighborList[n+3*Np]%Np;
			m3 = Den[nn];
			nn = neighborList[n+2*Np]%Np;
			m4 = Den[nn];		
			nn = neighborList[n+5*Np]%Np;
			m5 = Den[nn];
			nn = neighborList[n+4*Np]%Np;
			m6 = Den[nn];		
			nn = neighborList[n+7*Np]%Np;
			m7 = Den[nn];
			nn = neighborList[n+6*Np]%Np;
			m8 = Den[nn];		
			nn = neighborList[n+9*Np]%Np;
			m9 = Den[nn];
			nn = neighborList[n+8*Np]%Np;
			m10 = Den[nn];		
			nn = neighborList[n+11*Np]%Np;
			m11 = Den[nn];
			nn = neighborList[n+10*Np]%Np;
			m12 = Den[nn];		
			nn = neighborList[n+13*Np]%Np;
			m13 = Den[nn];
			nn = neighborList[n+12*Np]%Np;
			m14 = Den[nn];		
			nn = neighborList[n+15*Np]%Np;
			m15 = Den[nn];
			nn = neighborList[n+14*Np]%Np;
			m16 = Den[nn];		
			nn = neighborList[n+17*Np]%Np;
			m17 = Den[nn];
			nn = neighborList[n+16*Np]%Np;
			m18 = Den[nn];					

			//............Compute the Color Gradient...................................
			nx = 1.f/6.f*(m1-m2+0.5*(m7-m8+m9-m10+m11-m12+m13-m14));
			ny = 1.f/6.f*(m3-m4+0.5*(m7-m8-m9+m10+m15-m16+m17-m18));
			nz = 1.f/6.f*(m5-m6+0.5*(m11-m12-m13+m14+m15-m16-m17+m18));
			
			DenGrad[n] = nx;
			DenGrad[Np+n] = ny;
			DenGrad[2*Np+n] = nz;
		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_GreyscaleColor_Laplacian(int *neighborList, double *Den, double *DenLap, int start, int finish, int Np){

	int n,nn;
	// distributions
	double m1,m2,m3,m4,m5,m6,m7,m8,m9;
	double m10,m11,m12,m13,m14,m15,m16,m17,m18;
	double lap;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {
//			nn = neighborList[n+Np]%Np;
//			m1 = Den[nn]*int(n!=nn);
//			nn = neighborList[n]%Np;
//			m2 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+3*Np]%Np;
//			m3 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+2*Np]%Np;
//			m4 = Den[nn]*int(n!=nn);		
//			nn = neighborList[n+5*Np]%Np;
//			m5 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+4*Np]%Np;
//			m6 = Den[nn]*int(n!=nn);		
//			nn = neighborList[n+7*Np]%Np;
//			m7 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+6*Np]%Np;
//			m8 = Den[nn]*int(n!=nn);		
//			nn = neighborList[n+9*Np]%Np;
//			m9 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+8*Np]%Np;
//			m10 = Den[nn]*int(n!=nn);		
//			nn = neighborList[n+11*Np]%Np;
//			m11 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+10*Np]%Np;
//			m12 = Den[nn]*int(n!=nn);		
//			nn = neighborList[n+13*Np]%Np;
//			m13 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+12*Np]%Np;
//			m14 = Den[nn]*int(n!=nn);		
//			nn = neighborList[n+15*Np]%Np;
//			m15 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+14*Np]%Np;
//			m16 = Den[nn]*int(n!=nn);		
//			nn = neighborList[n+17*Np]%Np;
//			m17 = Den[nn]*int(n!=nn);
//			nn = neighborList[n+16*Np]%Np;
//			m18 = Den[nn]*int(n!=nn);					
			
			nn = neighborList[n+Np]%Np;
			m1 = Den[nn];
			nn = neighborList[n]%Np;
			m2 = Den[nn];
			nn = neighborList[n+3*Np]%Np;
			m3 = Den[nn];
			nn = neighborList[n+2*Np]%Np;
			m4 = Den[nn];		
			nn = neighborList[n+5*Np]%Np;
			m5 = Den[nn];
			nn = neighborList[n+4*Np]%Np;
			m6 = Den[nn];		
			nn = neighborList[n+7*Np]%Np;
			m7 = Den[nn];
			nn = neighborList[n+6*Np]%Np;
			m8 = Den[nn];		
			nn = neighborList[n+9*Np]%Np;
			m9 = Den[nn];
			nn = neighborList[n+8*Np]%Np;
			m10 = Den[nn];		
			nn = neighborList[n+11*Np]%Np;
			m11 = Den[nn];
			nn = neighborList[n+10*Np]%Np;
			m12 = Den[nn];		
			nn = neighborList[n+13*Np]%Np;
			m13 = Den[nn];
			nn = neighborList[n+12*Np]%Np;
			m14 = Den[nn];		
			nn = neighborList[n+15*Np]%Np;
			m15 = Den[nn];
			nn = neighborList[n+14*Np]%Np;
			m16 = Den[nn];		
			nn = neighborList[n+17*Np]%Np;
			m17 = Den[nn];
			nn = neighborList[n+16*Np]%Np;
			m18 = Den[nn];					


            lap = 1.f/3.f*(m1+m2+m3+m4+m5+m6-6*Den[n]+0.5*(m7+m8+m9+m10+m11+m12+m13+m14+m15+m16+m17+m18-12*Den[n]));
			DenLap[n] = lap;
		}
	}
}

__global__  void dvc_ScaLBL_D3Q19_GreyscaleColor_PressureTensor(int *neighborList, double *Phi, double *PressTensor, double *PhiLap,
      		     double kappaA,double kappaB,double lambdaA,double lambdaB, int start, int finish, int Np){
	//**GreyscaleColor model related parameters:
	//kappaA, kappaB: characterize interfacial tension
	//lambdaA, lambdaB: characterize bulk free energy 
	//nA: concentration of liquid 1; 
	//nB: concentration of liquid 2;
	//nA = 0.5*(1+phi/chi)
	//nB = 0.5*(1-phi/chi)
	//nA+nB=1
	//chi: a scaling factor, is set to 1.0 for now.

	int nn,n;
	double m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
	double m3,m5,m7;
    double nx,ny,nz;//Color gradient
    double nA,nB;//ELBM parameters: concentration of liquid 1 and 2
    double phi;//phase field
    double pb;//thermodynamic bulk fluid pressure
    double Lphi;//Laplacian of phase field
    double C;//squared magnitude of the gradient of phase field
    double chi = 1.0;//legacy ELBM parameter, scale the phase field; may be useful in the future;
    double kappa = 0.25*(kappaA+kappaB)/(chi*chi);//the effective surface tension coefficient
    double Pxx,Pyy,Pzz,Pxy,Pyz,Pxz;//Pressure tensor

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
		//........Get 1-D index for this thread....................
		n =  S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;
		if (n<finish) {

//			nn = neighborList[n+Np]%Np;
//			m1 = Phi[nn]*int(n!=nn);
//			nn = neighborList[n]%Np;
//			m2 = Phi[nn]*int(n!=nn);
//			nn = neighborList[n+3*Np]%Np;
//			m3 = Phi[nn]*int(n!=nn);
//			nn = neighborList[n+2*Np]%Np;
//			m4 = Phi[nn]*int(n!=nn);		
//			nn = neighborList[n+5*Np]%Np;
//			m5 = Phi[nn]*int(n!=nn);
//			nn = neighborList[n+4*Np]%Np;
//			m6 = Phi[nn]*int(n!=nn);		
//			nn = neighborList[n+7*Np]%Np;
//			m7 = Phi[nn]*int(n!=nn);
//			nn = neighborList[n+6*Np]%Np;
//			m8 = Phi[nn]*int(n!=nn);		
//			nn = neighborList[n+9*Np]%Np;
//			m9 = Phi[nn]*int(n!=nn);
//			nn = neighborList[n+8*Np]%Np;
//			m10 = Phi[nn]*int(n!=nn);		
//			nn = neighborList[n+11*Np]%Np;
//			m11 = Phi[nn]*int(n!=nn);
//			nn = neighborList[n+10*Np]%Np;
//			m12 = Phi[nn]*int(n!=nn);		
//			nn = neighborList[n+13*Np]%Np;
//			m13 = Phi[nn]*int(n!=nn);
//			nn = neighborList[n+12*Np]%Np;
//			m14 = Phi[nn]*int(n!=nn);		
//			nn = neighborList[n+15*Np]%Np;
//			m15 = Phi[nn]*int(n!=nn);
//			nn = neighborList[n+14*Np]%Np;
//			m16 = Phi[nn]*int(n!=nn);		
//			nn = neighborList[n+17*Np]%Np;
//			m17 = Phi[nn]*int(n!=nn);
//			nn = neighborList[n+16*Np]%Np;
//			m18 = Phi[nn]*int(n!=nn);					

			nn = neighborList[n+Np]%Np;
			m1 = Phi[nn];
			nn = neighborList[n]%Np;
			m2 = Phi[nn];
			nn = neighborList[n+3*Np]%Np;
			m3 = Phi[nn];
			nn = neighborList[n+2*Np]%Np;
			m4 = Phi[nn];		
			nn = neighborList[n+5*Np]%Np;
			m5 = Phi[nn];
			nn = neighborList[n+4*Np]%Np;
			m6 = Phi[nn];		
			nn = neighborList[n+7*Np]%Np;
			m7 = Phi[nn];
			nn = neighborList[n+6*Np]%Np;
			m8 = Phi[nn];		
			nn = neighborList[n+9*Np]%Np;
			m9 = Phi[nn];
			nn = neighborList[n+8*Np]%Np;
			m10 = Phi[nn];		
			nn = neighborList[n+11*Np]%Np;
			m11 = Phi[nn];
			nn = neighborList[n+10*Np]%Np;
			m12 = Phi[nn];		
			nn = neighborList[n+13*Np]%Np;
			m13 = Phi[nn];
			nn = neighborList[n+12*Np]%Np;
			m14 = Phi[nn];		
			nn = neighborList[n+15*Np]%Np;
			m15 = Phi[nn];
			nn = neighborList[n+14*Np]%Np;
			m16 = Phi[nn];		
			nn = neighborList[n+17*Np]%Np;
			m17 = Phi[nn];
			nn = neighborList[n+16*Np]%Np;
			m18 = Phi[nn];					

			//............Compute the Color Gradient...................................
			nx = 1.f/6.f*(m1-m2+0.5*(m7-m8+m9-m10+m11-m12+m13-m14));
			ny = 1.f/6.f*(m3-m4+0.5*(m7-m8-m9+m10+m15-m16+m17-m18));
			nz = 1.f/6.f*(m5-m6+0.5*(m11-m12-m13+m14+m15-m16-m17+m18));
			C = nx*nx+ny*ny+nz*nz;
			// Laplacian of phase field
			//Lphi = 0.3333333333333333*(m1+m2+m3+m4+m5+m6)+
			//		0.16666666666666666*(m7+m8+m9+m10+m11+m12+m13+m14+m15+m16+m17+m18) - 4.0*phi;
            phi = Phi[n];
            Lphi = 1.f/3.f*(m1+m2+m3+m4+m5+m6-6*phi+0.5*(m7+m8+m9+m10+m11+m12+m13+m14+m15+m16+m17+m18-12*phi));

			//bulk pressure p_b
			nA = 0.5*(1.0+phi/chi);
			nB = 0.5*(1.0-phi/chi);
            pb = -((1.0-nA)*(1.0-nA)*nA*nA*lambdaA)*0.5 - ((1.0-nB)*(1.0-nB)*nB*nB*lambdaB)*0.5 + 
                (nA - nB)*chi*(((0.5*nA-1.5*nA*nA+nA*nA*nA)*lambdaA)/chi - ((0.5*nB-1.5*nB*nB+nB*nB*nB)*lambdaB)/chi);

			//Pressure tensors
			Pxx=pb-kappa*phi*Lphi-0.5*kappa*C + kappa*nx*nx ;
			Pyy=pb-kappa*phi*Lphi-0.5*kappa*C + kappa*ny*ny ;
			Pzz=pb-kappa*phi*Lphi-0.5*kappa*C + kappa*nz*nz ;
			Pxy= kappa*nx*ny;
			Pyz= kappa*ny*nz;
			Pxz= kappa*nx*nz;

			//...Store the Pressure Tensors....................
			PressTensor[n+0*Np] = Pxx;
			PressTensor[n+1*Np] = Pyy;
			PressTensor[n+2*Np] = Pzz;
			PressTensor[n+3*Np] = Pxy;
			PressTensor[n+4*Np] = Pyz;
			PressTensor[n+5*Np] = Pxz;
			//...............................................

			//...Store the Laplacian of phase field....................
			PhiLap[n]=Lphi;
		}
	}
}


extern "C" void ScaLBL_D3Q19_AAeven_GreyscaleColor(double *dist, double *Aq, double *Bq, double *Den,
                double *DenGradA, double *DenGradB, double *SolidForce, int start, int finish, int Np,
                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double Gsc, double Gx, double Gy, double Gz,
                double *Poros,double *Perm, double *Velocity,double *Pressure){

    dvc_ScaLBL_D3Q19_AAeven_GreyscaleColor<<<NBLOCKS,NTHREADS >>>(dist, Aq, Bq, Den, DenGradA, DenGradB, SolidForce, start, finish, Np,
                                                                 tauA, tauB, tauA_eff, tauB_eff, rhoA, rhoB, Gsc, Gx, Gy, Gz, Poros, Perm, Velocity, Pressure);

    hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_GreyscaleColor: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAodd_GreyscaleColor(int *neighborList, double *dist, double *Aq, double *Bq, double *Den,
                double *DenGradA, double *DenGradB, double *SolidForce, int start, int finish, int Np,
                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double Gsc, double Gx, double Gy, double Gz,
                double *Poros,double *Perm, double *Velocity,double *Pressure){

    dvc_ScaLBL_D3Q19_AAodd_GreyscaleColor<<<NBLOCKS,NTHREADS >>>(neighborList, dist, Aq, Bq, Den, DenGradA, DenGradB, SolidForce, start, finish, Np,
                                                                 tauA, tauB, tauA_eff, tauB_eff, rhoA, rhoB, Gsc, Gx, Gy, Gz, Poros, Perm, Velocity, Pressure);

    hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_GreyscaleColor: %s \n",hipGetErrorString(err));
	}
}

//extern "C" void ScaLBL_D3Q19_AAeven_GreyscaleColorChem(double *dist, double *Aq, double *Bq, double *Den,double *SolidForce, int start, int finish, int Np,
//                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double gamma,double kappaA,double kappaB,double lambdaA,double lambdaB,
//                double Gx, double Gy, double Gz,
//                double *Poros,double *Perm, double *Velocity,double *Pressure,double *PressureGrad,double *PressTensorGrad,double *PhiLap){
//
//    dvc_ScaLBL_D3Q19_AAeven_GreyscaleColorChem<<<NBLOCKS,NTHREADS >>>(dist, Aq, Bq, Den, SolidForce, start, finish, Np,
//                                                                 tauA, tauB, tauA_eff, tauB_eff, rhoA, rhoB, gamma,kappaA,kappaB,lambdaA,lambdaB, Gx, Gy, Gz, Poros, Perm, Velocity, Pressure,PressureGrad,PressTensorGrad,PhiLap);
//
//    cudaError_t err = cudaGetLastError();
//	if (cudaSuccess != err){
//		printf("CUDA error in ScaLBL_D3Q19_AAeven_GreyscaleColorChem: %s \n",cudaGetErrorString(err));
//	}
//}
//
//extern "C" void ScaLBL_D3Q19_AAodd_GreyscaleColorChem(int *neighborList, double *dist, double *Aq, double *Bq, double *Den,double *SolidForce, int start, int finish, int Np,
//                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double gamma,double kappaA,double kappaB,double lambdaA,double lambdaB,
//                double Gx, double Gy, double Gz,
//                double *Poros,double *Perm, double *Velocity,double *Pressure,double *PressureGrad,double *PressTensorGrad,double *PhiLap){
//
//    dvc_ScaLBL_D3Q19_AAodd_GreyscaleColorChem<<<NBLOCKS,NTHREADS >>>(neighborList, dist, Aq, Bq, Den, SolidForce, start, finish, Np,
//                                                                 tauA, tauB, tauA_eff, tauB_eff, rhoA, rhoB, gamma,kappaA,kappaB,lambdaA,lambdaB, Gx, Gy, Gz, 
//                                                                 Poros, Perm, Velocity, Pressure,PressureGrad,PressTensorGrad,PhiLap);
//
//    cudaError_t err = cudaGetLastError();
//	if (cudaSuccess != err){
//		printf("CUDA error in ScaLBL_D3Q19_AAodd_GreyscaleColorChem: %s \n",cudaGetErrorString(err));
//	}
//}

extern "C" void ScaLBL_D3Q19_AAeven_GreyscaleColorChem(double *dist, double *Cq, double *Phi, double *Den,double *SolidForce, int start, int finish, int Np,
                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double gamma,double kappaA,double kappaB,double lambdaA,double lambdaB,
                double Gx, double Gy, double Gz,
                double *Poros,double *Perm, double *Velocity,double *Pressure,double *PressureGrad,double *PressTensorGrad,double *PhiLap){

    dvc_ScaLBL_D3Q19_AAeven_GreyscaleColorChem<<<NBLOCKS,NTHREADS >>>(dist, Cq, Phi, Den, SolidForce, start, finish, Np,
                                                                 tauA, tauB, tauA_eff, tauB_eff, rhoA, rhoB, gamma,kappaA,kappaB,lambdaA,lambdaB, Gx, Gy, Gz, Poros, Perm, Velocity, Pressure,PressureGrad,PressTensorGrad,PhiLap);

    hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_GreyscaleColorChem: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAodd_GreyscaleColorChem(int *neighborList, double *dist, double *Cq, double *Phi, double *Den,double *SolidForce, int start, int finish, int Np,
                double tauA,double tauB,double tauA_eff,double tauB_eff,double rhoA,double rhoB,double gamma,double kappaA,double kappaB,double lambdaA,double lambdaB,
                double Gx, double Gy, double Gz,
                double *Poros,double *Perm, double *Velocity,double *Pressure,double *PressureGrad,double *PressTensorGrad,double *PhiLap){

    dvc_ScaLBL_D3Q19_AAodd_GreyscaleColorChem<<<NBLOCKS,NTHREADS >>>(neighborList, dist, Cq, Phi, Den, SolidForce, start, finish, Np,
                                                                 tauA, tauB, tauA_eff, tauB_eff, rhoA, rhoB, gamma,kappaA,kappaB,lambdaA,lambdaB, Gx, Gy, Gz, 
                                                                 Poros, Perm, Velocity, Pressure,PressureGrad,PressTensorGrad,PhiLap);

    hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAodd_GreyscaleColorChem: %s \n",hipGetErrorString(err));
	}
}

//extern "C" void ScaLBL_D3Q7_GreyColorIMRT_Init(double *Den, double *Aq, double *Bq, double *Phi, int start, int finish, int Np){
//	dvc_ScaLBL_D3Q7_GreyColorIMRT_Init<<<NBLOCKS,NTHREADS >>>(Den, Aq, Bq, Phi, start, finish, Np);
//	cudaError_t err = cudaGetLastError();
//	if (cudaSuccess != err){
//		printf("CUDA error in ScaLBL_D3Q7_GreyColorIMRT_Init: %s \n",cudaGetErrorString(err));
//	}
//}

extern "C" void ScaLBL_D3Q7_GreyColorIMRT_Init(double *Den, double *Cq, double *PhiLap, double gamma, double kappaA, double kappaB, double lambdaA, double lambdaB, int start, int finish, int Np){
	dvc_ScaLBL_D3Q7_GreyColorIMRT_Init<<<NBLOCKS,NTHREADS >>>(Den, Cq, PhiLap,gamma,kappaA,kappaB,lambdaA,lambdaB, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_GreyColorIMRT_Init: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_GreyColorIMRT_Init(double *dist, double *Den, double rhoA, double rhoB, int Np){
	dvc_ScaLBL_D3Q19_GreyColorIMRT_Init<<<NBLOCKS,NTHREADS >>>(dist,Den,rhoA,rhoB,Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_GreyColorIMRT_Init: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q7_AAodd_GreyscaleColorDensity(int *NeighborList, double *Aq, double *Bq, double *Den, double *Phi, int start, int finish, int Np){

	dvc_ScaLBL_D3Q7_AAodd_GreyscaleColorDensity<<<NBLOCKS,NTHREADS >>>(NeighborList, Aq, Bq, Den, Phi, start, finish, Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_AAodd_GreyscaleColorDensity: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q7_AAeven_GreyscaleColorDensity(double *Aq, double *Bq, double *Den, double *Phi, int start, int finish, int Np){

	dvc_ScaLBL_D3Q7_AAeven_GreyscaleColorDensity<<<NBLOCKS,NTHREADS >>>(Aq, Bq, Den, Phi, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_AAeven_GreyscaleColorDensity: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q7_AAodd_GreyscaleColorPhi(int *NeighborList, double *Cq, double *Den, double *Phi, int start, int finish, int Np){

	dvc_ScaLBL_D3Q7_AAodd_GreyscaleColorPhi<<<NBLOCKS,NTHREADS >>>(NeighborList, Cq, Den, Phi, start, finish, Np);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_AAodd_GreyscaleColorPhi: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q7_AAeven_GreyscaleColorPhi(double *Cq, double *Den, double *Phi, int start, int finish, int Np){

	dvc_ScaLBL_D3Q7_AAeven_GreyscaleColorPhi<<<NBLOCKS,NTHREADS >>>(Cq, Den, Phi, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q7_AAeven_GreyscaleColorPhi: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_GreyscaleColor_Gradient(int *neighborList, double *Den, double *DenGrad, int start, int finish, int Np){

	dvc_ScaLBL_D3Q19_GreyscaleColor_Gradient<<<NBLOCKS,NTHREADS >>>(neighborList, Den, DenGrad, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_GreyscaleColor_Gradient: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_GreyscaleColor_Laplacian(int *neighborList, double *Den, double *DenLap, int start, int finish, int Np){
	dvc_ScaLBL_D3Q19_GreyscaleColor_Laplacian<<<NBLOCKS,NTHREADS >>>(neighborList, Den, DenLap, start, finish, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_GreyscaleColor_Laplacian: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_GreyscaleColor_Pressure(double *dist, double *Den, double *Porosity,double *Velocity,
                double *Pressure, double rhoA,double rhoB, int Np){

	dvc_ScaLBL_D3Q19_GreyscaleColor_Pressure<<<NBLOCKS,NTHREADS >>>(dist, Den, Porosity, Velocity, Pressure, rhoA, rhoB, Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_GreyscaleColor_Pressure: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_GreyscaleColor_PressureTensor(int *neighborList, double *Phi, double *PressTensor, double *PhiLap,
      		     double kappaA,double kappaB,double lambdaA,double lambdaB, int start, int finish, int Np){
	dvc_ScaLBL_D3Q19_GreyscaleColor_PressureTensor<<<NBLOCKS,NTHREADS >>>(neighborList,Phi,PressTensor,PhiLap,kappaA,kappaB,lambdaA,lambdaB,start,finish,Np);
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_GreyscaleColor_PressureTensor: %s \n",hipGetErrorString(err));
	}
}
