/*
  Copyright 2013--2018 James E. McClure, Virginia Polytechnic & State University

  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS 1024
#define NTHREADS 256

__global__ void dvc_ScaLBL_D3Q19_AAeven_BGK(double *dist, int start, int finish, int Np, double rlx, double Fx, double Fy, double Fz){
	int n;
	// conserved momemnts
	double rho,ux,uy,uz,uu;
	// non-conserved moments
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
	    //........Get 1-D index for this thread....................
	    		  n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;

			    if ( n<finish ){
		// q=0
		f0 = dist[n];
		f1 = dist[2*Np+n];
		f2 = dist[1*Np+n];
		f3 = dist[4*Np+n];
		f4 = dist[3*Np+n];
		f5 = dist[6*Np+n];
		f6 = dist[5*Np+n];
		f7 = dist[8*Np+n];
		f8 = dist[7*Np+n];
		f9 = dist[10*Np+n];
		f10 = dist[9*Np+n];
		f11 = dist[12*Np+n];
		f12 = dist[11*Np+n];
		f13 = dist[14*Np+n];
		f14 = dist[13*Np+n];
		f15 = dist[16*Np+n];
		f16 = dist[15*Np+n];
		f17 = dist[18*Np+n];
		f18 = dist[17*Np+n];

		rho = f0+f2+f1+f4+f3+f6+f5+f8+f7+f10+f9+f12+f11+f14+f13+f16+f15+f18+f17;
		ux = f1-f2+f7-f8+f9-f10+f11-f12+f13-f14;
		uy = f3-f4+f7-f8-f9+f10+f15-f16+f17-f18;
		uz = f5-f6+f11-f12-f13+f14+f15-f16-f17+f18;
		uu = 1.5*(ux*ux+uy*uy+uz*uz);

		// q=0
		dist[n] = f0*(1.0-rlx)+rlx*0.3333333333333333*(1.0-uu);

		// q = 1
		dist[1*Np+n] = f1*(1.0-rlx) + rlx*0.05555555555555555*(rho + 3.0*ux + 4.5*ux*ux - uu) + 0.16666666*Fx;

		// q=2
		dist[2*Np+n] = f2*(1.0-rlx) + rlx*0.05555555555555555*(rho - 3.0*ux + 4.5*ux*ux - uu)-  0.16666666*Fx;

		// q = 3
		dist[3*Np+n] = f3*(1.0-rlx) +
				rlx*0.05555555555555555*(rho + 3.0*uy + 4.5*uy*uy - uu) + 0.16666666*Fy;

		// q = 4
		dist[4*Np+n] = f4*(1.0-rlx) + 
				rlx*0.05555555555555555*(rho - 3.0*uy + 4.5*uy*uy - uu)- 0.16666666*Fy;

		// q = 5
		dist[5*Np+n] = f5*(1.0-rlx) + 
				rlx*0.05555555555555555*(rho + 3.0*uz + 4.5*uz*uz - uu) + 0.16666666*Fz;

		// q = 6
		dist[6*Np+n] = f6*(1.0-rlx) + 
				rlx*0.05555555555555555*(rho - 3.0*uz + 4.5*uz*uz - uu) - 0.16666666*Fz;

		// q = 7
		dist[7*Np+n] = f7*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho + 3.0*(ux+uy) + 4.5*(ux+uy)*(ux+uy) - uu) + 0.08333333333*(Fx+Fy);

		// q = 8
		dist[8*Np+n] = f8*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho - 3.0*(ux+uy) + 4.5*(ux+uy)*(ux+uy) - uu) - 0.08333333333*(Fx+Fy);

		// q = 9
		dist[9*Np+n] = f9*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho + 3.0*(ux-uy) + 4.5*(ux-uy)*(ux-uy) - uu) + 0.08333333333*(Fx-Fy);

		// q = 10
		dist[10*Np+n] = f10*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho - 3.0*(ux-uy) + 4.5*(ux-uy)*(ux-uy) - uu) - 0.08333333333*(Fx-Fy);

		// q = 11
		dist[11*Np+n] = f11*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho + 3.0*(ux+uz) + 4.5*(ux+uz)*(ux+uz) - uu) + 0.08333333333*(Fx+Fz);

		// q = 12
		dist[12*Np+n] = f12*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho - 3.0*(ux+uz) + 4.5*(ux+uz)*(ux+uz) - uu)  - 0.08333333333*(Fx+Fz);

		// q = 13
		dist[13*Np+n] = f13*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho + 3.0*(ux-uz) + 4.5*(ux-uz)*(ux-uz) - uu) + 0.08333333333*(Fx-Fz);

		// q= 14
		dist[14*Np+n] = f14*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho - 3.0*(ux-uz) + 4.5*(ux-uz)*(ux-uz) - uu)- 0.08333333333*(Fx-Fz);

		// q = 15
		dist[15*Np+n] = f15*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho + 3.0*(uy+uz) + 4.5*(uy+uz)*(uy+uz) - uu) + 0.08333333333*(Fy+Fz);

		// q = 16
		dist[16*Np+n] = f16*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho - 3.0*(uy+uz) + 4.5*(uy+uz)*(uy+uz) - uu) - 0.08333333333*(Fy+Fz);

		// q = 17
		dist[17*Np+n] = f17*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho + 3.0*(uy-uz) + 4.5*(uy-uz)*(uy-uz) - uu) + 0.08333333333*(Fy-Fz);

		// q = 18
		dist[18*Np+n] = f18*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho - 3.0*(uy-uz) + 4.5*(uy-uz)*(uy-uz) - uu) - 0.08333333333*(Fy-Fz);

		//........................................................................
		}
	}
}

__global__ void dvc_ScaLBL_D3Q19_AAodd_BGK(int *neighborList, double *dist, int start, int finish, int Np, double rlx, double Fx, double Fy, double Fz){
	int n;
	// conserved momemnts
	double rho,ux,uy,uz,uu;
	// non-conserved moments
	double f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int nr1,nr2,nr3,nr4,nr5,nr6,nr7,nr8,nr9,nr10,nr11,nr12,nr13,nr14,nr15,nr16,nr17,nr18;

	int S = Np/NBLOCKS/NTHREADS + 1;
	for (int s=0; s<S; s++){
	    //........Get 1-D index for this thread....................
	    		  n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x + start;

			    if ( n<finish ){		
		// q=0
		f0 = dist[n];
		// q=1
		nr1 = neighborList[n]; // neighbor 2 ( > 10Np => odd part of dist)
		f1 = dist[nr1]; // reading the f1 data into register fq

		nr2 = neighborList[n+Np]; // neighbor 1 ( < 10Np => even part of dist)
		f2 = dist[nr2];  // reading the f2 data into register fq

		// q=3
		nr3 = neighborList[n+2*Np]; // neighbor 4
		f3 = dist[nr3];

		// q = 4
		nr4 = neighborList[n+3*Np]; // neighbor 3
		f4 = dist[nr4];

		// q=5
		nr5 = neighborList[n+4*Np];
		f5 = dist[nr5];

		// q = 6
		nr6 = neighborList[n+5*Np];
		f6 = dist[nr6];
		
		// q=7
		nr7 = neighborList[n+6*Np];
		f7 = dist[nr7];

		// q = 8
		nr8 = neighborList[n+7*Np];
		f8 = dist[nr8];

		// q=9
		nr9 = neighborList[n+8*Np];
		f9 = dist[nr9];

		// q = 10
		nr10 = neighborList[n+9*Np];
		f10 = dist[nr10];

		// q=11
		nr11 = neighborList[n+10*Np];
		f11 = dist[nr11];

		// q=12
		nr12 = neighborList[n+11*Np];
		f12 = dist[nr12];

		// q=13
		nr13 = neighborList[n+12*Np];
		f13 = dist[nr13];

		// q=14
		nr14 = neighborList[n+13*Np];
		f14 = dist[nr14];

		// q=15
		nr15 = neighborList[n+14*Np];
		f15 = dist[nr15];

		// q=16
		nr16 = neighborList[n+15*Np];
		f16 = dist[nr16];

		// q=17
		//fq = dist[18*Np+n];
		nr17 = neighborList[n+16*Np];
		f17 = dist[nr17];

		// q=18
		nr18 = neighborList[n+17*Np];
		f18 = dist[nr18];

		rho = f0+f2+f1+f4+f3+f6+f5+f8+f7+f10+f9+f12+f11+f14+f13+f16+f15+f18+f17;
		ux = f1-f2+f7-f8+f9-f10+f11-f12+f13-f14;
		uy = f3-f4+f7-f8-f9+f10+f15-f16+f17-f18;
		uz = f5-f6+f11-f12-f13+f14+f15-f16-f17+f18;
		uu = 1.5*(ux*ux+uy*uy+uz*uz);

		// q=0
		dist[n] = f0*(1.0-rlx)+rlx*0.3333333333333333*(1.0-uu);

		// q = 1
		dist[nr2] = f1*(1.0-rlx) + rlx*0.05555555555555555*(rho + 3.0*ux + 4.5*ux*ux - uu) + 0.16666666*Fx;

		// q=2
		dist[nr1] = f2*(1.0-rlx) + rlx*0.05555555555555555*(rho - 3.0*ux + 4.5*ux*ux - uu)-  0.16666666*Fx;

		// q = 3
		dist[nr4] = f3*(1.0-rlx) +
				rlx*0.05555555555555555*(rho + 3.0*uy + 4.5*uy*uy - uu) + 0.16666666*Fy;

		// q = 4
		dist[nr3] = f4*(1.0-rlx) + 
				rlx*0.05555555555555555*(rho - 3.0*uy + 4.5*uy*uy - uu)- 0.16666666*Fy;

		// q = 5
		dist[nr6] = f5*(1.0-rlx) + 
				rlx*0.05555555555555555*(rho + 3.0*uz + 4.5*uz*uz - uu) + 0.16666666*Fz;

		// q = 6
		dist[nr5] = f6*(1.0-rlx) + 
				rlx*0.05555555555555555*(rho - 3.0*uz + 4.5*uz*uz - uu) - 0.16666666*Fz;

		// q = 7
		dist[nr8] = f7*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho + 3.0*(ux+uy) + 4.5*(ux+uy)*(ux+uy) - uu) + 0.08333333333*(Fx+Fy);

		// q = 8
		dist[nr7] = f8*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho - 3.0*(ux+uy) + 4.5*(ux+uy)*(ux+uy) - uu) - 0.08333333333*(Fx+Fy);

		// q = 9
		dist[nr10] = f9*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho + 3.0*(ux-uy) + 4.5*(ux-uy)*(ux-uy) - uu) + 0.08333333333*(Fx-Fy);

		// q = 10
		dist[nr9] = f10*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho - 3.0*(ux-uy) + 4.5*(ux-uy)*(ux-uy) - uu) - 0.08333333333*(Fx-Fy);

		// q = 11
		dist[nr12] = f11*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho + 3.0*(ux+uz) + 4.5*(ux+uz)*(ux+uz) - uu) + 0.08333333333*(Fx+Fz);

		// q = 12
		dist[nr11] = f12*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho - 3.0*(ux+uz) + 4.5*(ux+uz)*(ux+uz) - uu)  - 0.08333333333*(Fx+Fz);

		// q = 13
		dist[nr14] = f13*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho + 3.0*(ux-uz) + 4.5*(ux-uz)*(ux-uz) - uu) + 0.08333333333*(Fx-Fz);

		// q= 14
		dist[nr13] = f14*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho - 3.0*(ux-uz) + 4.5*(ux-uz)*(ux-uz) - uu)- 0.08333333333*(Fx-Fz);

		// q = 15
		dist[nr16] = f15*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho + 3.0*(uy+uz) + 4.5*(uy+uz)*(uy+uz) - uu) + 0.08333333333*(Fy+Fz);

		// q = 16
		dist[nr15] = f16*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho - 3.0*(uy+uz) + 4.5*(uy+uz)*(uy+uz) - uu) - 0.08333333333*(Fy+Fz);

		// q = 17
		dist[nr18] = f17*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho + 3.0*(uy-uz) + 4.5*(uy-uz)*(uy-uz) - uu) + 0.08333333333*(Fy-Fz);

		// q = 18
		dist[nr17] = f18*(1.0-rlx) + 
				rlx*0.02777777777777778*(rho - 3.0*(uy-uz) + 4.5*(uy-uz)*(uy-uz) - uu) - 0.08333333333*(Fy-Fz);
		}
	}
}

extern "C" void ScaLBL_D3Q19_AAeven_BGK(double *dist, int start, int finish, int Np, double rlx, double Fx, double Fy, double Fz){
	
    dvc_ScaLBL_D3Q19_AAeven_BGK<<<NBLOCKS,NTHREADS >>>(dist,start,finish,Np,rlx,Fx,Fy,Fz);

    hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_BGK: %s \n",hipGetErrorString(err));
	}
}

extern "C" void ScaLBL_D3Q19_AAodd_BGK(int *neighborList, double *dist, int start, int finish, int Np, double rlx, double Fx, double Fy, double Fz){
    dvc_ScaLBL_D3Q19_AAodd_BGK<<<NBLOCKS,NTHREADS >>>(neighborList,dist,start,finish,Np,rlx,Fx,Fy,Fz);

    hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		printf("CUDA error in ScaLBL_D3Q19_AAeven_BGK: %s \n",hipGetErrorString(err));
	}
}